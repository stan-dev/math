#include "hip/hip_runtime.h"
/* ----------------------------------------------------------------- 
 * Programmer(s): Slaven Peles @ LLNL                               
 * -----------------------------------------------------------------
 * LLNS Copyright Start
 * Copyright (c) 2014, Lawrence Livermore National Security
 * This work was performed under the auspices of the U.S. Department 
 * of Energy by Lawrence Livermore National Laboratory in part under 
 * Contract W-7405-Eng-48 and in part under Contract DE-AC52-07NA27344.
 * Produced at the Lawrence Livermore National Laboratory.
 * All rights reserved.
 * For details, see the LICENSE file.
 * LLNS Copyright End
 * -----------------------------------------------------------------*/

#include <stdio.h>
#include <stdlib.h>

#include <nvector/raja/Vector.hpp>
#include <RAJA/RAJA.hpp>

#define ZERO   RCONST(0.0)
#define HALF   RCONST(0.5)
#define ONE    RCONST(1.0)
#define ONEPT5 RCONST(1.5)

extern "C" {

using namespace sunrajavec;

static constexpr sunindextype zeroIdx = 0;

/* ----------------------------------------------------------------
 * Returns vector type ID. Used to identify vector implementation
 * from abstract N_Vector interface.
 */
N_Vector_ID N_VGetVectorID_Raja(N_Vector v)
{
  return SUNDIALS_NVEC_RAJA;
}

N_Vector N_VNewEmpty_Raja(sunindextype length)
{
  N_Vector v;
  N_Vector_Ops ops;
  N_VectorContent_Raja content;

  /* Create vector */
  v = NULL;
  v = (N_Vector) malloc(sizeof *v);
  if (v == NULL) return(NULL);
  
  /* Create vector operation structure */
  ops = NULL;
  ops = (N_Vector_Ops) malloc(sizeof(struct _generic_N_Vector_Ops));
  if (ops == NULL) { free(v); return(NULL); }

  ops->nvgetvectorid     = N_VGetVectorID_Raja;
  ops->nvclone           = N_VClone_Raja;
  ops->nvcloneempty      = N_VCloneEmpty_Raja;
  ops->nvdestroy         = N_VDestroy_Raja;
  ops->nvspace           = N_VSpace_Raja;
  ops->nvgetarraypointer = NULL; //N_VGetArrayPointer_Raja;
  ops->nvsetarraypointer = NULL; //N_VSetArrayPointer_Raja;
  ops->nvlinearsum       = N_VLinearSum_Raja;
  ops->nvconst           = N_VConst_Raja;
  ops->nvprod            = N_VProd_Raja;
  ops->nvdiv             = N_VDiv_Raja;
  ops->nvscale           = N_VScale_Raja;
  ops->nvabs             = N_VAbs_Raja;
  ops->nvinv             = N_VInv_Raja;
  ops->nvaddconst        = N_VAddConst_Raja;
  ops->nvdotprod         = N_VDotProd_Raja;
  ops->nvmaxnorm         = N_VMaxNorm_Raja;
  ops->nvwrmsnormmask    = N_VWrmsNormMask_Raja;
  ops->nvwrmsnorm        = N_VWrmsNorm_Raja;
  ops->nvmin             = N_VMin_Raja;
  ops->nvwl2norm         = N_VWL2Norm_Raja;
  ops->nvl1norm          = N_VL1Norm_Raja;
  ops->nvcompare         = N_VCompare_Raja;
  ops->nvinvtest         = N_VInvTest_Raja;
  ops->nvconstrmask      = N_VConstrMask_Raja;
  ops->nvminquotient     = N_VMinQuotient_Raja;

  /* Create content */
  content = NULL;

  /* Attach content and ops */
  v->content = content;
  v->ops     = ops;

  return(v);
}

    
N_Vector N_VNew_Raja(sunindextype length)
{
  N_Vector v;

  v = NULL;
  v = N_VNewEmpty_Raja(length);
  if (v == NULL) return(NULL);

  v->content = new Vector<realtype, sunindextype>(length);

  return(v);
}


N_Vector N_VMake_Raja(N_VectorContent_Raja c)
{
  N_Vector v;
  Vector<realtype, sunindextype>* x = static_cast<Vector<realtype, sunindextype>*>(c);
  sunindextype length = x->size();

  v = NULL;
  v = N_VNewEmpty_Raja(length);
  if (v == NULL) return(NULL);

  v->content = c;

  return(v);
}


/* ----------------------------------------------------------------------------
 * Function to create an array of new RAJA-based vectors.
 */

N_Vector *N_VCloneVectorArray_Raja(int count, N_Vector w)
{
  N_Vector *vs;
  int j;

  if (count <= 0) return(NULL);

  vs = NULL;
  vs = (N_Vector *) malloc(count * sizeof(N_Vector));
  if(vs == NULL) return(NULL);

  for (j = 0; j < count; j++) {
    vs[j] = NULL;
    vs[j] = N_VClone_Raja(w);
    if (vs[j] == NULL) {
      N_VDestroyVectorArray_Raja(vs, j-1);
      return(NULL);
    }
  }

  return(vs);
}

/* ----------------------------------------------------------------------------
 * Function to create an array of new RAJA-based vectors with NULL data array.
 */

N_Vector *N_VCloneVectorArrayEmpty_Raja(int count, N_Vector w)
{
  N_Vector *vs;
  int j;

  if (count <= 0) return(NULL);

  vs = NULL;
  vs = (N_Vector *) malloc(count * sizeof(N_Vector));
  if(vs == NULL) return(NULL);

  for (j = 0; j < count; j++) {
    vs[j] = NULL;
    vs[j] = N_VCloneEmpty_Raja(w);
    if (vs[j] == NULL) {
      N_VDestroyVectorArray_Raja(vs, j-1);
      return(NULL);
    }
  }

  return(vs);
}

/* ----------------------------------------------------------------------------
 * Function to free an array created with N_VCloneVectorArray_Raja
 */

void N_VDestroyVectorArray_Raja(N_Vector *vs, int count)
{
  int j;

  for (j = 0; j < count; j++) N_VDestroy_Raja(vs[j]);

  free(vs); vs = NULL;

  return;
}

/* -----------------------------------------------------------------
 * Function to return the length of the vector.
 */
sunindextype N_VGetLength_Raja(N_Vector v)
{
  Vector<realtype, sunindextype>* xd = static_cast<Vector<realtype, sunindextype>*>(v->content);
  return xd->size();
}

/* ----------------------------------------------------------------------------
 * Return pointer to the raw host data
 */

realtype *N_VGetHostArrayPointer_Raja(N_Vector x)
{
  Vector<realtype, sunindextype>* xv = static_cast<Vector<realtype, sunindextype>*>(x->content);
  return (xv->host());
}

/* ----------------------------------------------------------------------------
 * Return pointer to the raw device data
 */

realtype *N_VGetDeviceArrayPointer_Raja(N_Vector x)
{
  Vector<realtype, sunindextype>* xv = static_cast<Vector<realtype, sunindextype>*>(x->content);
  return (xv->device());
}

/* ----------------------------------------------------------------------------
 * Copy vector data to the device
 */

void N_VCopyToDevice_Raja(N_Vector x)
{
  Vector<realtype, sunindextype>* xv = static_cast<Vector<realtype, sunindextype>*>(x->content);
  xv->copyToDev();
}

/* ----------------------------------------------------------------------------
 * Copy vector data from the device to the host
 */

void N_VCopyFromDevice_Raja(N_Vector x)
{
  Vector<realtype, sunindextype>* xv = static_cast<Vector<realtype, sunindextype>*>(x->content);
  xv->copyFromDev();
}

/* ----------------------------------------------------------------------------
 * Function to print the a serial vector to stdout
 */

void N_VPrint_Raja(N_Vector X)
{
  N_VPrintFile_Raja(X, stdout);
}

/* ----------------------------------------------------------------------------
 * Function to print the a serial vector to outfile
 */

void N_VPrintFile_Raja(N_Vector X, FILE *outfile)
{
  const realtype *xd = getDevData<realtype, sunindextype>(X);
  const sunindextype N = getSize<realtype, sunindextype>(X);
  sunindextype i;

  for (i = 0; i < N; ++i) {
#if defined(SUNDIALS_EXTENDED_PRECISION)
    fprintf(outfile, "%35.32Lg\n", xd[i]);
#elif defined(SUNDIALS_DOUBLE_PRECISION)
    fprintf(outfile, "%19.16g\n", xd[i]);
#else
    fprintf(outfile, "%11.8g\n", xd[i]);
#endif
  }
  fprintf(outfile, "\n");

  return;
}

/*
 * -----------------------------------------------------------------
 * implementation of vector operations
 * -----------------------------------------------------------------
 */

N_Vector N_VCloneEmpty_Raja(N_Vector w)
{
  N_Vector v;
  N_Vector_Ops ops;

  if (w == NULL) return(NULL);

  /* Create vector */
  v = NULL;
  v = (N_Vector) malloc(sizeof *v);
  if (v == NULL) return(NULL);

  /* Create vector operation structure */
  ops = NULL;
  ops = (N_Vector_Ops) malloc(sizeof(struct _generic_N_Vector_Ops));
  if (ops == NULL) { free(v); return(NULL); }

  ops->nvgetvectorid     = w->ops->nvgetvectorid;
  ops->nvclone           = w->ops->nvclone;
  ops->nvcloneempty      = w->ops->nvcloneempty;
  ops->nvdestroy         = w->ops->nvdestroy;
  ops->nvspace           = w->ops->nvspace;
  ops->nvgetarraypointer = w->ops->nvgetarraypointer;
  ops->nvsetarraypointer = w->ops->nvsetarraypointer;
  ops->nvlinearsum       = w->ops->nvlinearsum;
  ops->nvconst           = w->ops->nvconst;
  ops->nvprod            = w->ops->nvprod;
  ops->nvdiv             = w->ops->nvdiv;
  ops->nvscale           = w->ops->nvscale;
  ops->nvabs             = w->ops->nvabs;
  ops->nvinv             = w->ops->nvinv;
  ops->nvaddconst        = w->ops->nvaddconst;
  ops->nvdotprod         = w->ops->nvdotprod;
  ops->nvmaxnorm         = w->ops->nvmaxnorm;
  ops->nvwrmsnormmask    = w->ops->nvwrmsnormmask;
  ops->nvwrmsnorm        = w->ops->nvwrmsnorm;
  ops->nvmin             = w->ops->nvmin;
  ops->nvwl2norm         = w->ops->nvwl2norm;
  ops->nvl1norm          = w->ops->nvl1norm;
  ops->nvcompare         = w->ops->nvcompare;
  ops->nvinvtest         = w->ops->nvinvtest;
  ops->nvconstrmask      = w->ops->nvconstrmask;
  ops->nvminquotient     = w->ops->nvminquotient;

  /* Create content */
  v->content = NULL;
  v->ops  = ops;

  return(v);
}

N_Vector N_VClone_Raja(N_Vector w)
{
  N_Vector v;
  Vector<realtype, sunindextype>* wdat = static_cast<Vector<realtype, sunindextype>*>(w->content);
  Vector<realtype, sunindextype>* vdat = new Vector<realtype, sunindextype>(*wdat);
  v = NULL;
  v = N_VCloneEmpty_Raja(w);
  if (v == NULL) return(NULL);

  v->content = vdat;

  return(v);
}


void N_VDestroy_Raja(N_Vector v)
{
  Vector<realtype, sunindextype>* x = static_cast<Vector<realtype, sunindextype>*>(v->content);
  if (x != NULL) {
    delete x;
    v->content = NULL;
  }

  free(v->ops); v->ops = NULL;
  free(v); v = NULL;

  return;
}

void N_VSpace_Raja(N_Vector X, sunindextype *lrw, sunindextype *liw)
{
    *lrw = getSize<realtype, sunindextype>(X);
    *liw = 1;
}

void N_VConst_Raja(realtype c, N_Vector Z)
{
  const sunindextype N = getSize<realtype, sunindextype>(Z);
  realtype *zdata = getDevData<realtype, sunindextype>(Z);

  RAJA::forall<RAJA::cuda_exec<256> >(zeroIdx, N, [=] __device__(sunindextype i) {
     zdata[i] = c;
  });
}

void N_VLinearSum_Raja(realtype a, N_Vector X, realtype b, N_Vector Y, N_Vector Z)
{
  const realtype *xdata = getDevData<realtype, sunindextype>(X);
  const realtype *ydata = getDevData<realtype, sunindextype>(Y);
  const sunindextype N = getSize<realtype, sunindextype>(X);
  realtype *zdata = getDevData<realtype, sunindextype>(Z);

  RAJA::forall<RAJA::cuda_exec<256> >(zeroIdx, N, [=] __device__(sunindextype i) {
     zdata[i] = a*xdata[i] + b*ydata[i];
  });
}

void N_VProd_Raja(N_Vector X, N_Vector Y, N_Vector Z)
{
  const realtype *xdata = getDevData<realtype, sunindextype>(X);
  const realtype *ydata = getDevData<realtype, sunindextype>(Y);
  const sunindextype N = getSize<realtype, sunindextype>(X);
  realtype *zdata = getDevData<realtype, sunindextype>(Z);

  RAJA::forall<RAJA::cuda_exec<256> >(zeroIdx, N, [=] __device__(sunindextype i) {
     zdata[i] = xdata[i] * ydata[i];
  });
}

void N_VDiv_Raja(N_Vector X, N_Vector Y, N_Vector Z)
{
  const realtype *xdata = getDevData<realtype, sunindextype>(X);
  const realtype *ydata = getDevData<realtype, sunindextype>(Y);
  const sunindextype N = getSize<realtype, sunindextype>(X);
  realtype *zdata = getDevData<realtype, sunindextype>(Z);

  RAJA::forall<RAJA::cuda_exec<256> >(zeroIdx, N, [=] __device__(sunindextype i) {
     zdata[i] = xdata[i] / ydata[i];
  });
}

void N_VScale_Raja(realtype c, N_Vector X, N_Vector Z)
{
  const realtype *xdata = getDevData<realtype, sunindextype>(X);
  const sunindextype N = getSize<realtype, sunindextype>(X);
  realtype *zdata = getDevData<realtype, sunindextype>(Z);

  RAJA::forall<RAJA::cuda_exec<256> >(zeroIdx, N, [=] __device__(sunindextype i) {
     zdata[i] = c * xdata[i];
  });
}

void N_VAbs_Raja(N_Vector X, N_Vector Z)
{
  const realtype *xdata = getDevData<realtype, sunindextype>(X);
  const sunindextype N = getSize<realtype, sunindextype>(X);
  realtype *zdata = getDevData<realtype, sunindextype>(Z);

  RAJA::forall<RAJA::cuda_exec<256> >(zeroIdx, N, [=] __device__(sunindextype i) {
     zdata[i] = abs(xdata[i]);
  });
}

void N_VInv_Raja(N_Vector X, N_Vector Z)
{
  const realtype *xdata = getDevData<realtype, sunindextype>(X);
  const sunindextype N = getSize<realtype, sunindextype>(X);
  realtype *zdata = getDevData<realtype, sunindextype>(Z);

  RAJA::forall<RAJA::cuda_exec<256> >(zeroIdx, N, [=] __device__(sunindextype i) {
     zdata[i] = RCONST(1.0) / xdata[i];
  });
}

void N_VAddConst_Raja(N_Vector X, realtype b, N_Vector Z)
{
  const realtype *xdata = getDevData<realtype, sunindextype>(X);
  const sunindextype N = getSize<realtype, sunindextype>(X);
  realtype *zdata = getDevData<realtype, sunindextype>(Z);

  RAJA::forall<RAJA::cuda_exec<256> >(zeroIdx, N, [=] __device__(sunindextype i) {
     zdata[i] = xdata[i] + b;
  });
}

realtype N_VDotProd_Raja(N_Vector X, N_Vector Y)
{
  const realtype *xdata = getDevData<realtype, sunindextype>(X);
  const realtype *ydata = getDevData<realtype, sunindextype>(Y);
  const sunindextype N = getSize<realtype, sunindextype>(X);

  RAJA::ReduceSum<RAJA::cuda_reduce<128>, realtype> gpu_result(0.0);
  RAJA::forall<RAJA::cuda_exec<128> >(zeroIdx, N, [=] __device__(sunindextype i) {
    gpu_result += xdata[i] * ydata[i] ;
  });

  return static_cast<realtype>(gpu_result);
}

realtype N_VMaxNorm_Raja(N_Vector X)
{
  const realtype *xdata = getDevData<realtype, sunindextype>(X);
  const sunindextype N = getSize<realtype, sunindextype>(X);

  RAJA::ReduceMax<RAJA::cuda_reduce<128>, realtype> gpu_result(0.0);
  RAJA::forall<RAJA::cuda_exec<128> >(zeroIdx, N, [=] __device__(sunindextype i) {
    gpu_result.max(abs(xdata[i]));
  });

  return static_cast<realtype>(gpu_result);
}

realtype N_VWrmsNorm_Raja(N_Vector X, N_Vector W)
{
  const realtype *xdata = getDevData<realtype, sunindextype>(X);
  const realtype *wdata = getDevData<realtype, sunindextype>(W);
  const sunindextype N = getSize<realtype, sunindextype>(X);

  RAJA::ReduceSum<RAJA::cuda_reduce<128>, realtype> gpu_result(0.0);
  RAJA::forall<RAJA::cuda_exec<128> >(zeroIdx, N, [=] __device__(sunindextype i) {
    gpu_result += (xdata[i] * wdata[i] * xdata[i] * wdata[i]);
  });

  return std::sqrt(static_cast<realtype>(gpu_result)/N);
}

realtype N_VWrmsNormMask_Raja(N_Vector X, N_Vector W, N_Vector ID)
{
  const realtype *xdata = getDevData<realtype, sunindextype>(X);
  const realtype *wdata = getDevData<realtype, sunindextype>(W);
  const realtype *iddata = getDevData<realtype, sunindextype>(ID);
  const sunindextype N = getSize<realtype, sunindextype>(X);

  RAJA::ReduceSum<RAJA::cuda_reduce<128>, realtype> gpu_result(0.0);
  RAJA::forall<RAJA::cuda_exec<128> >(zeroIdx, N, [=] __device__(sunindextype i) {
    gpu_result += (xdata[i] * wdata[i] * xdata[i] * wdata[i] * iddata[i]);
  });

  return std::sqrt(static_cast<realtype>(gpu_result)/N);
}

realtype N_VMin_Raja(N_Vector X)
{
  const realtype *xdata = getDevData<realtype, sunindextype>(X);
  const sunindextype N = getSize<realtype, sunindextype>(X);

  RAJA::ReduceMin<RAJA::cuda_reduce<128>, realtype> gpu_result(std::numeric_limits<realtype>::max());
  RAJA::forall<RAJA::cuda_exec<128> >(zeroIdx, N, [=] __device__(sunindextype i) {
    gpu_result.min(xdata[i]);
  });

  return static_cast<realtype>(gpu_result);
}

realtype N_VWL2Norm_Raja(N_Vector X, N_Vector W)
{
  const realtype *xdata = getDevData<realtype, sunindextype>(X);
  const realtype *wdata = getDevData<realtype, sunindextype>(W);
  const sunindextype N = getSize<realtype, sunindextype>(X);

  RAJA::ReduceSum<RAJA::cuda_reduce<128>, realtype> gpu_result(0.0);
  RAJA::forall<RAJA::cuda_exec<128> >(zeroIdx, N, [=] __device__(sunindextype i) {
    gpu_result += (xdata[i] * wdata[i] * xdata[i] * wdata[i]);
  });

  return std::sqrt(static_cast<realtype>(gpu_result));
}

realtype N_VL1Norm_Raja(N_Vector X)
{
  const realtype *xdata = getDevData<realtype, sunindextype>(X);
  const sunindextype N = getSize<realtype, sunindextype>(X);

  RAJA::ReduceSum<RAJA::cuda_reduce<128>, realtype> gpu_result(0.0);
  RAJA::forall<RAJA::cuda_exec<128> >(zeroIdx, N, [=] __device__(sunindextype i) {
    gpu_result += (abs(xdata[i]));
  });

  return static_cast<realtype>(gpu_result);
}

void N_VCompare_Raja(realtype c, N_Vector X, N_Vector Z)
{
  const realtype *xdata = getDevData<realtype, sunindextype>(X);
  const sunindextype N = getSize<realtype, sunindextype>(X);
  realtype *zdata = getDevData<realtype, sunindextype>(Z);

  RAJA::forall<RAJA::cuda_exec<256> >(zeroIdx, N, [=] __device__(sunindextype i) {
     zdata[i] = abs(xdata[i]) >= c ? ONE : ZERO;
  });
}

booleantype N_VInvTest_Raja(N_Vector x, N_Vector z)
{
  const realtype *xdata = getDevData<realtype, sunindextype>(x);
  const sunindextype N = getSize<realtype, sunindextype>(x);
  realtype *zdata = getDevData<realtype, sunindextype>(z);

  RAJA::ReduceSum<RAJA::cuda_reduce<128>, realtype> gpu_result(ZERO);
  RAJA::forall<RAJA::cuda_exec<128> >(zeroIdx, N, [=] __device__(sunindextype i) {
    if (xdata[i] == ZERO) {
      gpu_result += ONE;
    } else {
      zdata[i] = ONE/xdata[i];
    }
  });

  return (static_cast<realtype>(gpu_result) < HALF);
}

booleantype N_VConstrMask_Raja(N_Vector c, N_Vector x, N_Vector m)
{
  const realtype *cdata = getDevData<realtype, sunindextype>(c);
  const realtype *xdata = getDevData<realtype, sunindextype>(x);
  const sunindextype N = getSize<realtype, sunindextype>(x);
  realtype *mdata = getDevData<realtype, sunindextype>(m);

  RAJA::ReduceSum<RAJA::cuda_reduce<128>, realtype> gpu_result(ZERO);
  RAJA::forall<RAJA::cuda_exec<128> >(zeroIdx, N, [=] __device__(sunindextype i) {
    bool test = (abs(cdata[i]) > ONEPT5 && cdata[i]*xdata[i] <= ZERO) ||
                (abs(cdata[i]) > HALF   && cdata[i]*xdata[i] <  ZERO);
    mdata[i] = test ? ONE : ZERO;
    gpu_result += mdata[i];
  });

  return (static_cast<realtype>(gpu_result) < HALF);
}

realtype N_VMinQuotient_Raja(N_Vector num, N_Vector denom)
{
  const realtype *ndata = getDevData<realtype, sunindextype>(num);
  const realtype *ddata = getDevData<realtype, sunindextype>(denom);
  const sunindextype N = getSize<realtype, sunindextype>(num);

  RAJA::ReduceMin<RAJA::cuda_reduce<128>, realtype> gpu_result(std::numeric_limits<realtype>::max());
  RAJA::forall<RAJA::cuda_exec<128> >(zeroIdx, N, [=] __device__(sunindextype i) {
    if (ddata[i] != ZERO)
      gpu_result.min(ndata[i]/ddata[i]);
  });

  return (static_cast<realtype>(gpu_result));
}


} // extern "C"
