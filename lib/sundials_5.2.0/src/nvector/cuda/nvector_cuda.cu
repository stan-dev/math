#include "hip/hip_runtime.h"
/* -----------------------------------------------------------------
 * Programmer(s): Slaven Peles, and Cody J. Balos @ LLNL
 * -----------------------------------------------------------------
 * SUNDIALS Copyright Start
 * Copyright (c) 2002-2020, Lawrence Livermore National Security
 * and Southern Methodist University.
 * All rights reserved.
 *
 * See the top-level LICENSE and NOTICE files for details.
 *
 * SPDX-License-Identifier: BSD-3-Clause
 * SUNDIALS Copyright End
 * -----------------------------------------------------------------
 * This is the implementation file for a CUDA implementation
 * of the NVECTOR package.
 * -----------------------------------------------------------------*/

#include <stdio.h>
#include <stdlib.h>
#include <cmath>

#include <nvector/cuda/Vector.hpp>
#include <nvector/cuda/VectorKernels.cuh>
#include <nvector/cuda/VectorArrayKernels.cuh>

#define ZERO   RCONST(0.0)
#define HALF   RCONST(0.5)
#define ONE    RCONST(1.0)
#define ONEPT5 RCONST(1.5)

extern "C" {

using namespace suncudavec;

/*
 * Type definitions
 */

typedef suncudavec::Vector<realtype, sunindextype> vector_type;
typedef suncudavec::ThreadPartitioning<realtype, sunindextype> part_type;

/* ----------------------------------------------------------------
 * Returns vector type ID. Used to identify vector implementation
 * from abstract N_Vector interface.
 */
N_Vector_ID N_VGetVectorID_Cuda(N_Vector v)
{
  return SUNDIALS_NVEC_CUDA;
}

N_Vector N_VNewEmpty_Cuda()
{
  N_Vector v;

  /* Create vector */
  v = NULL;
  v = N_VNewEmpty();
  if (v == NULL) return(NULL);

  /* Attach operations */

  /* constructors, destructors, and utility operations */
  v->ops->nvgetvectorid     = N_VGetVectorID_Cuda;
  v->ops->nvclone           = N_VClone_Cuda;
  v->ops->nvcloneempty      = N_VCloneEmpty_Cuda;
  v->ops->nvdestroy         = N_VDestroy_Cuda;
  v->ops->nvspace           = N_VSpace_Cuda;
  v->ops->nvgetlength       = N_VGetLength_Cuda;

  /* standard vector operations */
  v->ops->nvlinearsum    = N_VLinearSum_Cuda;
  v->ops->nvconst        = N_VConst_Cuda;
  v->ops->nvprod         = N_VProd_Cuda;
  v->ops->nvdiv          = N_VDiv_Cuda;
  v->ops->nvscale        = N_VScale_Cuda;
  v->ops->nvabs          = N_VAbs_Cuda;
  v->ops->nvinv          = N_VInv_Cuda;
  v->ops->nvaddconst     = N_VAddConst_Cuda;
  v->ops->nvdotprod      = N_VDotProd_Cuda;
  v->ops->nvmaxnorm      = N_VMaxNorm_Cuda;
  v->ops->nvmin          = N_VMin_Cuda;
  v->ops->nvl1norm       = N_VL1Norm_Cuda;
  v->ops->nvinvtest      = N_VInvTest_Cuda;
  v->ops->nvconstrmask   = N_VConstrMask_Cuda;
  v->ops->nvminquotient  = N_VMinQuotient_Cuda;
  v->ops->nvwrmsnormmask = N_VWrmsNormMask_Cuda;
  v->ops->nvwrmsnorm     = N_VWrmsNorm_Cuda;
  v->ops->nvwl2norm      = N_VWL2Norm_Cuda;
  v->ops->nvcompare      = N_VCompare_Cuda;

  /* fused and vector array operations are disabled (NULL) by default */

  /* local reduction operations */
  v->ops->nvdotprodlocal     = N_VDotProd_Cuda;
  v->ops->nvmaxnormlocal     = N_VMaxNorm_Cuda;
  v->ops->nvminlocal         = N_VMin_Cuda;
  v->ops->nvl1normlocal      = N_VL1Norm_Cuda;
  v->ops->nvinvtestlocal     = N_VInvTest_Cuda;
  v->ops->nvconstrmasklocal  = N_VConstrMask_Cuda;
  v->ops->nvminquotientlocal = N_VMinQuotient_Cuda;
  v->ops->nvwsqrsumlocal     = N_VWSqrSumLocal_Cuda;
  v->ops->nvwsqrsummasklocal = N_VWSqrSumMaskLocal_Cuda;

  return(v);
}

N_Vector N_VNew_Cuda(sunindextype length)
{
  N_Vector v;

  v = NULL;
  v = N_VNewEmpty_Cuda();
  if (v == NULL) return(NULL);

  v->content = new vector_type(length);

  return(v);
}

N_Vector N_VNewManaged_Cuda(sunindextype length)
{
  N_Vector v;

  v = NULL;
  v = N_VNewEmpty_Cuda();
  if (v == NULL) return(NULL);

  /* if using managed memory, we can attach an operation for
     nvgetarraypointer since the host and device pointers are the same */
  v->ops->nvgetarraypointer = N_VGetHostArrayPointer_Cuda;

  /* create suncudavec::Vector with managed memory */
  v->content = new vector_type(length, true);

  return(v);
}

N_Vector N_VMake_Cuda(sunindextype length, realtype *h_vdata, realtype *d_vdata)
{
  N_Vector v;

  if (h_vdata == NULL || d_vdata == NULL) return(NULL);

  v = NULL;
  v = N_VNewEmpty_Cuda();
  if (v == NULL) return(NULL);

  /* create suncudavec::Vector using the user-provided data arrays */
  v->content = new vector_type(length, false, false, h_vdata, d_vdata);

  return(v);
}

N_Vector N_VMakeManaged_Cuda(sunindextype length, realtype *vdata)
{
  N_Vector v;

  if (vdata == NULL) return(NULL);

  v = NULL;
  v = N_VNewEmpty_Cuda();
  if (v == NULL) return(NULL);

  /* if using managed memory, we can attach an operation for
     nvgetarraypointer since the host and device pointers are the same */
  v->ops->nvgetarraypointer = N_VGetHostArrayPointer_Cuda;

  /* create suncudavec::Vector with managed memory using the user-provided data arrays */
  v->content = new vector_type(length, true, false, vdata, vdata);

  return(v);
}

N_Vector N_VMakeWithManagedAllocator_Cuda(sunindextype length,
                                          void* (*allocfn)(size_t),
                                          void (*freefn)(void*))
{
  N_Vector v;

  v = NULL;
  v = N_VNewEmpty_Cuda();
  if (v == NULL) return(NULL);

  /* if using managed memory, we can attach an operation for
     nvgetarraypointer since the host and device pointers are the same */
  v->ops->nvgetarraypointer = N_VGetHostArrayPointer_Cuda;

  /* create suncudavec::Vector with a custom allocator/deallocator */
  v->content = new vector_type(length, allocfn, freefn, true);

  return(v);
}

/* -----------------------------------------------------------------
 * Function to return the global length of the vector.
 */
sunindextype N_VGetLength_Cuda(N_Vector v)
{
  vector_type* xd = static_cast<vector_type*>(v->content);
  return (xd->size());
}

/* ----------------------------------------------------------------------------
 * Return pointer to the raw host data
 */

realtype *N_VGetHostArrayPointer_Cuda(N_Vector x)
{
  vector_type* xv = static_cast<vector_type*>(x->content);
  return (xv->host());
}

/* ----------------------------------------------------------------------------
 * Return pointer to the raw device data
 */

realtype *N_VGetDeviceArrayPointer_Cuda(N_Vector x)
{
  vector_type* xv = static_cast<vector_type*>(x->content);
  return (xv->device());
}

/* ----------------------------------------------------------------------------
 * Return a flag indicating if the memory for the vector data is managed
 */
booleantype N_VIsManagedMemory_Cuda(N_Vector x)
{
  vector_type* xv = static_cast<vector_type*>(x->content);
  return (xv->isManaged());
}

/*
 * ----------------------------------------------------------------------------
 * Sets the hipStream_t to use for execution of the CUDA kernels.
 */
void N_VSetCudaStream_Cuda(N_Vector x, hipStream_t *stream)
{
  vector_type* xv = static_cast<vector_type*>(x->content);
  xv->partStream().setStream(*stream);
  xv->partReduce().setStream(*stream);
}

/* ----------------------------------------------------------------------------
 * Copy vector data to the device
 */

void N_VCopyToDevice_Cuda(N_Vector x)
{
  vector_type* xv = static_cast<vector_type*>(x->content);
  xv->copyToDev();
}

/* ----------------------------------------------------------------------------
 * Copy vector data from the device to the host
 */

void N_VCopyFromDevice_Cuda(N_Vector x)
{
  vector_type* xv = static_cast<vector_type*>(x->content);
  xv->copyFromDev();
}

/* ----------------------------------------------------------------------------
 * Function to print the a CUDA-based vector to stdout
 */

void N_VPrint_Cuda(N_Vector x)
{
  N_VPrintFile_Cuda(x, stdout);
}

/* ----------------------------------------------------------------------------
 * Function to print the a CUDA-based vector to outfile
 */

void N_VPrintFile_Cuda(N_Vector x, FILE *outfile)
{
  sunindextype i;
  vector_type* xd = static_cast<vector_type*>(x->content);

  for (i = 0; i < xd->size(); i++) {
#if defined(SUNDIALS_EXTENDED_PRECISION)
    fprintf(outfile, "%35.32Lg\n", xd->host()[i]);
#elif defined(SUNDIALS_DOUBLE_PRECISION)
    fprintf(outfile, "%19.16g\n", xd->host()[i]);
#else
    fprintf(outfile, "%11.8g\n", xd->host()[i]);
#endif
  }
  fprintf(outfile, "\n");

  return;
}


/*
 * -----------------------------------------------------------------
 * implementation of vector operations
 * -----------------------------------------------------------------
 */

N_Vector N_VCloneEmpty_Cuda(N_Vector w)
{
  N_Vector v;

  if (w == NULL) return(NULL);

  /* Create vector */
  v = NULL;
  v = N_VNewEmpty();
  if (v == NULL) return(NULL);

  /* Attach operations */
  if (N_VCopyOps(w, v)) { N_VDestroy(v); return(NULL); }

  return(v);
}

N_Vector N_VClone_Cuda(N_Vector w)
{
  N_Vector v;
  v = NULL;
  v = N_VCloneEmpty_Cuda(w);
  if (v == NULL) return(NULL);

  vector_type* wdat = static_cast<vector_type*>(w->content);
  vector_type* vdat = new vector_type(*wdat);

  v->content = vdat;

  return(v);
}


void N_VDestroy_Cuda(N_Vector v)
{
  if (v == NULL) return;

  vector_type* x = static_cast<vector_type*>(v->content);
  if (x != NULL) {
    delete x;
    v->content = NULL;
  }

  /* free ops and vector */
  if (v->ops != NULL) { free(v->ops); v->ops = NULL; }
  free(v); v = NULL;

  return;
}

void N_VSpace_Cuda(N_Vector X, sunindextype *lrw, sunindextype *liw)
{
  vector_type* x = static_cast<vector_type*>(X->content);
  *lrw = x->size();
  *liw = 2;
}

void N_VConst_Cuda(realtype a, N_Vector X)
{
  vector_type *xvec = static_cast<vector_type*>(X->content);
  setConst(a, *xvec);
}

void N_VLinearSum_Cuda(realtype a, N_Vector X, realtype b, N_Vector Y, N_Vector Z)
{
  const vector_type *xvec = static_cast<vector_type*>(X->content);
  const vector_type *yvec = static_cast<vector_type*>(Y->content);
  vector_type *zvec = static_cast<vector_type*>(Z->content);
  linearSum(a, *xvec, b, *yvec, *zvec);
}

void N_VProd_Cuda(N_Vector X, N_Vector Y, N_Vector Z)
{
  const vector_type *xvec = static_cast<vector_type*>(X->content);
  const vector_type *yvec = static_cast<vector_type*>(Y->content);
  vector_type *zvec = static_cast<vector_type*>(Z->content);
  prod(*xvec, *yvec, *zvec);
}

void N_VDiv_Cuda(N_Vector X, N_Vector Y, N_Vector Z)
{
  const vector_type *xvec = static_cast<vector_type*>(X->content);
  const vector_type *yvec = static_cast<vector_type*>(Y->content);
  vector_type *zvec = static_cast<vector_type*>(Z->content);
  div(*xvec, *yvec, *zvec);
}

void N_VScale_Cuda(realtype a, N_Vector X, N_Vector Z)
{
  const vector_type *xvec = static_cast<vector_type*>(X->content);
  vector_type *zvec = static_cast<vector_type*>(Z->content);
  scale(a, *xvec, *zvec);
}

void N_VAbs_Cuda(N_Vector X, N_Vector Z)
{
  const vector_type *xvec = static_cast<vector_type*>(X->content);
  vector_type *zvec = static_cast<vector_type*>(Z->content);
  absVal(*xvec, *zvec);
}

void N_VInv_Cuda(N_Vector X, N_Vector Z)
{
  const vector_type *xvec = static_cast<vector_type*>(X->content);
  vector_type *zvec = static_cast<vector_type*>(Z->content);
  inv(*xvec, *zvec);
}

void N_VAddConst_Cuda(N_Vector X, realtype b, N_Vector Z)
{
  const vector_type *xvec = static_cast<vector_type*>(X->content);
  vector_type *zvec = static_cast<vector_type*>(Z->content);
  addConst(b, *xvec, *zvec);
}

realtype N_VDotProd_Cuda(N_Vector X, N_Vector Y)
{
  const vector_type *xvec = static_cast<vector_type*>(X->content);
  const vector_type *yvec = static_cast<vector_type*>(Y->content);
  return(dotProd(*xvec, *yvec));
}

realtype N_VMaxNorm_Cuda(N_Vector X)
{
  const vector_type *xvec = static_cast<vector_type*>(X->content);
  return(maxNorm(*xvec));
}

realtype N_VWSqrSumLocal_Cuda(N_Vector X, N_Vector W)
{
  const vector_type *xvec = static_cast<vector_type*>(X->content);
  const vector_type *wvec = static_cast<vector_type*>(W->content);
  return(wL2NormSquare(*xvec, *wvec));
}

realtype N_VWrmsNorm_Cuda(N_Vector X, N_Vector W)
{
  const realtype sum = N_VWSqrSumLocal_Cuda(X, W);
  const vector_type *xvec = static_cast<vector_type*>(X->content);
  return std::sqrt(sum/xvec->size());
}

realtype N_VWSqrSumMaskLocal_Cuda(N_Vector X, N_Vector W, N_Vector Id)
{
  const vector_type *xvec = static_cast<vector_type*>(X->content);
  const vector_type *wvec = static_cast<vector_type*>(W->content);
  const vector_type *ivec = static_cast<vector_type*>(Id->content);
  return(wL2NormSquareMask(*xvec, *wvec, *ivec));
}

realtype N_VWrmsNormMask_Cuda(N_Vector X, N_Vector W, N_Vector Id)
{
  const realtype sum = N_VWSqrSumMaskLocal_Cuda(X, W, Id);
  const vector_type *xvec = static_cast<vector_type*>(X->content);
  return std::sqrt(sum/xvec->size());
}

realtype N_VMin_Cuda(N_Vector X)
{
  const vector_type *xvec = static_cast<vector_type*>(X->content);
  return(findMin(*xvec));
}

realtype N_VWL2Norm_Cuda(N_Vector X, N_Vector W)
{
  const realtype sum = N_VWSqrSumLocal_Cuda(X, W);
  return std::sqrt(sum);
}

realtype N_VL1Norm_Cuda(N_Vector X)
{
  const vector_type *xvec = static_cast<vector_type*>(X->content);
  return(L1Norm(*xvec));
}

void N_VCompare_Cuda(realtype c, N_Vector X, N_Vector Z)
{
  const vector_type *xvec = static_cast<vector_type*>(X->content);
  vector_type *zvec = static_cast<vector_type*>(Z->content);
  compare(c, *xvec, *zvec);
}

booleantype N_VInvTest_Cuda(N_Vector X, N_Vector Z)
{
  const vector_type *xvec = static_cast<vector_type*>(X->content);
  vector_type *zvec = static_cast<vector_type*>(Z->content);
  const realtype locmin = invTest(*xvec, *zvec);
  return (locmin < HALF);
}

booleantype N_VConstrMask_Cuda(N_Vector C, N_Vector X, N_Vector M)
{
  const vector_type *cvec = static_cast<vector_type*>(C->content);
  const vector_type *xvec = static_cast<vector_type*>(X->content);
  vector_type *mvec = static_cast<vector_type*>(M->content);
  const realtype locsum = constrMask(*cvec, *xvec, *mvec);
  return (locsum < HALF);
}

realtype N_VMinQuotient_Cuda(N_Vector num, N_Vector denom)
{
  const vector_type *numvec = static_cast<vector_type*>(num->content);
  const vector_type *denvec = static_cast<vector_type*>(denom->content);
  return(minQuotient(*numvec, *denvec));
}

/*
 * -----------------------------------------------------------------
 * fused vector operations
 * -----------------------------------------------------------------
 */

int N_VLinearCombination_Cuda(int nvec, realtype* c, N_Vector* X, N_Vector Z)
{
  hipError_t err;
  vector_type** Xv;
  vector_type*  Zv;

  Zv = static_cast<vector_type*>(Z->content);

  Xv = new vector_type*[nvec];
  for (int i=0; i<nvec; i++)
    Xv[i] = static_cast<vector_type*>(X[i]->content);

  err = linearCombination(nvec, c, Xv, Zv);

  delete[] Xv;

  return err == hipSuccess ? 0 : -1;
}

int N_VScaleAddMulti_Cuda(int nvec, realtype* c, N_Vector X, N_Vector* Y,
                           N_Vector* Z)
{
  hipError_t err;
  vector_type*  Xv;
  vector_type** Yv;
  vector_type** Zv;

  Xv = static_cast<vector_type*>(X->content);

  Yv = new vector_type*[nvec];
  for (int i=0; i<nvec; i++)
    Yv[i] = static_cast<vector_type*>(Y[i]->content);

  Zv = new vector_type*[nvec];
  for (int i=0; i<nvec; i++)
    Zv[i] = static_cast<vector_type*>(Z[i]->content);

  err = scaleAddMulti(nvec, c, Xv, Yv, Zv);

  delete[] Yv;
  delete[] Zv;

  return err == hipSuccess ? 0 : -1;
}


int N_VDotProdMulti_Cuda(int nvec, N_Vector x, N_Vector* Y, realtype* dotprods)
{
  hipError_t err;
  vector_type*  Xv;
  vector_type** Yv;

  Xv = static_cast<vector_type*>(x->content);

  Yv = new vector_type*[nvec];
  for (int i=0; i<nvec; i++)
    Yv[i] = static_cast<vector_type*>(Y[i]->content);

  err = dotProdMulti(nvec, Xv, Yv, dotprods);

  delete[] Yv;

  return err == hipSuccess ? 0 : -1;
}



/*
 * -----------------------------------------------------------------------------
 * vector array operations
 * -----------------------------------------------------------------------------
 */

int N_VLinearSumVectorArray_Cuda(int nvec, realtype a, N_Vector* X, realtype b,
                                 N_Vector* Y, N_Vector* Z)
{
  hipError_t err;
  vector_type** Xv;
  vector_type** Yv;
  vector_type** Zv;

  Xv = new vector_type*[nvec];
  for (int i=0; i<nvec; i++)
    Xv[i] = static_cast<vector_type*>(X[i]->content);

  Yv = new vector_type*[nvec];
  for (int i=0; i<nvec; i++)
    Yv[i] = static_cast<vector_type*>(Y[i]->content);

  Zv = new vector_type*[nvec];
  for (int i=0; i<nvec; i++)
    Zv[i] = static_cast<vector_type*>(Z[i]->content);

  err = linearSumVectorArray(nvec, a, Xv, b, Yv, Zv);

  delete[] Xv;
  delete[] Yv;
  delete[] Zv;

  return err == hipSuccess ? 0 : -1;
}


int N_VScaleVectorArray_Cuda(int nvec, realtype* c, N_Vector* X, N_Vector* Z)
{
  hipError_t err;
  vector_type** Xv;
  vector_type** Zv;

  Xv = new vector_type*[nvec];
  for (int i=0; i<nvec; i++)
    Xv[i] = static_cast<vector_type*>(X[i]->content);

  Zv = new vector_type*[nvec];
  for (int i=0; i<nvec; i++)
    Zv[i] = static_cast<vector_type*>(Z[i]->content);

  err = scaleVectorArray(nvec, c, Xv, Zv);

  delete[] Xv;
  delete[] Zv;

  return err == hipSuccess ? 0 : -1;
}


int N_VConstVectorArray_Cuda(int nvec, realtype c, N_Vector* Z)
{
  hipError_t err;
  vector_type** Zv;

  Zv = new vector_type*[nvec];
  for (int i=0; i<nvec; i++)
    Zv[i] = static_cast<vector_type*>(Z[i]->content);

  err = constVectorArray(nvec, c, Zv);

  delete[] Zv;

  return err == hipSuccess ? 0 : -1;
}


int N_VWrmsNormVectorArray_Cuda(int nvec, N_Vector* X, N_Vector* W,
                                realtype* norms)
{
  hipError_t err;
  const vector_type* xvec = static_cast<vector_type*>(X[0]->content);
  vector_type** Xv;
  vector_type** Wv;

  sunindextype N = xvec->size();

  Xv = new vector_type*[nvec];
  for (int k=0; k<nvec; k++)
    Xv[k] = static_cast<vector_type*>(X[k]->content);

  Wv = new vector_type*[nvec];
  for (int k=0; k<nvec; k++)
    Wv[k] = static_cast<vector_type*>(W[k]->content);

  err = wL2NormSquareVectorArray(nvec, Xv, Wv, norms);

  delete[] Xv;
  delete[] Wv;

  if (err != hipSuccess)  return(-1);

  for (int k=0; k<nvec; ++k)
    norms[k] = std::sqrt(norms[k]/N);

  return 0;
}


int N_VWrmsNormMaskVectorArray_Cuda(int nvec, N_Vector* X, N_Vector* W,
                                    N_Vector id, realtype* norms)
{
  hipError_t err;
  const vector_type* xvec = static_cast<vector_type*>(X[0]->content);
  vector_type** Xv;
  vector_type** Wv;
  vector_type*  IDv;

  sunindextype N = xvec->size();

  Xv = new vector_type*[nvec];
  for (int k=0; k<nvec; k++)
    Xv[k] = static_cast<vector_type*>(X[k]->content);

  Wv = new vector_type*[nvec];
  for (int k=0; k<nvec; k++)
    Wv[k] = static_cast<vector_type*>(W[k]->content);

  IDv = static_cast<vector_type*>(id->content);

  err = wL2NormSquareMaskVectorArray(nvec, Xv, Wv, IDv, norms);

  delete[] Xv;
  delete[] Wv;

  if (err != hipSuccess)  return(-1);

  for (int k=0; k<nvec; ++k)
    norms[k] = std::sqrt(norms[k]/N);

  return 0;
}


int N_VScaleAddMultiVectorArray_Cuda(int nvec, int nsum, realtype* c,
                                     N_Vector* X, N_Vector** Y, N_Vector** Z)
{
  hipError_t err;
  vector_type** Xv;
  vector_type** Yv;
  vector_type** Zv;

  Xv = new vector_type*[nvec];
  for (int k=0; k<nvec; k++)
    Xv[k] = static_cast<vector_type*>(X[k]->content);

  Yv = new vector_type*[nsum*nvec];
  for (int k=0; k<nvec; k++)
    for (int j=0; j<nsum; j++)
      Yv[k*nsum+j] = static_cast<vector_type*>(Y[j][k]->content);

  Zv = new vector_type*[nsum*nvec];
  for (int k=0; k<nvec; k++)
    for (int j=0; j<nsum; j++)
      Zv[k*nsum+j] = static_cast<vector_type*>(Z[j][k]->content);

  err = scaleAddMultiVectorArray(nvec, nsum, c, Xv, Yv, Zv);

  delete[] Xv;
  delete[] Yv;
  delete[] Zv;

  return err == hipSuccess ? 0 : -1;
}


int N_VLinearCombinationVectorArray_Cuda(int nvec, int nsum, realtype* c,
                                         N_Vector** X, N_Vector* Z)
{
  hipError_t err;
  vector_type** Xv;
  vector_type** Zv;

  Xv = new vector_type*[nsum*nvec];
  for (int k=0; k<nvec; k++)
    for (int j=0; j<nsum; j++)
      Xv[k*nsum+j] = static_cast<vector_type*>(X[j][k]->content);

  Zv = new vector_type*[nvec];
  for (int k=0; k<nvec; k++)
    Zv[k] = static_cast<vector_type*>(Z[k]->content);

  err = linearCombinationVectorArray(nvec, nsum, c, Xv, Zv);

  delete[] Xv;
  delete[] Zv;

  return err == hipSuccess ? 0 : -1;
}


/*
 * -----------------------------------------------------------------
 * Enable / Disable fused and vector array operations
 * -----------------------------------------------------------------
 */

int N_VEnableFusedOps_Cuda(N_Vector v, booleantype tf)
{
  /* check that vector is non-NULL */
  if (v == NULL) return(-1);

  /* check that ops structure is non-NULL */
  if (v->ops == NULL) return(-1);

  if (tf) {
    /* enable all fused vector operations */
    v->ops->nvlinearcombination = N_VLinearCombination_Cuda;
    v->ops->nvscaleaddmulti     = N_VScaleAddMulti_Cuda;
    v->ops->nvdotprodmulti      = N_VDotProdMulti_Cuda;
    /* enable all vector array operations */
    v->ops->nvlinearsumvectorarray         = N_VLinearSumVectorArray_Cuda;
    v->ops->nvscalevectorarray             = N_VScaleVectorArray_Cuda;
    v->ops->nvconstvectorarray             = N_VConstVectorArray_Cuda;
    v->ops->nvwrmsnormvectorarray          = N_VWrmsNormVectorArray_Cuda;
    v->ops->nvwrmsnormmaskvectorarray      = N_VWrmsNormMaskVectorArray_Cuda;
    v->ops->nvscaleaddmultivectorarray     = N_VScaleAddMultiVectorArray_Cuda;
    v->ops->nvlinearcombinationvectorarray = N_VLinearCombinationVectorArray_Cuda;
  } else {
    /* disable all fused vector operations */
    v->ops->nvlinearcombination = NULL;
    v->ops->nvscaleaddmulti     = NULL;
    v->ops->nvdotprodmulti      = NULL;
    /* disable all vector array operations */
    v->ops->nvlinearsumvectorarray         = NULL;
    v->ops->nvscalevectorarray             = NULL;
    v->ops->nvconstvectorarray             = NULL;
    v->ops->nvwrmsnormvectorarray          = NULL;
    v->ops->nvwrmsnormmaskvectorarray      = NULL;
    v->ops->nvscaleaddmultivectorarray     = NULL;
    v->ops->nvlinearcombinationvectorarray = NULL;
  }

  /* return success */
  return(0);
}


int N_VEnableLinearCombination_Cuda(N_Vector v, booleantype tf)
{
  /* check that vector is non-NULL */
  if (v == NULL) return(-1);

  /* check that ops structure is non-NULL */
  if (v->ops == NULL) return(-1);

  /* enable/disable operation */
  if (tf)
    v->ops->nvlinearcombination = N_VLinearCombination_Cuda;
  else
    v->ops->nvlinearcombination = NULL;

  /* return success */
  return(0);
}

int N_VEnableScaleAddMulti_Cuda(N_Vector v, booleantype tf)
{
  /* check that vector is non-NULL */
  if (v == NULL) return(-1);

  /* check that ops structure is non-NULL */
  if (v->ops == NULL) return(-1);

  /* enable/disable operation */
  if (tf)
    v->ops->nvscaleaddmulti = N_VScaleAddMulti_Cuda;
  else
    v->ops->nvscaleaddmulti = NULL;

  /* return success */
  return(0);
}

int N_VEnableDotProdMulti_Cuda(N_Vector v, booleantype tf)
{
  /* check that vector is non-NULL */
  if (v == NULL) return(-1);

  /* check that ops structure is non-NULL */
  if (v->ops == NULL) return(-1);

  /* enable/disable operation */
  if (tf)
    v->ops->nvdotprodmulti = N_VDotProdMulti_Cuda;
  else
    v->ops->nvdotprodmulti = NULL;

  /* return success */
  return(0);
}

int N_VEnableLinearSumVectorArray_Cuda(N_Vector v, booleantype tf)
{
  /* check that vector is non-NULL */
  if (v == NULL) return(-1);

  /* check that ops structure is non-NULL */
  if (v->ops == NULL) return(-1);

  /* enable/disable operation */
  if (tf)
    v->ops->nvlinearsumvectorarray = N_VLinearSumVectorArray_Cuda;
  else
    v->ops->nvlinearsumvectorarray = NULL;

  /* return success */
  return(0);
}

int N_VEnableScaleVectorArray_Cuda(N_Vector v, booleantype tf)
{
  /* check that vector is non-NULL */
  if (v == NULL) return(-1);

  /* check that ops structure is non-NULL */
  if (v->ops == NULL) return(-1);

  /* enable/disable operation */
  if (tf)
    v->ops->nvscalevectorarray = N_VScaleVectorArray_Cuda;
  else
    v->ops->nvscalevectorarray = NULL;

  /* return success */
  return(0);
}

int N_VEnableConstVectorArray_Cuda(N_Vector v, booleantype tf)
{
  /* check that vector is non-NULL */
  if (v == NULL) return(-1);

  /* check that ops structure is non-NULL */
  if (v->ops == NULL) return(-1);

  /* enable/disable operation */
  if (tf)
    v->ops->nvconstvectorarray = N_VConstVectorArray_Cuda;
  else
    v->ops->nvconstvectorarray = NULL;

  /* return success */
  return(0);
}

int N_VEnableWrmsNormVectorArray_Cuda(N_Vector v, booleantype tf)
{
  /* check that vector is non-NULL */
  if (v == NULL) return(-1);

  /* check that ops structure is non-NULL */
  if (v->ops == NULL) return(-1);

  /* enable/disable operation */
  if (tf)
    v->ops->nvwrmsnormvectorarray = N_VWrmsNormVectorArray_Cuda;
  else
    v->ops->nvwrmsnormvectorarray = NULL;

  /* return success */
  return(0);
}

int N_VEnableWrmsNormMaskVectorArray_Cuda(N_Vector v, booleantype tf)
{
  /* check that vector is non-NULL */
  if (v == NULL) return(-1);

  /* check that ops structure is non-NULL */
  if (v->ops == NULL) return(-1);

  /* enable/disable operation */
  if (tf)
    v->ops->nvwrmsnormmaskvectorarray = N_VWrmsNormMaskVectorArray_Cuda;
  else
    v->ops->nvwrmsnormmaskvectorarray = NULL;

  /* return success */
  return(0);
}

int N_VEnableScaleAddMultiVectorArray_Cuda(N_Vector v, booleantype tf)
{
  /* check that vector is non-NULL */
  if (v == NULL) return(-1);

  /* check that ops structure is non-NULL */
  if (v->ops == NULL) return(-1);

  /* enable/disable operation */
  if (tf)
    v->ops->nvscaleaddmultivectorarray = N_VScaleAddMultiVectorArray_Cuda;
  else
    v->ops->nvscaleaddmultivectorarray = NULL;

  /* return success */
  return(0);
}

int N_VEnableLinearCombinationVectorArray_Cuda(N_Vector v, booleantype tf)
{
  /* check that vector is non-NULL */
  if (v == NULL) return(-1);

  /* check that ops structure is non-NULL */
  if (v->ops == NULL) return(-1);

  /* enable/disable operation */
  if (tf)
    v->ops->nvlinearcombinationvectorarray = N_VLinearCombinationVectorArray_Cuda;
  else
    v->ops->nvlinearcombinationvectorarray = NULL;

  /* return success */
  return(0);
}

} // extern "C"
