#include "hip/hip_runtime.h"
/* -----------------------------------------------------------------
 * Programmer(s): Slaven Peles, Cody J. Balos @ LLNL
 * -----------------------------------------------------------------
 * SUNDIALS Copyright Start
 * Copyright (c) 2002-2020, Lawrence Livermore National Security
 * and Southern Methodist University.
 * All rights reserved.
 *
 * See the top-level LICENSE and NOTICE files for details.
 *
 * SPDX-License-Identifier: BSD-3-Clause
 * SUNDIALS Copyright End
 * -----------------------------------------------------------------
 * This is the implementation file for a RAJA+CUDA implementation
 * of the NVECTOR package.
 * -----------------------------------------------------------------*/

#include <stdio.h>
#include <stdlib.h>

#include <nvector/raja/Vector.hpp>
#include <RAJA/RAJA.hpp>


#define ZERO   RCONST(0.0)
#define HALF   RCONST(0.5)
#define ONE    RCONST(1.0)
#define ONEPT5 RCONST(1.5)

// RAJA defines
#define CUDA_BLOCK_SIZE 256
#define RAJA_NODE_TYPE RAJA::cuda_exec< CUDA_BLOCK_SIZE >
#define RAJA_REDUCE_TYPE RAJA::cuda_reduce
#define RAJA_LAMBDA [=] __device__

extern "C" {

using namespace sunrajavec;

// Type defines
typedef sunrajavec::Vector<realtype, sunindextype> vector_type;

// Static constants
static constexpr sunindextype zeroIdx = 0;

/* ----------------------------------------------------------------
 * Returns vector type ID. Used to identify vector implementation
 * from abstract N_Vector interface.
 */
N_Vector_ID N_VGetVectorID_Raja(N_Vector v)
{
  return SUNDIALS_NVEC_RAJA;
}

N_Vector N_VNewEmpty_Raja()
{
  N_Vector v;

  /* Create an empty vector object */
  v = NULL;
  v = N_VNewEmpty();
  if (v == NULL) return(NULL);

  /* Attach operations */

  /* constructors, destructors, and utility operations */
  v->ops->nvgetvectorid     = N_VGetVectorID_Raja;
  v->ops->nvclone           = N_VClone_Raja;
  v->ops->nvcloneempty      = N_VCloneEmpty_Raja;
  v->ops->nvdestroy         = N_VDestroy_Raja;
  v->ops->nvspace           = N_VSpace_Raja;
  v->ops->nvgetlength       = N_VGetLength_Raja;

  /* standard vector operations */
  v->ops->nvlinearsum    = N_VLinearSum_Raja;
  v->ops->nvconst        = N_VConst_Raja;
  v->ops->nvprod         = N_VProd_Raja;
  v->ops->nvdiv          = N_VDiv_Raja;
  v->ops->nvscale        = N_VScale_Raja;
  v->ops->nvabs          = N_VAbs_Raja;
  v->ops->nvinv          = N_VInv_Raja;
  v->ops->nvaddconst     = N_VAddConst_Raja;
  v->ops->nvdotprod      = N_VDotProd_Raja;
  v->ops->nvmaxnorm      = N_VMaxNorm_Raja;
  v->ops->nvmin          = N_VMin_Raja;
  v->ops->nvl1norm       = N_VL1Norm_Raja;
  v->ops->nvinvtest      = N_VInvTest_Raja;
  v->ops->nvconstrmask   = N_VConstrMask_Raja;
  v->ops->nvminquotient  = N_VMinQuotient_Raja;
  v->ops->nvwrmsnormmask = N_VWrmsNormMask_Raja;
  v->ops->nvwrmsnorm     = N_VWrmsNorm_Raja;
  v->ops->nvwl2norm      = N_VWL2Norm_Raja;
  v->ops->nvcompare      = N_VCompare_Raja;

  /* fused and vector array operations are disabled (NULL) by default */

  /* local reduction operations */
  v->ops->nvwsqrsumlocal     = N_VWSqrSumLocal_Raja;
  v->ops->nvwsqrsummasklocal = N_VWSqrSumMaskLocal_Raja;
  v->ops->nvdotprodlocal     = N_VDotProd_Raja;
  v->ops->nvmaxnormlocal     = N_VMaxNorm_Raja;
  v->ops->nvminlocal         = N_VMin_Raja;
  v->ops->nvl1normlocal      = N_VL1Norm_Raja;
  v->ops->nvinvtestlocal     = N_VInvTest_Raja;
  v->ops->nvconstrmasklocal  = N_VConstrMask_Raja;
  v->ops->nvminquotientlocal = N_VMinQuotient_Raja;

  return(v);
}

N_Vector N_VNew_Raja(sunindextype length)
{
  N_Vector v;

  v = NULL;
  v = N_VNewEmpty_Raja();
  if (v == NULL) return(NULL);

  v->content = new vector_type(length);

  return(v);
}

N_Vector N_VMake_Raja(N_VectorContent_Raja c)
{
  N_Vector v;
  vector_type* x = static_cast<vector_type*>(c);
  sunindextype length = x->size();

  v = NULL;
  v = N_VNewEmpty_Raja();
  if (v == NULL) return(NULL);

  v->content = c;

  return(v);
}


/* -----------------------------------------------------------------
 * Function to return the global length of the vector.
 */
sunindextype N_VGetLength_Raja(N_Vector v)
{
  vector_type* xd = static_cast<vector_type*>(v->content);
  return xd->size();
}

/* ----------------------------------------------------------------------------
 * Return pointer to the raw host data
 */

realtype *N_VGetHostArrayPointer_Raja(N_Vector x)
{
  vector_type* xv = static_cast<vector_type*>(x->content);
  return (xv->host());
}

/* ----------------------------------------------------------------------------
 * Return pointer to the raw device data
 */

realtype *N_VGetDeviceArrayPointer_Raja(N_Vector x)
{
  vector_type* xv = static_cast<vector_type*>(x->content);
  return (xv->device());
}

/* ----------------------------------------------------------------------------
 * Copy vector data to the device
 */

void N_VCopyToDevice_Raja(N_Vector x)
{
  vector_type* xv = static_cast<vector_type*>(x->content);
  xv->copyToDev();
}

/* ----------------------------------------------------------------------------
 * Copy vector data from the device to the host
 */

void N_VCopyFromDevice_Raja(N_Vector x)
{
  vector_type* xv = static_cast<vector_type*>(x->content);
  xv->copyFromDev();
}

/* ----------------------------------------------------------------------------
 * Function to print the a serial vector to stdout
 */

void N_VPrint_Raja(N_Vector X)
{
  N_VPrintFile_Raja(X, stdout);
}

/* ----------------------------------------------------------------------------
 * Function to print the a serial vector to outfile
 */

void N_VPrintFile_Raja(N_Vector X, FILE *outfile)
{
  const realtype *xd = N_VGetDeviceArrayPointer_Raja(X);
  const sunindextype N = N_VGetLength_Raja(X);
  sunindextype i;

  for (i = 0; i < N; ++i) {
#if defined(SUNDIALS_EXTENDED_PRECISION)
    fprintf(outfile, "%35.32Lg\n", xd[i]);
#elif defined(SUNDIALS_DOUBLE_PRECISION)
    fprintf(outfile, "%19.16g\n", xd[i]);
#else
    fprintf(outfile, "%11.8g\n", xd[i]);
#endif
  }
  fprintf(outfile, "\n");

  return;
}

/*
 * -----------------------------------------------------------------
 * implementation of vector operations
 * -----------------------------------------------------------------
 */

N_Vector N_VCloneEmpty_Raja(N_Vector w)
{
  N_Vector v;

  if (w == NULL) return(NULL);

  /* Create vector */
  v = NULL;
  v = N_VNewEmpty();
  if (v == NULL) return(NULL);

  /* Attach operations */
  if (N_VCopyOps(w, v)) { N_VDestroy(v); return(NULL); }

  return(v);
}

N_Vector N_VClone_Raja(N_Vector w)
{
  N_Vector v;
  v = NULL;
  v = N_VCloneEmpty_Raja(w);
  if (v == NULL) return(NULL);

  vector_type* wdat = static_cast<vector_type*>(w->content);
  vector_type* vdat = new vector_type(*wdat);

  v->content = vdat;

  return(v);
}


void N_VDestroy_Raja(N_Vector v)
{
  if (v == NULL) return;

  vector_type* x = static_cast<vector_type*>(v->content);
  if (x != NULL) {
    delete x;
    v->content = NULL;
  }

  /* free ops and vector */
  if (v->ops != NULL) { free(v->ops); v->ops = NULL; }
  free(v); v = NULL;

  return;
}

void N_VSpace_Raja(N_Vector X, sunindextype *lrw, sunindextype *liw)
{
  *lrw = N_VGetLength_Raja(X);
  *liw = 2;
}

void N_VConst_Raja(realtype c, N_Vector Z)
{
  const sunindextype N = N_VGetLength_Raja(Z);
  realtype *zdata = N_VGetDeviceArrayPointer_Raja(Z);

  RAJA::forall< RAJA_NODE_TYPE >(RAJA::RangeSegment(zeroIdx, N), RAJA_LAMBDA(sunindextype i) {
     zdata[i] = c;
  });
}

void N_VLinearSum_Raja(realtype a, N_Vector X, realtype b, N_Vector Y, N_Vector Z)
{
  const realtype *xdata = N_VGetDeviceArrayPointer_Raja(X);
  const realtype *ydata = N_VGetDeviceArrayPointer_Raja(Y);
  const sunindextype N = N_VGetLength_Raja(X);
  realtype *zdata = N_VGetDeviceArrayPointer_Raja(Z);

  RAJA::forall< RAJA_NODE_TYPE >(RAJA::RangeSegment(zeroIdx, N),
    RAJA_LAMBDA(sunindextype i) {
      zdata[i] = a*xdata[i] + b*ydata[i];
    }
  );
}

void N_VProd_Raja(N_Vector X, N_Vector Y, N_Vector Z)
{
  const realtype *xdata = N_VGetDeviceArrayPointer_Raja(X);
  const realtype *ydata = N_VGetDeviceArrayPointer_Raja(Y);
  const sunindextype N = N_VGetLength_Raja(X);
  realtype *zdata = N_VGetDeviceArrayPointer_Raja(Z);

  RAJA::forall< RAJA_NODE_TYPE >(RAJA::RangeSegment(zeroIdx, N),
    RAJA_LAMBDA(sunindextype i) {
      zdata[i] = xdata[i] * ydata[i];
    }
  );
}

void N_VDiv_Raja(N_Vector X, N_Vector Y, N_Vector Z)
{
  const realtype *xdata = N_VGetDeviceArrayPointer_Raja(X);
  const realtype *ydata = N_VGetDeviceArrayPointer_Raja(Y);
  const sunindextype N = N_VGetLength_Raja(X);
  realtype *zdata = N_VGetDeviceArrayPointer_Raja(Z);

  RAJA::forall< RAJA_NODE_TYPE >(RAJA::RangeSegment(zeroIdx, N),
    RAJA_LAMBDA(sunindextype i) {
      zdata[i] = xdata[i] / ydata[i];
    }
  );
}

void N_VScale_Raja(realtype c, N_Vector X, N_Vector Z)
{
  const realtype *xdata = N_VGetDeviceArrayPointer_Raja(X);
  const sunindextype N = N_VGetLength_Raja(X);
  realtype *zdata = N_VGetDeviceArrayPointer_Raja(Z);

  RAJA::forall<RAJA_NODE_TYPE >(RAJA::RangeSegment(zeroIdx, N),
    RAJA_LAMBDA(sunindextype i) {
      zdata[i] = c * xdata[i];
    }
  );
}

void N_VAbs_Raja(N_Vector X, N_Vector Z)
{
  const realtype *xdata = N_VGetDeviceArrayPointer_Raja(X);
  const sunindextype N = N_VGetLength_Raja(X);
  realtype *zdata = N_VGetDeviceArrayPointer_Raja(Z);

  RAJA::forall<RAJA_NODE_TYPE >(RAJA::RangeSegment(zeroIdx, N),
    RAJA_LAMBDA(sunindextype i) {
      zdata[i] = abs(xdata[i]);
    }
  );
}

void N_VInv_Raja(N_Vector X, N_Vector Z)
{
  const realtype *xdata = N_VGetDeviceArrayPointer_Raja(X);
  const sunindextype N = N_VGetLength_Raja(X);
  realtype *zdata = N_VGetDeviceArrayPointer_Raja(Z);

  RAJA::forall<RAJA_NODE_TYPE >(RAJA::RangeSegment(zeroIdx, N),
    RAJA_LAMBDA(sunindextype i) {
      zdata[i] = ONE / xdata[i];
    }
  );
}

void N_VAddConst_Raja(N_Vector X, realtype b, N_Vector Z)
{
  const realtype *xdata = N_VGetDeviceArrayPointer_Raja(X);
  const sunindextype N = N_VGetLength_Raja(X);
  realtype *zdata = N_VGetDeviceArrayPointer_Raja(Z);

  RAJA::forall< RAJA_NODE_TYPE >(RAJA::RangeSegment(zeroIdx, N),
    RAJA_LAMBDA(sunindextype i) {
      zdata[i] = xdata[i] + b;
    }
  );
}

realtype N_VDotProd_Raja(N_Vector X, N_Vector Y)
{
  const realtype *xdata = N_VGetDeviceArrayPointer_Raja(X);
  const realtype *ydata = N_VGetDeviceArrayPointer_Raja(Y);
  const sunindextype N = N_VGetLength_Raja(X);

  RAJA::ReduceSum< RAJA_REDUCE_TYPE, realtype> gpu_result(0.0);
  RAJA::forall< RAJA_NODE_TYPE >(RAJA::RangeSegment(zeroIdx, N),
    RAJA_LAMBDA(sunindextype i) {
      gpu_result += xdata[i] * ydata[i] ;
    }
  );

  return(static_cast<realtype>(gpu_result));
}

realtype N_VMaxNorm_Raja(N_Vector X)
{
  const realtype *xdata = N_VGetDeviceArrayPointer_Raja(X);
  const sunindextype N = N_VGetLength_Raja(X);

  RAJA::ReduceMax< RAJA_REDUCE_TYPE, realtype> gpu_result(0.0);
  RAJA::forall< RAJA_NODE_TYPE >(RAJA::RangeSegment(zeroIdx, N),
    RAJA_LAMBDA(sunindextype i) {
      gpu_result.max(abs(xdata[i]));
    }
  );

  return(static_cast<realtype>(gpu_result));
}

realtype N_VWSqrSumLocal_Raja(N_Vector X, N_Vector W)
{
  const realtype *xdata = N_VGetDeviceArrayPointer_Raja(X);
  const realtype *wdata = N_VGetDeviceArrayPointer_Raja(W);
  const sunindextype N = N_VGetLength_Raja(X);

  RAJA::ReduceSum< RAJA_REDUCE_TYPE, realtype> gpu_result(0.0);
  RAJA::forall< RAJA_NODE_TYPE >(RAJA::RangeSegment(zeroIdx, N),
    RAJA_LAMBDA(sunindextype i) {
      gpu_result += (xdata[i] * wdata[i] * xdata[i] * wdata[i]);
    }
  );

  return(static_cast<realtype>(gpu_result));
}

realtype N_VWrmsNorm_Raja(N_Vector X, N_Vector W)
{
  const realtype sum = N_VWSqrSumLocal_Raja(X, W);
  const sunindextype N = N_VGetLength_Raja(X);
  return std::sqrt(sum/N);
}

realtype N_VWSqrSumMaskLocal_Raja(N_Vector X, N_Vector W, N_Vector ID)
{
  const realtype *xdata = N_VGetDeviceArrayPointer_Raja(X);
  const realtype *wdata = N_VGetDeviceArrayPointer_Raja(W);
  const realtype *iddata = N_VGetDeviceArrayPointer_Raja(ID);
  const sunindextype N = N_VGetLength_Raja(X);

  RAJA::ReduceSum< RAJA_REDUCE_TYPE, realtype> gpu_result(0.0);
  RAJA::forall< RAJA_NODE_TYPE >(RAJA::RangeSegment(zeroIdx, N),
    RAJA_LAMBDA(sunindextype i) {
      if (iddata[i] > ZERO)
        gpu_result += (xdata[i] * wdata[i] * xdata[i] * wdata[i]);
    }
  );

  return(static_cast<realtype>(gpu_result));
}

realtype N_VWrmsNormMask_Raja(N_Vector X, N_Vector W, N_Vector ID)
{
  const realtype sum = N_VWSqrSumMaskLocal_Raja(X, W, ID);
  const sunindextype N = N_VGetLength_Raja(X);
  return std::sqrt(sum/N);
}

realtype N_VMin_Raja(N_Vector X)
{
  const realtype *xdata = N_VGetDeviceArrayPointer_Raja(X);
  const sunindextype N = N_VGetLength_Raja(X);

  RAJA::ReduceMin< RAJA_REDUCE_TYPE, realtype> gpu_result(std::numeric_limits<realtype>::max());
  RAJA::forall< RAJA_NODE_TYPE >(RAJA::RangeSegment(zeroIdx, N),
    RAJA_LAMBDA(sunindextype i) {
      gpu_result.min(xdata[i]);
    }
  );

  return(static_cast<realtype>(gpu_result));
}

realtype N_VWL2Norm_Raja(N_Vector X, N_Vector W)
{
  return std::sqrt(N_VWSqrSumLocal_Raja(X, W));
}

realtype N_VL1Norm_Raja(N_Vector X)
{
  const realtype *xdata = N_VGetDeviceArrayPointer_Raja(X);
  const sunindextype N = N_VGetLength_Raja(X);

  RAJA::ReduceSum< RAJA_REDUCE_TYPE, realtype> gpu_result(0.0);
  RAJA::forall< RAJA_NODE_TYPE >(RAJA::RangeSegment(zeroIdx, N),
    RAJA_LAMBDA(sunindextype i) {
      gpu_result += (abs(xdata[i]));
    }
  );

  return(static_cast<realtype>(gpu_result));
}

void N_VCompare_Raja(realtype c, N_Vector X, N_Vector Z)
{
  const realtype *xdata = N_VGetDeviceArrayPointer_Raja(X);
  const sunindextype N = N_VGetLength_Raja(X);
  realtype *zdata = N_VGetDeviceArrayPointer_Raja(Z);

  RAJA::forall< RAJA_NODE_TYPE >(RAJA::RangeSegment(zeroIdx, N),
    RAJA_LAMBDA(sunindextype i) {
      zdata[i] = abs(xdata[i]) >= c ? ONE : ZERO;
    }
  );
}

booleantype N_VInvTest_Raja(N_Vector x, N_Vector z)
{
  const realtype *xdata = N_VGetDeviceArrayPointer_Raja(x);
  const sunindextype N = N_VGetLength_Raja(x);
  realtype *zdata = N_VGetDeviceArrayPointer_Raja(z);

  RAJA::ReduceSum< RAJA_REDUCE_TYPE, realtype> gpu_result(ZERO);
  RAJA::forall< RAJA_NODE_TYPE >(RAJA::RangeSegment(zeroIdx, N),
    RAJA_LAMBDA(sunindextype i) {
      if (xdata[i] == ZERO) {
        gpu_result += ONE;
      } else {
        zdata[i] = ONE/xdata[i];
      }
    }
  );
  realtype minimum = static_cast<realtype>(gpu_result);
  return (minimum < HALF);
}

booleantype N_VConstrMask_Raja(N_Vector c, N_Vector x, N_Vector m)
{
  const realtype *cdata = N_VGetDeviceArrayPointer_Raja(c);
  const realtype *xdata = N_VGetDeviceArrayPointer_Raja(x);
  const sunindextype N = N_VGetLength_Raja(x);
  realtype *mdata = N_VGetDeviceArrayPointer_Raja(m);

  RAJA::ReduceSum< RAJA_REDUCE_TYPE, realtype> gpu_result(ZERO);
  RAJA::forall< RAJA_NODE_TYPE >(RAJA::RangeSegment(zeroIdx, N),
    RAJA_LAMBDA(sunindextype i) {
      bool test = (abs(cdata[i]) > ONEPT5 && cdata[i]*xdata[i] <= ZERO) ||
                  (abs(cdata[i]) > HALF   && cdata[i]*xdata[i] <  ZERO);
      mdata[i] = test ? ONE : ZERO;
      gpu_result += mdata[i];
    }
  );

  realtype sum = static_cast<realtype>(gpu_result);
  return(sum < HALF);
}

realtype N_VMinQuotient_Raja(N_Vector num, N_Vector denom)
{
  const realtype *ndata = N_VGetDeviceArrayPointer_Raja(num);
  const realtype *ddata = N_VGetDeviceArrayPointer_Raja(denom);
  const sunindextype N = N_VGetLength_Raja(num);

  RAJA::ReduceMin< RAJA_REDUCE_TYPE, realtype> gpu_result(std::numeric_limits<realtype>::max());
  RAJA::forall< RAJA_NODE_TYPE >(RAJA::RangeSegment(zeroIdx, N),
    RAJA_LAMBDA(sunindextype i) {
      if (ddata[i] != ZERO)
        gpu_result.min(ndata[i]/ddata[i]);
    }
  );
  return(static_cast<realtype>(gpu_result));
}


/*
 * -----------------------------------------------------------------------------
 * fused vector operations
 * -----------------------------------------------------------------------------
 */

int N_VLinearCombination_Raja(int nvec, realtype* c, N_Vector* X, N_Vector z)
{
  hipError_t  err;

  sunindextype N = N_VGetLength_Raja(z);
  realtype* d_zd = N_VGetDeviceArrayPointer_Raja(z);

  // Copy c array to device
  realtype* d_c;
  err = hipMalloc((void**) &d_c, nvec*sizeof(realtype));
  if (err != hipSuccess) return hipGetLastError();
  err = hipMemcpy(d_c, c, nvec*sizeof(realtype), hipMemcpyHostToDevice);
  if (err != hipSuccess) return hipGetLastError();

  // Create array of device pointers on host
  realtype** h_Xd = new realtype*[nvec];
  for (int j=0; j<nvec; j++)
    h_Xd[j] = N_VGetDeviceArrayPointer_Raja(X[j]);

  // Copy array of device pointers to device from host
  realtype** d_Xd;
  err = hipMalloc((void**) &d_Xd, nvec*sizeof(realtype*));
  if (err != hipSuccess) return hipGetLastError();
  err = hipMemcpy(d_Xd, h_Xd, nvec*sizeof(realtype*), hipMemcpyHostToDevice);
  if (err != hipSuccess) return hipGetLastError();

  RAJA::forall< RAJA_NODE_TYPE >(RAJA::RangeSegment(zeroIdx, N),
    RAJA_LAMBDA(sunindextype i) {
      d_zd[i] = d_c[0] * d_Xd[0][i];
      for (int j=1; j<nvec; j++)
        d_zd[i] += d_c[j] * d_Xd[j][i];
    }
  );

  // Free host array
  delete[] h_Xd;

  // Free device arrays
  err = hipFree(d_c);
  if (err != hipSuccess) return hipGetLastError();
  err = hipFree(d_Xd);
  if (err != hipSuccess) return hipGetLastError();

  return(0);
}


int N_VScaleAddMulti_Raja(int nvec, realtype* c, N_Vector x, N_Vector* Y, N_Vector* Z)
{
  hipError_t err;

  sunindextype N = N_VGetLength_Raja(x);
  realtype* d_xd = N_VGetDeviceArrayPointer_Raja(x);

  // Copy c array to device
  realtype* d_c;
  err = hipMalloc((void**) &d_c, nvec*sizeof(realtype));
  if (err != hipSuccess) return hipGetLastError();
  err = hipMemcpy(d_c, c, nvec*sizeof(realtype), hipMemcpyHostToDevice);
  if (err != hipSuccess) return hipGetLastError();

  // Create array of device pointers on host
  realtype** h_Yd = new realtype*[nvec];
  for (int j=0; j<nvec; j++)
    h_Yd[j] = N_VGetDeviceArrayPointer_Raja(Y[j]);

  realtype** h_Zd = new realtype*[nvec];
  for (int j=0; j<nvec; j++)
    h_Zd[j] = N_VGetDeviceArrayPointer_Raja(Z[j]);

  // Copy array of device pointers to device from host
  realtype** d_Yd;
  err = hipMalloc((void**) &d_Yd, nvec*sizeof(realtype*));
  if (err != hipSuccess) return hipGetLastError();
  err = hipMemcpy(d_Yd, h_Yd, nvec*sizeof(realtype*), hipMemcpyHostToDevice);
  if (err != hipSuccess) return hipGetLastError();

  realtype** d_Zd;
  err = hipMalloc((void**) &d_Zd, nvec*sizeof(realtype*));
  if (err != hipSuccess) return hipGetLastError();
  err = hipMemcpy(d_Zd, h_Zd, nvec*sizeof(realtype*), hipMemcpyHostToDevice);
  if (err != hipSuccess) return hipGetLastError();

  RAJA::forall< RAJA_NODE_TYPE >(RAJA::RangeSegment(zeroIdx, N),
    RAJA_LAMBDA(sunindextype i) {
      for (int j=0; j<nvec; j++)
        d_Zd[j][i] = d_c[j] * d_xd[i] + d_Yd[j][i];
    }
  );

  // Free host array
  delete[] h_Yd;
  delete[] h_Zd;

  // Free device arrays
  err = hipFree(d_c);
  if (err != hipSuccess) return hipGetLastError();
  err = hipFree(d_Yd);
  if (err != hipSuccess) return hipGetLastError();
  err = hipFree(d_Zd);
  if (err != hipSuccess) return hipGetLastError();

  return(0);
}


/*
 * -----------------------------------------------------------------------------
 * vector array operations
 * -----------------------------------------------------------------------------
 */

int N_VLinearSumVectorArray_Raja(int nvec,
                                 realtype a, N_Vector* X,
                                 realtype b, N_Vector* Y,
                                 N_Vector* Z)
{
  hipError_t err;

  sunindextype N = N_VGetLength_Raja(Z[0]);

  // Create array of device pointers on host
  realtype** h_Xd = new realtype*[nvec];
  for (int j=0; j<nvec; j++)
    h_Xd[j] = N_VGetDeviceArrayPointer_Raja(X[j]);

  realtype** h_Yd = new realtype*[nvec];
  for (int j=0; j<nvec; j++)
    h_Yd[j] = N_VGetDeviceArrayPointer_Raja(Y[j]);

  realtype** h_Zd = new realtype*[nvec];
  for (int j=0; j<nvec; j++)
    h_Zd[j] = N_VGetDeviceArrayPointer_Raja(Z[j]);

  // Copy array of device pointers to device from host
  realtype** d_Xd;
  err = hipMalloc((void**) &d_Xd, nvec*sizeof(realtype*));
  if (err != hipSuccess) return hipGetLastError();
  err = hipMemcpy(d_Xd, h_Xd, nvec*sizeof(realtype*), hipMemcpyHostToDevice);
  if (err != hipSuccess) return hipGetLastError();

  realtype** d_Yd;
  err = hipMalloc((void**) &d_Yd, nvec*sizeof(realtype*));
  if (err != hipSuccess) return hipGetLastError();
  err = hipMemcpy(d_Yd, h_Yd, nvec*sizeof(realtype*), hipMemcpyHostToDevice);
  if (err != hipSuccess) return hipGetLastError();

  realtype** d_Zd;
  err = hipMalloc((void**) &d_Zd, nvec*sizeof(realtype*));
  if (err != hipSuccess) return hipGetLastError();
  err = hipMemcpy(d_Zd, h_Zd, nvec*sizeof(realtype*), hipMemcpyHostToDevice);
  if (err != hipSuccess) return hipGetLastError();

  RAJA::forall< RAJA_NODE_TYPE >(RAJA::RangeSegment(zeroIdx, N),
    RAJA_LAMBDA(sunindextype i) {
      for (int j=0; j<nvec; j++)
        d_Zd[j][i] = a * d_Xd[j][i] + b * d_Yd[j][i];
    }
  );

  // Free host array
  delete[] h_Xd;
  delete[] h_Yd;
  delete[] h_Zd;

  // Free device arrays
  err = hipFree(d_Xd);
  if (err != hipSuccess) return hipGetLastError();
  err = hipFree(d_Yd);
  if (err != hipSuccess) return hipGetLastError();
  err = hipFree(d_Zd);
  if (err != hipSuccess) return hipGetLastError();

  return(0);
}


int N_VScaleVectorArray_Raja(int nvec, realtype* c, N_Vector* X, N_Vector* Z)
{
  hipError_t err;

  sunindextype N = N_VGetLength_Raja(Z[0]);

  // Copy c array to device
  realtype* d_c;
  err = hipMalloc((void**) &d_c, nvec*sizeof(realtype));
  if (err != hipSuccess) return hipGetLastError();
  err = hipMemcpy(d_c, c, nvec*sizeof(realtype), hipMemcpyHostToDevice);
  if (err != hipSuccess) return hipGetLastError();

  // Create array of device pointers on host
  realtype** h_Xd = new realtype*[nvec];
  for (int j=0; j<nvec; j++)
    h_Xd[j] = N_VGetDeviceArrayPointer_Raja(X[j]);

  realtype** h_Zd = new realtype*[nvec];
  for (int j=0; j<nvec; j++)
    h_Zd[j] = N_VGetDeviceArrayPointer_Raja(Z[j]);

  // Copy array of device pointers to device from host
  realtype** d_Xd;
  err = hipMalloc((void**) &d_Xd, nvec*sizeof(realtype*));
  if (err != hipSuccess) return hipGetLastError();
  err = hipMemcpy(d_Xd, h_Xd, nvec*sizeof(realtype*), hipMemcpyHostToDevice);
  if (err != hipSuccess) return hipGetLastError();

  realtype** d_Zd;
  err = hipMalloc((void**) &d_Zd, nvec*sizeof(realtype*));
  if (err != hipSuccess) return hipGetLastError();
  err = hipMemcpy(d_Zd, h_Zd, nvec*sizeof(realtype*), hipMemcpyHostToDevice);
  if (err != hipSuccess) return hipGetLastError();

  RAJA::forall< RAJA_NODE_TYPE >(RAJA::RangeSegment(zeroIdx, N),
    RAJA_LAMBDA(sunindextype i) {
      for (int j=0; j<nvec; j++)
        d_Zd[j][i] = d_c[j] * d_Xd[j][i];
    }
  );

  // Free host array
  delete[] h_Xd;
  delete[] h_Zd;

  // Free device arrays
  err = hipFree(d_Xd);
  if (err != hipSuccess) return hipGetLastError();
  err = hipFree(d_Zd);
  if (err != hipSuccess) return hipGetLastError();

  return(0);
}


int N_VConstVectorArray_Raja(int nvec, realtype c, N_Vector* Z)
{
  hipError_t err;

  sunindextype N = N_VGetLength_Raja(Z[0]);

  // Create array of device pointers on host
  realtype** h_Zd = new realtype*[nvec];
  for (int j=0; j<nvec; j++)
    h_Zd[j] = N_VGetDeviceArrayPointer_Raja(Z[j]);

  // Copy array of device pointers to device from host
  realtype** d_Zd;
  err = hipMalloc((void**) &d_Zd, nvec*sizeof(realtype*));
  if (err != hipSuccess) return hipGetLastError();
  err = hipMemcpy(d_Zd, h_Zd, nvec*sizeof(realtype*), hipMemcpyHostToDevice);
  if (err != hipSuccess) return hipGetLastError();

  RAJA::forall< RAJA_NODE_TYPE >(RAJA::RangeSegment(zeroIdx, N),
    RAJA_LAMBDA(sunindextype i) {
      for (int j=0; j<nvec; j++)
        d_Zd[j][i] = c;
    }
  );

  // Free host array
  delete[] h_Zd;

  // Free device arrays
  err = hipFree(d_Zd);
  if (err != hipSuccess) return hipGetLastError();

  return(0);
}


int N_VScaleAddMultiVectorArray_Raja(int nvec, int nsum, realtype* c,
                                     N_Vector* X, N_Vector** Y, N_Vector** Z)
{
  hipError_t err;

  sunindextype N = N_VGetLength_Raja(X[0]);

  // Copy c array to device
  realtype* d_c;
  err = hipMalloc((void**) &d_c, nsum*sizeof(realtype));
  if (err != hipSuccess) return hipGetLastError();
  err = hipMemcpy(d_c, c, nsum*sizeof(realtype), hipMemcpyHostToDevice);
  if (err != hipSuccess) return hipGetLastError();

  // Create array of device pointers on host
  realtype** h_Xd = new realtype*[nvec];
  for (int j=0; j<nvec; j++)
    h_Xd[j] = N_VGetDeviceArrayPointer_Raja(X[j]);

  realtype** h_Yd = new realtype*[nsum*nvec];
  for (int j=0; j<nvec; j++)
    for (int k=0; k<nsum; k++)
      h_Yd[j*nsum+k] = N_VGetDeviceArrayPointer_Raja(Y[k][j]);

  realtype** h_Zd = new realtype*[nsum*nvec];
  for (int j=0; j<nvec; j++)
    for (int k=0; k<nsum; k++)
      h_Zd[j*nsum+k] = N_VGetDeviceArrayPointer_Raja(Z[k][j]);

  // Copy array of device pointers to device from host
  realtype** d_Xd;
  err = hipMalloc((void**) &d_Xd, nvec*sizeof(realtype*));
  if (err != hipSuccess) return hipGetLastError();
  err = hipMemcpy(d_Xd, h_Xd, nvec*sizeof(realtype*), hipMemcpyHostToDevice);
  if (err != hipSuccess) return hipGetLastError();

  realtype** d_Yd;
  err = hipMalloc((void**) &d_Yd, nsum*nvec*sizeof(realtype*));
  if (err != hipSuccess) return hipGetLastError();
  err = hipMemcpy(d_Yd, h_Yd, nsum*nvec*sizeof(realtype*), hipMemcpyHostToDevice);
  if (err != hipSuccess) return hipGetLastError();

  realtype** d_Zd;
  err = hipMalloc((void**) &d_Zd, nsum*nvec*sizeof(realtype*));
  if (err != hipSuccess) return hipGetLastError();
  err = hipMemcpy(d_Zd, h_Zd, nsum*nvec*sizeof(realtype*), hipMemcpyHostToDevice);
  if (err != hipSuccess) return hipGetLastError();

  RAJA::forall< RAJA_NODE_TYPE >(RAJA::RangeSegment(zeroIdx, N),
    RAJA_LAMBDA(sunindextype i) {
      for (int j=0; j<nvec; j++)
        for (int k=0; k<nsum; k++)
          d_Zd[j*nsum+k][i] = d_c[k] * d_Xd[j][i] + d_Yd[j*nsum+k][i];
    }
  );

  // Free host array
  delete[] h_Xd;
  delete[] h_Yd;
  delete[] h_Zd;

  // Free device arrays
  err = hipFree(d_Xd);
  if (err != hipSuccess) return hipGetLastError();
  err = hipFree(d_Yd);
  if (err != hipSuccess) return hipGetLastError();
  err = hipFree(d_Zd);
  if (err != hipSuccess) return hipGetLastError();

  return(0);
}


int N_VLinearCombinationVectorArray_Raja(int nvec, int nsum, realtype* c,
                                         N_Vector** X, N_Vector* Z)
{
  hipError_t err;

  sunindextype N = N_VGetLength_Raja(Z[0]);

  // Copy c array to device
  realtype* d_c;
  err = hipMalloc((void**) &d_c, nsum*sizeof(realtype));
  if (err != hipSuccess) return hipGetLastError();
  err = hipMemcpy(d_c, c, nsum*sizeof(realtype), hipMemcpyHostToDevice);
  if (err != hipSuccess) return hipGetLastError();

  // Create array of device pointers on host
  realtype** h_Xd = new realtype*[nsum*nvec];
  for (int j=0; j<nvec; j++)
    for (int k=0; k<nsum; k++)
      h_Xd[j*nsum+k] = N_VGetDeviceArrayPointer_Raja(X[k][j]);

  realtype** h_Zd = new realtype*[nvec];
  for (int j=0; j<nvec; j++)
    h_Zd[j] = N_VGetDeviceArrayPointer_Raja(Z[j]);

  // Copy array of device pointers to device from host
  realtype** d_Xd;
  err = hipMalloc((void**) &d_Xd, nsum*nvec*sizeof(realtype*));
  if (err != hipSuccess) return hipGetLastError();
  err = hipMemcpy(d_Xd, h_Xd, nsum*nvec*sizeof(realtype*), hipMemcpyHostToDevice);
  if (err != hipSuccess) return hipGetLastError();

  realtype** d_Zd;
  err = hipMalloc((void**) &d_Zd, nvec*sizeof(realtype*));
  if (err != hipSuccess) return hipGetLastError();
  err = hipMemcpy(d_Zd, h_Zd, nvec*sizeof(realtype*), hipMemcpyHostToDevice);
  if (err != hipSuccess) return hipGetLastError();

  RAJA::forall< RAJA_NODE_TYPE >(RAJA::RangeSegment(zeroIdx, N),
    RAJA_LAMBDA(sunindextype i) {
      for (int j=0; j<nvec; j++) {
        d_Zd[j][i] = d_c[0] * d_Xd[j*nsum][i];
        for (int k=1; k<nsum; k++) {
          d_Zd[j][i] += d_c[k] * d_Xd[j*nsum+k][i];
        }
      }
    }
  );

  // Free host array
  delete[] h_Xd;
  delete[] h_Zd;

  // Free device arrays
  err = hipFree(d_Xd);
  if (err != hipSuccess) return hipGetLastError();
  err = hipFree(d_Zd);
  if (err != hipSuccess) return hipGetLastError();

  return(0);
}


/*
 * -----------------------------------------------------------------
 * Enable / Disable fused and vector array operations
 * -----------------------------------------------------------------
 */

int N_VEnableFusedOps_Raja(N_Vector v, booleantype tf)
{
  /* check that vector is non-NULL */
  if (v == NULL) return(-1);

  /* check that ops structure is non-NULL */
  if (v->ops == NULL) return(-1);

  if (tf) {
    /* enable all fused vector operations */
    v->ops->nvlinearcombination = N_VLinearCombination_Raja;
    v->ops->nvscaleaddmulti     = N_VScaleAddMulti_Raja;
    v->ops->nvdotprodmulti      = NULL;
    /* enable all vector array operations */
    v->ops->nvlinearsumvectorarray         = N_VLinearSumVectorArray_Raja;
    v->ops->nvscalevectorarray             = N_VScaleVectorArray_Raja;
    v->ops->nvconstvectorarray             = N_VConstVectorArray_Raja;
    v->ops->nvwrmsnormvectorarray          = NULL;
    v->ops->nvwrmsnormmaskvectorarray      = NULL;
    v->ops->nvscaleaddmultivectorarray     = N_VScaleAddMultiVectorArray_Raja;
    v->ops->nvlinearcombinationvectorarray = N_VLinearCombinationVectorArray_Raja;
  } else {
    /* disable all fused vector operations */
    v->ops->nvlinearcombination = NULL;
    v->ops->nvscaleaddmulti     = NULL;
    v->ops->nvdotprodmulti      = NULL;
    /* disable all vector array operations */
    v->ops->nvlinearsumvectorarray         = NULL;
    v->ops->nvscalevectorarray             = NULL;
    v->ops->nvconstvectorarray             = NULL;
    v->ops->nvwrmsnormvectorarray          = NULL;
    v->ops->nvwrmsnormmaskvectorarray      = NULL;
    v->ops->nvscaleaddmultivectorarray     = NULL;
    v->ops->nvlinearcombinationvectorarray = NULL;
  }

  /* return success */
  return(0);
}


int N_VEnableLinearCombination_Raja(N_Vector v, booleantype tf)
{
  /* check that vector is non-NULL */
  if (v == NULL) return(-1);

  /* check that ops structure is non-NULL */
  if (v->ops == NULL) return(-1);

  /* enable/disable operation */
  if (tf)
    v->ops->nvlinearcombination = N_VLinearCombination_Raja;
  else
    v->ops->nvlinearcombination = NULL;

  /* return success */
  return(0);
}

int N_VEnableScaleAddMulti_Raja(N_Vector v, booleantype tf)
{
  /* check that vector is non-NULL */
  if (v == NULL) return(-1);

  /* check that ops structure is non-NULL */
  if (v->ops == NULL) return(-1);

  /* enable/disable operation */
  if (tf)
    v->ops->nvscaleaddmulti = N_VScaleAddMulti_Raja;
  else
    v->ops->nvscaleaddmulti = NULL;

  /* return success */
  return(0);
}

int N_VEnableLinearSumVectorArray_Raja(N_Vector v, booleantype tf)
{
  /* check that vector is non-NULL */
  if (v == NULL) return(-1);

  /* check that ops structure is non-NULL */
  if (v->ops == NULL) return(-1);

  /* enable/disable operation */
  if (tf)
    v->ops->nvlinearsumvectorarray = N_VLinearSumVectorArray_Raja;
  else
    v->ops->nvlinearsumvectorarray = NULL;

  /* return success */
  return(0);
}

int N_VEnableScaleVectorArray_Raja(N_Vector v, booleantype tf)
{
  /* check that vector is non-NULL */
  if (v == NULL) return(-1);

  /* check that ops structure is non-NULL */
  if (v->ops == NULL) return(-1);

  /* enable/disable operation */
  if (tf)
    v->ops->nvscalevectorarray = N_VScaleVectorArray_Raja;
  else
    v->ops->nvscalevectorarray = NULL;

  /* return success */
  return(0);
}

int N_VEnableConstVectorArray_Raja(N_Vector v, booleantype tf)
{
  /* check that vector is non-NULL */
  if (v == NULL) return(-1);

  /* check that ops structure is non-NULL */
  if (v->ops == NULL) return(-1);

  /* enable/disable operation */
  if (tf)
    v->ops->nvconstvectorarray = N_VConstVectorArray_Raja;
  else
    v->ops->nvconstvectorarray = NULL;

  /* return success */
  return(0);
}

int N_VEnableScaleAddMultiVectorArray_Raja(N_Vector v, booleantype tf)
{
  /* check that vector is non-NULL */
  if (v == NULL) return(-1);

  /* check that ops structure is non-NULL */
  if (v->ops == NULL) return(-1);

  /* enable/disable operation */
  if (tf)
    v->ops->nvscaleaddmultivectorarray = N_VScaleAddMultiVectorArray_Raja;
  else
    v->ops->nvscaleaddmultivectorarray = NULL;

  /* return success */
  return(0);
}

int N_VEnableLinearCombinationVectorArray_Raja(N_Vector v, booleantype tf)
{
  /* check that vector is non-NULL */
  if (v == NULL) return(-1);

  /* check that ops structure is non-NULL */
  if (v->ops == NULL) return(-1);

  /* enable/disable operation */
  if (tf)
    v->ops->nvlinearcombinationvectorarray = N_VLinearCombinationVectorArray_Raja;
  else
    v->ops->nvlinearcombinationvectorarray = NULL;

  /* return success */
  return(0);
}

} // extern "C"
