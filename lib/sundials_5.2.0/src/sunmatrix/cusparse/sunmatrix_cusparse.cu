/*
 * -----------------------------------------------------------------
 * Programmer(s): Cody J. Balos @ LLNL
 * -----------------------------------------------------------------
 * SUNDIALS Copyright Start
 * Copyright (c) 2002-2020, Lawrence Livermore National Security
 * and Southern Methodist University.
 * All rights reserved.
 *
 * See the top-level LICENSE and NOTICE files for details.
 *
 * SPDX-License-Identifier: BSD-3-Clause
 * SUNDIALS Copyright End
 * -----------------------------------------------------------------
 * This is the header file is for the cuSPARSE implementation of the
 * SUNMATRIX module.
 * -----------------------------------------------------------------
 */

#include <stdio.h>
#include <stdlib.h>

#include <nvector/nvector_cuda.h>
#include <sunmatrix/sunmatrix_cusparse.h>

#include "sundials_cuda.h"
#include "sundials_debug.h"
#include "cusparse_kernels.cuh"

/* Use the namespace for the kernels */
using namespace sundials::device::sunmatrix_cusparse;

/* Constants */
#define ZERO RCONST(0.0)
#define ONE  RCONST(1.0)

#define MAX_THREAD_PER_BLOCK(val) ( (val > 16*CUDA_WARP_SIZE) ? (16*CUDA_WARP_SIZE) : (val) )

/* Private function prototypes */
static booleantype SMCompatible_cuSparse(SUNMatrix A, SUNMatrix B);
static SUNMatrix SUNMatrix_cuSparse_NewEmpty();

/* Macros for handling the different function names based on precision */
#if defined(SUNDIALS_DOUBLE_PRECISION)
#define cusparseXcsrmv hipsparseDcsrmv
#elif defined(SUNDIALS_SINGLE_PRECISION)
#define cusparseXcsrmv hipsparseScsrmv
#endif

/* Content accessor macros */
#define SMCU_CONTENT_S(A)     ( (SUNMatrix_Content_cuSparse)(A->content) )
#define SMCU_ROWS_S(A)        ( SMCU_CONTENT_S(A)->M )
#define SMCU_COLUMNS_S(A)     ( SMCU_CONTENT_S(A)->N )
#define SMCU_NNZ_S(A)         ( SMCU_CONTENT_S(A)->NNZ )
#define SMCU_NBLOCKS_S(A)     ( SMCU_CONTENT_S(A)->nblocks )
#define SMCU_BLOCKROWS_S(A)   ( SMCU_CONTENT_S(A)->blockrows )
#define SMCU_BLOCKCOLS_S(A)   ( SMCU_CONTENT_S(A)->blockcols )
#define SMCU_BLOCKNNZ_S(A)    ( SMCU_CONTENT_S(A)->blocknnz )
#define SMCU_NP_S(A)          ( SMCU_CONTENT_S(A)->NP )
#define SMCU_SPARSETYPE_S(A)  ( SMCU_CONTENT_S(A)->sparse_type )
#define SMCU_OWNDATA_S(A)     ( SMCU_CONTENT_S(A)->own_data )
#define SMCU_DATA_S(A)        ( SMCU_CONTENT_S(A)->data )
#define SMCU_INDEXVALS_S(A)   ( SMCU_CONTENT_S(A)->colind )
#define SMCU_INDEXPTRS_S(A)   ( SMCU_CONTENT_S(A)->rowptrs )
#define SMCU_MATDESCR_S(A)    ( SMCU_CONTENT_S(A)->mat_descr )
#define SMCU_CUSPHANDLE_S(A)  ( SMCU_CONTENT_S(A)->cusp_handle )
#define SMCU_FIXEDPATTERN_S(A)( SMCU_CONTENT_S(A)->fixed_pattern )


/* ------------------------------------------------------------------
 * Constructors.
 * ------------------------------------------------------------------ */


SUNMatrix SUNMatrix_cuSparse_NewCSR(int M, int N, int NNZ, hipsparseHandle_t cusp)
{
  /* return with NULL matrix on illegal input */
  if ( (M <= 0) || (N <= 0) || (NNZ < 0) )
  {
    SUNDIALS_DEBUG_PRINT("ERROR in SUNMatrix_NewCSR_cuSparse: illegal value(s) for M, N, or NNZ\n");
    return NULL;
  }

  SUNMatrix A = SUNMatrix_cuSparse_NewEmpty();
  if (A == NULL)
  {
    SUNDIALS_DEBUG_PRINT("ERROR in SUNMatrix_NewCSR_cuSparse: SUNMatrix_cuSparse_NewEmpty returned NULL\n");
    return NULL;
  }

  /* Allocate device memory for the matrix */
  int *d_colind, *d_rowptr;
  realtype *d_values;

  d_colind = NULL;
  d_rowptr = NULL;
  d_values = NULL;

  hipError_t cuerr;
  cuerr = hipMalloc((void **) &d_colind, sizeof(*d_colind) * NNZ);
  if (!SUNDIALS_CUDA_VERIFY(cuerr))
  {
    SUNMatDestroy(A);
    return NULL;
  }
  cuerr = hipMalloc((void **) &d_rowptr, sizeof(*d_rowptr) * (M+1));
  if (!SUNDIALS_CUDA_VERIFY(cuerr))
  {
    SUNMatDestroy(A);
    hipFree(d_colind);
    return NULL;
  }
  cuerr = hipMalloc((void **) &d_values, sizeof(*d_values) * NNZ);
  if (!SUNDIALS_CUDA_VERIFY(cuerr))
  {
    SUNMatDestroy(A);
    hipFree(d_colind);
    hipFree(d_rowptr);
    return NULL;
  }

  /* Choose sensible defaults */
  hipsparseStatus_t cusparse_status = HIPSPARSE_STATUS_SUCCESS;
  hipsparseMatDescr_t mat_descr;
  cusparse_status = hipsparseCreateMatDescr(&mat_descr);
  if (!SUNDIALS_CUSPARSE_VERIFY(cusparse_status))
  {
    hipFree(d_rowptr); hipFree(d_colind);
    hipFree(d_values); SUNMatDestroy(A);
    return NULL;
  }

  cusparse_status = hipsparseSetMatType(mat_descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
  if (!SUNDIALS_CUSPARSE_VERIFY(cusparse_status))
  {
    hipFree(d_rowptr); hipFree(d_colind);
    hipFree(d_values); SUNMatDestroy(A);
    hipsparseDestroyMatDescr(mat_descr);
    return NULL;
  }

  cusparse_status = hipsparseSetMatIndexBase(mat_descr, HIPSPARSE_INDEX_BASE_ZERO);
  if (!SUNDIALS_CUSPARSE_VERIFY(cusparse_status))
  {
    hipFree(d_rowptr); hipFree(d_colind);
    hipFree(d_values); SUNMatDestroy(A);
    hipsparseDestroyMatDescr(mat_descr);
    return NULL;
  }

  /* Fill the content */
  SMCU_CONTENT_S(A)->M             = M;
  SMCU_CONTENT_S(A)->N             = N;
  SMCU_CONTENT_S(A)->NNZ           = NNZ;
  SMCU_CONTENT_S(A)->nblocks       = 1;
  SMCU_CONTENT_S(A)->blockrows     = M;
  SMCU_CONTENT_S(A)->blockcols     = N;
  SMCU_CONTENT_S(A)->blocknnz      = NNZ;
  SMCU_CONTENT_S(A)->own_data      = SUNTRUE;
  SMCU_CONTENT_S(A)->sparse_type   = SUNMAT_CUSPARSE_CSR;
  SMCU_CONTENT_S(A)->colind        = d_colind;
  SMCU_CONTENT_S(A)->rowptrs       = d_rowptr;
  SMCU_CONTENT_S(A)->data          = d_values;
  SMCU_CONTENT_S(A)->mat_descr     = mat_descr;
  SMCU_CONTENT_S(A)->cusp_handle   = cusp;
  SMCU_CONTENT_S(A)->fixed_pattern = SUNFALSE;

  return A;
}


SUNMatrix SUNMatrix_cuSparse_MakeCSR(hipsparseMatDescr_t mat_descr, int M, int N, int NNZ,
                                     int *rowptrs , int *colind , realtype *data,
                                     hipsparseHandle_t cusp)
{
  /* return with NULL matrix on illegal input */
  if ( (M <= 0) || (N <= 0) || (NNZ < 0) )
  {
    SUNDIALS_DEBUG_PRINT("ERROR in SUNMatrix_MakeCSR_cuSparse: illegal value(s) for M, N, or NNZ\n");
    return NULL;
  }

  if ( (rowptrs == NULL) || (colind == NULL) || (data == NULL) )
  {
    SUNDIALS_DEBUG_PRINT("ERROR in SUNMatrix_MakeCSR_cuSparse: rowptrs, colind, or data is NULL\n");
    return NULL;
  }

  if (hipsparseGetMatIndexBase(mat_descr) != HIPSPARSE_INDEX_BASE_ZERO)
  {
    SUNDIALS_DEBUG_PRINT("ERROR in SUNMatrix_MakeCSR_cuSparse: the hipsparseMatDescr_t must have index base HIPSPARSE_INDEX_BASE_ZERO\n");
    return NULL;
  }

  SUNMatrix A = SUNMatrix_cuSparse_NewEmpty();
  if (A == NULL)
  {
    SUNDIALS_DEBUG_PRINT("ERROR in SUNMatrix_MakeCSR_cuSparse: SUNMatrix_cuSparse_NewEmpty returned NULL\n");
    return NULL;
  }

  /* Fill content */
  SMCU_CONTENT_S(A)->M             = M;
  SMCU_CONTENT_S(A)->N             = N;
  SMCU_CONTENT_S(A)->NNZ           = NNZ;
  SMCU_CONTENT_S(A)->nblocks       = 1;
  SMCU_CONTENT_S(A)->blockrows     = M;
  SMCU_CONTENT_S(A)->blockcols     = N;
  SMCU_CONTENT_S(A)->blocknnz      = NNZ;
  SMCU_CONTENT_S(A)->own_data      = SUNFALSE;
  SMCU_CONTENT_S(A)->sparse_type   = SUNMAT_CUSPARSE_CSR;
  SMCU_CONTENT_S(A)->colind        = colind;
  SMCU_CONTENT_S(A)->rowptrs       = rowptrs;
  SMCU_CONTENT_S(A)->data          = data;
  SMCU_CONTENT_S(A)->mat_descr     = mat_descr;
  SMCU_CONTENT_S(A)->cusp_handle   = cusp;
  SMCU_CONTENT_S(A)->fixed_pattern = SUNFALSE;

  return A;
}


SUNMatrix SUNMatrix_cuSparse_NewBlockCSR(int nblocks, int blockrows, int blockcols, int blocknnz, hipsparseHandle_t cusp)
{
  int M, N, NNZ;

  /* Return with NULL matrix on illegal input */
  if (blockrows != blockcols)
  {
    SUNDIALS_DEBUG_PRINT("ERROR in SUNMatrix_cuSparse_NewBlockCSR: matrix must be square for the BCSR format\n");
    return NULL;
  }

  M   = nblocks * blockrows;
  N   = M;
  NNZ = nblocks * blocknnz;

  /* Return with NULL matrix on illegal input */
  if ( (M <= 0) || (N <= 0) || (NNZ < 0) )
  {
    SUNDIALS_DEBUG_PRINT("ERROR in SUNMatrix_cuSparse_NewBlockCSR: illegal value(s) for M, N, or NNZ\n");
    return NULL;
  }

  /* Allocate the SUNMatrix object */
  SUNMatrix A = SUNMatrix_cuSparse_NewEmpty();
  if (A == NULL)
  {
    SUNDIALS_DEBUG_PRINT("ERROR in SUNMatrix_cuSparse_NewBlockCSR: SUNMatrix_cuSparse_NewEmpty returned NULL\n");
    return NULL;
  }

  /* Allocate device memory for the matrix */
  int *d_colind, *d_rowptr;
  realtype *d_values;

  d_colind = NULL;
  d_rowptr = NULL;
  d_values = NULL;

  hipError_t cuerr;
  cuerr = hipMalloc((void **) &d_colind, sizeof(*d_colind) * blocknnz);
  if (!SUNDIALS_CUDA_VERIFY(cuerr))
  {
    SUNMatDestroy(A);
    return NULL;
  }
  cuerr = hipMalloc((void **) &d_rowptr, sizeof(*d_rowptr) * (blockrows + 1));
  if (!SUNDIALS_CUDA_VERIFY(cuerr))
  {
    SUNMatDestroy(A);
    hipFree(d_colind);
    return NULL;
  }
  cuerr = hipMalloc((void **) &d_values, sizeof(*d_values) * blocknnz * nblocks);
  if (!SUNDIALS_CUDA_VERIFY(cuerr))
  {
    SUNMatDestroy(A);
    hipFree(d_colind);
    hipFree(d_rowptr);
    return NULL;
  }

  /* Choose sensible defaults */
  hipsparseStatus_t cusparse_status = HIPSPARSE_STATUS_SUCCESS;
  hipsparseMatDescr_t mat_descr;
  cusparse_status = hipsparseCreateMatDescr(&mat_descr);
  if (!SUNDIALS_CUSPARSE_VERIFY(cusparse_status))
  {
    hipFree(d_rowptr); hipFree(d_colind);
    hipFree(d_values); SUNMatDestroy(A);
    return NULL;
  }

  cusparse_status = hipsparseSetMatType(mat_descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
  if (!SUNDIALS_CUSPARSE_VERIFY(cusparse_status))
  {
    hipFree(d_rowptr); hipFree(d_colind);
    hipFree(d_values); SUNMatDestroy(A);
    hipsparseDestroyMatDescr(mat_descr);
    return NULL;
  }

  cusparse_status = hipsparseSetMatIndexBase(mat_descr, HIPSPARSE_INDEX_BASE_ZERO);
  if (!SUNDIALS_CUSPARSE_VERIFY(cusparse_status))
  {
    hipFree(d_rowptr); hipFree(d_colind);
    hipFree(d_values); SUNMatDestroy(A);
    hipsparseDestroyMatDescr(mat_descr);
    return NULL;
  }

  /* Fill the content */
  SMCU_CONTENT_S(A)->M             = M;
  SMCU_CONTENT_S(A)->N             = N;
  SMCU_CONTENT_S(A)->NNZ           = NNZ;
  SMCU_CONTENT_S(A)->nblocks       = nblocks;
  SMCU_CONTENT_S(A)->blockrows     = blockrows;
  SMCU_CONTENT_S(A)->blockcols     = blockrows;
  SMCU_CONTENT_S(A)->blocknnz      = blocknnz;
  SMCU_CONTENT_S(A)->own_data      = SUNTRUE;
  SMCU_CONTENT_S(A)->sparse_type   = SUNMAT_CUSPARSE_BCSR;
  SMCU_CONTENT_S(A)->colind        = d_colind;
  SMCU_CONTENT_S(A)->rowptrs       = d_rowptr;
  SMCU_CONTENT_S(A)->data          = d_values;
  SMCU_CONTENT_S(A)->mat_descr     = mat_descr;
  SMCU_CONTENT_S(A)->cusp_handle   = cusp;
  SMCU_CONTENT_S(A)->fixed_pattern = SUNFALSE;

  return A;
}

/* ------------------------------------------------------------------
 * Implementation specific routines.
 * ------------------------------------------------------------------ */

int SUNMatrix_cuSparse_SparseType(SUNMatrix A)
{
  if (SUNMatGetID(A) == SUNMATRIX_CUSPARSE)
    return SMCU_SPARSETYPE_S(A);
  else
    return SUNMAT_ILL_INPUT;
}

int SUNMatrix_cuSparse_Rows(SUNMatrix A)
{
  if (SUNMatGetID(A) == SUNMATRIX_CUSPARSE)
    return SMCU_ROWS_S(A);
  else
    return SUNMAT_ILL_INPUT;
}

int SUNMatrix_cuSparse_Columns(SUNMatrix A)
{
  if (SUNMatGetID(A) == SUNMATRIX_CUSPARSE)
    return SMCU_COLUMNS_S(A);
  else
    return SUNMAT_ILL_INPUT;
}

int SUNMatrix_cuSparse_NNZ(SUNMatrix A)
{
  if (SUNMatGetID(A) == SUNMATRIX_CUSPARSE)
    return SMCU_NNZ_S(A);
  else
    return SUNMAT_ILL_INPUT;
}

int* SUNMatrix_cuSparse_IndexPointers(SUNMatrix A)
{
  if (SUNMatGetID(A) == SUNMATRIX_CUSPARSE)
    return SMCU_INDEXPTRS_S(A);
  else
    return NULL;
}

int* SUNMatrix_cuSparse_IndexValues(SUNMatrix A)
{
  if (SUNMatGetID(A) == SUNMATRIX_CUSPARSE)
    return SMCU_INDEXVALS_S(A);
  else
    return NULL;
}

realtype* SUNMatrix_cuSparse_Data(SUNMatrix A)
{
  if (SUNMatGetID(A) == SUNMATRIX_CUSPARSE)
    return SMCU_DATA_S(A);
  else
    return NULL;
}

int SUNMatrix_cuSparse_NumBlocks(SUNMatrix A)
{
  if (SUNMatGetID(A) == SUNMATRIX_CUSPARSE)
    return SMCU_NBLOCKS_S(A);
  else
    return SUNMAT_ILL_INPUT;
}

int SUNMatrix_cuSparse_BlockRows(SUNMatrix A)
{
  if (SUNMatGetID(A) == SUNMATRIX_CUSPARSE)
    return SMCU_BLOCKROWS_S(A);
  else
    return SUNMAT_ILL_INPUT;
}

int SUNMatrix_cuSparse_BlockColumns(SUNMatrix A)
{
  if (SUNMatGetID(A) == SUNMATRIX_CUSPARSE)
    return SMCU_BLOCKCOLS_S(A);
  else
    return SUNMAT_ILL_INPUT;
}

int SUNMatrix_cuSparse_BlockNNZ(SUNMatrix A)
{
  if (SUNMatGetID(A) == SUNMATRIX_CUSPARSE)
    return SMCU_BLOCKNNZ_S(A);
  else
    return SUNMAT_ILL_INPUT;
}

realtype* SUNMatrix_cuSparse_BlockData(SUNMatrix A, int blockidx)
{
  realtype *matdata;
  int offset;

  if (SUNMatGetID(A) != SUNMATRIX_CUSPARSE)
    return NULL;

  if (blockidx >= SMCU_NBLOCKS_S(A))
    return NULL;

  matdata = SMCU_DATA_S(A);
  offset = SMCU_BLOCKNNZ_S(A)*blockidx;

  return (&matdata[offset]);
}

hipsparseMatDescr_t SUNMatrix_cuSparse_MatDescr(SUNMatrix A)
{
  if (SUNMatGetID(A) == SUNMATRIX_CUSPARSE)
    return SMCU_MATDESCR_S(A);
  else
    return NULL;
}

int SUNMatrix_cuSparse_SetFixedPattern(SUNMatrix A, booleantype yesno)
{
  if (SUNMatGetID(A) != SUNMATRIX_CUSPARSE)
    return SUNMAT_ILL_INPUT;

  SMCU_FIXEDPATTERN_S(A) = yesno;

  return SUNMAT_SUCCESS;
}


int SUNMatrix_cuSparse_CopyToDevice(SUNMatrix dA, realtype* h_data,
                                    int* h_idxptrs, int* h_idxvals)
{
  hipError_t cuerr;
  hipStream_t stream;
  hipsparseStatus_t cusparse_status;
  int nidxvals, nidxptrs;

  if (SUNMatGetID(dA) != SUNMATRIX_CUSPARSE)
    return SUNMAT_ILL_INPUT;

  cusparse_status = hipsparseGetStream(SMCU_CUSPHANDLE_S(dA), &stream);
  if (!SUNDIALS_CUSPARSE_VERIFY(cusparse_status)) return SUNMAT_OPERATION_FAIL;

  if (h_data != NULL)
  {
    cuerr = hipMemcpyAsync(SMCU_DATA_S(dA), h_data,
                            SMCU_NNZ_S(dA)*sizeof(realtype),
                            hipMemcpyHostToDevice, stream);
    if (!SUNDIALS_CUDA_VERIFY(cuerr)) return SUNMAT_OPERATION_FAIL;
  }

  switch(SMCU_SPARSETYPE_S(dA))
  {
    case SUNMAT_CUSPARSE_CSR:
      nidxptrs = SMCU_ROWS_S(dA)+1;
      nidxvals = SMCU_NNZ_S(dA);
      break;
    case SUNMAT_CUSPARSE_BCSR:
      nidxptrs = SMCU_BLOCKROWS_S(dA)+1;
      nidxvals = SMCU_BLOCKNNZ_S(dA);
      break;
    default:
      SUNDIALS_DEBUG_PRINT("ERROR in SUNMatrix_cuSparse_CopyToDevice: unrecognized sparse type\n");
      return SUNMAT_ILL_INPUT;
  }

  if (h_idxptrs != NULL)
  {
    cuerr = hipMemcpyAsync(SMCU_INDEXPTRS_S(dA), h_idxptrs,
                            nidxptrs*sizeof(int),
                            hipMemcpyHostToDevice, stream);
    if (!SUNDIALS_CUDA_VERIFY(cuerr)) return SUNMAT_OPERATION_FAIL;
  }

  if (h_idxvals != NULL)
  {
    cuerr = hipMemcpyAsync(SMCU_INDEXVALS_S(dA), h_idxvals,
                            nidxvals*sizeof(int),
                            hipMemcpyHostToDevice, stream);
    if (!SUNDIALS_CUDA_VERIFY(cuerr)) return SUNMAT_OPERATION_FAIL;
  }

  return SUNMAT_SUCCESS;
}


int SUNMatrix_cuSparse_CopyFromDevice(SUNMatrix dA, realtype* h_data,
                                      int* h_idxptrs, int* h_idxvals)
{
  hipError_t cuerr;
  hipStream_t stream;
  hipsparseStatus_t cusparse_status;
  int nidxvals, nidxptrs;

  if (SUNMatGetID(dA) != SUNMATRIX_CUSPARSE)
    return SUNMAT_ILL_INPUT;

  cusparse_status = hipsparseGetStream(SMCU_CUSPHANDLE_S(dA), &stream);
  if (!SUNDIALS_CUSPARSE_VERIFY(cusparse_status)) return SUNMAT_OPERATION_FAIL;

  if (h_data != NULL)
  {
    cuerr = hipMemcpyAsync(h_data, SMCU_DATA_S(dA),
                            SMCU_NNZ_S(dA)*sizeof(realtype),
                            hipMemcpyDeviceToHost, stream);
    if (!SUNDIALS_CUDA_VERIFY(cuerr)) return SUNMAT_OPERATION_FAIL;
  }

  switch(SMCU_SPARSETYPE_S(dA))
  {
    case SUNMAT_CUSPARSE_CSR:
      nidxptrs = SMCU_ROWS_S(dA)+1;
      nidxvals = SMCU_NNZ_S(dA);
    case SUNMAT_CUSPARSE_BCSR:
      nidxptrs = SMCU_BLOCKROWS_S(dA)+1;
      nidxvals = SMCU_BLOCKNNZ_S(dA);
  }

  if (h_idxptrs != NULL)
  {
    cuerr = hipMemcpyAsync(h_idxptrs, SMCU_INDEXPTRS_S(dA),
                            nidxptrs*sizeof(int),
                            hipMemcpyDeviceToHost, stream);
    if (!SUNDIALS_CUDA_VERIFY(cuerr)) return SUNMAT_OPERATION_FAIL;
  }

  if (h_idxvals != NULL)
  {
    cuerr = hipMemcpyAsync(h_idxvals, SMCU_INDEXVALS_S(dA),
                            nidxvals*sizeof(int),
                            hipMemcpyDeviceToHost, stream);
    if (!SUNDIALS_CUDA_VERIFY(cuerr)) return SUNMAT_OPERATION_FAIL;
  }

  return SUNMAT_SUCCESS;
}

/*
 * -----------------------------------------------------------------
 * implementation of matrix operations
 * -----------------------------------------------------------------
 */


SUNMatrix_ID SUNMatGetID_cuSparse(SUNMatrix A)
{
  return SUNMATRIX_CUSPARSE;
}

/* Returns a new matrix allocated to have the same structure as A,
   but it does not copy any nonzeros, column vals, or row pointers. */
SUNMatrix SUNMatClone_cuSparse(SUNMatrix A)
{
  SUNMatrix B;

  switch (SMCU_SPARSETYPE_S(A))
  {
    case SUNMAT_CUSPARSE_CSR:
      B = SUNMatrix_cuSparse_NewCSR(SMCU_ROWS_S(A), SMCU_COLUMNS_S(A), SMCU_NNZ_S(A),
                                    SMCU_CUSPHANDLE_S(A));
      break;
    case SUNMAT_CUSPARSE_BCSR:
      B = SUNMatrix_cuSparse_NewBlockCSR(SMCU_NBLOCKS_S(A), SMCU_BLOCKROWS_S(A), SMCU_BLOCKCOLS_S(A),
                                         SMCU_BLOCKNNZ_S(A), SMCU_CUSPHANDLE_S(A));
      break;
    default:
      SUNDIALS_DEBUG_PRINT("ERROR in SUNMatClone_cuSparse: sparse type not recognized\n");
      B = NULL;
  }

  SMCU_FIXEDPATTERN_S(B) = SMCU_FIXEDPATTERN_S(A);

  return B;
}


/* Deallocates the SUNMatrix object and all data it owns */
void SUNMatDestroy_cuSparse(SUNMatrix A)
{
  if (A == NULL) return;

  /* free content */
  if (A->content != NULL)
  {
    if (SMCU_OWNDATA_S(A))
    {
      /* free data array */
      if (SMCU_DATA_S(A))
      {
        hipFree(SMCU_DATA_S(A));
        SMCU_DATA_S(A) = NULL;
      }

      /* free index values array */
      if (SMCU_INDEXVALS_S(A))
      {
        hipFree(SMCU_INDEXVALS_S(A));
        SMCU_INDEXVALS_S(A) = NULL;
      }

      /* free index pointers array */
      if (SMCU_INDEXPTRS_S(A))
      {
        hipFree(SMCU_INDEXPTRS_S(A));
        SMCU_INDEXPTRS_S(A) = NULL;
      }

      /* free hipsparseMatDescr_t */
      hipsparseDestroyMatDescr(SMCU_MATDESCR_S(A));
    }

    /* free content struct */
    free(A->content);
    A->content = NULL;
  }

  /* free ops and matrix */
  if (A->ops) { free(A->ops); A->ops = NULL; }
  free(A); A = NULL;

  return;
}


/* Performs A_ij = 0 */
int SUNMatZero_cuSparse(SUNMatrix A)
{
  hipError_t cuerr;
  hipStream_t stream;

  hipsparseGetStream(SMCU_CUSPHANDLE_S(A), &stream);

  /* set all data to zero */
  cuerr = hipMemsetAsync(SMCU_DATA_S(A), 0, SMCU_NNZ_S(A)*sizeof(realtype), stream);
  if (!SUNDIALS_CUDA_VERIFY(cuerr)) return SUNMAT_OPERATION_FAIL;

  /* set all rowptrs to zero unless the sparsity pattern is fixed */
  if (!SMCU_FIXEDPATTERN_S(A))
  {
    cuerr = hipMemsetAsync(SMCU_INDEXPTRS_S(A), 0,
                            (SMCU_BLOCKROWS_S(A)+1)*sizeof(int),
                            stream);
    if (!SUNDIALS_CUDA_VERIFY(cuerr)) return SUNMAT_OPERATION_FAIL;

    /* set all colind to zero */
    cuerr = hipMemsetAsync(SMCU_INDEXVALS_S(A), 0,
                            SMCU_BLOCKNNZ_S(A)*sizeof(int),
                            stream);
    if (!SUNDIALS_CUDA_VERIFY(cuerr)) return SUNMAT_OPERATION_FAIL;
  }

  return SUNMAT_SUCCESS;
}


/* Copies the nonzeros, column vals, and row pointers into dst */
int SUNMatCopy_cuSparse(SUNMatrix src, SUNMatrix dst)
{
  hipError_t cuerr;
  hipStream_t stream;

  /* Verify that src and dst are compatible */
  if (!SMCompatible_cuSparse(src, dst))
    return SUNMAT_ILL_INPUT;

  hipsparseGetStream(SMCU_CUSPHANDLE_S(src), &stream);

  /* Ensure that dst is allocated with at least as
     much memory as we have nonzeros in src */
  if (SMCU_NNZ_S(dst) < SMCU_NNZ_S(src))
  {
    SUNDIALS_DEBUG_PRINT("ERROR in SUNMatCopy_cuSparse: the destination matrix has less nonzeros than the source\n");
    return SUNMAT_ILL_INPUT;
  }

  /* Zero out dst so that copy works correctly */
  if (SUNMatZero_cuSparse(dst) != SUNMAT_SUCCESS)
  {
    SUNDIALS_DEBUG_PRINT("ERROR in SUNMatCopy_cuSparse: SUNMatZero_cuSparse failed\n");
    return SUNMAT_OPERATION_FAIL;
  }

  /* Copy the data over */
  cuerr = hipMemcpyAsync(SMCU_DATA_S(dst), SMCU_DATA_S(src),
                          SMCU_NNZ_S(src)*sizeof(*SMCU_DATA_S(src)),
                          hipMemcpyDeviceToDevice, stream);
  if (!SUNDIALS_CUDA_VERIFY(cuerr)) return SUNMAT_OPERATION_FAIL;

  /* Copy the row pointers over */
  cuerr = hipMemcpyAsync(SMCU_INDEXPTRS_S(dst), SMCU_INDEXPTRS_S(src),
                          (SMCU_BLOCKROWS_S(src)+1)*sizeof(*SMCU_INDEXPTRS_S(src)),
                          hipMemcpyDeviceToDevice, stream);
  if (!SUNDIALS_CUDA_VERIFY(cuerr)) return SUNMAT_OPERATION_FAIL;

  /* Copy the column indices over */
  cuerr = hipMemcpyAsync(SMCU_INDEXVALS_S(dst), SMCU_INDEXVALS_S(src),
                          SMCU_BLOCKNNZ_S(src)*sizeof(*SMCU_INDEXVALS_S(src)),
                          hipMemcpyDeviceToDevice, stream);
  if (!SUNDIALS_CUDA_VERIFY(cuerr)) return SUNMAT_OPERATION_FAIL;

  return SUNMAT_SUCCESS;
}


/* Performs A = cA + I. Requires the diagonal to be allocated already. */
int SUNMatScaleAddI_cuSparse(realtype c, SUNMatrix A)
{
  hipStream_t stream;
  hipsparseStatus_t cusparse_status;

  cusparse_status = hipsparseGetStream(SMCU_CUSPHANDLE_S(A), &stream);
  if (!SUNDIALS_CUSPARSE_VERIFY(cusparse_status)) return SUNMAT_OPERATION_FAIL;

  unsigned threadsPerBlock, gridSize;
  switch (SMCU_SPARSETYPE_S(A))
  {
    case SUNMAT_CUSPARSE_CSR:
      /* Choose the grid size to be the number of rows in the matrix,
        and then choose threadsPerBlock to be a multiple of the warp size
        that results in enough threads to have one per 2 columns. */
        threadsPerBlock = MAX_THREAD_PER_BLOCK(CUDA_WARP_SIZE*(SMCU_COLUMNS_S(A)/2 + CUDA_WARP_SIZE - 1)/CUDA_WARP_SIZE);
        gridSize = SMCU_ROWS_S(A);

      {
#ifdef SUNDIALS_CUDA_KERNEL_TIMING
        hipEvent_t start, stop;
        float milliseconds = 0;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);
#endif

        scaleAddIKernelCSR<realtype, int>
          <<<gridSize, threadsPerBlock, 0, stream>>>(SMCU_ROWS_S(A),
                                                     c,
                                                     SMCU_DATA_S(A),
                                                     SMCU_INDEXPTRS_S(A),
                                                     SMCU_INDEXVALS_S(A));

#ifdef SUNDIALS_CUDA_KERNEL_TIMING
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);
        fprintf(stdout, 
                "[performance] scaleAddIKernelCSR runtime (s): %22.15e\n",
                milliseconds/1000.0);
        /* scaleAddIKernelCSR reads 1 real, writes 1 real, reads 3 ints */
        fprintf(stdout,
                "[performance] scaleAddIKernelCSR effective bandwidth (GB/s): %f\n",
                (SMCU_NNZ_S(A)*(2*sizeof(realtype) + sizeof(int)) + 2*SMCU_ROWS_S(A)*sizeof(int))/milliseconds/1e6);
#endif
      }

      break;
    case SUNMAT_CUSPARSE_BCSR:
      /* Choose the grid size to be the number of blocks in the matrix,
         and then choose threadsPerBlock to be a multiple of the warp size
         that results in enough threads to have one per row of the block. */
      threadsPerBlock = MAX_THREAD_PER_BLOCK(CUDA_WARP_SIZE*(SMCU_BLOCKROWS_S(A) + CUDA_WARP_SIZE - 1)/CUDA_WARP_SIZE);
      gridSize = SMCU_NBLOCKS_S(A);

      {
#ifdef SUNDIALS_CUDA_KERNEL_TIMING
        hipEvent_t start, stop;
        float milliseconds = 0;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);
#endif

        scaleAddIKernelBCSR<realtype, int>
          <<<gridSize, threadsPerBlock, 0, stream>>>(SMCU_BLOCKROWS_S(A),
                                                     SMCU_NBLOCKS_S(A),
                                                     SMCU_BLOCKNNZ_S(A),
                                                     c,
                                                     SMCU_DATA_S(A),
                                                     SMCU_INDEXPTRS_S(A),
                                                     SMCU_INDEXVALS_S(A));

#ifdef SUNDIALS_CUDA_KERNEL_TIMING
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);
        fprintf(stdout, 
                "[performance] scaleAddIKernelBCSR runtime (s): %22.15e\n",
                milliseconds/1000.0);
        /* scaleAddIKernelBCSR reads 1 real, writes 1 real, reads 3 ints */
        fprintf(stdout,
                "[performance] scaleAddIKernelBCSR effective bandwidth (GB/s): %f\n",
                (SMCU_NNZ_S(A)*(2*sizeof(realtype) + sizeof(int)) + 2*SMCU_ROWS_S(A)*sizeof(int))/milliseconds/1e6);
#endif
      }
      break;
    default:
      SUNDIALS_DEBUG_PRINT("ERROR in SUNMatScaleAddI_cuSparse: sparse type not recognized\n");
      return SUNMAT_ILL_INPUT;
  }

#ifdef SUNDIALS_DEBUG_CUDA_LASTERROR
  hipDeviceSynchronize();
  if (!SUNDIALS_CUDA_VERIFY(hipGetLastError())) return SUNMAT_OPERATION_FAIL;
#endif

  return SUNMAT_SUCCESS;
}


/* Performs A = cA + B */
int SUNMatScaleAdd_cuSparse(realtype c, SUNMatrix A, SUNMatrix B)
{
  hipStream_t stream;
  hipsparseStatus_t cusparse_status;

  if (!SMCompatible_cuSparse(A, B))
  {
    SUNDIALS_DEBUG_PRINT("ERROR in SUNMatScaleAdd_cuSparse: SUNMatScaleAdd_cuSparse failed\n");
    return SUNMAT_ILL_INPUT;
  }

  cusparse_status = hipsparseGetStream(SMCU_CUSPHANDLE_S(A), &stream);
  if (!SUNDIALS_CUSPARSE_VERIFY(cusparse_status)) return SUNMAT_OPERATION_FAIL;

  unsigned threadsPerBlock, gridSize;
  switch (SMCU_SPARSETYPE_S(A))
  {
    case SUNMAT_CUSPARSE_CSR:
      /* Choose the grid size to be the number of rows in the matrix,
        and then choose threadsPerBlock to be a multiple of the warp size
        that results in enough threads to have one per 2 columns. */
      threadsPerBlock = MAX_THREAD_PER_BLOCK(CUDA_WARP_SIZE*(SMCU_COLUMNS_S(A)/2 + CUDA_WARP_SIZE - 1)/CUDA_WARP_SIZE);
      gridSize = SMCU_ROWS_S(A);
     
      {
#ifdef SUNDIALS_CUDA_KERNEL_TIMING
        hipEvent_t start, stop;
        float milliseconds = 0;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);
#endif

        scaleAddKernelCSR<realtype, int>
          <<<gridSize, threadsPerBlock, 0, stream>>>(SMCU_NNZ_S(A),
                                                     c,
                                                     SMCU_DATA_S(A),
                                                     SMCU_DATA_S(B));

#ifdef SUNDIALS_CUDA_KERNEL_TIMING
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);
        fprintf(stdout, 
                "[performance] scaleAddKernelCSR runtime (s): %22.15e\n",
                milliseconds/1000.0);
        /* scaleAddKernelCSR reads 2 realtype, and writes 1 realtype */
        fprintf(stdout,
                "[performance] scaleAddKernelCSR effective bandwidth (GB/s): %f\n",
                SMCU_NNZ_S(A)*sizeof(realtype)*3/milliseconds/1e6);
#endif
      }

      break;
    case SUNMAT_CUSPARSE_BCSR:
      /* Choose the grid size to be the number of blocks in the matrix,
         and then choose threadsPerBlock to be a multiple of the warp size
         that results in enough threads to have one per row of the block. */
      threadsPerBlock = MAX_THREAD_PER_BLOCK(CUDA_WARP_SIZE*(SMCU_BLOCKROWS_S(A) + CUDA_WARP_SIZE - 1)/CUDA_WARP_SIZE);
      gridSize = SMCU_NBLOCKS_S(A);

      {
#ifdef SUNDIALS_CUDA_KERNEL_TIMING
        hipEvent_t start, stop;
        float milliseconds = 0;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);
#endif

        scaleAddKernelCSR<realtype, int>
          <<<gridSize, threadsPerBlock, 0, stream>>>(SMCU_NNZ_S(A),
                                                     c,
                                                     SMCU_DATA_S(A),
                                                     SMCU_DATA_S(B));

#ifdef SUNDIALS_CUDA_KERNEL_TIMING
          hipEventRecord(stop);
          hipEventSynchronize(stop);
          hipEventElapsedTime(&milliseconds, start, stop);
          fprintf(stdout, 
                  "[performance] scaleAddKernelCSR (BCSR format) runtime (s): %22.15e\n",
                  milliseconds/1000.0);
          /* scaleAddKernelCSR reads 2 realtype, and writes 1 realtype */
          fprintf(stdout,
                  "[performance] scaleAddKernelCSR (BCSR format) effective bandwidth (GB/s): %f\n",
                  SMCU_NNZ_S(A)*sizeof(realtype)*3/milliseconds/1e6);
#endif
      }

      break;
    default:
      SUNDIALS_DEBUG_PRINT("ERROR in SUNMatScaleAdd_cuSparse: sparse type not recognized\n");
      return SUNMAT_ILL_INPUT;
  }

#ifdef SUNDIALS_DEBUG_CUDA_LASTERROR
  hipDeviceSynchronize();
  if (!SUNDIALS_CUDA_VERIFY(hipGetLastError())) return SUNMAT_OPERATION_FAIL;
#endif

  return SUNMAT_SUCCESS;
}


/* Perform y = Ax */
int SUNMatMatvec_cuSparse(SUNMatrix A, N_Vector x, N_Vector y)
{
  /* Verify that the dimensions of A, x, and y agree */
  if ( (SMCU_COLUMNS_S(A) != N_VGetLength(x)) ||
       (SMCU_ROWS_S(A) != N_VGetLength(y)) )
  {
    SUNDIALS_DEBUG_PRINT("ERROR in SUNMatMatvec_cuSparse: dimensions do not agree\n");
    return SUNMAT_ILL_INPUT;
  }

  realtype *d_xdata = N_VGetDeviceArrayPointer_Cuda(x);
  realtype *d_ydata = N_VGetDeviceArrayPointer_Cuda(y);

  if (SMCU_SPARSETYPE_S(A) == SUNMAT_CUSPARSE_CSR)
  {
    const realtype one = ONE;
    hipsparseStatus_t cusparse_status;

    /* Zero result vector */
    N_VConst(ZERO, y);

    {
#ifdef SUNDIALS_CUDA_KERNEL_TIMING
      hipEvent_t start, stop;
      float milliseconds = 0;
      hipEventCreate(&start);
      hipEventCreate(&stop);
      hipEventRecord(start);
#endif

      cusparse_status = cusparseXcsrmv(SMCU_CUSPHANDLE_S(A),
                                       HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                       SMCU_ROWS_S(A),
                                       SMCU_COLUMNS_S(A),
                                       SMCU_NNZ_S(A),
                                       &one,
                                       SMCU_MATDESCR_S(A),
                                       SMCU_DATA_S(A),
                                       SMCU_INDEXPTRS_S(A),
                                       SMCU_INDEXVALS_S(A),
                                       d_xdata,
                                       &one,
                                       d_ydata);

#ifdef SUNDIALS_CUDA_KERNEL_TIMING
          hipEventRecord(stop);
          hipEventSynchronize(stop);
          hipEventElapsedTime(&milliseconds, start, stop);
          fprintf(stdout, 
                  "[performance] cusparseXcsrmv untime (s): %22.15e\n",
                  milliseconds/1000.0);
#endif
    }

    if (!SUNDIALS_CUSPARSE_VERIFY(cusparse_status)) return SUNMAT_OPERATION_FAIL;
  }
  else if (SMCU_SPARSETYPE_S(A) == SUNMAT_CUSPARSE_BCSR)
  {
    hipStream_t stream;
    hipsparseStatus_t cusparse_status;
    unsigned gridSize, threadsPerBlock;

    cusparse_status = hipsparseGetStream(SMCU_CUSPHANDLE_S(A), &stream);
    if (!SUNDIALS_CUSPARSE_VERIFY(cusparse_status)) return SUNMAT_OPERATION_FAIL;

    /* Choose the grid size to be the number of blocks in the matrix,
       and then choose threadsPerBlock to be a multiple of the warp size
       that results in enough threads to have one per row of the block. */
    threadsPerBlock = MAX_THREAD_PER_BLOCK(CUDA_WARP_SIZE*(SMCU_BLOCKROWS_S(A) + CUDA_WARP_SIZE - 1)/CUDA_WARP_SIZE);
    gridSize = SMCU_NBLOCKS_S(A);

    {
#ifdef SUNDIALS_CUDA_KERNEL_TIMING
        hipEvent_t start, stop;
        float milliseconds = 0;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);
#endif

      matvecBCSR<realtype, int>
        <<<gridSize, threadsPerBlock, 0, stream>>>(SMCU_BLOCKROWS_S(A),
                                                   SMCU_NBLOCKS_S(A),
                                                   SMCU_BLOCKNNZ_S(A),
                                                   SMCU_DATA_S(A),
                                                   SMCU_INDEXPTRS_S(A),
                                                   SMCU_INDEXVALS_S(A),
                                                   d_xdata,
                                                   d_ydata);

#ifdef SUNDIALS_CUDA_KERNEL_TIMING
      hipEventRecord(stop);
      hipEventSynchronize(stop);
      hipEventElapsedTime(&milliseconds, start, stop);
      fprintf(stdout, 
              "[performance] matvecBCSR runtime (s): %22.15e\n",
              milliseconds/1000.0);
      fprintf(stdout,
              "[performance] matvecBCSR effective bandwidth (GB/s): %f\n",
              (SMCU_NNZ_S(A)*(sizeof(realtype)*4 + sizeof(int)) + 2*SMCU_ROWS_S(A)*sizeof(int))/milliseconds/1e6);
#endif

    }

#ifdef SUNDIALS_DEBUG_CUDA_LASTERROR
    hipDeviceSynchronize();
    if (!SUNDIALS_CUDA_VERIFY(hipGetLastError())) return SUNMAT_OPERATION_FAIL;
#endif
  }
  else
  {
    SUNDIALS_DEBUG_PRINT("ERROR in SUNMatMatvec_cuSparse: sparse type not recognized\n");
    return SUNMAT_ILL_INPUT;
  }

  return SUNMAT_SUCCESS;
}


/*
 * =================================================================
 * private functions
 * =================================================================
 */


/* -----------------------------------------------------------------
 * Function to check compatibility of two sparse SUNMatrix objects
 */
static booleantype SMCompatible_cuSparse(SUNMatrix A, SUNMatrix B)
{
  /* both matrices must be sparse */
  if ( (SUNMatGetID(A) != SUNMATRIX_CUSPARSE) ||
       (SUNMatGetID(B) != SUNMATRIX_CUSPARSE) )
    return SUNFALSE;

  /* both matrices must have the same shape and sparsity type */
  if (SMCU_ROWS_S(A) != SMCU_ROWS_S(B))
    return SUNFALSE;
  if (SMCU_COLUMNS_S(A) != SMCU_COLUMNS_S(B))
    return SUNFALSE;
  if (SMCU_SPARSETYPE_S(A) != SMCU_SPARSETYPE_S(B))
    return SUNFALSE;

  return SUNTRUE;
}

/* -----------------------------------------------------------------
 * Function to create empty SUNMatrix with ops attached and
 * the content structure allocated.
 */
SUNMatrix SUNMatrix_cuSparse_NewEmpty()
{
  /* Create an empty matrix object */
  SUNMatrix A = NULL;
  A = SUNMatNewEmpty();
  if (A == NULL)
  {
    SUNDIALS_DEBUG_PRINT("ERROR in SUNMatrix_cuSparse_NewEmpty: SUNMatNewEmpty failed\n");
    return NULL;
  }

  /* Attach operations */
  A->ops->getid     = SUNMatGetID_cuSparse;
  A->ops->clone     = SUNMatClone_cuSparse;
  A->ops->destroy   = SUNMatDestroy_cuSparse;
  A->ops->zero      = SUNMatZero_cuSparse;
  A->ops->copy      = SUNMatCopy_cuSparse;
  A->ops->scaleadd  = SUNMatScaleAdd_cuSparse;
  A->ops->scaleaddi = SUNMatScaleAddI_cuSparse;
  A->ops->matvec    = SUNMatMatvec_cuSparse;

  /* Create content */
  SUNMatrix_Content_cuSparse content = NULL;
  content = (SUNMatrix_Content_cuSparse) malloc(sizeof *content);
  if (content == NULL)
  {
    SUNDIALS_DEBUG_PRINT("ERROR in SUNMatrix_cuSparse_NewEmpty: failed to malloc content\n");
    SUNMatDestroy(A);
    return NULL;
  }

  /* Attach content */
  A->content = content;

  return A;
}
