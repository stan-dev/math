#include "hip/hip_runtime.h"
/* -----------------------------------------------------------------
 * Programmer(s): Slaven Peles, and Cody J. Balos @ LLNL
 * -----------------------------------------------------------------
 * SUNDIALS Copyright Start
 * Copyright (c) 2002-2021, Lawrence Livermore National Security
 * and Southern Methodist University.
 * All rights reserved.
 *
 * See the top-level LICENSE and NOTICE files for details.
 *
 * SPDX-License-Identifier: BSD-3-Clause
 * SUNDIALS Copyright End
 * -----------------------------------------------------------------
 * This is the implementation file for a CUDA implementation
 * of the NVECTOR package.
 * -----------------------------------------------------------------*/

#include <cstdio>
#include <cstdlib>
#include <cmath>
#include <limits>

#include <nvector/nvector_cuda.h>
#include "VectorArrayKernels.cuh"
#include "VectorKernels.cuh"

#include "sundials_cuda.h"
#include "sundials_debug.h"

#define ZERO RCONST(0.0)
#define HALF RCONST(0.5)

extern "C" {

using namespace sundials;
using namespace sundials::nvector_cuda;

/*
 * Macro definitions
 */

#define NVEC_CUDA_CONTENT(x)  ((N_VectorContent_Cuda)(x->content))
#define NVEC_CUDA_PRIVATE(x)  ((N_PrivateVectorContent_Cuda)(NVEC_CUDA_CONTENT(x)->priv))
#define NVEC_CUDA_MEMSIZE(x)  (NVEC_CUDA_CONTENT(x)->length * sizeof(realtype))
#define NVEC_CUDA_MEMHELP(x)  (NVEC_CUDA_CONTENT(x)->mem_helper)
#define NVEC_CUDA_HDATAp(x)   ((realtype*) NVEC_CUDA_CONTENT(x)->host_data->ptr)
#define NVEC_CUDA_DDATAp(x)   ((realtype*) NVEC_CUDA_CONTENT(x)->device_data->ptr)
#define NVEC_CUDA_HBUFFERp(x) ((realtype*) NVEC_CUDA_PRIVATE(x)->reduce_buffer_host->ptr)
#define NVEC_CUDA_DBUFFERp(x) ((realtype*) NVEC_CUDA_PRIVATE(x)->reduce_buffer_dev->ptr)
#define NVEC_CUDA_STREAM(x)   (NVEC_CUDA_CONTENT(x)->stream_exec_policy->stream())


/*
 * Private structure definition
 */

struct _N_PrivateVectorContent_Cuda
{
  booleantype     use_managed_mem;               /* indicates if the data pointers and buffer pointers are managed memory */
  size_t          reduce_buffer_allocated_bytes; /* current size of the reduction buffer */
  SUNMemory       reduce_buffer_dev;             /* device buffer used for reductions */
  SUNMemory       reduce_buffer_host;            /* host buffer used for reductions */
};

typedef struct _N_PrivateVectorContent_Cuda *N_PrivateVectorContent_Cuda;

/*
 * Private function definitions
 */

static int AllocateData(N_Vector v);
static int InitializeReductionBuffer(N_Vector v, const realtype value);
static void FreeReductionBuffer(N_Vector v);
static int CopyReductionBufferFromDevice(N_Vector v, size_t n = 1);
static int GetKernelParameters(N_Vector v, booleantype reduction, size_t& grid, size_t& block,
                               size_t& shMemSize, hipStream_t& stream, size_t n = 0);
static void PostKernelLaunch();

/*
 * Private functions needed for N_VMakeWithManagedAllocator_Cuda
 * backwards compatibility.
 */

/* DEPRECATION NOTICE: The 4 functions below can be removed once
   N_VMakeWithManagedAllocator_Cuda (deprecated) is removed in the
   next major release. The UserAllocHelper struct can also be removed. */

/* Struct that we use to pack up the user
   provided alloc and free functions. */
typedef struct _UserAllocHelper
{
  void*  (*userallocfn)(size_t);
  void   (*userfreefn)(void*);
} UserAllocHelper;

static int UserAlloc(SUNMemoryHelper helper, SUNMemory* memptr,
                     size_t memsize, SUNMemoryType mem_type)
{
  UserAllocHelper* ua = (UserAllocHelper*) helper->content;
  SUNMemory mem = SUNMemoryNewEmpty();

  mem->type = SUNMEMTYPE_UVM;
  mem->ptr  = ua->userallocfn(memsize);
  mem->own  = SUNTRUE;
  if (mem->ptr == NULL)
  {
    SUNDIALS_DEBUG_PRINT("ERROR in UserAlloc: user provided alloc failed\n");
    free(mem);
    return(-1);
  }

  *memptr = mem;
  return(0);
}

static int UserDealloc(SUNMemoryHelper helper, SUNMemory mem)
{
  UserAllocHelper* ua = (UserAllocHelper*) helper->content;
  if (mem->own)
  {
    ua->userfreefn(mem->ptr);
    mem->ptr = NULL;
  }
  free(mem);
  return(0);
}

static SUNMemoryHelper HelperClone(SUNMemoryHelper helper)
{
  UserAllocHelper* uaclone;
  UserAllocHelper* ua = (UserAllocHelper*) helper->content;
  SUNMemoryHelper hclone = SUNMemoryHelper_NewEmpty();

  SUNMemoryHelper_CopyOps(helper, hclone);

  uaclone = (UserAllocHelper*) malloc(sizeof(UserAllocHelper));
  uaclone->userallocfn = ua->userallocfn;
  uaclone->userfreefn  = ua->userfreefn;

  hclone->content = uaclone;

  return(hclone);
}

static int HelperDestroy(SUNMemoryHelper helper)
{
  free(helper->content);
  helper->content = NULL;
  free(helper->ops);
  free(helper);
  return(0);
}

N_Vector N_VNewEmpty_Cuda()
{
  N_Vector v;

  /* Create vector */
  v = NULL;
  v = N_VNewEmpty();
  if (v == NULL) return(NULL);

  /* Attach operations */

  /* constructors, destructors, and utility operations */
  v->ops->nvgetvectorid           = N_VGetVectorID_Cuda;
  v->ops->nvclone                 = N_VClone_Cuda;
  v->ops->nvcloneempty            = N_VCloneEmpty_Cuda;
  v->ops->nvdestroy               = N_VDestroy_Cuda;
  v->ops->nvspace                 = N_VSpace_Cuda;
  v->ops->nvgetlength             = N_VGetLength_Cuda;
  v->ops->nvgetarraypointer       = N_VGetHostArrayPointer_Cuda;
  v->ops->nvgetdevicearraypointer = N_VGetDeviceArrayPointer_Cuda;
  v->ops->nvsetarraypointer       = N_VSetHostArrayPointer_Cuda;

  /* standard vector operations */
  v->ops->nvlinearsum    = N_VLinearSum_Cuda;
  v->ops->nvconst        = N_VConst_Cuda;
  v->ops->nvprod         = N_VProd_Cuda;
  v->ops->nvdiv          = N_VDiv_Cuda;
  v->ops->nvscale        = N_VScale_Cuda;
  v->ops->nvabs          = N_VAbs_Cuda;
  v->ops->nvinv          = N_VInv_Cuda;
  v->ops->nvaddconst     = N_VAddConst_Cuda;
  v->ops->nvdotprod      = N_VDotProd_Cuda;
  v->ops->nvmaxnorm      = N_VMaxNorm_Cuda;
  v->ops->nvmin          = N_VMin_Cuda;
  v->ops->nvl1norm       = N_VL1Norm_Cuda;
  v->ops->nvinvtest      = N_VInvTest_Cuda;
  v->ops->nvconstrmask   = N_VConstrMask_Cuda;
  v->ops->nvminquotient  = N_VMinQuotient_Cuda;
  v->ops->nvwrmsnormmask = N_VWrmsNormMask_Cuda;
  v->ops->nvwrmsnorm     = N_VWrmsNorm_Cuda;
  v->ops->nvwl2norm      = N_VWL2Norm_Cuda;
  v->ops->nvcompare      = N_VCompare_Cuda;

  /* fused and vector array operations are disabled (NULL) by default */

  /* local reduction operations */
  v->ops->nvdotprodlocal     = N_VDotProd_Cuda;
  v->ops->nvmaxnormlocal     = N_VMaxNorm_Cuda;
  v->ops->nvminlocal         = N_VMin_Cuda;
  v->ops->nvl1normlocal      = N_VL1Norm_Cuda;
  v->ops->nvinvtestlocal     = N_VInvTest_Cuda;
  v->ops->nvconstrmasklocal  = N_VConstrMask_Cuda;
  v->ops->nvminquotientlocal = N_VMinQuotient_Cuda;
  v->ops->nvwsqrsumlocal     = N_VWSqrSumLocal_Cuda;
  v->ops->nvwsqrsummasklocal = N_VWSqrSumMaskLocal_Cuda;

  /* XBraid interface operations */
  v->ops->nvbufsize   = N_VBufSize_Cuda;
  v->ops->nvbufpack   = N_VBufPack_Cuda;
  v->ops->nvbufunpack = N_VBufUnpack_Cuda;

  /* print operation for debugging */
  v->ops->nvprint     = N_VPrint_Cuda;
  v->ops->nvprintfile = N_VPrintFile_Cuda;

  /* Create content */

  v->content = (N_VectorContent_Cuda) malloc(sizeof(_N_VectorContent_Cuda));
  if (v->content == NULL)
  {
    N_VDestroy(v);
    return(NULL);
  }

  NVEC_CUDA_CONTENT(v)->priv = malloc(sizeof(_N_PrivateVectorContent_Cuda));
  if (NVEC_CUDA_CONTENT(v)->priv == NULL)
  {
    N_VDestroy(v);
    return(NULL);
  }

  NVEC_CUDA_CONTENT(v)->length                        = 0;
  NVEC_CUDA_CONTENT(v)->host_data                     = NULL;
  NVEC_CUDA_CONTENT(v)->device_data                   = NULL;
  NVEC_CUDA_CONTENT(v)->stream_exec_policy            = NULL;
  NVEC_CUDA_CONTENT(v)->reduce_exec_policy            = NULL;
  NVEC_CUDA_CONTENT(v)->mem_helper                    = NULL;
  NVEC_CUDA_CONTENT(v)->own_helper                    = SUNFALSE;
  NVEC_CUDA_CONTENT(v)->own_exec                      = SUNTRUE;
  NVEC_CUDA_PRIVATE(v)->use_managed_mem               = SUNFALSE;
  NVEC_CUDA_PRIVATE(v)->reduce_buffer_dev             = NULL;
  NVEC_CUDA_PRIVATE(v)->reduce_buffer_host            = NULL;
  NVEC_CUDA_PRIVATE(v)->reduce_buffer_allocated_bytes = 0;

  return(v);
}

N_Vector N_VNew_Cuda(sunindextype length)
{
  N_Vector v;

  v = NULL;
  v = N_VNewEmpty_Cuda();
  if (v == NULL) return(NULL);

  NVEC_CUDA_CONTENT(v)->length                        = length;
  NVEC_CUDA_CONTENT(v)->host_data                     = NULL;
  NVEC_CUDA_CONTENT(v)->device_data                   = NULL;
  NVEC_CUDA_CONTENT(v)->mem_helper                    = SUNMemoryHelper_Cuda();
  NVEC_CUDA_CONTENT(v)->stream_exec_policy            = new CudaThreadDirectExecPolicy(256);
  NVEC_CUDA_CONTENT(v)->reduce_exec_policy            = new CudaBlockReduceExecPolicy(256);
  NVEC_CUDA_CONTENT(v)->own_helper                    = SUNTRUE;
  NVEC_CUDA_CONTENT(v)->own_exec                      = SUNTRUE;
  NVEC_CUDA_PRIVATE(v)->use_managed_mem               = SUNFALSE;
  NVEC_CUDA_PRIVATE(v)->reduce_buffer_dev             = NULL;
  NVEC_CUDA_PRIVATE(v)->reduce_buffer_host            = NULL;
  NVEC_CUDA_PRIVATE(v)->reduce_buffer_allocated_bytes = 0;

  if (NVEC_CUDA_MEMHELP(v) == NULL)
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VNew_Cuda: memory helper is NULL\n");
    N_VDestroy(v);
    return(NULL);
  }

  if (AllocateData(v))
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VNew_Cuda: AllocateData returned nonzero\n");
    N_VDestroy(v);
    return(NULL);
  }

  return(v);
}

N_Vector N_VNewWithMemHelp_Cuda(sunindextype length, booleantype use_managed_mem, SUNMemoryHelper helper)
{
  N_Vector v;

  if (helper == NULL)
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VNewWithMemHelp_Cuda: helper is NULL\n");
    return(NULL);
  }

  if (!SUNMemoryHelper_ImplementsRequiredOps(helper))
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VNewWithMemHelp_Cuda: helper doesn't implement all required ops\n");
    return(NULL);
  }

  v = NULL;
  v = N_VNewEmpty_Cuda();
  if (v == NULL) return(NULL);

  NVEC_CUDA_CONTENT(v)->length                        = length;
  NVEC_CUDA_CONTENT(v)->host_data                     = NULL;
  NVEC_CUDA_CONTENT(v)->device_data                   = NULL;
  NVEC_CUDA_CONTENT(v)->mem_helper                    = helper;
  NVEC_CUDA_CONTENT(v)->stream_exec_policy            = new CudaThreadDirectExecPolicy(256);
  NVEC_CUDA_CONTENT(v)->reduce_exec_policy            = new CudaBlockReduceExecPolicy(256);
  NVEC_CUDA_CONTENT(v)->own_helper                    = SUNFALSE;
  NVEC_CUDA_CONTENT(v)->own_exec                      = SUNTRUE;
  NVEC_CUDA_PRIVATE(v)->use_managed_mem               = use_managed_mem;
  NVEC_CUDA_PRIVATE(v)->reduce_buffer_dev             = NULL;
  NVEC_CUDA_PRIVATE(v)->reduce_buffer_host            = NULL;
  NVEC_CUDA_PRIVATE(v)->reduce_buffer_allocated_bytes = 0;

  if (AllocateData(v))
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VNewWithMemHelp_Cuda: AllocateData returned nonzero\n");
    N_VDestroy(v);
    return(NULL);
  }

  return(v);
}

N_Vector N_VNewManaged_Cuda(sunindextype length)
{
  N_Vector v;

  v = NULL;
  v = N_VNewEmpty_Cuda();
  if (v == NULL) return(NULL);

  NVEC_CUDA_CONTENT(v)->length                        = length;
  NVEC_CUDA_CONTENT(v)->host_data                     = NULL;
  NVEC_CUDA_CONTENT(v)->device_data                   = NULL;
  NVEC_CUDA_CONTENT(v)->stream_exec_policy            = new CudaThreadDirectExecPolicy(256);
  NVEC_CUDA_CONTENT(v)->reduce_exec_policy            = new CudaBlockReduceExecPolicy(256);
  NVEC_CUDA_CONTENT(v)->mem_helper                    = SUNMemoryHelper_Cuda();
  NVEC_CUDA_CONTENT(v)->own_helper                    = SUNTRUE;
  NVEC_CUDA_CONTENT(v)->own_exec                      = SUNTRUE;
  NVEC_CUDA_PRIVATE(v)->use_managed_mem               = SUNTRUE;
  NVEC_CUDA_PRIVATE(v)->reduce_buffer_dev             = NULL;
  NVEC_CUDA_PRIVATE(v)->reduce_buffer_host            = NULL;
  NVEC_CUDA_PRIVATE(v)->reduce_buffer_allocated_bytes = 0;

  if (NVEC_CUDA_MEMHELP(v) == NULL)
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VNewManaged_Cuda: memory helper is NULL\n");
    N_VDestroy(v);
    return(NULL);
  }

  if (AllocateData(v))
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VNewManaged_Cuda: AllocateData returned nonzero\n");
    N_VDestroy(v);
    return(NULL);
  }

  return(v);
}

N_Vector N_VMake_Cuda(sunindextype length, realtype *h_vdata, realtype *d_vdata)
{
  N_Vector v;

  if (h_vdata == NULL || d_vdata == NULL) return(NULL);

  v = NULL;
  v = N_VNewEmpty_Cuda();
  if (v == NULL) return(NULL);

  NVEC_CUDA_CONTENT(v)->length                        = length;
  NVEC_CUDA_CONTENT(v)->host_data                     = SUNMemoryHelper_Wrap(h_vdata, SUNMEMTYPE_HOST);
  NVEC_CUDA_CONTENT(v)->device_data                   = SUNMemoryHelper_Wrap(d_vdata, SUNMEMTYPE_DEVICE);
  NVEC_CUDA_CONTENT(v)->stream_exec_policy            = new CudaThreadDirectExecPolicy(256);
  NVEC_CUDA_CONTENT(v)->reduce_exec_policy            = new CudaBlockReduceExecPolicy(256);
  NVEC_CUDA_CONTENT(v)->mem_helper                    = SUNMemoryHelper_Cuda();
  NVEC_CUDA_CONTENT(v)->own_helper                    = SUNTRUE;
  NVEC_CUDA_CONTENT(v)->own_exec                      = SUNTRUE;
  NVEC_CUDA_PRIVATE(v)->use_managed_mem               = SUNFALSE;
  NVEC_CUDA_PRIVATE(v)->reduce_buffer_dev             = NULL;
  NVEC_CUDA_PRIVATE(v)->reduce_buffer_host            = NULL;
  NVEC_CUDA_PRIVATE(v)->reduce_buffer_allocated_bytes = 0;

  if (NVEC_CUDA_MEMHELP(v) == NULL)
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VMake_Cuda: memory helper is NULL\n");
    N_VDestroy(v);
    return(NULL);
  }

  if (NVEC_CUDA_CONTENT(v)->device_data == NULL ||
      NVEC_CUDA_CONTENT(v)->host_data == NULL)
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VMake_Cuda: SUNMemoryHelper_Wrap returned NULL\n");
    N_VDestroy(v);
    return(NULL);
  }

  return(v);
}

N_Vector N_VMakeManaged_Cuda(sunindextype length, realtype *vdata)
{
  N_Vector v;

  if (vdata == NULL) return(NULL);

  v = NULL;
  v = N_VNewEmpty_Cuda();
  if (v == NULL) return(NULL);

  NVEC_CUDA_CONTENT(v)->length                        = length;
  NVEC_CUDA_CONTENT(v)->host_data                     = SUNMemoryHelper_Wrap(vdata, SUNMEMTYPE_UVM);
  NVEC_CUDA_CONTENT(v)->device_data                   = SUNMemoryHelper_Alias(NVEC_CUDA_CONTENT(v)->host_data);
  NVEC_CUDA_CONTENT(v)->stream_exec_policy            = new CudaThreadDirectExecPolicy(256);
  NVEC_CUDA_CONTENT(v)->reduce_exec_policy            = new CudaBlockReduceExecPolicy(256);
  NVEC_CUDA_CONTENT(v)->mem_helper                    = SUNMemoryHelper_Cuda();
  NVEC_CUDA_CONTENT(v)->own_helper                    = SUNTRUE;
  NVEC_CUDA_CONTENT(v)->own_exec                      = SUNTRUE;
  NVEC_CUDA_PRIVATE(v)->use_managed_mem               = SUNTRUE;
  NVEC_CUDA_PRIVATE(v)->reduce_buffer_dev             = NULL;
  NVEC_CUDA_PRIVATE(v)->reduce_buffer_host            = NULL;
  NVEC_CUDA_PRIVATE(v)->reduce_buffer_allocated_bytes = 0;

  if (NVEC_CUDA_MEMHELP(v) == NULL)
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VMakeManaged_Cuda: memory helper is NULL\n");
    N_VDestroy(v);
    return(NULL);
  }

  if (NVEC_CUDA_CONTENT(v)->device_data == NULL ||
      NVEC_CUDA_CONTENT(v)->host_data == NULL)
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VMakeManaged_Cuda: SUNMemoryHelper_Wrap returned NULL\n");
    N_VDestroy(v);
    return(NULL);
  }

  return(v);
}

N_Vector N_VMakeWithManagedAllocator_Cuda(sunindextype length,
                                          void* (*allocfn)(size_t),
                                          void (*freefn)(void*))
{
  UserAllocHelper* ua;
  N_Vector v;

  v = NULL;
  v = N_VNewEmpty_Cuda();
  if (v == NULL) return(NULL);

  NVEC_CUDA_CONTENT(v)->length                        = length;
  NVEC_CUDA_CONTENT(v)->host_data                     = NULL;
  NVEC_CUDA_CONTENT(v)->device_data                   = NULL;
  NVEC_CUDA_CONTENT(v)->stream_exec_policy            = new CudaThreadDirectExecPolicy(256);
  NVEC_CUDA_CONTENT(v)->reduce_exec_policy            = new CudaBlockReduceExecPolicy(256);
  NVEC_CUDA_CONTENT(v)->mem_helper                    = SUNMemoryHelper_Cuda();
  NVEC_CUDA_CONTENT(v)->own_helper                    = SUNTRUE;
  NVEC_CUDA_CONTENT(v)->own_exec                      = SUNTRUE;
  NVEC_CUDA_PRIVATE(v)->use_managed_mem               = SUNTRUE;
  NVEC_CUDA_PRIVATE(v)->reduce_buffer_dev             = NULL;
  NVEC_CUDA_PRIVATE(v)->reduce_buffer_host            = NULL;
  NVEC_CUDA_PRIVATE(v)->reduce_buffer_allocated_bytes = 0;

  if (NVEC_CUDA_MEMHELP(v) == NULL)
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VMakeWithManagedAllocator_Cuda: memory helper is NULL\n");
    N_VDestroy(v);
    return(NULL);
  }

  ua = (UserAllocHelper*) malloc(sizeof(UserAllocHelper));
  ua->userallocfn                    = allocfn;
  ua->userfreefn                     = freefn;
  NVEC_CUDA_MEMHELP(v)->content      = (void*) ua;
  NVEC_CUDA_MEMHELP(v)->ops->alloc   = UserAlloc;
  NVEC_CUDA_MEMHELP(v)->ops->dealloc = UserDealloc;
  NVEC_CUDA_MEMHELP(v)->ops->clone   = HelperClone;
  NVEC_CUDA_MEMHELP(v)->ops->destroy = HelperDestroy;

  if (AllocateData(v))
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VMakeWithManagedAllocator_Cuda: AllocateData returned nonzero\n");
    N_VDestroy(v);
    return(NULL);
  }

  return(v);
}

/* ----------------------------------------------------------------------------
 * Set pointer to the raw host data. Does not free the existing pointer.
 */

void N_VSetHostArrayPointer_Cuda(realtype* h_vdata, N_Vector v)
{
  if (N_VIsManagedMemory_Cuda(v))
  {
    if (NVEC_CUDA_CONTENT(v)->host_data)
    {
      NVEC_CUDA_CONTENT(v)->host_data->ptr = (void*) h_vdata;
      NVEC_CUDA_CONTENT(v)->device_data->ptr = (void*) h_vdata;
    }
    else
    {
      NVEC_CUDA_CONTENT(v)->host_data = SUNMemoryHelper_Wrap((void*) h_vdata, SUNMEMTYPE_UVM);
      NVEC_CUDA_CONTENT(v)->device_data = SUNMemoryHelper_Alias(NVEC_CUDA_CONTENT(v)->host_data);
    }
  }
  else
  {
    if (NVEC_CUDA_CONTENT(v)->host_data)
    {
      NVEC_CUDA_CONTENT(v)->host_data->ptr = (void*) h_vdata;
    }
    else
    {
      NVEC_CUDA_CONTENT(v)->host_data = SUNMemoryHelper_Wrap((void*) h_vdata, SUNMEMTYPE_HOST);
    }
  }
}

/* ----------------------------------------------------------------------------
 * Set pointer to the raw device data
 */

void N_VSetDeviceArrayPointer_Cuda(realtype* d_vdata, N_Vector v)
{
  if (N_VIsManagedMemory_Cuda(v))
  {
    if (NVEC_CUDA_CONTENT(v)->device_data)
    {
      NVEC_CUDA_CONTENT(v)->device_data->ptr = (void*) d_vdata;
      NVEC_CUDA_CONTENT(v)->host_data->ptr = (void*) d_vdata;
    }
    else
    {
      NVEC_CUDA_CONTENT(v)->device_data = SUNMemoryHelper_Wrap((void*) d_vdata, SUNMEMTYPE_UVM);
      NVEC_CUDA_CONTENT(v)->host_data = SUNMemoryHelper_Alias(NVEC_CUDA_CONTENT(v)->device_data);
    }
  }
  else
  {
    if (NVEC_CUDA_CONTENT(v)->device_data)
    {
      NVEC_CUDA_CONTENT(v)->device_data->ptr = (void*) d_vdata;
    }
    else
    {
      NVEC_CUDA_CONTENT(v)->device_data = SUNMemoryHelper_Wrap((void*) d_vdata, SUNMEMTYPE_DEVICE);
    }
  }
}

/* ----------------------------------------------------------------------------
 * Return a flag indicating if the memory for the vector data is managed
 */
booleantype N_VIsManagedMemory_Cuda(N_Vector x)
{
  return NVEC_CUDA_PRIVATE(x)->use_managed_mem;
}

int N_VSetKernelExecPolicy_Cuda(N_Vector x,
                                SUNCudaExecPolicy* stream_exec_policy,
                                SUNCudaExecPolicy* reduce_exec_policy)
{
  if (x == NULL || stream_exec_policy == NULL || reduce_exec_policy == NULL)
    return(-1);

  if (NVEC_CUDA_CONTENT(x)->own_exec)
  {
    delete NVEC_CUDA_CONTENT(x)->stream_exec_policy;
    delete NVEC_CUDA_CONTENT(x)->reduce_exec_policy;
  }

  NVEC_CUDA_CONTENT(x)->stream_exec_policy = stream_exec_policy;
  NVEC_CUDA_CONTENT(x)->reduce_exec_policy = reduce_exec_policy;
  NVEC_CUDA_CONTENT(x)->own_exec = SUNFALSE;

  return(0);
}

/*
 * ----------------------------------------------------------------------------
 * DEPRECATED: will be removed in SUNDIALS v6.
 * Sets the hipStream_t to use for execution of the CUDA kernels.
 */
void N_VSetCudaStream_Cuda(N_Vector x, hipStream_t *stream)
{
  const CudaExecPolicy* xs = NVEC_CUDA_CONTENT(x)->stream_exec_policy;
  const CudaExecPolicy* xr = NVEC_CUDA_CONTENT(x)->reduce_exec_policy;
  CudaThreadDirectExecPolicy* s =
    new CudaThreadDirectExecPolicy(xs->blockSize(), *stream);
  CudaBlockReduceExecPolicy* r =
    new CudaBlockReduceExecPolicy(xr->blockSize(), xr->gridSize(), *stream);
  N_VSetKernelExecPolicy_Cuda(x, s, r);
  NVEC_CUDA_CONTENT(x)->own_exec = SUNTRUE;
}

/* ----------------------------------------------------------------------------
 * Copy vector data to the device
 */

void N_VCopyToDevice_Cuda(N_Vector x)
{
  int copy_fail;

  copy_fail = SUNMemoryHelper_CopyAsync(NVEC_CUDA_MEMHELP(x),
                                        NVEC_CUDA_CONTENT(x)->device_data,
                                        NVEC_CUDA_CONTENT(x)->host_data,
                                        NVEC_CUDA_MEMSIZE(x),
                                        (void*) NVEC_CUDA_STREAM(x));

  if (copy_fail)
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VCopyToDevice_Cuda: SUNMemoryHelper_CopyAsync returned nonzero\n");
  }

  /* we synchronize with respect to the host, but only in this stream */
  SUNDIALS_CUDA_VERIFY(hipStreamSynchronize(*NVEC_CUDA_STREAM(x)));
}

/* ----------------------------------------------------------------------------
 * Copy vector data from the device to the host
 */

void N_VCopyFromDevice_Cuda(N_Vector x)
{
  int copy_fail;

  copy_fail = SUNMemoryHelper_CopyAsync(NVEC_CUDA_MEMHELP(x),
                                        NVEC_CUDA_CONTENT(x)->host_data,
                                        NVEC_CUDA_CONTENT(x)->device_data,
                                        NVEC_CUDA_MEMSIZE(x),
                                        (void*) NVEC_CUDA_STREAM(x));

  if (copy_fail)
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VCopyFromDevice_Cuda: SUNMemoryHelper_CopyAsync returned nonzero\n");
  }

  /* we synchronize with respect to the host, but only in this stream */
  SUNDIALS_CUDA_VERIFY(hipStreamSynchronize(*NVEC_CUDA_STREAM(x)));
}

/* ----------------------------------------------------------------------------
 * Function to print the a CUDA-based vector to stdout
 */

void N_VPrint_Cuda(N_Vector x)
{
  N_VPrintFile_Cuda(x, stdout);
}

/* ----------------------------------------------------------------------------
 * Function to print the a CUDA-based vector to outfile
 */

void N_VPrintFile_Cuda(N_Vector x, FILE *outfile)
{
  sunindextype i;

  for (i = 0; i < NVEC_CUDA_CONTENT(x)->length; i++) {
#if defined(SUNDIALS_EXTENDED_PRECISION)
    fprintf(outfile, "%35.32Lg\n", NVEC_CUDA_HDATAp(x)[i]);
#elif defined(SUNDIALS_DOUBLE_PRECISION)
    fprintf(outfile, "%19.16g\n", NVEC_CUDA_HDATAp(x)[i]);
#else
    fprintf(outfile, "%11.8g\n", NVEC_CUDA_HDATAp(x)[i]);
#endif
  }
  fprintf(outfile, "\n");

  return;
}


/*
 * -----------------------------------------------------------------
 * implementation of vector operations
 * -----------------------------------------------------------------
 */

N_Vector N_VCloneEmpty_Cuda(N_Vector w)
{
  N_Vector v;

  if (w == NULL) return(NULL);

  /* Create vector */
  v = NULL;
  v = N_VNewEmpty_Cuda();
  if (v == NULL) return(NULL);

  /* Attach operations */
  if (N_VCopyOps(w, v)) { N_VDestroy(v); return(NULL); }

  /* Set content */
  NVEC_CUDA_CONTENT(v)->length                        = NVEC_CUDA_CONTENT(w)->length;
  NVEC_CUDA_CONTENT(v)->host_data                     = NULL;
  NVEC_CUDA_CONTENT(v)->device_data                   = NULL;
  NVEC_CUDA_CONTENT(v)->mem_helper                    = NULL;
  NVEC_CUDA_CONTENT(v)->own_exec                      = SUNTRUE;
  NVEC_CUDA_PRIVATE(v)->use_managed_mem               = NVEC_CUDA_PRIVATE(w)->use_managed_mem;
  NVEC_CUDA_PRIVATE(v)->reduce_buffer_dev             = NULL;
  NVEC_CUDA_PRIVATE(v)->reduce_buffer_host            = NULL;
  NVEC_CUDA_PRIVATE(v)->reduce_buffer_allocated_bytes = 0;

  return(v);
}

N_Vector N_VClone_Cuda(N_Vector w)
{
  N_Vector v;

  v = NULL;
  v = N_VCloneEmpty_Cuda(w);
  if (v == NULL) return(NULL);

  NVEC_CUDA_MEMHELP(v) = SUNMemoryHelper_Clone(NVEC_CUDA_MEMHELP(w));
  NVEC_CUDA_CONTENT(v)->own_helper = SUNTRUE;
  NVEC_CUDA_CONTENT(v)->stream_exec_policy = NVEC_CUDA_CONTENT(w)->stream_exec_policy->clone();
  NVEC_CUDA_CONTENT(v)->reduce_exec_policy = NVEC_CUDA_CONTENT(w)->reduce_exec_policy->clone();

  if (NVEC_CUDA_MEMHELP(v) == NULL)
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VClone_Cuda: SUNMemoryHelper_Clone returned NULL\n");
    N_VDestroy(v);
    return(NULL);
  }

  if (AllocateData(v))
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VClone_Cuda: AllocateData returned nonzero\n");
    N_VDestroy(v);
    return(NULL);
  }

  return(v);
}

void N_VDestroy_Cuda(N_Vector v)
{
  N_VectorContent_Cuda vc;
  N_PrivateVectorContent_Cuda vcp;

  if (v == NULL) return;

  /* free ops structure */
  if (v->ops != NULL)
  {
    free(v->ops);
    v->ops = NULL;
  }

  /* extract content */
  vc = NVEC_CUDA_CONTENT(v);
  if (vc == NULL)
  {
    free(v);
    v = NULL;
    return;
  }

  /* free private content */
  vcp = (N_PrivateVectorContent_Cuda) vc->priv;
  if (vcp != NULL)
  {
    /* free items in private content */
    FreeReductionBuffer(v);
    free(vcp);
    vc->priv = NULL;
  }

  /* free items in content */
  if (vc->own_exec)
  {
    delete vc->stream_exec_policy;
    vc->stream_exec_policy = NULL;
    delete vc->reduce_exec_policy;
    vc->reduce_exec_policy = NULL;
  }

  if (NVEC_CUDA_MEMHELP(v))
  {
    SUNMemoryHelper_Dealloc(NVEC_CUDA_MEMHELP(v), vc->host_data);
    vc->host_data = NULL;
    SUNMemoryHelper_Dealloc(NVEC_CUDA_MEMHELP(v), vc->device_data);
    vc->device_data = NULL;
    if (vc->own_helper) SUNMemoryHelper_Destroy(vc->mem_helper);
    vc->mem_helper = NULL;
  }

  /* free content struct */
  free(vc);

  /* free vector */
  free(v);

  return;
}

void N_VSpace_Cuda(N_Vector X, sunindextype *lrw, sunindextype *liw)
{
  *lrw = NVEC_CUDA_CONTENT(X)->length;
  *liw = 2;
}

void N_VConst_Cuda(realtype a, N_Vector X)
{
  size_t grid, block, shMemSize;
  hipStream_t stream;

  GetKernelParameters(X, false, grid, block, shMemSize, stream);
  setConstKernel<<<grid, block, shMemSize, stream>>>
  (
    a,
    NVEC_CUDA_DDATAp(X),
    NVEC_CUDA_CONTENT(X)->length
  );
  PostKernelLaunch();
}

void N_VLinearSum_Cuda(realtype a, N_Vector X, realtype b, N_Vector Y, N_Vector Z)
{
  size_t grid, block, shMemSize;
  hipStream_t stream;

  GetKernelParameters(X, false, grid, block, shMemSize, stream);
  linearSumKernel<<<grid, block, shMemSize, stream>>>
  (
    a,
    NVEC_CUDA_DDATAp(X),
    b,
    NVEC_CUDA_DDATAp(Y),
    NVEC_CUDA_DDATAp(Z),
    NVEC_CUDA_CONTENT(X)->length
  );
  PostKernelLaunch();
}

void N_VProd_Cuda(N_Vector X, N_Vector Y, N_Vector Z)
{
  size_t grid, block, shMemSize;
  hipStream_t stream;

  GetKernelParameters(X, false, grid, block, shMemSize, stream);
  prodKernel<<<grid, block, shMemSize, stream>>>
  (
    NVEC_CUDA_DDATAp(X),
    NVEC_CUDA_DDATAp(Y),
    NVEC_CUDA_DDATAp(Z),
    NVEC_CUDA_CONTENT(X)->length
  );
  PostKernelLaunch();
}

void N_VDiv_Cuda(N_Vector X, N_Vector Y, N_Vector Z)
{
  size_t grid, block, shMemSize;
  hipStream_t stream;

  GetKernelParameters(X, false, grid, block, shMemSize, stream);
  divKernel<<<grid, block, shMemSize, stream>>>
  (
    NVEC_CUDA_DDATAp(X),
    NVEC_CUDA_DDATAp(Y),
    NVEC_CUDA_DDATAp(Z),
    NVEC_CUDA_CONTENT(X)->length
  );
  PostKernelLaunch();
}

void N_VScale_Cuda(realtype a, N_Vector X, N_Vector Z)
{
  size_t grid, block, shMemSize;
  hipStream_t stream;

  GetKernelParameters(X, false, grid, block, shMemSize, stream);
  scaleKernel<<<grid, block, shMemSize, stream>>>
  (
    a,
    NVEC_CUDA_DDATAp(X),
    NVEC_CUDA_DDATAp(Z),
    NVEC_CUDA_CONTENT(X)->length
  );
  PostKernelLaunch();
}

void N_VAbs_Cuda(N_Vector X, N_Vector Z)
{
  size_t grid, block, shMemSize;
  hipStream_t stream;

  GetKernelParameters(X, false, grid, block, shMemSize, stream);
  absKernel<<<grid, block, shMemSize, stream>>>
  (
    NVEC_CUDA_DDATAp(X),
    NVEC_CUDA_DDATAp(Z),
    NVEC_CUDA_CONTENT(X)->length
  );
  PostKernelLaunch();
}

void N_VInv_Cuda(N_Vector X, N_Vector Z)
{
  size_t grid, block, shMemSize;
  hipStream_t stream;

  GetKernelParameters(X, false, grid, block, shMemSize, stream);
  invKernel<<<grid, block, shMemSize, stream>>>
  (
    NVEC_CUDA_DDATAp(X),
    NVEC_CUDA_DDATAp(Z),
    NVEC_CUDA_CONTENT(X)->length
  );
  PostKernelLaunch();
}

void N_VAddConst_Cuda(N_Vector X, realtype b, N_Vector Z)
{
  size_t grid, block, shMemSize;
  hipStream_t stream;

  GetKernelParameters(X, false, grid, block, shMemSize, stream);
  addConstKernel<<<grid, block, shMemSize, stream>>>
  (
    b,
    NVEC_CUDA_DDATAp(X),
    NVEC_CUDA_DDATAp(Z),
    NVEC_CUDA_CONTENT(X)->length
  );
  PostKernelLaunch();
}

realtype N_VDotProd_Cuda(N_Vector X, N_Vector Y)
{
  size_t grid, block, shMemSize;
  hipStream_t stream;

  if (InitializeReductionBuffer(X, ZERO))
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VDotProd_Cuda: InitializeReductionBuffer returned nonzero\n");
  }

  GetKernelParameters(X, true, grid, block, shMemSize, stream);
  dotProdKernel<realtype, sunindextype><<<grid, block, shMemSize, stream>>>
  (
    NVEC_CUDA_DDATAp(X),
    NVEC_CUDA_DDATAp(Y),
    NVEC_CUDA_DBUFFERp(X),
    NVEC_CUDA_CONTENT(X)->length
  );
  PostKernelLaunch();

  // Get result from the GPU
  CopyReductionBufferFromDevice(X);
  realtype gpu_result = NVEC_CUDA_HBUFFERp(X)[0];

  return gpu_result;
}

realtype N_VMaxNorm_Cuda(N_Vector X)
{
  size_t grid, block, shMemSize;
  hipStream_t stream;

  if (InitializeReductionBuffer(X, ZERO))
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VMaxNorm_Cuda: InitializeReductionBuffer returned nonzero\n");
  }

  GetKernelParameters(X, true, grid, block, shMemSize, stream);
  maxNormKernel<realtype, sunindextype><<<grid, block, shMemSize, stream>>>
  (
    NVEC_CUDA_DDATAp(X),
    NVEC_CUDA_DBUFFERp(X),
    NVEC_CUDA_CONTENT(X)->length
  );
  PostKernelLaunch();

  // Finish reduction on CPU if there are less than two blocks of data left.
  CopyReductionBufferFromDevice(X);
  realtype gpu_result = NVEC_CUDA_HBUFFERp(X)[0];

  return gpu_result;
}

realtype N_VWSqrSumLocal_Cuda(N_Vector X, N_Vector W)
{
  size_t grid, block, shMemSize;
  hipStream_t stream;

  if (InitializeReductionBuffer(X, ZERO))
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VWSqrSumLocal_Cuda: InitializeReductionBuffer returned nonzero\n");
  }

  GetKernelParameters(X, true, grid, block, shMemSize, stream);
  wL2NormSquareKernel<realtype, sunindextype><<<grid, block, shMemSize, stream>>>
  (
    NVEC_CUDA_DDATAp(X),
    NVEC_CUDA_DDATAp(W),
    NVEC_CUDA_DBUFFERp(X),
    NVEC_CUDA_CONTENT(X)->length
  );
  PostKernelLaunch();

  // Get result from the GPU
  CopyReductionBufferFromDevice(X);
  realtype gpu_result = NVEC_CUDA_HBUFFERp(X)[0];

  return gpu_result;
}

realtype N_VWrmsNorm_Cuda(N_Vector X, N_Vector W)
{
  const realtype sum = N_VWSqrSumLocal_Cuda(X, W);
  return std::sqrt(sum/NVEC_CUDA_CONTENT(X)->length);
}

realtype N_VWSqrSumMaskLocal_Cuda(N_Vector X, N_Vector W, N_Vector Id)
{
  size_t grid, block, shMemSize;
  hipStream_t stream;

  if (InitializeReductionBuffer(X, ZERO))
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VWSqrSumMaskLocal_Cuda: InitializeReductionBuffer returned nonzero\n");
  }

  GetKernelParameters(X, true, grid, block, shMemSize, stream);
  wL2NormSquareMaskKernel<realtype, sunindextype><<<grid, block, shMemSize, stream>>>
  (
    NVEC_CUDA_DDATAp(X),
    NVEC_CUDA_DDATAp(W),
    NVEC_CUDA_DDATAp(Id),
    NVEC_CUDA_DBUFFERp(X),
    NVEC_CUDA_CONTENT(X)->length
  );
  PostKernelLaunch();

  // Get result from the GPU
  CopyReductionBufferFromDevice(X);
  realtype gpu_result = NVEC_CUDA_HBUFFERp(X)[0];

  return gpu_result;
}

realtype N_VWrmsNormMask_Cuda(N_Vector X, N_Vector W, N_Vector Id)
{
  const realtype sum = N_VWSqrSumMaskLocal_Cuda(X, W, Id);
  return std::sqrt(sum/NVEC_CUDA_CONTENT(X)->length);
}

realtype N_VMin_Cuda(N_Vector X)
{
  const realtype maxVal = std::numeric_limits<realtype>::max();

  size_t grid, block, shMemSize;
  hipStream_t stream;

  if (InitializeReductionBuffer(X, maxVal))
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VMin_Cuda: InitializeReductionBuffer returned nonzero\n");
  }

  GetKernelParameters(X, true, grid, block, shMemSize, stream);
  findMinKernel<realtype, sunindextype><<<grid, block, shMemSize, stream>>>
  (
    maxVal,
    NVEC_CUDA_DDATAp(X),
    NVEC_CUDA_DBUFFERp(X),
    NVEC_CUDA_CONTENT(X)->length
  );
  PostKernelLaunch();

  // Get result from the GPU
  CopyReductionBufferFromDevice(X);
  realtype gpu_result = NVEC_CUDA_HBUFFERp(X)[0];

  return gpu_result;
}

realtype N_VWL2Norm_Cuda(N_Vector X, N_Vector W)
{
  const realtype sum = N_VWSqrSumLocal_Cuda(X, W);
  return std::sqrt(sum);
}

realtype N_VL1Norm_Cuda(N_Vector X)
{
  size_t grid, block, shMemSize;
  hipStream_t stream;

  if (InitializeReductionBuffer(X, ZERO))
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VL1Norm_Cuda: InitializeReductionBuffer returned nonzero\n");
  }

  GetKernelParameters(X, true, grid, block, shMemSize, stream);
  L1NormKernel<realtype, sunindextype><<<grid, block, shMemSize, stream>>>
  (
    NVEC_CUDA_DDATAp(X),
    NVEC_CUDA_DBUFFERp(X),
    NVEC_CUDA_CONTENT(X)->length
  );
  PostKernelLaunch();

  // Get result from the GPU
  CopyReductionBufferFromDevice(X);
  realtype gpu_result = NVEC_CUDA_HBUFFERp(X)[0];

  return gpu_result;
}

void N_VCompare_Cuda(realtype c, N_Vector X, N_Vector Z)
{
  size_t grid, block, shMemSize;
  hipStream_t stream;

  GetKernelParameters(X, false, grid, block, shMemSize, stream);
  compareKernel<<<grid, block, shMemSize, stream>>>
  (
    c,
    NVEC_CUDA_DDATAp(X),
    NVEC_CUDA_DDATAp(Z),
    NVEC_CUDA_CONTENT(X)->length
  );
  PostKernelLaunch();
}

booleantype N_VInvTest_Cuda(N_Vector X, N_Vector Z)
{
  size_t grid, block, shMemSize;
  hipStream_t stream;

  if (InitializeReductionBuffer(X, ZERO))
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VInvTest_Cuda: InitializeReductionBuffer returned nonzero\n");
  }

  GetKernelParameters(X, true, grid, block, shMemSize, stream);
  invTestKernel<realtype, sunindextype><<<grid, block, shMemSize, stream>>>
  (
    NVEC_CUDA_DDATAp(X),
    NVEC_CUDA_DDATAp(Z),
    NVEC_CUDA_DBUFFERp(X),
    NVEC_CUDA_CONTENT(X)->length
  );
  PostKernelLaunch();

  // Get result from the GPU
  CopyReductionBufferFromDevice(X);
  realtype gpu_result = NVEC_CUDA_HBUFFERp(X)[0];

  return (gpu_result < HALF);
}

booleantype N_VConstrMask_Cuda(N_Vector C, N_Vector X, N_Vector M)
{
  size_t grid, block, shMemSize;
  hipStream_t stream;

  if (InitializeReductionBuffer(X, ZERO))
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VConstrMask_Cuda: InitializeReductionBuffer returned nonzero\n");
  }

  GetKernelParameters(X, true, grid, block, shMemSize, stream);
  constrMaskKernel<realtype, sunindextype><<<grid, block, shMemSize, stream>>>
  (
    NVEC_CUDA_DDATAp(C),
    NVEC_CUDA_DDATAp(X),
    NVEC_CUDA_DDATAp(M),
    NVEC_CUDA_DBUFFERp(X),
    NVEC_CUDA_CONTENT(X)->length
  );
  PostKernelLaunch();

  // Get result from the GPU
  CopyReductionBufferFromDevice(X);
  realtype gpu_result = NVEC_CUDA_HBUFFERp(X)[0];

  return (gpu_result < HALF);
}

realtype N_VMinQuotient_Cuda(N_Vector num, N_Vector denom)
{
  // Starting value for min reduction
  const realtype maxVal = std::numeric_limits<realtype>::max();
  size_t grid, block, shMemSize;
  hipStream_t stream;

  if (InitializeReductionBuffer(num, maxVal))
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VMinQuotient_Cuda: InitializeReductionBuffer returned nonzero\n");
  }

  GetKernelParameters(num, true, grid, block, shMemSize, stream);
  minQuotientKernel<realtype, sunindextype><<<grid, block, shMemSize, stream>>>
  (
    maxVal,
    NVEC_CUDA_DDATAp(num),
    NVEC_CUDA_DDATAp(denom),
    NVEC_CUDA_DBUFFERp(num),
    NVEC_CUDA_CONTENT(num)->length
  );
  PostKernelLaunch();

  // Get result from the GPU
  CopyReductionBufferFromDevice(num);
  realtype gpu_result = NVEC_CUDA_HBUFFERp(num)[0];

  return gpu_result;
}


/*
 * -----------------------------------------------------------------
 * fused vector operations
 * -----------------------------------------------------------------
 */

int N_VLinearCombination_Cuda(int nvec, realtype* c, N_Vector* X, N_Vector Z)
{
  hipError_t err;

  // Copy c array to device
  realtype* d_c;
  err = hipMalloc((void**) &d_c, nvec*sizeof(realtype));
  if (!SUNDIALS_CUDA_VERIFY(err)) return(-1);
  err = hipMemcpy(d_c, c, nvec*sizeof(realtype), hipMemcpyHostToDevice);
  if (!SUNDIALS_CUDA_VERIFY(err)) return(-1);

  // Create array of device pointers on host
  realtype** h_Xd = new realtype*[nvec];
  for (int i=0; i<nvec; i++)
    h_Xd[i] = NVEC_CUDA_DDATAp(X[i]);

  // Copy array of device pointers to device from host
  realtype** d_Xd;
  err = hipMalloc((void**) &d_Xd, nvec*sizeof(realtype*));
  if (!SUNDIALS_CUDA_VERIFY(err)) return(-1);
  err = hipMemcpy(d_Xd, h_Xd, nvec*sizeof(realtype*), hipMemcpyHostToDevice);
  if (!SUNDIALS_CUDA_VERIFY(err)) return(-1);

  // Set kernel parameters and launch
  size_t grid, block, shMemSize;
  hipStream_t stream;

  if (GetKernelParameters(X[0], false, grid, block, shMemSize, stream)) return(-1);
  linearCombinationKernel<<<grid, block, shMemSize, stream>>>
  (
    nvec,
    d_c,
    d_Xd,
    NVEC_CUDA_DDATAp(Z),
    NVEC_CUDA_CONTENT(Z)->length
  );
  PostKernelLaunch();

  // Free host array
  delete[] h_Xd;

  // Free device arrays
  err = hipFree(d_c);
  if (!SUNDIALS_CUDA_VERIFY(err)) return(-1);
  err = hipFree(d_Xd);
  if (!SUNDIALS_CUDA_VERIFY(err)) return(-1);

  return(0);
}

int N_VScaleAddMulti_Cuda(int nvec, realtype* c, N_Vector X, N_Vector* Y,
                          N_Vector* Z)
{
  hipError_t err;

  // Copy c array to device
  realtype* d_c;
  err = hipMalloc((void**) &d_c, nvec*sizeof(realtype));
  if (!SUNDIALS_CUDA_VERIFY(err)) return(-1);
  err = hipMemcpy(d_c, c, nvec*sizeof(realtype), hipMemcpyHostToDevice);
  if (!SUNDIALS_CUDA_VERIFY(err)) return(-1);

  // Create array of device pointers on host
  realtype** h_Yd = new realtype*[nvec];
  for (int i=0; i<nvec; i++)
    h_Yd[i] = NVEC_CUDA_DDATAp(Y[i]);

  realtype** h_Zd = new realtype*[nvec];
  for (int i=0; i<nvec; i++)
    h_Zd[i] = NVEC_CUDA_DDATAp(Z[i]);

  // Copy array of device pointers to device from host
  realtype** d_Yd;
  err = hipMalloc((void**) &d_Yd, nvec*sizeof(realtype*));
  if (!SUNDIALS_CUDA_VERIFY(err)) return(-1);
  err = hipMemcpy(d_Yd, h_Yd, nvec*sizeof(realtype*), hipMemcpyHostToDevice);
  if (!SUNDIALS_CUDA_VERIFY(err)) return(-1);

  realtype** d_Zd;
  err = hipMalloc((void**) &d_Zd, nvec*sizeof(realtype*));
  if (!SUNDIALS_CUDA_VERIFY(err)) return(-1);
  err = hipMemcpy(d_Zd, h_Zd, nvec*sizeof(realtype*), hipMemcpyHostToDevice);
  if (!SUNDIALS_CUDA_VERIFY(err)) return(-1);

  // Set kernel parameters
  size_t grid, block, shMemSize;
  hipStream_t stream;

  if (GetKernelParameters(X, false, grid, block, shMemSize, stream)) return(-1);
  scaleAddMultiKernel<<<grid, block, shMemSize, stream>>>
  (
    nvec,
    d_c,
    NVEC_CUDA_DDATAp(X),
    d_Yd,
    d_Zd,
    NVEC_CUDA_CONTENT(X)->length
  );
  PostKernelLaunch();

  // Free host array
  delete[] h_Yd;
  delete[] h_Zd;

  // Free device arrays
  err = hipFree(d_c);
  if (!SUNDIALS_CUDA_VERIFY(err)) return(-1);
  err = hipFree(d_Yd);
  if (!SUNDIALS_CUDA_VERIFY(err)) return(-1);
  err = hipFree(d_Zd);
  if (!SUNDIALS_CUDA_VERIFY(err)) return(-1);

  return(0);
}

int N_VDotProdMulti_Cuda(int nvec, N_Vector X, N_Vector* Y, realtype* dots)
{
  hipError_t err;

  // Create array of device pointers on host
  realtype** h_Yd = new realtype*[nvec];
  for (int i=0; i<nvec; i++)
    h_Yd[i] = NVEC_CUDA_DDATAp(Y[i]);

  // Copy array of device pointers to device from host
  realtype** d_Yd;
  err = hipMalloc((void**) &d_Yd, nvec*sizeof(realtype*));
  if (!SUNDIALS_CUDA_VERIFY(err)) return(-1);
  err = hipMemcpy(d_Yd, h_Yd, nvec*sizeof(realtype*), hipMemcpyHostToDevice);
  if (!SUNDIALS_CUDA_VERIFY(err)) return(-1);

  // Set kernel parameters
  size_t grid, block, shMemSize;
  hipStream_t stream;

  if (GetKernelParameters(X, false, grid, block, shMemSize, stream)) return(-1);
  grid = nvec;

  // Allocate reduction buffer on device
  realtype* d_buff;
  err = hipMalloc((void**) &d_buff, grid*sizeof(realtype));
  if (!SUNDIALS_CUDA_VERIFY(err)) return(-1);
  err = hipMemsetAsync(d_buff, 0, grid*sizeof(realtype));
  if (!SUNDIALS_CUDA_VERIFY(err)) return(-1);

  dotProdMultiKernel<realtype, sunindextype><<<grid, block, shMemSize, stream>>>
  (
    nvec,
    NVEC_CUDA_DDATAp(X),
    d_Yd,
    d_buff,
    NVEC_CUDA_CONTENT(X)->length
  );
  PostKernelLaunch();

  // Copy GPU result to the cpu.
  err = hipMemcpy(dots, d_buff, grid*sizeof(realtype), hipMemcpyDeviceToHost);
  if (!SUNDIALS_CUDA_VERIFY(err)) return(-1);

  // Free host array
  delete[] h_Yd;

  // Free device arrays
  err = hipFree(d_Yd);
  if (!SUNDIALS_CUDA_VERIFY(err)) return(-1);
  err = hipFree(d_buff);
  if (!SUNDIALS_CUDA_VERIFY(err)) return(-1);

  return(0);
}


/*
 * -----------------------------------------------------------------------------
 * vector array operations
 * -----------------------------------------------------------------------------
 */

int N_VLinearSumVectorArray_Cuda(int nvec, realtype a, N_Vector* X, realtype b,
                                 N_Vector* Y, N_Vector* Z)
{
  hipError_t err;

  // Create array of device pointers on host
  realtype** h_Xd = new realtype*[nvec];
  for (int i=0; i<nvec; i++)
    h_Xd[i] = NVEC_CUDA_DDATAp(X[i]);

  realtype** h_Yd = new realtype*[nvec];
  for (int i=0; i<nvec; i++)
    h_Yd[i] = NVEC_CUDA_DDATAp(Y[i]);

  realtype** h_Zd = new realtype*[nvec];
  for (int i=0; i<nvec; i++)
    h_Zd[i] = NVEC_CUDA_DDATAp(Z[i]);

  // Copy array of device pointers to device from host
  realtype** d_Xd;
  err = hipMalloc((void**) &d_Xd, nvec*sizeof(realtype*));
  if (!SUNDIALS_CUDA_VERIFY(err)) return(-1);
  err = hipMemcpy(d_Xd, h_Xd, nvec*sizeof(realtype*), hipMemcpyHostToDevice);
  if (!SUNDIALS_CUDA_VERIFY(err)) return(-1);

  realtype** d_Yd;
  err = hipMalloc((void**) &d_Yd, nvec*sizeof(realtype*));
  if (!SUNDIALS_CUDA_VERIFY(err)) return(-1);
  err = hipMemcpy(d_Yd, h_Yd, nvec*sizeof(realtype*), hipMemcpyHostToDevice);
  if (!SUNDIALS_CUDA_VERIFY(err)) return(-1);

  realtype** d_Zd;
  err = hipMalloc((void**) &d_Zd, nvec*sizeof(realtype*));
  if (!SUNDIALS_CUDA_VERIFY(err)) return(-1);
  err = hipMemcpy(d_Zd, h_Zd, nvec*sizeof(realtype*), hipMemcpyHostToDevice);
  if (!SUNDIALS_CUDA_VERIFY(err)) return(-1);

  // Set kernel parameters
  size_t grid, block, shMemSize;
  hipStream_t stream;

  if (GetKernelParameters(Z[0], false, grid, block, shMemSize, stream)) return(-1);
  linearSumVectorArrayKernel<<<grid, block, shMemSize, stream>>>
  (
    nvec,
    a,
    d_Xd,
    b,
    d_Yd,
    d_Zd,
    NVEC_CUDA_CONTENT(Z[0])->length
  );
  PostKernelLaunch();

  // Free host array
  delete[] h_Xd;
  delete[] h_Yd;
  delete[] h_Zd;

  // Free device arrays
  err = hipFree(d_Xd);
  if (!SUNDIALS_CUDA_VERIFY(err)) return(-1);
  err = hipFree(d_Yd);
  if (!SUNDIALS_CUDA_VERIFY(err)) return(-1);
  err = hipFree(d_Zd);
  if (!SUNDIALS_CUDA_VERIFY(err)) return(-1);

  return(0);
}

int N_VScaleVectorArray_Cuda(int nvec, realtype* c, N_Vector* X, N_Vector* Z)
{
  hipError_t err;

  // Copy c array to device
  realtype* d_c;
  err = hipMalloc((void**) &d_c, nvec*sizeof(realtype));
  if (!SUNDIALS_CUDA_VERIFY(err)) return(-1);
  err = hipMemcpy(d_c, c, nvec*sizeof(realtype), hipMemcpyHostToDevice);
  if (!SUNDIALS_CUDA_VERIFY(err)) return(-1);

  // Create array of device pointers on host
  realtype** h_Xd = new realtype*[nvec];
  for (int i=0; i<nvec; i++)
    h_Xd[i] = NVEC_CUDA_DDATAp(X[i]);

  realtype** h_Zd = new realtype*[nvec];
  for (int i=0; i<nvec; i++)
    h_Zd[i] = NVEC_CUDA_DDATAp(Z[i]);

  // Copy array of device pointers to device from host
  realtype** d_Xd;
  err = hipMalloc((void**) &d_Xd, nvec*sizeof(realtype*));
  if (!SUNDIALS_CUDA_VERIFY(err)) return(-1);
  err = hipMemcpy(d_Xd, h_Xd, nvec*sizeof(realtype*), hipMemcpyHostToDevice);
  if (!SUNDIALS_CUDA_VERIFY(err)) return(-1);

  realtype** d_Zd;
  err = hipMalloc((void**) &d_Zd, nvec*sizeof(realtype*));
  if (!SUNDIALS_CUDA_VERIFY(err)) return(-1);
  err = hipMemcpy(d_Zd, h_Zd, nvec*sizeof(realtype*), hipMemcpyHostToDevice);
  if (!SUNDIALS_CUDA_VERIFY(err)) return(-1);

  // Set kernel parameters
  size_t grid, block, shMemSize;
  hipStream_t stream;

  if (GetKernelParameters(Z[0], false, grid, block, shMemSize, stream)) return(-1);
  scaleVectorArrayKernel<<<grid, block, shMemSize, stream>>>
  (
    nvec,
    d_c,
    d_Xd,
    d_Zd,
    NVEC_CUDA_CONTENT(Z[0])->length
  );
  PostKernelLaunch();

  // Free host array
  delete[] h_Xd;
  delete[] h_Zd;

  // Free device arrays
  err = hipFree(d_c);
  if (!SUNDIALS_CUDA_VERIFY(err)) return(-1);
  err = hipFree(d_Xd);
  if (!SUNDIALS_CUDA_VERIFY(err)) return(-1);
  err = hipFree(d_Zd);
  if (!SUNDIALS_CUDA_VERIFY(err)) return(-1);

  return(0);
}

int N_VConstVectorArray_Cuda(int nvec, realtype c, N_Vector* Z)
{
  hipError_t err;

  // Create array of device pointers on host
  realtype** h_Zd = new realtype*[nvec];
  for (int i=0; i<nvec; i++)
    h_Zd[i] = NVEC_CUDA_DDATAp(Z[i]);

  // Copy array of device pointers to device from host
  realtype** d_Zd;
  err = hipMalloc((void**) &d_Zd, nvec*sizeof(realtype*));
  if (!SUNDIALS_CUDA_VERIFY(err)) return(-1);
  err = hipMemcpy(d_Zd, h_Zd, nvec*sizeof(realtype*), hipMemcpyHostToDevice);
  if (!SUNDIALS_CUDA_VERIFY(err)) return(-1);

  // Set kernel parameters
  size_t grid, block, shMemSize;
  hipStream_t stream;

  if (GetKernelParameters(Z[0], false, grid, block, shMemSize, stream)) return(-1);
  constVectorArrayKernel<<<grid, block, shMemSize, stream>>>
  (
    nvec,
    c,
    d_Zd,
    NVEC_CUDA_CONTENT(Z[0])->length
  );
  PostKernelLaunch();

  // Free host array
  delete[] h_Zd;

  // Free device arrays
  err = hipFree(d_Zd);
  if (!SUNDIALS_CUDA_VERIFY(err)) return(-1);

  return(0);
}

int N_VWrmsNormVectorArray_Cuda(int nvec, N_Vector* X, N_Vector* W,
                                realtype* norms)
{
  hipError_t err;

  // Create array of device pointers on host
  realtype** h_Xd = new realtype*[nvec];
  for (int i=0; i<nvec; i++)
    h_Xd[i] = NVEC_CUDA_DDATAp(X[i]);
  realtype** h_Wd = new realtype*[nvec];
  for (int i=0; i<nvec; i++)
    h_Wd[i] = NVEC_CUDA_DDATAp(W[i]);

  // Copy array of device pointers to device from host
  realtype** d_Xd;
  err = hipMalloc((void**) &d_Xd, nvec*sizeof(realtype*));
  if (!SUNDIALS_CUDA_VERIFY(err)) return(-1);
  err = hipMemcpy(d_Xd, h_Xd, nvec*sizeof(realtype*), hipMemcpyHostToDevice);
  if (!SUNDIALS_CUDA_VERIFY(err)) return(-1);

  realtype** d_Wd;
  err = hipMalloc((void**) &d_Wd, nvec*sizeof(realtype*));
  if (!SUNDIALS_CUDA_VERIFY(err)) return(-1);
  err = hipMemcpy(d_Wd, h_Wd, nvec*sizeof(realtype*), hipMemcpyHostToDevice);
  if (!SUNDIALS_CUDA_VERIFY(err)) return(-1);

  // Set kernel parameters
  size_t grid, block, shMemSize;
  hipStream_t stream;

  if (GetKernelParameters(X[0], true, grid, block, shMemSize, stream)) return(-1);
  grid = nvec;

  // Allocate reduction buffer on device
  realtype* d_buff;
  err = hipMalloc((void**) &d_buff, grid*sizeof(realtype));
  if (!SUNDIALS_CUDA_VERIFY(err)) return(-1);
  err = hipMemsetAsync(d_buff, 0, grid*sizeof(realtype));
  if (!SUNDIALS_CUDA_VERIFY(err)) return(-1);

  wL2NormSquareVectorArrayKernel<realtype, sunindextype><<<grid, block, shMemSize, stream>>>
  (
    nvec,
    d_Xd,
    d_Wd,
    d_buff,
    NVEC_CUDA_CONTENT(X[0])->length
  );
  PostKernelLaunch();

  // Copy GPU result to the cpu.
  err = hipMemcpy(norms, d_buff, grid*sizeof(realtype), hipMemcpyDeviceToHost);
  if (!SUNDIALS_CUDA_VERIFY(err)) return(-1);

  // Finish computation
  for (int k=0; k<nvec; ++k)
    norms[k] = std::sqrt(norms[k]/NVEC_CUDA_CONTENT(X[0])->length);

  // Free host array
  delete[] h_Xd;
  delete[] h_Wd;

  // Free device arrays
  err = hipFree(d_Xd);
  if (!SUNDIALS_CUDA_VERIFY(err)) return(-1);
  err = hipFree(d_Wd);
  if (!SUNDIALS_CUDA_VERIFY(err)) return(-1);
  err = hipFree(d_buff);
  if (!SUNDIALS_CUDA_VERIFY(err)) return(-1);

  return(0);
}

int N_VWrmsNormMaskVectorArray_Cuda(int nvec, N_Vector* X, N_Vector* W,
                                    N_Vector id, realtype* norms)
{
  hipError_t err;

  // Create array of device pointers on host
  realtype** h_Xd = new realtype*[nvec];
  for (int i=0; i<nvec; i++)
    h_Xd[i] = NVEC_CUDA_DDATAp(X[i]);

  realtype** h_Wd = new realtype*[nvec];
  for (int i=0; i<nvec; i++)
    h_Wd[i] = NVEC_CUDA_DDATAp(W[i]);

  // Copy array of device pointers to device from host
  realtype** d_Xd;
  err = hipMalloc((void**) &d_Xd, nvec*sizeof(realtype*));
  if (!SUNDIALS_CUDA_VERIFY(err)) return(-1);
  err = hipMemcpy(d_Xd, h_Xd, nvec*sizeof(realtype*), hipMemcpyHostToDevice);
  if (!SUNDIALS_CUDA_VERIFY(err)) return(-1);

  realtype** d_Wd;
  err = hipMalloc((void**) &d_Wd, nvec*sizeof(realtype*));
  if (!SUNDIALS_CUDA_VERIFY(err)) return(-1);
  err = hipMemcpy(d_Wd, h_Wd, nvec*sizeof(realtype*), hipMemcpyHostToDevice);
  if (!SUNDIALS_CUDA_VERIFY(err)) return(-1);

  // Set kernel parameters
  size_t grid, block, shMemSize;
  hipStream_t stream;

  if (GetKernelParameters(X[0], true, grid, block, shMemSize, stream)) return(-1);
  grid = nvec;

  // Allocate reduction buffer on device
  realtype* d_buff;
  err = hipMalloc((void**) &d_buff, grid*sizeof(realtype));
  if (!SUNDIALS_CUDA_VERIFY(err)) return(-1);
  err = hipMemsetAsync(d_buff, 0, grid*sizeof(realtype));
  if (!SUNDIALS_CUDA_VERIFY(err)) return(-1);

  wL2NormSquareMaskVectorArrayKernel<realtype, sunindextype><<<grid, block, shMemSize, stream>>>
  (
    nvec,
    d_Xd,
    d_Wd,
    NVEC_CUDA_DDATAp(id),
    d_buff,
    NVEC_CUDA_CONTENT(X[0])->length
  );
  PostKernelLaunch();

  // Copy GPU result to the cpu.
  err = hipMemcpy(norms, d_buff, grid*sizeof(realtype), hipMemcpyDeviceToHost);
  if (!SUNDIALS_CUDA_VERIFY(err)) return(-1);

  // Finish computation
  for (int k=0; k<nvec; ++k)
    norms[k] = std::sqrt(norms[k]/NVEC_CUDA_CONTENT(X[0])->length);

  // Free host array
  delete[] h_Xd;
  delete[] h_Wd;

  // Free device arrays
  err = hipFree(d_Xd);
  if (!SUNDIALS_CUDA_VERIFY(err)) return(-1);
  err = hipFree(d_Wd);
  if (!SUNDIALS_CUDA_VERIFY(err)) return(-1);
  err = hipFree(d_buff);
  if (!SUNDIALS_CUDA_VERIFY(err)) return(-1);

  return(0);
}

int N_VScaleAddMultiVectorArray_Cuda(int nvec, int nsum, realtype* c,
                                     N_Vector* X, N_Vector** Y, N_Vector** Z)
{
  hipError_t err;

  // Copy c array to device
  realtype* d_c;
  err = hipMalloc((void**) &d_c, nsum*sizeof(realtype));
  if (!SUNDIALS_CUDA_VERIFY(err)) return(-1);
  err = hipMemcpy(d_c, c, nsum*sizeof(realtype), hipMemcpyHostToDevice);
  if (!SUNDIALS_CUDA_VERIFY(err)) return(-1);

  // Create array of device pointers on host
  realtype** h_Xd = new realtype*[nvec];
  for (int i=0; i<nvec; i++)
    h_Xd[i] = NVEC_CUDA_DDATAp(X[i]);

  realtype** h_Yd = new realtype*[nsum*nvec];
  for (int j=0; j<nvec; j++)
    for (int i=0; i<nsum; i++)
      h_Yd[j*nsum+i] = NVEC_CUDA_DDATAp(Y[i][j]);

  realtype** h_Zd = new realtype*[nsum*nvec];
  for (int j=0; j<nvec; j++)
    for (int i=0; i<nsum; i++)
      h_Zd[j*nsum+i] = NVEC_CUDA_DDATAp(Z[i][j]);

  // Copy array of device pointers to device from host
  realtype** d_Xd;
  err = hipMalloc((void**) &d_Xd, nvec*sizeof(realtype*));
  if (!SUNDIALS_CUDA_VERIFY(err)) return(-1);
  err = hipMemcpy(d_Xd, h_Xd, nvec*sizeof(realtype*), hipMemcpyHostToDevice);
  if (!SUNDIALS_CUDA_VERIFY(err)) return(-1);

  realtype** d_Yd;
  err = hipMalloc((void**) &d_Yd, nsum*nvec*sizeof(realtype*));
  if (!SUNDIALS_CUDA_VERIFY(err)) return(-1);
  err = hipMemcpy(d_Yd, h_Yd, nsum*nvec*sizeof(realtype*), hipMemcpyHostToDevice);
  if (!SUNDIALS_CUDA_VERIFY(err)) return(-1);

  realtype** d_Zd;
  err = hipMalloc((void**) &d_Zd, nsum*nvec*sizeof(realtype*));
  if (!SUNDIALS_CUDA_VERIFY(err)) return(-1);
  err = hipMemcpy(d_Zd, h_Zd, nsum*nvec*sizeof(realtype*), hipMemcpyHostToDevice);
  if (!SUNDIALS_CUDA_VERIFY(err)) return(-1);

  // Set kernel parameters
  size_t grid, block, shMemSize;
  hipStream_t stream;

  if (GetKernelParameters(Z[0][0], false, grid, block, shMemSize, stream)) return(-1);
  scaleAddMultiVectorArrayKernel<<<grid, block, shMemSize, stream>>>
  (
    nvec,
    nsum,
    d_c,
    d_Xd,
    d_Yd,
    d_Zd,
    NVEC_CUDA_CONTENT(Z[0][0])->length
  );
  PostKernelLaunch();

  // Free host array
  delete[] h_Xd;
  delete[] h_Yd;
  delete[] h_Zd;

  // Free device arrays
  err = hipFree(d_c);
  if (!SUNDIALS_CUDA_VERIFY(err)) return(-1);
  err = hipFree(d_Xd);
  if (!SUNDIALS_CUDA_VERIFY(err)) return(-1);
  err = hipFree(d_Yd);
  if (!SUNDIALS_CUDA_VERIFY(err)) return(-1);
  err = hipFree(d_Zd);
  if (!SUNDIALS_CUDA_VERIFY(err)) return(-1);

  return(0);
}

int N_VLinearCombinationVectorArray_Cuda(int nvec, int nsum, realtype* c,
                                         N_Vector** X, N_Vector* Z)
{
  hipError_t err;

  // Copy c array to device
  realtype* d_c;
  err = hipMalloc((void**) &d_c, nsum*sizeof(realtype));
  if (!SUNDIALS_CUDA_VERIFY(err)) return(-1);
  err = hipMemcpy(d_c, c, nsum*sizeof(realtype), hipMemcpyHostToDevice);
  if (!SUNDIALS_CUDA_VERIFY(err)) return(-1);

  // Create array of device pointers on host
  realtype** h_Xd = new realtype*[nsum*nvec];
  for (int j=0; j<nvec; j++)
    for (int i=0; i<nsum; i++)
      h_Xd[j*nsum+i] = NVEC_CUDA_DDATAp(X[i][j]);

  realtype** h_Zd = new realtype*[nvec];
  for (int i=0; i<nvec; i++)
    h_Zd[i] = NVEC_CUDA_DDATAp(Z[i]);

  // Copy array of device pointers to device from host
  realtype** d_Xd;
  err = hipMalloc((void**) &d_Xd, nsum*nvec*sizeof(realtype*));
  if (!SUNDIALS_CUDA_VERIFY(err)) return(-1);
  err = hipMemcpy(d_Xd, h_Xd, nsum*nvec*sizeof(realtype*), hipMemcpyHostToDevice);
  if (!SUNDIALS_CUDA_VERIFY(err)) return(-1);

  realtype** d_Zd;
  err = hipMalloc((void**) &d_Zd, nvec*sizeof(realtype*));
  if (!SUNDIALS_CUDA_VERIFY(err)) return(-1);
  err = hipMemcpy(d_Zd, h_Zd, nvec*sizeof(realtype*), hipMemcpyHostToDevice);
  if (!SUNDIALS_CUDA_VERIFY(err)) return(-1);

  // Set kernel parameters
  size_t grid, block, shMemSize;
  hipStream_t stream;

  if (GetKernelParameters(Z[0], false, grid, block, shMemSize, stream)) return(-1);
  linearCombinationVectorArrayKernel<<<grid, block, shMemSize, stream>>>
  (
    nvec,
    nsum,
    d_c,
    d_Xd,
    d_Zd,
    NVEC_CUDA_CONTENT(Z[0])->length
  );
  PostKernelLaunch();

  // Free host array
  delete[] h_Xd;
  delete[] h_Zd;

  // Free device arrays
  err = hipFree(d_c);
  if (!SUNDIALS_CUDA_VERIFY(err)) return(-1);
  err = hipFree(d_Xd);
  if (!SUNDIALS_CUDA_VERIFY(err)) return(-1);
  err = hipFree(d_Zd);
  if (!SUNDIALS_CUDA_VERIFY(err)) return(-1);

  return hipGetLastError();
}


/*
 * -----------------------------------------------------------------
 * OPTIONAL XBraid interface operations
 * -----------------------------------------------------------------
 */


int N_VBufSize_Cuda(N_Vector x, sunindextype *size)
{
  if (x == NULL) return(-1);
  *size = (sunindextype)NVEC_CUDA_MEMSIZE(x);
  return(0);
}


int N_VBufPack_Cuda(N_Vector x, void *buf)
{
  int copy_fail = 0;
  hipError_t cuerr;

  if (x == NULL || buf == NULL) return(-1);

  SUNMemory buf_mem = SUNMemoryHelper_Wrap(buf, SUNMEMTYPE_HOST);
  if (buf_mem == NULL) return(-1);

  copy_fail = SUNMemoryHelper_CopyAsync(NVEC_CUDA_MEMHELP(x),
                                        buf_mem,
                                        NVEC_CUDA_CONTENT(x)->device_data,
                                        NVEC_CUDA_MEMSIZE(x),
                                        (void*) NVEC_CUDA_STREAM(x));

  /* we synchronize with respect to the host, but only in this stream */
  cuerr = hipStreamSynchronize(*NVEC_CUDA_STREAM(x));

  SUNMemoryHelper_Dealloc(NVEC_CUDA_MEMHELP(x), buf_mem);

  return (!SUNDIALS_CUDA_VERIFY(cuerr) || copy_fail ? -1 : 0);
}


int N_VBufUnpack_Cuda(N_Vector x, void *buf)
{
  int copy_fail = 0;
  hipError_t cuerr;

  if (x == NULL || buf == NULL) return(-1);

  SUNMemory buf_mem = SUNMemoryHelper_Wrap(buf, SUNMEMTYPE_HOST);
  if (buf_mem == NULL) return(-1);

  copy_fail = SUNMemoryHelper_CopyAsync(NVEC_CUDA_MEMHELP(x),
                                        NVEC_CUDA_CONTENT(x)->device_data,
                                        buf_mem,
                                        NVEC_CUDA_MEMSIZE(x),
                                        (void*) NVEC_CUDA_STREAM(x));

  /* we synchronize with respect to the host, but only in this stream */
  cuerr = hipStreamSynchronize(*NVEC_CUDA_STREAM(x));

  SUNMemoryHelper_Dealloc(NVEC_CUDA_MEMHELP(x), buf_mem);

  return (!SUNDIALS_CUDA_VERIFY(cuerr) || copy_fail ? -1 : 0);
}


/*
 * -----------------------------------------------------------------
 * Enable / Disable fused and vector array operations
 * -----------------------------------------------------------------
 */

int N_VEnableFusedOps_Cuda(N_Vector v, booleantype tf)
{
  /* check that vector is non-NULL */
  if (v == NULL) return(-1);

  /* check that ops structure is non-NULL */
  if (v->ops == NULL) return(-1);

  if (tf)
  {
    /* enable all fused vector operations */
    v->ops->nvlinearcombination = N_VLinearCombination_Cuda;
    v->ops->nvscaleaddmulti     = N_VScaleAddMulti_Cuda;
    v->ops->nvdotprodmulti      = N_VDotProdMulti_Cuda;
    /* enable all vector array operations */
    v->ops->nvlinearsumvectorarray         = N_VLinearSumVectorArray_Cuda;
    v->ops->nvscalevectorarray             = N_VScaleVectorArray_Cuda;
    v->ops->nvconstvectorarray             = N_VConstVectorArray_Cuda;
    v->ops->nvwrmsnormvectorarray          = N_VWrmsNormVectorArray_Cuda;
    v->ops->nvwrmsnormmaskvectorarray      = N_VWrmsNormMaskVectorArray_Cuda;
    v->ops->nvscaleaddmultivectorarray     = N_VScaleAddMultiVectorArray_Cuda;
    v->ops->nvlinearcombinationvectorarray = N_VLinearCombinationVectorArray_Cuda;
  }
  else
  {
    /* disable all fused vector operations */
    v->ops->nvlinearcombination = NULL;
    v->ops->nvscaleaddmulti     = NULL;
    v->ops->nvdotprodmulti      = NULL;
    /* disable all vector array operations */
    v->ops->nvlinearsumvectorarray         = NULL;
    v->ops->nvscalevectorarray             = NULL;
    v->ops->nvconstvectorarray             = NULL;
    v->ops->nvwrmsnormvectorarray          = NULL;
    v->ops->nvwrmsnormmaskvectorarray      = NULL;
    v->ops->nvscaleaddmultivectorarray     = NULL;
    v->ops->nvlinearcombinationvectorarray = NULL;
  }

  /* return success */
  return(0);
}

int N_VEnableLinearCombination_Cuda(N_Vector v, booleantype tf)
{
  /* check that vector is non-NULL */
  if (v == NULL) return(-1);

  /* check that ops structure is non-NULL */
  if (v->ops == NULL) return(-1);

  /* enable/disable operation */
  if (tf)
    v->ops->nvlinearcombination = N_VLinearCombination_Cuda;
  else
    v->ops->nvlinearcombination = NULL;

  /* return success */
  return(0);
}

int N_VEnableScaleAddMulti_Cuda(N_Vector v, booleantype tf)
{
  /* check that vector is non-NULL */
  if (v == NULL) return(-1);

  /* check that ops structure is non-NULL */
  if (v->ops == NULL) return(-1);

  /* enable/disable operation */
  if (tf)
    v->ops->nvscaleaddmulti = N_VScaleAddMulti_Cuda;
  else
    v->ops->nvscaleaddmulti = NULL;

  /* return success */
  return(0);
}

int N_VEnableDotProdMulti_Cuda(N_Vector v, booleantype tf)
{
  /* check that vector is non-NULL */
  if (v == NULL) return(-1);

  /* check that ops structure is non-NULL */
  if (v->ops == NULL) return(-1);

  /* enable/disable operation */
  if (tf)
    v->ops->nvdotprodmulti = N_VDotProdMulti_Cuda;
  else
    v->ops->nvdotprodmulti = NULL;

  /* return success */
  return(0);
}

int N_VEnableLinearSumVectorArray_Cuda(N_Vector v, booleantype tf)
{
  /* check that vector is non-NULL */
  if (v == NULL) return(-1);

  /* check that ops structure is non-NULL */
  if (v->ops == NULL) return(-1);

  /* enable/disable operation */
  if (tf)
    v->ops->nvlinearsumvectorarray = N_VLinearSumVectorArray_Cuda;
  else
    v->ops->nvlinearsumvectorarray = NULL;

  /* return success */
  return(0);
}

int N_VEnableScaleVectorArray_Cuda(N_Vector v, booleantype tf)
{
  /* check that vector is non-NULL */
  if (v == NULL) return(-1);

  /* check that ops structure is non-NULL */
  if (v->ops == NULL) return(-1);

  /* enable/disable operation */
  if (tf)
    v->ops->nvscalevectorarray = N_VScaleVectorArray_Cuda;
  else
    v->ops->nvscalevectorarray = NULL;

  /* return success */
  return(0);
}

int N_VEnableConstVectorArray_Cuda(N_Vector v, booleantype tf)
{
  /* check that vector is non-NULL */
  if (v == NULL) return(-1);

  /* check that ops structure is non-NULL */
  if (v->ops == NULL) return(-1);

  /* enable/disable operation */
  if (tf)
    v->ops->nvconstvectorarray = N_VConstVectorArray_Cuda;
  else
    v->ops->nvconstvectorarray = NULL;

  /* return success */
  return(0);
}

int N_VEnableWrmsNormVectorArray_Cuda(N_Vector v, booleantype tf)
{
  /* check that vector is non-NULL */
  if (v == NULL) return(-1);

  /* check that ops structure is non-NULL */
  if (v->ops == NULL) return(-1);

  /* enable/disable operation */
  if (tf)
    v->ops->nvwrmsnormvectorarray = N_VWrmsNormVectorArray_Cuda;
  else
    v->ops->nvwrmsnormvectorarray = NULL;

  /* return success */
  return(0);
}

int N_VEnableWrmsNormMaskVectorArray_Cuda(N_Vector v, booleantype tf)
{
  /* check that vector is non-NULL */
  if (v == NULL) return(-1);

  /* check that ops structure is non-NULL */
  if (v->ops == NULL) return(-1);

  /* enable/disable operation */
  if (tf)
    v->ops->nvwrmsnormmaskvectorarray = N_VWrmsNormMaskVectorArray_Cuda;
  else
    v->ops->nvwrmsnormmaskvectorarray = NULL;

  /* return success */
  return(0);
}

int N_VEnableScaleAddMultiVectorArray_Cuda(N_Vector v, booleantype tf)
{
  /* check that vector is non-NULL */
  if (v == NULL) return(-1);

  /* check that ops structure is non-NULL */
  if (v->ops == NULL) return(-1);

  /* enable/disable operation */
  if (tf)
    v->ops->nvscaleaddmultivectorarray = N_VScaleAddMultiVectorArray_Cuda;
  else
    v->ops->nvscaleaddmultivectorarray = NULL;

  /* return success */
  return(0);
}

int N_VEnableLinearCombinationVectorArray_Cuda(N_Vector v, booleantype tf)
{
  /* check that vector is non-NULL */
  if (v == NULL) return(-1);

  /* check that ops structure is non-NULL */
  if (v->ops == NULL) return(-1);

  /* enable/disable operation */
  if (tf)
    v->ops->nvlinearcombinationvectorarray = N_VLinearCombinationVectorArray_Cuda;
  else
    v->ops->nvlinearcombinationvectorarray = NULL;

  /* return success */
  return(0);
}

/*
 * Private helper functions.
 */

int AllocateData(N_Vector v)
{
  int alloc_fail = 0;
  N_VectorContent_Cuda vc = NVEC_CUDA_CONTENT(v);
  N_PrivateVectorContent_Cuda vcp = NVEC_CUDA_PRIVATE(v);

  if (N_VGetLength_Cuda(v) == 0) return(0);

  if (vcp->use_managed_mem)
  {
    alloc_fail = SUNMemoryHelper_Alloc(NVEC_CUDA_MEMHELP(v), &(vc->device_data),
                                       NVEC_CUDA_MEMSIZE(v), SUNMEMTYPE_UVM);
    if (alloc_fail)
    {
      SUNDIALS_DEBUG_PRINT("ERROR in AllocateData: SUNMemoryHelper_Alloc failed for SUNMEMTYPE_UVM\n");
    }
    vc->host_data = SUNMemoryHelper_Alias(vc->device_data);
  }
  else
  {
    alloc_fail = SUNMemoryHelper_Alloc(NVEC_CUDA_MEMHELP(v), &(vc->host_data),
                                       NVEC_CUDA_MEMSIZE(v), SUNMEMTYPE_HOST);
    if (alloc_fail)
    {
      SUNDIALS_DEBUG_PRINT("ERROR in AllocateData: SUNMemoryHelper_Alloc failed to alloc SUNMEMTYPE_HOST\n");
    }

    alloc_fail = SUNMemoryHelper_Alloc(NVEC_CUDA_MEMHELP(v), &(vc->device_data),
                                       NVEC_CUDA_MEMSIZE(v), SUNMEMTYPE_DEVICE);
    if (alloc_fail)
    {
      SUNDIALS_DEBUG_PRINT("ERROR in AllocateData: SUNMemoryHelper_Alloc failed to alloc SUNMEMTYPE_DEVICE\n");
    }
  }

  return(alloc_fail ? -1 : 0);
}

/*
 * Initializes the internal buffer used for reductions.
 * If the buffer is already allocated, it will only be reallocated
 * if it is no longer large enough. This may occur if the length
 * of the vector is increased. The buffer is initialized to the
 * value given.
 */
int InitializeReductionBuffer(N_Vector v, const realtype value)
{
  int alloc_fail = 0, copy_fail = 0;
  size_t bytes = sizeof(realtype);
  booleantype need_to_allocate = SUNFALSE;
  N_PrivateVectorContent_Cuda vcp = NVEC_CUDA_PRIVATE(v);
  SUNMemory value_mem = SUNMemoryHelper_Wrap((void*) &value, SUNMEMTYPE_HOST);

  /* we allocate if the existing reduction buffer is not large enough */
  if (vcp->reduce_buffer_allocated_bytes < bytes)
  {
    FreeReductionBuffer(v);
    need_to_allocate = SUNTRUE;
  }

  if (need_to_allocate)
  {
    alloc_fail = SUNMemoryHelper_Alloc(NVEC_CUDA_MEMHELP(v),
                                       &(vcp->reduce_buffer_host), bytes,
                                       SUNMEMTYPE_PINNED);
    if (alloc_fail)
    {
      SUNDIALS_DEBUG_PRINT("WARNING in InitializeReductionBuffer: SUNMemoryHelper_Alloc failed to alloc SUNMEMTYPE_PINNED, using SUNMEMTYPE_HOST instead\n");

      /* try to allocate just plain host memory instead */
      alloc_fail = SUNMemoryHelper_Alloc(NVEC_CUDA_MEMHELP(v),
                                         &(vcp->reduce_buffer_host), bytes,
                                         SUNMEMTYPE_HOST);
      if (alloc_fail)
      {
        SUNDIALS_DEBUG_PRINT("ERROR in InitializeReductionBuffer: SUNMemoryHelper_Alloc failed to alloc SUNMEMTYPE_HOST\n");
      }
    }
    alloc_fail = SUNMemoryHelper_Alloc(NVEC_CUDA_MEMHELP(v),
                                       &(vcp->reduce_buffer_dev), bytes,
                                       SUNMEMTYPE_DEVICE);
    if (alloc_fail)
    {
      SUNDIALS_DEBUG_PRINT("ERROR in InitializeReductionBuffer: SUNMemoryHelper_Alloc failed to alloc SUNMEMTYPE_DEVICE\n");
    }
  }

  if (!alloc_fail)
  {
    /* store the size of the buffer */
    vcp->reduce_buffer_allocated_bytes = bytes;

    /* initialize the memory with the value */
    copy_fail = SUNMemoryHelper_CopyAsync(NVEC_CUDA_MEMHELP(v),
                                          vcp->reduce_buffer_dev, value_mem,
                                          bytes, (void*) NVEC_CUDA_STREAM(v));

    if (copy_fail)
    {
      SUNDIALS_DEBUG_PRINT("ERROR in InitializeReductionBuffer: SUNMemoryHelper_CopyAsync failed\n");
    }
  }

  SUNMemoryHelper_Dealloc(NVEC_CUDA_MEMHELP(v), value_mem);
  return((alloc_fail || copy_fail) ? -1 : 0);
}

/* Free the reduction buffer
 */
void FreeReductionBuffer(N_Vector v)
{
  N_PrivateVectorContent_Cuda vcp = NVEC_CUDA_PRIVATE(v);

  if (vcp == NULL) return;

  if (vcp->reduce_buffer_dev != NULL)
    SUNMemoryHelper_Dealloc(NVEC_CUDA_MEMHELP(v), vcp->reduce_buffer_dev);
  vcp->reduce_buffer_dev  = NULL;
  if (vcp->reduce_buffer_host != NULL)
    SUNMemoryHelper_Dealloc(NVEC_CUDA_MEMHELP(v), vcp->reduce_buffer_host);
  vcp->reduce_buffer_host = NULL;
}

/* Copy the reduction buffer from the device to the host.
 */
int CopyReductionBufferFromDevice(N_Vector v, size_t n)
{
  int copy_fail;
  hipError_t cuerr;

  copy_fail = SUNMemoryHelper_CopyAsync(NVEC_CUDA_MEMHELP(v),
                                        NVEC_CUDA_PRIVATE(v)->reduce_buffer_host,
                                        NVEC_CUDA_PRIVATE(v)->reduce_buffer_dev,
                                        n*sizeof(realtype),
                                        (void*) NVEC_CUDA_STREAM(v));

  if (copy_fail)
  {
    SUNDIALS_DEBUG_PRINT("ERROR in CopyReductionBufferFromDevice: SUNMemoryHelper_CopyAsync returned nonzero\n");
  }

  /* we synchronize with respect to the host, but only in this stream */
  cuerr = hipStreamSynchronize(*NVEC_CUDA_STREAM(v));
  return (!SUNDIALS_CUDA_VERIFY(cuerr) || copy_fail ? -1 : 0);
}

/* Get the kernel launch parameters based on the kernel type (reduction or not),
 * using the appropriate kernel execution policy.
 */
static int GetKernelParameters(N_Vector v, booleantype reduction, size_t& grid,
                               size_t& block, size_t& shMemSize,
                               hipStream_t& stream, size_t n)
{
  n = (n == 0) ? NVEC_CUDA_CONTENT(v)->length : n;
  if (reduction)
  {
    SUNCudaExecPolicy* reduce_exec_policy = NVEC_CUDA_CONTENT(v)->reduce_exec_policy;
    grid      = reduce_exec_policy->gridSize(n);
    block     = reduce_exec_policy->blockSize();
    shMemSize = 0;
    stream    = *(reduce_exec_policy->stream());
    if (block % CUDA_WARP_SIZE)
    {
#ifdef SUNDIALS_DEBUG
      throw std::runtime_error("the block size must be a multiple must be of CUDA warp size");
#endif
      return(-1);
    }
  }
  else
  {
    SUNCudaExecPolicy* stream_exec_policy = NVEC_CUDA_CONTENT(v)->stream_exec_policy;
    grid      = stream_exec_policy->gridSize(n);
    block     = stream_exec_policy->blockSize();
    shMemSize = 0;
    stream    = *(stream_exec_policy->stream());
  }

  if (grid == 0)
  {
#ifdef SUNDIALS_DEBUG
    throw std::runtime_error("the grid size must be > 0");
#endif
    return(-1);
  }
  if (block == 0)
  {
#ifdef SUNDIALS_DEBUG
    throw std::runtime_error("the block size must be > 0");
#endif
    return(-1);
  }

  return(0);
}

/* Should be called after a kernel launch.
 * If SUNDIALS_DEBUG_CUDA_LASTERROR is not defined, then the function does nothing.
 * If it is defined, the function will synchronize and check the last CUDA error.
 */
void PostKernelLaunch()
{
#ifdef SUNDIALS_DEBUG_CUDA_LASTERROR
  hipDeviceSynchronize();
  SUNDIALS_CUDA_VERIFY(hipGetLastError());
#endif
}


} // extern "C"
