#include "hip/hip_runtime.h"
/* -----------------------------------------------------------------
 * Programmer(s): David J. Gardner @ LLNL
 * -----------------------------------------------------------------
 * SUNDIALS Copyright Start
 * Copyright (c) 2002-2022, Lawrence Livermore National Security
 * and Southern Methodist University.
 * All rights reserved.
 *
 * See the top-level LICENSE and NOTICE files for details.
 *
 * SPDX-License-Identifier: BSD-3-Clause
 * SUNDIALS Copyright End
 * -----------------------------------------------------------------
 * This is the testing routine to check the performance of the
 * NVECTOR CUDA module implementation.
 * -----------------------------------------------------------------*/

#include <stdio.h>
#include <stdlib.h>
#include <stddef.h>

#include <sundials/sundials_types.h>
#include <nvector/nvector_cuda.h>
#include <sundials/sundials_math.h>
#include "test_nvector_performance.h"

/* private functions */
static int InitializeClearCache(int cachesize);
static int FinalizeClearCache();

/* private data for clearing cache */
static sunindextype N;    /* data length */
static realtype* h_data;  /* host data   */
static realtype* h_sum;   /* host sum    */
static realtype* d_data;  /* device data */
static realtype* d_sum;   /* device sum  */
static int blocksPerGrid;

/* cuda reduction kernel to clearing cache between tests */
__global__
void ClearCacheKernel(sunindextype N, realtype* data, realtype* out)
{
  __shared__ realtype shared[256];

  int sharedidx = blockIdx.x;
  int tid = blockIdx.x * blockDim.x + threadIdx.x;

  realtype tmp = 0;
  while (tid < N) {
    tmp += data[tid];
    tid += blockDim.x * gridDim.x;
  }
  shared[sharedidx] = tmp;
  __syncthreads();

  /* assues blockDim is a power of 2 */
  int i = blockDim.x/2;
  while (i != 0) {
    if (sharedidx < i)
      shared[sharedidx] += shared[sharedidx + i];
    __syncthreads();
    i /= 2;
  }

  if (sharedidx == 0)
    out[sharedidx] = shared[0];
}

/* ----------------------------------------------------------------------
 * Main NVector Testing Routine
 * --------------------------------------------------------------------*/
int main(int argc, char *argv[])
{
  SUNContext   ctx = NULL;  /* SUNDIALS context */
  N_Vector     X   = NULL;  /* test vector      */
  sunindextype veclen;      /* vector length    */

  int print_timing;    /* output timings     */
  int ntests;          /* number of tests    */
  int nvecs;           /* number of tests    */
  int nsums;           /* number of sums     */
  int cachesize;       /* size of cache (MB) */
  int flag;            /* return flag        */

  printf("\nStart Tests\n");
  printf("Vector Name: Cuda\n");

  /* check input and set vector length */
  if (argc < 7){
    printf("ERROR: SIX (6) arguments required: ");
    printf("<vector length> <number of vectors> <number of sums> <number of tests> ");
    printf("<cache size (MB)> <print timing>\n");
    return(-1);
  }

  veclen = atol(argv[1]);
  if (veclen <= 0) {
    printf("ERROR: length of vector must be a positive integer \n");
    return(-1);
  }

  nvecs = atol(argv[2]);
  if (nvecs <= 0) {
    printf("ERROR: number of vectors must be a positive integer \n");
    return(-1);
  }

  nsums = atol(argv[3]);
  if (nsums <= 0) {
    printf("ERROR: number of sums must be a positive integer \n");
    return(-1);
  }

  ntests = atol(argv[4]);
  if (ntests <= 0) {
    printf("ERROR: number of tests must be a positive integer \n");
    return(-1);
  }

  cachesize = atol(argv[5]);
  if (cachesize < 0) {
    printf("ERROR: cache size (MB) must be a non-negative integer \n");
    return(-1);
  }
  InitializeClearCache(cachesize);

  print_timing = atoi(argv[6]);
  SetTiming(print_timing, 0);

  printf("\nRunning with: \n");
  printf("  vector length         %ld \n", (long int) veclen);
  printf("  max number of vectors %d  \n", nvecs);
  printf("  max number of sums    %d  \n", nsums);
  printf("  number of tests       %d  \n", ntests);
  printf("  timing on/off         %d  \n", print_timing);

  flag = SUNContext_Create(NULL, &ctx);
  if (flag) return flag;

  /* Create vectors */
  X = N_VNew_Cuda(veclen, ctx);

  /* run tests */
  if (print_timing) printf("\n\n standard operations:\n");
  if (print_timing) PrintTableHeader(1);
  flag = Test_N_VLinearSum(X, veclen, ntests);
  flag = Test_N_VConst(X, veclen, ntests);
  flag = Test_N_VProd(X, veclen, ntests);
  flag = Test_N_VDiv(X, veclen, ntests);
  flag = Test_N_VScale(X, veclen, ntests);
  flag = Test_N_VAbs(X, veclen, ntests);
  flag = Test_N_VInv(X, veclen, ntests);
  flag = Test_N_VAddConst(X, veclen, ntests);
  flag = Test_N_VDotProd(X, veclen, ntests);
  flag = Test_N_VMaxNorm(X, veclen, ntests);
  flag = Test_N_VWrmsNorm(X, veclen, ntests);
  flag = Test_N_VWrmsNormMask(X, veclen, ntests);
  flag = Test_N_VMin(X, veclen, ntests);
  flag = Test_N_VWL2Norm(X, veclen, ntests);
  flag = Test_N_VL1Norm(X, veclen, ntests);
  flag = Test_N_VCompare(X, veclen, ntests);
  flag = Test_N_VInvTest(X, veclen, ntests);
  flag = Test_N_VConstrMask(X, veclen, ntests);
  flag = Test_N_VMinQuotient(X, veclen, ntests);

  if (print_timing) printf("\n\n fused operations 1: nvecs= %d\n", nvecs);
  if (print_timing) PrintTableHeader(2);
  flag = Test_N_VLinearCombination(X, veclen, nvecs, ntests);
  flag = Test_N_VScaleAddMulti(X, veclen, nvecs, ntests);
  flag = Test_N_VDotProdMulti(X, veclen, nvecs, ntests);
  flag = Test_N_VLinearSumVectorArray(X, veclen, nvecs, ntests);
  flag = Test_N_VScaleVectorArray(X, veclen, nvecs, ntests);
  flag = Test_N_VConstVectorArray(X, veclen, nvecs, ntests);
  flag = Test_N_VWrmsNormVectorArray(X, veclen, nvecs, ntests);
  flag = Test_N_VWrmsNormMaskVectorArray(X, veclen, nvecs, ntests);

  if (print_timing) printf("\n\n fused operations 2: nvecs= %d nsums= %d\n", nvecs, nsums);
  if (print_timing) PrintTableHeader(2);
  flag = Test_N_VScaleAddMultiVectorArray(X, veclen, nvecs, nsums, ntests);
  flag = Test_N_VLinearCombinationVectorArray(X, veclen, nvecs, nsums, ntests);

  /* Free vectors */
  N_VDestroy(X);

  FinalizeClearCache();

  flag = SUNContext_Free(&ctx);
  if (flag) return flag;

  printf("\nFinished Tests\n");

  return(flag);
}


/* ----------------------------------------------------------------------
 * Functions required by testing routines to fill vector data
 * --------------------------------------------------------------------*/

/* random data between lower and upper */
void N_VRand(N_Vector Xvec, sunindextype Xlen, realtype lower, realtype upper)
{
  rand_realtype(N_VGetHostArrayPointer_Cuda(Xvec), Xlen, lower, upper);
  N_VCopyToDevice_Cuda(Xvec);
}

/* series of 0 and 1 */
void N_VRandZeroOne(N_Vector Xvec, sunindextype Xlen)
{
  rand_realtype_zero_one(N_VGetHostArrayPointer_Cuda(Xvec), Xlen);
  N_VCopyToDevice_Cuda(Xvec);
}

/* random values for constraint array */
void N_VRandConstraints(N_Vector Xvec, sunindextype Xlen)
{
  rand_realtype_constraints(N_VGetHostArrayPointer_Cuda(Xvec), Xlen);
  N_VCopyToDevice_Cuda(Xvec);
}


/* ----------------------------------------------------------------------
 * Functions required for MPI or GPU testing
 * --------------------------------------------------------------------*/

void collect_times(N_Vector X, double *times, int ntimes)
{
  /* not running with MPI, just return */
  return;
}

void sync_device(N_Vector x)
{
  hipDeviceSynchronize();
  return;
}


/* ----------------------------------------------------------------------
 * Functions required for clearing cache
 * --------------------------------------------------------------------*/

static int InitializeClearCache(int cachesize)
{
  hipError_t err;     /* cuda error flag     */
  size_t      nbytes;  /* cache size in bytes */

  /* determine size of vector to clear cache, N = ceil(2 * nbytes/realtype) */
  nbytes = (size_t) (2 * cachesize * 1024 * 1024);
  N = (sunindextype) ((nbytes + sizeof(realtype) - 1)/sizeof(realtype));

  /* allocate host data */
  blocksPerGrid = SUNMIN(32,(N+255)/256);

  h_data = (realtype*) malloc(N*sizeof(realtype));
  h_sum  = (realtype*) malloc(blocksPerGrid*sizeof(realtype));

  /* allocate device data */
  err = hipMalloc((void**) &d_data, N*sizeof(realtype));
  if (err != hipSuccess) {
    fprintf(stderr,"Failed to allocate device vector (error code %d )!\n",err);
    return(-1);
  }

  err = hipMalloc((void**) &d_sum, blocksPerGrid*sizeof(realtype));
  if (err != hipSuccess) {
    fprintf(stderr,"Failed to allocate device vector (error code %d )!\n",err);
    return(-1);
  }

  /* fill host vector with random data and copy to device */
  rand_realtype(h_data, N, RCONST(-1.0), RCONST(1.0));

  err = hipMemcpy(d_data, h_data, N*sizeof(realtype), hipMemcpyHostToDevice);
  if (err != hipSuccess) {
    fprintf(stderr,"Failed to copy data from host to device (error code %d )!\n",err);
    return(-1);
  }

  return(0);
}

static int FinalizeClearCache()
{
  hipError_t err;  /* cuda error flag */

  free(h_data);
  free(h_sum);

  err = hipFree(d_data);
  if (err != hipSuccess) {
    fprintf(stderr,"Failed to free device data (error code %d )!\n",err);
    return(-1);
  }

  err = hipFree(d_sum);
  if (err != hipSuccess) {
    fprintf(stderr,"Failed to free device data (error code %d )!\n",err);
    return(-1);
  }

  return(0);
}

void ClearCache()
{
  /* call cuda kernel to clear the cache */
  ClearCacheKernel<<<SUNMIN(32,(N+255)/256), 256>>>(N, d_data, d_sum);
  hipMemcpy(h_sum, d_sum, blocksPerGrid*sizeof(realtype), hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  return;
}
