#include "hip/hip_runtime.h"
/* -----------------------------------------------------------------
 * Programmer(s): Slaven Peles, and Cody J. Balos @ LLNL
 * -----------------------------------------------------------------
 * SUNDIALS Copyright Start
 * Copyright (c) 2002-2022, Lawrence Livermore National Security
 * and Southern Methodist University.
 * All rights reserved.
 *
 * See the top-level LICENSE and NOTICE files for details.
 *
 * SPDX-License-Identifier: BSD-3-Clause
 * SUNDIALS Copyright End
 * -----------------------------------------------------------------
 * This is the implementation file for a CUDA implementation
 * of the NVECTOR package.
 * -----------------------------------------------------------------*/

#include <cstdio>
#include <cstdlib>
#include <cmath>
#include <limits>
#include <iostream>

#include <nvector/nvector_cuda.h>
#include "VectorArrayKernels.cuh"
#include "VectorKernels.cuh"

#include "sundials_cuda.h"
#include "sundials_debug.h"

#define ZERO RCONST(0.0)
#define HALF RCONST(0.5)

using namespace sundials;
using namespace sundials::cuda;
using namespace sundials::cuda::impl;

/*
 * Private function definitions
 */

// Allocate vector data
static int AllocateData(N_Vector v);

// Reduction buffer functions
static int InitializeDeviceCounter(N_Vector v);
static int FreeDeviceCounter(N_Vector v);
static int InitializeReductionBuffer(N_Vector v, realtype value, size_t n = 1);
static void FreeReductionBuffer(N_Vector v);
static int CopyReductionBufferFromDevice(N_Vector v, size_t n = 1);

// Fused operation buffer functions
static int FusedBuffer_Init(N_Vector v, int nreal, int nptr);
static int FusedBuffer_CopyRealArray(N_Vector v, realtype *r_data, int nval,
                                     realtype **shortcut);
static int FusedBuffer_CopyPtrArray1D(N_Vector v, N_Vector *X, int nvec,
                                      realtype ***shortcut);
static int FusedBuffer_CopyPtrArray2D(N_Vector v, N_Vector **X, int nvec,
                                      int nsum, realtype ***shortcut);
static int FusedBuffer_CopyToDevice(N_Vector v);
static int FusedBuffer_Free(N_Vector v);

// Kernel launch parameters
static int GetKernelParameters(N_Vector v, booleantype reduction, size_t& grid, size_t& block,
                               size_t& shMemSize, hipStream_t& stream, size_t n = 0);
static int GetKernelParameters(N_Vector v, booleantype reduction, size_t& grid, size_t& block,
                                size_t& shMemSize, hipStream_t& stream, bool& atomic, size_t n = 0);
static void PostKernelLaunch();

/*
 * Macro definitions
 */

// Macros to access vector content
#define NVEC_CUDA_CONTENT(x)  ((N_VectorContent_Cuda)(x->content))
#define NVEC_CUDA_MEMSIZE(x)  (NVEC_CUDA_CONTENT(x)->length * sizeof(realtype))
#define NVEC_CUDA_MEMHELP(x)  (NVEC_CUDA_CONTENT(x)->mem_helper)
#define NVEC_CUDA_HDATAp(x)   ((realtype*) NVEC_CUDA_CONTENT(x)->host_data->ptr)
#define NVEC_CUDA_DDATAp(x)   ((realtype*) NVEC_CUDA_CONTENT(x)->device_data->ptr)
#define NVEC_CUDA_STREAM(x)   (NVEC_CUDA_CONTENT(x)->stream_exec_policy->stream())

// Macros to access vector private content
#define NVEC_CUDA_PRIVATE(x)   ((N_PrivateVectorContent_Cuda)(NVEC_CUDA_CONTENT(x)->priv))
#define NVEC_CUDA_HBUFFERp(x)  ((realtype*) NVEC_CUDA_PRIVATE(x)->reduce_buffer_host->ptr)
#define NVEC_CUDA_DBUFFERp(x)  ((realtype*) NVEC_CUDA_PRIVATE(x)->reduce_buffer_dev->ptr)
#define NVEC_CUDA_DCOUNTERp(x) ((unsigned int*) NVEC_CUDA_PRIVATE(x)->device_counter->ptr)

/*
 * Private structure definition
 */

struct _N_PrivateVectorContent_Cuda
{
  booleantype use_managed_mem; /* do data pointers use managed memory */

  // reduction workspace
  SUNMemory device_counter;      // device memory for a counter (used in LDS reductions)
  SUNMemory reduce_buffer_dev;   // device memory for reductions
  SUNMemory reduce_buffer_host;  // host memory for reductions
  size_t    reduce_buffer_bytes; // current size of reduction buffers

  // fused op workspace
  SUNMemory fused_buffer_dev;    // device memory for fused ops
  SUNMemory fused_buffer_host;   // host memory for fused ops
  size_t    fused_buffer_bytes;  // current size of the buffers
  size_t    fused_buffer_offset; // current offset into the buffer
};

typedef struct _N_PrivateVectorContent_Cuda *N_PrivateVectorContent_Cuda;

/* Default policies to clone */
ThreadDirectExecPolicy DEFAULT_STREAMING_EXECPOLICY(256);
BlockReduceAtomicExecPolicy DEFAULT_REDUCTION_EXECPOLICY(256);

extern "C" {

N_Vector N_VNewEmpty_Cuda(SUNContext sunctx)
{
  N_Vector v;

  /* Create vector */
  v = NULL;
  v = N_VNewEmpty(sunctx);
  if (v == NULL) return(NULL);

  /* Attach operations */

  /* constructors, destructors, and utility operations */
  v->ops->nvgetvectorid           = N_VGetVectorID_Cuda;
  v->ops->nvclone                 = N_VClone_Cuda;
  v->ops->nvcloneempty            = N_VCloneEmpty_Cuda;
  v->ops->nvdestroy               = N_VDestroy_Cuda;
  v->ops->nvspace                 = N_VSpace_Cuda;
  v->ops->nvgetlength             = N_VGetLength_Cuda;
  v->ops->nvgetarraypointer       = N_VGetHostArrayPointer_Cuda;
  v->ops->nvgetdevicearraypointer = N_VGetDeviceArrayPointer_Cuda;
  v->ops->nvsetarraypointer       = N_VSetHostArrayPointer_Cuda;

  /* standard vector operations */
  v->ops->nvlinearsum    = N_VLinearSum_Cuda;
  v->ops->nvconst        = N_VConst_Cuda;
  v->ops->nvprod         = N_VProd_Cuda;
  v->ops->nvdiv          = N_VDiv_Cuda;
  v->ops->nvscale        = N_VScale_Cuda;
  v->ops->nvabs          = N_VAbs_Cuda;
  v->ops->nvinv          = N_VInv_Cuda;
  v->ops->nvaddconst     = N_VAddConst_Cuda;
  v->ops->nvdotprod      = N_VDotProd_Cuda;
  v->ops->nvmaxnorm      = N_VMaxNorm_Cuda;
  v->ops->nvmin          = N_VMin_Cuda;
  v->ops->nvl1norm       = N_VL1Norm_Cuda;
  v->ops->nvinvtest      = N_VInvTest_Cuda;
  v->ops->nvconstrmask   = N_VConstrMask_Cuda;
  v->ops->nvminquotient  = N_VMinQuotient_Cuda;
  v->ops->nvwrmsnormmask = N_VWrmsNormMask_Cuda;
  v->ops->nvwrmsnorm     = N_VWrmsNorm_Cuda;
  v->ops->nvwl2norm      = N_VWL2Norm_Cuda;
  v->ops->nvcompare      = N_VCompare_Cuda;

  /* fused and vector array operations are disabled (NULL) by default */

  /* local reduction operations */
  v->ops->nvdotprodlocal     = N_VDotProd_Cuda;
  v->ops->nvmaxnormlocal     = N_VMaxNorm_Cuda;
  v->ops->nvminlocal         = N_VMin_Cuda;
  v->ops->nvl1normlocal      = N_VL1Norm_Cuda;
  v->ops->nvinvtestlocal     = N_VInvTest_Cuda;
  v->ops->nvconstrmasklocal  = N_VConstrMask_Cuda;
  v->ops->nvminquotientlocal = N_VMinQuotient_Cuda;
  v->ops->nvwsqrsumlocal     = N_VWSqrSumLocal_Cuda;
  v->ops->nvwsqrsummasklocal = N_VWSqrSumMaskLocal_Cuda;

  /* single buffer reduction operations */
  v->ops->nvdotprodmultilocal = N_VDotProdMulti_Cuda;

  /* XBraid interface operations */
  v->ops->nvbufsize   = N_VBufSize_Cuda;
  v->ops->nvbufpack   = N_VBufPack_Cuda;
  v->ops->nvbufunpack = N_VBufUnpack_Cuda;

  /* print operation for debugging */
  v->ops->nvprint     = N_VPrint_Cuda;
  v->ops->nvprintfile = N_VPrintFile_Cuda;

  /* Create content */

  v->content = (N_VectorContent_Cuda) malloc(sizeof(_N_VectorContent_Cuda));
  if (v->content == NULL)
  {
    N_VDestroy(v);
    return(NULL);
  }

  NVEC_CUDA_CONTENT(v)->priv = malloc(sizeof(_N_PrivateVectorContent_Cuda));
  if (NVEC_CUDA_CONTENT(v)->priv == NULL)
  {
    N_VDestroy(v);
    return(NULL);
  }

  // Initialize content
  NVEC_CUDA_CONTENT(v)->length             = 0;
  NVEC_CUDA_CONTENT(v)->host_data          = NULL;
  NVEC_CUDA_CONTENT(v)->device_data        = NULL;
  NVEC_CUDA_CONTENT(v)->stream_exec_policy = NULL;
  NVEC_CUDA_CONTENT(v)->reduce_exec_policy = NULL;
  NVEC_CUDA_CONTENT(v)->mem_helper         = NULL;
  NVEC_CUDA_CONTENT(v)->own_helper         = SUNFALSE;

  // Initialize private content
  NVEC_CUDA_PRIVATE(v)->use_managed_mem      = SUNFALSE;
  NVEC_CUDA_PRIVATE(v)->device_counter       = NULL;
  NVEC_CUDA_PRIVATE(v)->reduce_buffer_dev    = NULL;
  NVEC_CUDA_PRIVATE(v)->reduce_buffer_host   = NULL;
  NVEC_CUDA_PRIVATE(v)->reduce_buffer_bytes  = 0;
  NVEC_CUDA_PRIVATE(v)->fused_buffer_dev     = NULL;
  NVEC_CUDA_PRIVATE(v)->fused_buffer_host    = NULL;
  NVEC_CUDA_PRIVATE(v)->fused_buffer_bytes   = 0;
  NVEC_CUDA_PRIVATE(v)->fused_buffer_offset  = 0;

  return(v);
}

N_Vector N_VNew_Cuda(sunindextype length, SUNContext sunctx)
{
  N_Vector v;

  v = NULL;
  v = N_VNewEmpty_Cuda(sunctx);
  if (v == NULL) return(NULL);

  NVEC_CUDA_CONTENT(v)->length             = length;
  NVEC_CUDA_CONTENT(v)->mem_helper         = SUNMemoryHelper_Cuda(sunctx);
  NVEC_CUDA_CONTENT(v)->stream_exec_policy = DEFAULT_STREAMING_EXECPOLICY.clone();
  NVEC_CUDA_CONTENT(v)->reduce_exec_policy = DEFAULT_REDUCTION_EXECPOLICY.clone();
  NVEC_CUDA_CONTENT(v)->own_helper         = SUNTRUE;
  NVEC_CUDA_PRIVATE(v)->use_managed_mem    = SUNFALSE;

  if (NVEC_CUDA_MEMHELP(v) == NULL)
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VNew_Cuda: memory helper is NULL\n");
    N_VDestroy(v);
    return(NULL);
  }

  if (AllocateData(v))
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VNew_Cuda: AllocateData returned nonzero\n");
    N_VDestroy(v);
    return(NULL);
  }

  return(v);
}

N_Vector N_VNewWithMemHelp_Cuda(sunindextype length, booleantype use_managed_mem, SUNMemoryHelper helper, SUNContext sunctx)
{
  N_Vector v;

  if (helper == NULL)
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VNewWithMemHelp_Cuda: helper is NULL\n");
    return(NULL);
  }

  if (!SUNMemoryHelper_ImplementsRequiredOps(helper))
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VNewWithMemHelp_Cuda: helper doesn't implement all required ops\n");
    return(NULL);
  }

  v = NULL;
  v = N_VNewEmpty_Cuda(sunctx);
  if (v == NULL) return(NULL);

  NVEC_CUDA_CONTENT(v)->length             = length;
  NVEC_CUDA_CONTENT(v)->mem_helper         = helper;
  NVEC_CUDA_CONTENT(v)->stream_exec_policy = DEFAULT_STREAMING_EXECPOLICY.clone();
  NVEC_CUDA_CONTENT(v)->reduce_exec_policy = DEFAULT_REDUCTION_EXECPOLICY.clone();
  NVEC_CUDA_CONTENT(v)->own_helper         = SUNFALSE;
  NVEC_CUDA_PRIVATE(v)->use_managed_mem    = use_managed_mem;

  if (AllocateData(v))
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VNewWithMemHelp_Cuda: AllocateData returned nonzero\n");
    N_VDestroy(v);
    return(NULL);
  }

  return(v);
}

N_Vector N_VNewManaged_Cuda(sunindextype length, SUNContext sunctx)
{
  N_Vector v;

  v = NULL;
  v = N_VNewEmpty_Cuda(sunctx);
  if (v == NULL) return(NULL);

  NVEC_CUDA_CONTENT(v)->length             = length;
  NVEC_CUDA_CONTENT(v)->stream_exec_policy = DEFAULT_STREAMING_EXECPOLICY.clone();
  NVEC_CUDA_CONTENT(v)->reduce_exec_policy = DEFAULT_REDUCTION_EXECPOLICY.clone();
  NVEC_CUDA_CONTENT(v)->mem_helper         = SUNMemoryHelper_Cuda(sunctx);
  NVEC_CUDA_CONTENT(v)->own_helper         = SUNTRUE;
  NVEC_CUDA_PRIVATE(v)->use_managed_mem    = SUNTRUE;

  if (NVEC_CUDA_MEMHELP(v) == NULL)
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VNewManaged_Cuda: memory helper is NULL\n");
    N_VDestroy(v);
    return(NULL);
  }

  if (AllocateData(v))
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VNewManaged_Cuda: AllocateData returned nonzero\n");
    N_VDestroy(v);
    return(NULL);
  }

  return(v);
}

N_Vector N_VMake_Cuda(sunindextype length, realtype *h_vdata, realtype *d_vdata, SUNContext sunctx)
{
  N_Vector v;

  if (h_vdata == NULL || d_vdata == NULL) return(NULL);

  v = NULL;
  v = N_VNewEmpty_Cuda(sunctx);
  if (v == NULL) return(NULL);

  NVEC_CUDA_CONTENT(v)->length             = length;
  NVEC_CUDA_CONTENT(v)->host_data          = SUNMemoryHelper_Wrap(h_vdata, SUNMEMTYPE_HOST);
  NVEC_CUDA_CONTENT(v)->device_data        = SUNMemoryHelper_Wrap(d_vdata, SUNMEMTYPE_DEVICE);
  NVEC_CUDA_CONTENT(v)->stream_exec_policy = DEFAULT_STREAMING_EXECPOLICY.clone();
  NVEC_CUDA_CONTENT(v)->reduce_exec_policy = DEFAULT_REDUCTION_EXECPOLICY.clone();
  NVEC_CUDA_CONTENT(v)->mem_helper         = SUNMemoryHelper_Cuda(sunctx);
  NVEC_CUDA_CONTENT(v)->own_helper         = SUNTRUE;
  NVEC_CUDA_PRIVATE(v)->use_managed_mem    = SUNFALSE;

  if (NVEC_CUDA_MEMHELP(v) == NULL)
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VMake_Cuda: memory helper is NULL\n");
    N_VDestroy(v);
    return(NULL);
  }

  if (NVEC_CUDA_CONTENT(v)->device_data == NULL ||
      NVEC_CUDA_CONTENT(v)->host_data == NULL)
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VMake_Cuda: SUNMemoryHelper_Wrap returned NULL\n");
    N_VDestroy(v);
    return(NULL);
  }

  return(v);
}

N_Vector N_VMakeManaged_Cuda(sunindextype length, realtype *vdata, SUNContext sunctx)
{
  N_Vector v;

  if (vdata == NULL) return(NULL);

  v = NULL;
  v = N_VNewEmpty_Cuda(sunctx);
  if (v == NULL) return(NULL);

  NVEC_CUDA_CONTENT(v)->length             = length;
  NVEC_CUDA_CONTENT(v)->host_data          = SUNMemoryHelper_Wrap(vdata, SUNMEMTYPE_UVM);
  NVEC_CUDA_CONTENT(v)->device_data        = SUNMemoryHelper_Alias(NVEC_CUDA_CONTENT(v)->host_data);
  NVEC_CUDA_CONTENT(v)->stream_exec_policy = DEFAULT_STREAMING_EXECPOLICY.clone();
  NVEC_CUDA_CONTENT(v)->reduce_exec_policy = DEFAULT_REDUCTION_EXECPOLICY.clone();
  NVEC_CUDA_CONTENT(v)->mem_helper         = SUNMemoryHelper_Cuda(sunctx);
  NVEC_CUDA_CONTENT(v)->own_helper         = SUNTRUE;
  NVEC_CUDA_PRIVATE(v)->use_managed_mem    = SUNTRUE;

  if (NVEC_CUDA_MEMHELP(v) == NULL)
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VMakeManaged_Cuda: memory helper is NULL\n");
    N_VDestroy(v);
    return(NULL);
  }

  if (NVEC_CUDA_CONTENT(v)->device_data == NULL ||
      NVEC_CUDA_CONTENT(v)->host_data == NULL)
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VMakeManaged_Cuda: SUNMemoryHelper_Wrap returned NULL\n");
    N_VDestroy(v);
    return(NULL);
  }

  return(v);
}

/* ----------------------------------------------------------------------------
 * Set pointer to the raw host data. Does not free the existing pointer.
 */

void N_VSetHostArrayPointer_Cuda(realtype* h_vdata, N_Vector v)
{
  if (N_VIsManagedMemory_Cuda(v))
  {
    if (NVEC_CUDA_CONTENT(v)->host_data)
    {
      NVEC_CUDA_CONTENT(v)->host_data->ptr = (void*) h_vdata;
      NVEC_CUDA_CONTENT(v)->device_data->ptr = (void*) h_vdata;
    }
    else
    {
      NVEC_CUDA_CONTENT(v)->host_data = SUNMemoryHelper_Wrap((void*) h_vdata, SUNMEMTYPE_UVM);
      NVEC_CUDA_CONTENT(v)->device_data = SUNMemoryHelper_Alias(NVEC_CUDA_CONTENT(v)->host_data);
    }
  }
  else
  {
    if (NVEC_CUDA_CONTENT(v)->host_data)
    {
      NVEC_CUDA_CONTENT(v)->host_data->ptr = (void*) h_vdata;
    }
    else
    {
      NVEC_CUDA_CONTENT(v)->host_data = SUNMemoryHelper_Wrap((void*) h_vdata, SUNMEMTYPE_HOST);
    }
  }
}

/* ----------------------------------------------------------------------------
 * Set pointer to the raw device data
 */

void N_VSetDeviceArrayPointer_Cuda(realtype* d_vdata, N_Vector v)
{
  if (N_VIsManagedMemory_Cuda(v))
  {
    if (NVEC_CUDA_CONTENT(v)->device_data)
    {
      NVEC_CUDA_CONTENT(v)->device_data->ptr = (void*) d_vdata;
      NVEC_CUDA_CONTENT(v)->host_data->ptr = (void*) d_vdata;
    }
    else
    {
      NVEC_CUDA_CONTENT(v)->device_data = SUNMemoryHelper_Wrap((void*) d_vdata, SUNMEMTYPE_UVM);
      NVEC_CUDA_CONTENT(v)->host_data = SUNMemoryHelper_Alias(NVEC_CUDA_CONTENT(v)->device_data);
    }
  }
  else
  {
    if (NVEC_CUDA_CONTENT(v)->device_data)
    {
      NVEC_CUDA_CONTENT(v)->device_data->ptr = (void*) d_vdata;
    }
    else
    {
      NVEC_CUDA_CONTENT(v)->device_data = SUNMemoryHelper_Wrap((void*) d_vdata, SUNMEMTYPE_DEVICE);
    }
  }
}

/* ----------------------------------------------------------------------------
 * Return a flag indicating if the memory for the vector data is managed
 */

booleantype N_VIsManagedMemory_Cuda(N_Vector x)
{
  return NVEC_CUDA_PRIVATE(x)->use_managed_mem;
}

int N_VSetKernelExecPolicy_Cuda(N_Vector x,
                                SUNCudaExecPolicy* stream_exec_policy,
                                SUNCudaExecPolicy* reduce_exec_policy)
{
  if (x == NULL) return(-1);

  /* Delete the old policies */
  delete NVEC_CUDA_CONTENT(x)->stream_exec_policy;
  delete NVEC_CUDA_CONTENT(x)->reduce_exec_policy;

  /* Reset the policy if it is null */

  if (stream_exec_policy == NULL)
    NVEC_CUDA_CONTENT(x)->stream_exec_policy = DEFAULT_STREAMING_EXECPOLICY.clone();
  else
    NVEC_CUDA_CONTENT(x)->stream_exec_policy = stream_exec_policy->clone();

  if (reduce_exec_policy == NULL)
    NVEC_CUDA_CONTENT(x)->reduce_exec_policy = DEFAULT_REDUCTION_EXECPOLICY.clone();
  else
    NVEC_CUDA_CONTENT(x)->reduce_exec_policy = reduce_exec_policy->clone();

  return(0);
}

/* ----------------------------------------------------------------------------
 * Copy vector data to the device
 */

void N_VCopyToDevice_Cuda(N_Vector x)
{
  int copy_fail;

  copy_fail = SUNMemoryHelper_CopyAsync(NVEC_CUDA_MEMHELP(x),
                                        NVEC_CUDA_CONTENT(x)->device_data,
                                        NVEC_CUDA_CONTENT(x)->host_data,
                                        NVEC_CUDA_MEMSIZE(x),
                                        (void*) NVEC_CUDA_STREAM(x));

  if (copy_fail)
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VCopyToDevice_Cuda: SUNMemoryHelper_CopyAsync returned nonzero\n");
  }

  /* we synchronize with respect to the host, but only in this stream */
  SUNDIALS_CUDA_VERIFY(hipStreamSynchronize(*NVEC_CUDA_STREAM(x)));
}

/* ----------------------------------------------------------------------------
 * Copy vector data from the device to the host
 */

void N_VCopyFromDevice_Cuda(N_Vector x)
{
  int copy_fail;

  copy_fail = SUNMemoryHelper_CopyAsync(NVEC_CUDA_MEMHELP(x),
                                        NVEC_CUDA_CONTENT(x)->host_data,
                                        NVEC_CUDA_CONTENT(x)->device_data,
                                        NVEC_CUDA_MEMSIZE(x),
                                        (void*) NVEC_CUDA_STREAM(x));

  if (copy_fail)
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VCopyFromDevice_Cuda: SUNMemoryHelper_CopyAsync returned nonzero\n");
  }

  /* we synchronize with respect to the host, but only in this stream */
  SUNDIALS_CUDA_VERIFY(hipStreamSynchronize(*NVEC_CUDA_STREAM(x)));
}

/* ----------------------------------------------------------------------------
 * Function to print the a CUDA-based vector to stdout
 */

void N_VPrint_Cuda(N_Vector x)
{
  N_VPrintFile_Cuda(x, stdout);
}

/* ----------------------------------------------------------------------------
 * Function to print the a CUDA-based vector to outfile
 */

void N_VPrintFile_Cuda(N_Vector x, FILE *outfile)
{
  sunindextype i;

#ifdef SUNDIALS_DEBUG_PRINTVEC
  N_VCopyFromDevice_Cuda(x);
#endif

  for (i = 0; i < NVEC_CUDA_CONTENT(x)->length; i++) {
#if defined(SUNDIALS_EXTENDED_PRECISION)
    fprintf(outfile, "%35.32Le\n", NVEC_CUDA_HDATAp(x)[i]);
#elif defined(SUNDIALS_DOUBLE_PRECISION)
    fprintf(outfile, "%19.16e\n", NVEC_CUDA_HDATAp(x)[i]);
#else
    fprintf(outfile, "%11.8e\n", NVEC_CUDA_HDATAp(x)[i]);
#endif
  }
  fprintf(outfile, "\n");

  return;
}


/*
 * -----------------------------------------------------------------
 * implementation of vector operations
 * -----------------------------------------------------------------
 */

N_Vector N_VCloneEmpty_Cuda(N_Vector w)
{
  N_Vector v;

  if (w == NULL) return(NULL);

  /* Create vector */
  v = NULL;
  v = N_VNewEmpty_Cuda(w->sunctx);
  if (v == NULL) return(NULL);

  /* Attach operations */
  if (N_VCopyOps(w, v)) { N_VDestroy(v); return(NULL); }

  /* Set content */
  NVEC_CUDA_CONTENT(v)->length          = NVEC_CUDA_CONTENT(w)->length;
  NVEC_CUDA_PRIVATE(v)->use_managed_mem = NVEC_CUDA_PRIVATE(w)->use_managed_mem;

  return(v);
}

N_Vector N_VClone_Cuda(N_Vector w)
{
  N_Vector v;

  v = NULL;
  v = N_VCloneEmpty_Cuda(w);
  if (v == NULL) return(NULL);

  NVEC_CUDA_MEMHELP(v) = SUNMemoryHelper_Clone(NVEC_CUDA_MEMHELP(w));
  NVEC_CUDA_CONTENT(v)->own_helper = SUNTRUE;
  NVEC_CUDA_CONTENT(v)->stream_exec_policy = NVEC_CUDA_CONTENT(w)->stream_exec_policy->clone();
  NVEC_CUDA_CONTENT(v)->reduce_exec_policy = NVEC_CUDA_CONTENT(w)->reduce_exec_policy->clone();

  if (NVEC_CUDA_MEMHELP(v) == NULL)
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VClone_Cuda: SUNMemoryHelper_Clone returned NULL\n");
    N_VDestroy(v);
    return(NULL);
  }

  if (AllocateData(v))
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VClone_Cuda: AllocateData returned nonzero\n");
    N_VDestroy(v);
    return(NULL);
  }

  return(v);
}

void N_VDestroy_Cuda(N_Vector v)
{
  N_VectorContent_Cuda vc;
  N_PrivateVectorContent_Cuda vcp;

  if (v == NULL) return;

  /* free ops structure */
  if (v->ops != NULL)
  {
    free(v->ops);
    v->ops = NULL;
  }

  /* extract content */
  vc = NVEC_CUDA_CONTENT(v);
  if (vc == NULL)
  {
    free(v);
    v = NULL;
    return;
  }

  /* free private content */
  vcp = (N_PrivateVectorContent_Cuda) vc->priv;
  if (vcp != NULL)
  {
    /* free items in private content */
    FreeDeviceCounter(v);
    FreeReductionBuffer(v);
    FusedBuffer_Free(v);
    free(vcp);
    vc->priv = NULL;
  }

  /* free items in content */
  if (NVEC_CUDA_MEMHELP(v))
  {
    SUNMemoryHelper_Dealloc(NVEC_CUDA_MEMHELP(v), vc->host_data, (void*) NVEC_CUDA_STREAM(v));
    vc->host_data = NULL;
    SUNMemoryHelper_Dealloc(NVEC_CUDA_MEMHELP(v), vc->device_data, (void*) NVEC_CUDA_STREAM(v));
    vc->device_data = NULL;
    if (vc->own_helper) SUNMemoryHelper_Destroy(vc->mem_helper);
    vc->mem_helper = NULL;
  }

  /* we can delete the exec policies now that we are done with the streams */
  delete vc->stream_exec_policy;
  delete vc->reduce_exec_policy;

  /* free content struct */
  free(vc);

  /* free vector */
  free(v);

  return;
}

void N_VSpace_Cuda(N_Vector X, sunindextype *lrw, sunindextype *liw)
{
  *lrw = NVEC_CUDA_CONTENT(X)->length;
  *liw = 2;
}

void N_VConst_Cuda(realtype a, N_Vector X)
{
  size_t grid, block, shMemSize;
  hipStream_t stream;

  if (GetKernelParameters(X, false, grid, block, shMemSize, stream))
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VConst_Cuda: GetKernelParameters returned nonzero\n");
  }

  setConstKernel<<<grid, block, shMemSize, stream>>>
  (
    a,
    NVEC_CUDA_DDATAp(X),
    NVEC_CUDA_CONTENT(X)->length
  );
  PostKernelLaunch();
}

void N_VLinearSum_Cuda(realtype a, N_Vector X, realtype b, N_Vector Y, N_Vector Z)
{
  size_t grid, block, shMemSize;
  hipStream_t stream;

  if (GetKernelParameters(X, false, grid, block, shMemSize, stream))
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VLinearSum_Cuda: GetKernelParameters returned nonzero\n");
  }

  linearSumKernel<<<grid, block, shMemSize, stream>>>
  (
    a,
    NVEC_CUDA_DDATAp(X),
    b,
    NVEC_CUDA_DDATAp(Y),
    NVEC_CUDA_DDATAp(Z),
    NVEC_CUDA_CONTENT(X)->length
  );
  PostKernelLaunch();
}

void N_VProd_Cuda(N_Vector X, N_Vector Y, N_Vector Z)
{
  size_t grid, block, shMemSize;
  hipStream_t stream;

  if (GetKernelParameters(X, false, grid, block, shMemSize, stream))
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VProd_Cuda: GetKernelParameters returned nonzero\n");
  }


  prodKernel<<<grid, block, shMemSize, stream>>>
  (
    NVEC_CUDA_DDATAp(X),
    NVEC_CUDA_DDATAp(Y),
    NVEC_CUDA_DDATAp(Z),
    NVEC_CUDA_CONTENT(X)->length
  );
  PostKernelLaunch();
}

void N_VDiv_Cuda(N_Vector X, N_Vector Y, N_Vector Z)
{
  size_t grid, block, shMemSize;
  hipStream_t stream;

  if (GetKernelParameters(X, false, grid, block, shMemSize, stream))
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VDiv_Cuda: GetKernelParameters returned nonzero\n");
  }

  divKernel<<<grid, block, shMemSize, stream>>>
  (
    NVEC_CUDA_DDATAp(X),
    NVEC_CUDA_DDATAp(Y),
    NVEC_CUDA_DDATAp(Z),
    NVEC_CUDA_CONTENT(X)->length
  );
  PostKernelLaunch();
}

void N_VScale_Cuda(realtype a, N_Vector X, N_Vector Z)
{
  size_t grid, block, shMemSize;
  hipStream_t stream;

  if (GetKernelParameters(X, false, grid, block, shMemSize, stream))
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VScale_Cuda: GetKernelParameters returned nonzero\n");
  }

  scaleKernel<<<grid, block, shMemSize, stream>>>
  (
    a,
    NVEC_CUDA_DDATAp(X),
    NVEC_CUDA_DDATAp(Z),
    NVEC_CUDA_CONTENT(X)->length
  );
  PostKernelLaunch();
}

void N_VAbs_Cuda(N_Vector X, N_Vector Z)
{
  size_t grid, block, shMemSize;
  hipStream_t stream;

  if (GetKernelParameters(X, false, grid, block, shMemSize, stream))
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VAbs_Cuda: GetKernelParameters returned nonzero\n");
  }

  absKernel<<<grid, block, shMemSize, stream>>>
  (
    NVEC_CUDA_DDATAp(X),
    NVEC_CUDA_DDATAp(Z),
    NVEC_CUDA_CONTENT(X)->length
  );
  PostKernelLaunch();
}

void N_VInv_Cuda(N_Vector X, N_Vector Z)
{
  size_t grid, block, shMemSize;
  hipStream_t stream;

  if (GetKernelParameters(X, false, grid, block, shMemSize, stream))
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VInv_Cuda: GetKernelParameters returned nonzero\n");
  }

  invKernel<<<grid, block, shMemSize, stream>>>
  (
    NVEC_CUDA_DDATAp(X),
    NVEC_CUDA_DDATAp(Z),
    NVEC_CUDA_CONTENT(X)->length
  );
  PostKernelLaunch();
}

void N_VAddConst_Cuda(N_Vector X, realtype b, N_Vector Z)
{
  size_t grid, block, shMemSize;
  hipStream_t stream;

  if (GetKernelParameters(X, false, grid, block, shMemSize, stream))
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VAddConst_Cuda: GetKernelParameters returned nonzero\n");
  }

  addConstKernel<<<grid, block, shMemSize, stream>>>
  (
    b,
    NVEC_CUDA_DDATAp(X),
    NVEC_CUDA_DDATAp(Z),
    NVEC_CUDA_CONTENT(X)->length
  );
  PostKernelLaunch();
}

realtype N_VDotProd_Cuda(N_Vector X, N_Vector Y)
{
  bool atomic;
  size_t grid, block, shMemSize;
  hipStream_t stream;

  realtype gpu_result = ZERO;

  if (GetKernelParameters(X, true, grid, block, shMemSize, stream, atomic))
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VDotProd_Cuda: GetKernelParameters returned nonzero\n");
  }

  // When using atomic reductions, we only need one output value
  const size_t buffer_size = atomic ? 1 : grid;
  if (InitializeReductionBuffer(X, gpu_result, buffer_size))
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VDotProd_Cuda: InitializeReductionBuffer returned nonzero\n");
  }

  if (atomic)
  {
    dotProdKernel<realtype, sunindextype, GridReducerAtomic><<<grid, block, shMemSize, stream>>>
    (
      NVEC_CUDA_DDATAp(X),
      NVEC_CUDA_DDATAp(Y),
      NVEC_CUDA_DBUFFERp(X),
      NVEC_CUDA_CONTENT(X)->length,
      nullptr
    );
  }
  else
  {
    dotProdKernel<realtype, sunindextype, GridReducerLDS><<<grid, block, shMemSize, stream>>>
    (
      NVEC_CUDA_DDATAp(X),
      NVEC_CUDA_DDATAp(Y),
      NVEC_CUDA_DBUFFERp(X),
      NVEC_CUDA_CONTENT(X)->length,
      NVEC_CUDA_DCOUNTERp(X)
    );
  }
  PostKernelLaunch();

  // Get result from the GPU
  CopyReductionBufferFromDevice(X);
  gpu_result = NVEC_CUDA_HBUFFERp(X)[0];

  return gpu_result;
}

realtype N_VMaxNorm_Cuda(N_Vector X)
{
  bool atomic;
  size_t grid, block, shMemSize;
  hipStream_t stream;

  realtype gpu_result = ZERO;

  if (GetKernelParameters(X, true, grid, block, shMemSize, stream, atomic))
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VMaxNorm_Cuda: GetKernelParameters returned nonzero\n");
  }

  // When using atomic reductions, we only need one output value
  const size_t buffer_size = atomic ? 1 : grid;
  if (InitializeReductionBuffer(X, gpu_result, buffer_size))
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VMaxNorm_Cuda: InitializeReductionBuffer returned nonzero\n");
  }

  if (atomic)
  {
    maxNormKernel<realtype, sunindextype, GridReducerAtomic><<<grid, block, shMemSize, stream>>>
    (
      NVEC_CUDA_DDATAp(X),
      NVEC_CUDA_DBUFFERp(X),
      NVEC_CUDA_CONTENT(X)->length,
      nullptr
    );
  }
  else
  {
    maxNormKernel<realtype, sunindextype, GridReducerLDS><<<grid, block, shMemSize, stream>>>
    (
      NVEC_CUDA_DDATAp(X),
      NVEC_CUDA_DBUFFERp(X),
      NVEC_CUDA_CONTENT(X)->length,
      NVEC_CUDA_DCOUNTERp(X)
    );
  }

  PostKernelLaunch();

  // Finish reduction on CPU if there are less than two blocks of data left.
  CopyReductionBufferFromDevice(X);
  gpu_result = NVEC_CUDA_HBUFFERp(X)[0];

  return gpu_result;
}

realtype N_VWSqrSumLocal_Cuda(N_Vector X, N_Vector W)
{
  bool atomic;
  size_t grid, block, shMemSize;
  hipStream_t stream;

  realtype gpu_result = ZERO;

  if (GetKernelParameters(X, true, grid, block, shMemSize, stream, atomic))
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VWSqrSumLocal_Cuda: GetKernelParameters returned nonzero\n");
  }

  const size_t buffer_size = atomic ? 1 : grid;
  if (InitializeReductionBuffer(X, gpu_result, buffer_size))
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VWSqrSumLocal_Cuda: InitializeReductionBuffer returned nonzero\n");
  }

  if (atomic)
  {
    wL2NormSquareKernel<realtype, sunindextype, GridReducerAtomic><<<grid, block, shMemSize, stream>>>
    (
      NVEC_CUDA_DDATAp(X),
      NVEC_CUDA_DDATAp(W),
      NVEC_CUDA_DBUFFERp(X),
      NVEC_CUDA_CONTENT(X)->length,
      nullptr
    );
  }
  else
  {
    wL2NormSquareKernel<realtype, sunindextype, GridReducerLDS><<<grid, block, shMemSize, stream>>>
    (
      NVEC_CUDA_DDATAp(X),
      NVEC_CUDA_DDATAp(W),
      NVEC_CUDA_DBUFFERp(X),
      NVEC_CUDA_CONTENT(X)->length,
      NVEC_CUDA_DCOUNTERp(X)
    );
  }

  PostKernelLaunch();

  // Get result from the GPU
  CopyReductionBufferFromDevice(X);
  gpu_result = NVEC_CUDA_HBUFFERp(X)[0];

  return gpu_result;
}

realtype N_VWrmsNorm_Cuda(N_Vector X, N_Vector W)
{
  const realtype sum = N_VWSqrSumLocal_Cuda(X, W);
  return std::sqrt(sum/NVEC_CUDA_CONTENT(X)->length);
}

realtype N_VWSqrSumMaskLocal_Cuda(N_Vector X, N_Vector W, N_Vector Id)
{
  bool atomic;
  size_t grid, block, shMemSize;
  hipStream_t stream;

  realtype gpu_result = ZERO;

  if (GetKernelParameters(X, true, grid, block, shMemSize, stream, atomic))
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VWSqrSumMaskLocal_Cuda: GetKernelParameters returned nonzero\n");
  }

  const size_t buffer_size = atomic ? 1 : grid;
  if (InitializeReductionBuffer(X, gpu_result, buffer_size))
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VWSqrSumMaskLocal_Cuda: InitializeReductionBuffer returned nonzero\n");
  }

  if (atomic)
  {
    wL2NormSquareMaskKernel<realtype, sunindextype, GridReducerAtomic><<<grid, block, shMemSize, stream>>>
    (
      NVEC_CUDA_DDATAp(X),
      NVEC_CUDA_DDATAp(W),
      NVEC_CUDA_DDATAp(Id),
      NVEC_CUDA_DBUFFERp(X),
      NVEC_CUDA_CONTENT(X)->length,
      nullptr
    );
  }
  else
  {
    wL2NormSquareMaskKernel<realtype, sunindextype, GridReducerLDS><<<grid, block, shMemSize, stream>>>
    (
      NVEC_CUDA_DDATAp(X),
      NVEC_CUDA_DDATAp(W),
      NVEC_CUDA_DDATAp(Id),
      NVEC_CUDA_DBUFFERp(X),
      NVEC_CUDA_CONTENT(X)->length,
      NVEC_CUDA_DCOUNTERp(X)
    );
  }

  PostKernelLaunch();

  // Get result from the GPU
  CopyReductionBufferFromDevice(X);
  gpu_result = NVEC_CUDA_HBUFFERp(X)[0];

  return gpu_result;
}

realtype N_VWrmsNormMask_Cuda(N_Vector X, N_Vector W, N_Vector Id)
{
  const realtype sum = N_VWSqrSumMaskLocal_Cuda(X, W, Id);
  return std::sqrt(sum/NVEC_CUDA_CONTENT(X)->length);
}

realtype N_VMin_Cuda(N_Vector X)
{
  bool atomic;
  size_t grid, block, shMemSize;
  hipStream_t stream;

  realtype gpu_result = std::numeric_limits<realtype>::max();

  if (GetKernelParameters(X, true, grid, block, shMemSize, stream, atomic))
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VMin_Cuda: GetKernelParameters returned nonzero\n");
  }

  const size_t buffer_size = atomic ? 1 : grid;
  if (InitializeReductionBuffer(X, gpu_result, buffer_size))
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VMin_Cuda: InitializeReductionBuffer returned nonzero\n");
  }

  if (atomic)
  {
    findMinKernel<realtype, sunindextype, GridReducerAtomic><<<grid, block, shMemSize, stream>>>
    (
      gpu_result,
      NVEC_CUDA_DDATAp(X),
      NVEC_CUDA_DBUFFERp(X),
      NVEC_CUDA_CONTENT(X)->length,
      nullptr
    );
  }
  else
  {
    findMinKernel<realtype, sunindextype, GridReducerLDS><<<grid, block, shMemSize, stream>>>
    (
      gpu_result,
      NVEC_CUDA_DDATAp(X),
      NVEC_CUDA_DBUFFERp(X),
      NVEC_CUDA_CONTENT(X)->length,
      NVEC_CUDA_DCOUNTERp(X)
    );
  }

  PostKernelLaunch();

  // Get result from the GPU
  CopyReductionBufferFromDevice(X);
  gpu_result = NVEC_CUDA_HBUFFERp(X)[0];

  return gpu_result;
}

realtype N_VWL2Norm_Cuda(N_Vector X, N_Vector W)
{
  const realtype sum = N_VWSqrSumLocal_Cuda(X, W);
  return std::sqrt(sum);
}

realtype N_VL1Norm_Cuda(N_Vector X)
{
  bool atomic;
  size_t grid, block, shMemSize;
  hipStream_t stream;

  realtype gpu_result = ZERO;

  if (GetKernelParameters(X, true, grid, block, shMemSize, stream, atomic))
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VL1Norm_Cuda: GetKernelParameters returned nonzero\n");
  }

  const size_t buffer_size = atomic ? 1 : grid;
  if (InitializeReductionBuffer(X, gpu_result, buffer_size))
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VL1Norm_Cuda: InitializeReductionBuffer returned nonzero\n");
  }

  if (atomic)
  {
    L1NormKernel<realtype, sunindextype, GridReducerAtomic><<<grid, block, shMemSize, stream>>>
    (
      NVEC_CUDA_DDATAp(X),
      NVEC_CUDA_DBUFFERp(X),
      NVEC_CUDA_CONTENT(X)->length,
      nullptr
    );
  }
  else
  {
    L1NormKernel<realtype, sunindextype, GridReducerLDS><<<grid, block, shMemSize, stream>>>
    (
      NVEC_CUDA_DDATAp(X),
      NVEC_CUDA_DBUFFERp(X),
      NVEC_CUDA_CONTENT(X)->length,
      NVEC_CUDA_DCOUNTERp(X)
    );
  }

  PostKernelLaunch();

  // Get result from the GPU
  CopyReductionBufferFromDevice(X);
  gpu_result = NVEC_CUDA_HBUFFERp(X)[0];

  return gpu_result;
}

void N_VCompare_Cuda(realtype c, N_Vector X, N_Vector Z)
{
  size_t grid, block, shMemSize;
  hipStream_t stream;

  if (GetKernelParameters(X, false, grid, block, shMemSize, stream))
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VCompare_Cuda: GetKernelParameters returned nonzero\n");
  }

  compareKernel<<<grid, block, shMemSize, stream>>>
  (
    c,
    NVEC_CUDA_DDATAp(X),
    NVEC_CUDA_DDATAp(Z),
    NVEC_CUDA_CONTENT(X)->length
  );
  PostKernelLaunch();
}

booleantype N_VInvTest_Cuda(N_Vector X, N_Vector Z)
{
  bool atomic;
  size_t grid, block, shMemSize;
  hipStream_t stream;

  realtype gpu_result = ZERO;

  if (GetKernelParameters(X, true, grid, block, shMemSize, stream, atomic))
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VInvTest_Cuda: GetKernelParameters returned nonzero\n");
  }

  const size_t buffer_size = atomic ? 1 : grid;
  if (InitializeReductionBuffer(X, gpu_result, buffer_size))
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VInvTest_Cuda: InitializeReductionBuffer returned nonzero\n");
  }

  if (atomic)
  {
    invTestKernel<realtype, sunindextype, GridReducerAtomic><<<grid, block, shMemSize, stream>>>
    (
      NVEC_CUDA_DDATAp(X),
      NVEC_CUDA_DDATAp(Z),
      NVEC_CUDA_DBUFFERp(X),
      NVEC_CUDA_CONTENT(X)->length,
      nullptr
    );
  }
  else
  {
    invTestKernel<realtype, sunindextype, GridReducerLDS><<<grid, block, shMemSize, stream>>>
    (
      NVEC_CUDA_DDATAp(X),
      NVEC_CUDA_DDATAp(Z),
      NVEC_CUDA_DBUFFERp(X),
      NVEC_CUDA_CONTENT(X)->length,
      NVEC_CUDA_DCOUNTERp(X)
    );
  }

  PostKernelLaunch();

  // Get result from the GPU
  CopyReductionBufferFromDevice(X);
  gpu_result = NVEC_CUDA_HBUFFERp(X)[0];

  return (gpu_result < HALF);
}

booleantype N_VConstrMask_Cuda(N_Vector C, N_Vector X, N_Vector M)
{
  bool atomic;
  size_t grid, block, shMemSize;
  hipStream_t stream;

  realtype gpu_result = ZERO;

  if (GetKernelParameters(X, true, grid, block, shMemSize, stream, atomic))
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VConstrMask_Cuda: GetKernelParameters returned nonzero\n");
  }

  const size_t buffer_size = atomic ? 1 : grid;
  if (InitializeReductionBuffer(X, gpu_result, buffer_size))
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VConstrMask_Cuda: InitializeReductionBuffer returned nonzero\n");
  }

  if (atomic)
  {
    constrMaskKernel<realtype, sunindextype, GridReducerAtomic><<<grid, block, shMemSize, stream>>>
    (
      NVEC_CUDA_DDATAp(C),
      NVEC_CUDA_DDATAp(X),
      NVEC_CUDA_DDATAp(M),
      NVEC_CUDA_DBUFFERp(X),
      NVEC_CUDA_CONTENT(X)->length,
      nullptr
    );
  }
  else
  {
    constrMaskKernel<realtype, sunindextype, GridReducerLDS><<<grid, block, shMemSize, stream>>>
    (
      NVEC_CUDA_DDATAp(C),
      NVEC_CUDA_DDATAp(X),
      NVEC_CUDA_DDATAp(M),
      NVEC_CUDA_DBUFFERp(X),
      NVEC_CUDA_CONTENT(X)->length,
      NVEC_CUDA_DCOUNTERp(X)
    );
  }

  PostKernelLaunch();

  // Get result from the GPU
  CopyReductionBufferFromDevice(X);
  gpu_result = NVEC_CUDA_HBUFFERp(X)[0];

  return (gpu_result < HALF);
}

realtype N_VMinQuotient_Cuda(N_Vector num, N_Vector denom)
{
  bool atomic;
  size_t grid, block, shMemSize;
  hipStream_t stream;

  realtype gpu_result = std::numeric_limits<realtype>::max();;

  if (GetKernelParameters(num, true, grid, block, shMemSize, stream, atomic))
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VMinQuotient_Cuda: GetKernelParameters returned nonzero\n");
  }

  const size_t buffer_size = atomic ? 1 : grid;
  if (InitializeReductionBuffer(num, gpu_result, buffer_size))
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VMinQuotient_Cuda: InitializeReductionBuffer returned nonzero\n");
  }

  if (atomic)
  {
    minQuotientKernel<realtype, sunindextype, GridReducerAtomic><<<grid, block, shMemSize, stream>>>
    (
      gpu_result,
      NVEC_CUDA_DDATAp(num),
      NVEC_CUDA_DDATAp(denom),
      NVEC_CUDA_DBUFFERp(num),
      NVEC_CUDA_CONTENT(num)->length,
      nullptr
    );
  }
  else
  {
    minQuotientKernel<realtype, sunindextype, GridReducerLDS><<<grid, block, shMemSize, stream>>>
    (
      gpu_result,
      NVEC_CUDA_DDATAp(num),
      NVEC_CUDA_DDATAp(denom),
      NVEC_CUDA_DBUFFERp(num),
      NVEC_CUDA_CONTENT(num)->length,
      NVEC_CUDA_DCOUNTERp(num)
    );
  }

  PostKernelLaunch();

  // Get result from the GPU
  CopyReductionBufferFromDevice(num);
  gpu_result = NVEC_CUDA_HBUFFERp(num)[0];

  return gpu_result;
}


/*
 * -----------------------------------------------------------------
 * fused vector operations
 * -----------------------------------------------------------------
 */


int N_VLinearCombination_Cuda(int nvec, realtype* c, N_Vector* X, N_Vector z)
{
  // Fused op workspace shortcuts
  realtype*  cdata = NULL;
  realtype** xdata = NULL;

  // Setup the fused op workspace
  if (FusedBuffer_Init(z, nvec, nvec))
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VLinearCombination_Cuda: FusedBuffer_Init returned nonzero\n");
    return -1;
  }

  if (FusedBuffer_CopyRealArray(z, c, nvec, &cdata))
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VLinearCombination_Cuda: FusedBuffer_CopyRealArray returned nonzero\n");
    return -1;
  }

  if (FusedBuffer_CopyPtrArray1D(z, X, nvec, &xdata))
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VLinearCombination_Cuda: FusedBuffer_CopyPtrArray1D returned nonzero\n");
    return -1;
  }

  if (FusedBuffer_CopyToDevice(z))
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VLinearCombination_Cuda: FusedBuffer_CopyToDevice returned nonzero\n");
    return -1;
  }

  // Set kernel parameters and launch
  size_t grid, block, shMemSize;
  hipStream_t stream;

  if (GetKernelParameters(X[0], false, grid, block, shMemSize, stream))
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VLinearCombination_Cuda: GetKernelParameters returned nonzero\n");
    return -1;
  }

  linearCombinationKernel<<<grid, block, shMemSize, stream>>>
  (
    nvec,
    cdata,
    xdata,
    NVEC_CUDA_DDATAp(z),
    NVEC_CUDA_CONTENT(z)->length
  );
  PostKernelLaunch();

  return 0;
}


int N_VScaleAddMulti_Cuda(int nvec, realtype* c, N_Vector x, N_Vector* Y,
                          N_Vector* Z)
{
  // Shortcuts to the fused op workspace
  realtype*  cdata = NULL;
  realtype** ydata = NULL;
  realtype** zdata = NULL;

  // Setup the fused op workspace
  if (FusedBuffer_Init(x, nvec, 2 * nvec))
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VScaleAddMulti_Cuda: FusedBuffer_Init returned nonzero\n");
    return -1;
  }

  if (FusedBuffer_CopyRealArray(x, c, nvec, &cdata))
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VScaleAddMulti_Cuda: FusedBuffer_CopyRealArray returned nonzero\n");
    return -1;
  }

  if (FusedBuffer_CopyPtrArray1D(x, Y, nvec, &ydata))
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VScaleAddMulti_Cuda: FusedBuffer_CopyPtrArray1D returned nonzero\n");
    return -1;
  }

  if (FusedBuffer_CopyPtrArray1D(x, Z, nvec, &zdata))
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VScaleAddMulti_Cuda: FusedBuffer_CopyPtrArray1D returned nonzero\n");
    return -1;
  }

  if (FusedBuffer_CopyToDevice(x))
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VScaleAddMulti_Cuda: FusedBuffer_CopyToDevice returned nonzero\n");
    return -1;
  }

  // Set kernel parameters
  size_t grid, block, shMemSize;
  hipStream_t stream;

  if (GetKernelParameters(x, false, grid, block, shMemSize, stream))
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VScaleAddMulti_Cuda: GetKernelParameters returned nonzero\n");
    return -1;
  }

  scaleAddMultiKernel<<<grid, block, shMemSize, stream>>>
  (
    nvec,
    cdata,
    NVEC_CUDA_DDATAp(x),
    ydata,
    zdata,
    NVEC_CUDA_CONTENT(x)->length
  );
  PostKernelLaunch();

  return 0;
}


int N_VDotProdMulti_Cuda(int nvec, N_Vector x, N_Vector* Y, realtype* dots)
{
  // Fused op workspace shortcuts
  realtype** ydata = NULL;

  // Setup the fused op workspace
  if (FusedBuffer_Init(x, 0, nvec))
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VDotProdMulti_Cuda: FusedBuffer_Init returned nonzero\n");
    return -1;
  }

  if (FusedBuffer_CopyPtrArray1D(x, Y, nvec, &ydata))
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VDotProdMulti_Cuda: FusedBuffer_CopyPtrArray1D returned nonzero\n");
    return -1;
  }

  if (FusedBuffer_CopyToDevice(x))
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VDotProdMulti_Cuda: FusedBuffer_CopyToDevice returned nonzero\n");
    return -1;
  }

  // Set kernel parameters
  size_t grid, block, shMemSize;
  hipStream_t stream;

  if (GetKernelParameters(x, false, grid, block, shMemSize, stream))
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VDotProdMulti_Cuda: GetKernelParameters returned nonzero\n");
    return -1;
  }
  grid = nvec;

  if (InitializeReductionBuffer(x, ZERO, nvec))
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VDotProd_Cuda: InitializeReductionBuffer returned nonzero\n");
  }

  dotProdMultiKernel<realtype, sunindextype, GridReducerAtomic><<<grid, block, shMemSize, stream>>>
  (
    nvec,
    NVEC_CUDA_DDATAp(x),
    ydata,
    NVEC_CUDA_DBUFFERp(x),
    NVEC_CUDA_CONTENT(x)->length
  );

  PostKernelLaunch();

  // Get result from the GPU
  CopyReductionBufferFromDevice(x, nvec);
  for (int i = 0; i < nvec; ++i)
  {
    dots[i] = NVEC_CUDA_HBUFFERp(x)[i];
  }

  return 0;
}


/*
 * -----------------------------------------------------------------------------
 * vector array operations
 * -----------------------------------------------------------------------------
 */


int N_VLinearSumVectorArray_Cuda(int nvec,
                                 realtype a, N_Vector* X,
                                 realtype b, N_Vector* Y,
                                 N_Vector* Z)
{
  // Shortcuts to the fused op workspace
  realtype** xdata = NULL;
  realtype** ydata = NULL;
  realtype** zdata = NULL;

  // Setup the fused op workspace
  if (FusedBuffer_Init(Z[0], 0, 3 * nvec))
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VLinearSumVectorArray_Cuda: FusedBuffer_Init returned nonzero\n");
    return -1;
  }

  if (FusedBuffer_CopyPtrArray1D(Z[0], X, nvec, &xdata))
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VLinearSumVectorArray_Cuda: FusedBuffer_CopyPtrArray1D returned nonzero\n");
    return -1;
  }

  if (FusedBuffer_CopyPtrArray1D(Z[0], Y, nvec, &ydata))
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VLinearSumVectorArray_Cuda: FusedBuffer_CopyPtrArray1D returned nonzero\n");
    return -1;
  }

  if (FusedBuffer_CopyPtrArray1D(Z[0], Z, nvec, &zdata))
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VLinearSumVectorArray_Cuda: FusedBuffer_CopyPtrArray1D returned nonzero\n");
    return -1;
  }

  if (FusedBuffer_CopyToDevice(Z[0]))
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VLinaerSumVectorArray_Cuda: FusedBuffer_CopyToDevice returned nonzero\n");
    return -1;
  }

  // Set kernel parameters
  size_t grid, block, shMemSize;
  hipStream_t stream;

  if (GetKernelParameters(Z[0], false, grid, block, shMemSize, stream))
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VLinearSumVectorArray_Cuda: GetKernelParameters returned nonzero\n");
    return -1;
  }

  linearSumVectorArrayKernel<<<grid, block, shMemSize, stream>>>
  (
    nvec,
    a,
    xdata,
    b,
    ydata,
    zdata,
    NVEC_CUDA_CONTENT(Z[0])->length
  );
  PostKernelLaunch();

  return 0;
}


int N_VScaleVectorArray_Cuda(int nvec, realtype* c, N_Vector* X, N_Vector* Z)
{
  // Shortcuts to the fused op workspace arrays
  realtype*  cdata = NULL;
  realtype** xdata = NULL;
  realtype** zdata = NULL;

  // Setup the fused op workspace
  if (FusedBuffer_Init(Z[0], nvec, 2 * nvec))
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VScaleVectorArray_Cuda: FusedBuffer_Init returned nonzero\n");
    return -1;
  }

  if (FusedBuffer_CopyRealArray(Z[0], c, nvec, &cdata))
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VScaleVectorArray_Cuda: FusedBuffer_CopyRealArray returned nonzero\n");
    return -1;
  }

  if (FusedBuffer_CopyPtrArray1D(Z[0], X, nvec, &xdata))
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VScaleVectorArray_Cuda: FusedBuffer_CopyPtrArray1D returned nonzero\n");
    return -1;
  }

  if (FusedBuffer_CopyPtrArray1D(Z[0], Z, nvec, &zdata))
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VScaleVectorArray_Cuda: FusedBuffer_CopyPtrArray1D returned nonzero\n");
    return -1;
  }

  if (FusedBuffer_CopyToDevice(Z[0]))
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VScaleVectorArray_Cuda: FusedBuffer_CopyToDevice returned nonzero\n");
    return -1;
  }

  // Set kernel parameters
  size_t grid, block, shMemSize;
  hipStream_t stream;

  if (GetKernelParameters(Z[0], false, grid, block, shMemSize, stream))
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VScaleVectorArray_Cuda: GetKernelParameters returned nonzero\n");
    return -1;
  }

  scaleVectorArrayKernel<<<grid, block, shMemSize, stream>>>
  (
    nvec,
    cdata,
    xdata,
    zdata,
    NVEC_CUDA_CONTENT(Z[0])->length
  );
  PostKernelLaunch();

  return 0;
}


int N_VConstVectorArray_Cuda(int nvec, realtype c, N_Vector* Z)
{
  // Shortcuts to the fused op workspace arrays
  realtype** zdata = NULL;

  // Setup the fused op workspace
  if (FusedBuffer_Init(Z[0], 0, nvec))
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VConstVectorArray_Cuda: FusedBuffer_Init returned nonzero\n");
    return -1;
  }

  if (FusedBuffer_CopyPtrArray1D(Z[0], Z, nvec, &zdata))
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VConstVectorArray_Cuda: FusedBuffer_CopyPtrArray1D returned nonzero\n");
    return -1;
  }

  if (FusedBuffer_CopyToDevice(Z[0]))
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VConstVectorArray_Cuda: FusedBuffer_CopyToDevice returned nonzero\n");
    return -1;
  }

  // Set kernel parameters
  size_t grid, block, shMemSize;
  hipStream_t stream;

  if (GetKernelParameters(Z[0], false, grid, block, shMemSize, stream))
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VConstVectorArray_Cuda: GetKernelParameters returned nonzero\n");
    return -1;
  }

  constVectorArrayKernel<<<grid, block, shMemSize, stream>>>
  (
    nvec,
    c,
    zdata,
    NVEC_CUDA_CONTENT(Z[0])->length
  );
  PostKernelLaunch();

  return 0;
}


int N_VWrmsNormVectorArray_Cuda(int nvec, N_Vector* X, N_Vector* W,
                                realtype* norms)
{
  // Fused op workspace shortcuts
  realtype** xdata = NULL;
  realtype** wdata = NULL;

  // Setup the fused op workspace
  if (FusedBuffer_Init(W[0], 0, 2 * nvec))
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VWrmsNormVectorArray_Cuda: FusedBuffer_Init returned nonzero\n");
    return -1;
  }

  if (FusedBuffer_CopyPtrArray1D(W[0], X, nvec, &xdata))
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VWrmsNormVectorArray_Cuda: FusedBuffer_CopyPtrArray1D returned nonzero\n");
    return -1;
  }

  if (FusedBuffer_CopyPtrArray1D(W[0], W, nvec, &wdata))
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VWrmsNormVectorArray_Cuda: FusedBuffer_CopyPtrArray1D returned nonzero\n");
    return -1;
  }

  if (FusedBuffer_CopyToDevice(W[0]))
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VWrmsNormVectorArray_Cuda: FusedBuffer_CopyToDevice returned nonzero\n");
    return -1;
  }

  if (InitializeReductionBuffer(W[0], ZERO, nvec))
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VWrmsNormVectorArray_Cuda: InitializeReductionBuffer returned nonzero\n");
  }

  // Set kernel parameters
  size_t grid, block, shMemSize;
  hipStream_t stream;

  if (GetKernelParameters(W[0], true, grid, block, shMemSize, stream))
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VWrmsNormVectorArray_Cuda: GetKernelParameters returned nonzero\n");
    return -1;
  }
  grid = nvec;

  wL2NormSquareVectorArrayKernel<realtype, sunindextype, GridReducerAtomic><<<grid, block, shMemSize, stream>>>
  (
    nvec,
    xdata,
    wdata,
    NVEC_CUDA_DBUFFERp(W[0]),
    NVEC_CUDA_CONTENT(W[0])->length
  );
  PostKernelLaunch();

  // Get result from the GPU
  CopyReductionBufferFromDevice(W[0], nvec);
  for (int i = 0; i < nvec; ++i)
  {
    norms[i] = std::sqrt(NVEC_CUDA_HBUFFERp(W[0])[i] /
                         NVEC_CUDA_CONTENT(W[0])->length);
  }

  return 0;
}


int N_VWrmsNormMaskVectorArray_Cuda(int nvec, N_Vector* X, N_Vector* W,
                                    N_Vector id, realtype* norms)
{
  // Fused op workspace shortcuts
  realtype** xdata = NULL;
  realtype** wdata = NULL;

  // Setup the fused op workspace
  if (FusedBuffer_Init(W[0], 0, 2 * nvec))
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VWrmsNormVectorArray_Cuda: FusedBuffer_Init returned nonzero\n");
    return -1;
  }

  if (FusedBuffer_CopyPtrArray1D(W[0], X, nvec, &xdata))
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VWrmsNormVectorArray_Cuda: FusedBuffer_CopyPtrArray1D returned nonzero\n");
    return -1;
  }

  if (FusedBuffer_CopyPtrArray1D(W[0], W, nvec, &wdata))
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VWrmsNormVectorArray_Cuda: FusedBuffer_CopyPtrArray1D returned nonzero\n");
    return -1;
  }

  if (FusedBuffer_CopyToDevice(W[0]))
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VWrmsNormVectorArray_Cuda: FusedBuffer_CopyToDevice returned nonzero\n");
    return -1;
  }

  if (InitializeReductionBuffer(W[0], ZERO, nvec))
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VWrmsNormVectorArray_Cuda: InitializeReductionBuffer returned nonzero\n");
  }

  // Set kernel parameters
  size_t grid, block, shMemSize;
  hipStream_t stream;

  if (GetKernelParameters(W[0], true, grid, block, shMemSize, stream))
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VWrmsNormMaskVectorArray_Cuda: GetKernelParameters returned nonzero\n");
    return -1;
  }
  grid = nvec;

  wL2NormSquareMaskVectorArrayKernel<realtype, sunindextype, GridReducerAtomic><<<grid, block, shMemSize, stream>>>
  (
    nvec,
    xdata,
    wdata,
    NVEC_CUDA_DDATAp(id),
    NVEC_CUDA_DBUFFERp(W[0]),
    NVEC_CUDA_CONTENT(W[0])->length
  );
  PostKernelLaunch();

  // Get result from the GPU
  CopyReductionBufferFromDevice(W[0], nvec);
  for (int i = 0; i < nvec; ++i)
  {
    norms[i] = std::sqrt(NVEC_CUDA_HBUFFERp(W[0])[i] /
                         NVEC_CUDA_CONTENT(W[0])->length);
  }

  return 0;
}


int N_VScaleAddMultiVectorArray_Cuda(int nvec, int nsum, realtype* c,
                                     N_Vector* X, N_Vector** Y, N_Vector** Z)
{
  // Shortcuts to the fused op workspace
  realtype*  cdata = NULL;
  realtype** xdata = NULL;
  realtype** ydata = NULL;
  realtype** zdata = NULL;

  // Setup the fused op workspace
  if (FusedBuffer_Init(X[0], nsum, nvec + 2 * nvec * nsum))
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VScaleAddMultiArray_Cuda: FusedBuffer_Init returned nonzero\n");
    return -1;
  }

  if (FusedBuffer_CopyRealArray(X[0], c, nsum, &cdata))
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VScaleAddMultiArray_Cuda: FusedBuffer_CopyRealArray returned nonzero\n");
    return -1;
  }

  if (FusedBuffer_CopyPtrArray1D(X[0], X, nvec, &xdata))
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VScaleAddMultiVectorArray_Cuda: FusedBuffer_CopyPtrArray1D returned nonzero\n");
    return -1;
  }

  if (FusedBuffer_CopyPtrArray2D(X[0], Y, nvec, nsum, &ydata))
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VScaleAddMultiVectorArray_Cuda: FusedBuffer_CopyPtrArray2D returned nonzero\n");
    return -1;
  }

  if (FusedBuffer_CopyPtrArray2D(X[0], Z, nvec, nsum, &zdata))
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VScaleAddMultiVectorArray_Cuda: FusedBuffer_CopyPtrArray2D returned nonzero\n");
    return -1;
  }

  if (FusedBuffer_CopyToDevice(X[0]))
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VScaleVectorArray_Cuda: FusedBuffer_CopyToDevice returned nonzero\n");
    return -1;
  }

  // Set kernel parameters
  size_t grid, block, shMemSize;
  hipStream_t stream;

  if (GetKernelParameters(X[0], false, grid, block, shMemSize, stream))
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VScaleAddMultiVectorArray_Cuda: GetKernelParameters returned nonzero\n");
    return -1;
  }

  scaleAddMultiVectorArrayKernel<<<grid, block, shMemSize, stream>>>
  (
    nvec,
    nsum,
    cdata,
    xdata,
    ydata,
    zdata,
    NVEC_CUDA_CONTENT(X[0])->length
  );
  PostKernelLaunch();

  return 0;
}


int N_VLinearCombinationVectorArray_Cuda(int nvec, int nsum, realtype* c,
                                         N_Vector** X, N_Vector* Z)
{
  // Shortcuts to the fused op workspace arrays
  realtype*  cdata = NULL;
  realtype** xdata = NULL;
  realtype** zdata = NULL;

  // Setup the fused op workspace
  if (FusedBuffer_Init(Z[0], nsum, nvec + nvec * nsum))
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VLinearCombinationVectorArray_Cuda: FusedBuffer_Init returned nonzero\n");
    return -1;
  }

  if (FusedBuffer_CopyRealArray(Z[0], c, nsum, &cdata))
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VLinearCombinationVectorArray_Cuda: FusedBuffer_CopyRealArray returned nonzero\n");
    return -1;
  }

  if (FusedBuffer_CopyPtrArray2D(Z[0], X, nvec, nsum, &xdata))
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VLinearCombinationVectorArray_Cuda: FusedBuffer_CopyPtrArray2D returned nonzero\n");
    return -1;
  }

  if (FusedBuffer_CopyPtrArray1D(Z[0], Z, nvec, &zdata))
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VLinearCombinationVectorArray_Cuda: FusedBuffer_CopyPtrArray1D returned nonzero\n");
    return -1;
  }

  if (FusedBuffer_CopyToDevice(Z[0]))
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VLinearCombinationVectorArray_Cuda: FusedBuffer_CopyToDevice returned nonzero\n");
    return -1;
  }

  // Set kernel parameters
  size_t grid, block, shMemSize;
  hipStream_t stream;

  if (GetKernelParameters(Z[0], false, grid, block, shMemSize, stream))
  {
    SUNDIALS_DEBUG_PRINT("ERROR in N_VLinearCombinationVectorArray_Cuda: GetKernelParameters returned nonzero\n");
    return -1;
  }

  linearCombinationVectorArrayKernel<<<grid, block, shMemSize, stream>>>
  (
    nvec,
    nsum,
    cdata,
    xdata,
    zdata,
    NVEC_CUDA_CONTENT(Z[0])->length
  );
  PostKernelLaunch();

  return 0;
}


/*
 * -----------------------------------------------------------------
 * OPTIONAL XBraid interface operations
 * -----------------------------------------------------------------
 */


int N_VBufSize_Cuda(N_Vector x, sunindextype *size)
{
  if (x == NULL) return(-1);
  *size = (sunindextype)NVEC_CUDA_MEMSIZE(x);
  return(0);
}


int N_VBufPack_Cuda(N_Vector x, void *buf)
{
  int copy_fail = 0;
  hipError_t cuerr;

  if (x == NULL || buf == NULL) return(-1);

  SUNMemory buf_mem = SUNMemoryHelper_Wrap(buf, SUNMEMTYPE_HOST);
  if (buf_mem == NULL) return(-1);

  copy_fail = SUNMemoryHelper_CopyAsync(NVEC_CUDA_MEMHELP(x),
                                        buf_mem,
                                        NVEC_CUDA_CONTENT(x)->device_data,
                                        NVEC_CUDA_MEMSIZE(x),
                                        (void*) NVEC_CUDA_STREAM(x));

  /* we synchronize with respect to the host, but only in this stream */
  cuerr = hipStreamSynchronize(*NVEC_CUDA_STREAM(x));

  SUNMemoryHelper_Dealloc(NVEC_CUDA_MEMHELP(x), buf_mem, (void*) NVEC_CUDA_STREAM(x));

  return (!SUNDIALS_CUDA_VERIFY(cuerr) || copy_fail ? -1 : 0);
}


int N_VBufUnpack_Cuda(N_Vector x, void *buf)
{
  int copy_fail = 0;
  hipError_t cuerr;

  if (x == NULL || buf == NULL) return(-1);

  SUNMemory buf_mem = SUNMemoryHelper_Wrap(buf, SUNMEMTYPE_HOST);
  if (buf_mem == NULL) return(-1);

  copy_fail = SUNMemoryHelper_CopyAsync(NVEC_CUDA_MEMHELP(x),
                                        NVEC_CUDA_CONTENT(x)->device_data,
                                        buf_mem,
                                        NVEC_CUDA_MEMSIZE(x),
                                        (void*) NVEC_CUDA_STREAM(x));

  /* we synchronize with respect to the host, but only in this stream */
  cuerr = hipStreamSynchronize(*NVEC_CUDA_STREAM(x));

  SUNMemoryHelper_Dealloc(NVEC_CUDA_MEMHELP(x), buf_mem, (void*) NVEC_CUDA_STREAM(x));

  return (!SUNDIALS_CUDA_VERIFY(cuerr) || copy_fail ? -1 : 0);
}


/*
 * -----------------------------------------------------------------
 * Enable / Disable fused and vector array operations
 * -----------------------------------------------------------------
 */


int N_VEnableFusedOps_Cuda(N_Vector v, booleantype tf)
{
  /* check that vector is non-NULL */
  if (v == NULL) return(-1);

  /* check that ops structure is non-NULL */
  if (v->ops == NULL) return(-1);

  if (tf)
  {
    /* enable all fused vector operations */
    v->ops->nvlinearcombination = N_VLinearCombination_Cuda;
    v->ops->nvscaleaddmulti     = N_VScaleAddMulti_Cuda;
    v->ops->nvdotprodmulti      = N_VDotProdMulti_Cuda;
    /* enable all vector array operations */
    v->ops->nvlinearsumvectorarray         = N_VLinearSumVectorArray_Cuda;
    v->ops->nvscalevectorarray             = N_VScaleVectorArray_Cuda;
    v->ops->nvconstvectorarray             = N_VConstVectorArray_Cuda;
    v->ops->nvwrmsnormvectorarray          = N_VWrmsNormVectorArray_Cuda;
    v->ops->nvwrmsnormmaskvectorarray      = N_VWrmsNormMaskVectorArray_Cuda;
    v->ops->nvscaleaddmultivectorarray     = N_VScaleAddMultiVectorArray_Cuda;
    v->ops->nvlinearcombinationvectorarray = N_VLinearCombinationVectorArray_Cuda;
    /* enable single buffer reduction operations */
    v->ops->nvdotprodmultilocal = N_VDotProdMulti_Cuda;
  }
  else
  {
    /* disable all fused vector operations */
    v->ops->nvlinearcombination = NULL;
    v->ops->nvscaleaddmulti     = NULL;
    v->ops->nvdotprodmulti      = NULL;
    /* disable all vector array operations */
    v->ops->nvlinearsumvectorarray         = NULL;
    v->ops->nvscalevectorarray             = NULL;
    v->ops->nvconstvectorarray             = NULL;
    v->ops->nvwrmsnormvectorarray          = NULL;
    v->ops->nvwrmsnormmaskvectorarray      = NULL;
    v->ops->nvscaleaddmultivectorarray     = NULL;
    v->ops->nvlinearcombinationvectorarray = NULL;
    /* disable single buffer reduction operations */
    v->ops->nvdotprodmultilocal = NULL;
  }

  /* return success */
  return(0);
}

int N_VEnableLinearCombination_Cuda(N_Vector v, booleantype tf)
{
  if (v == NULL) return -1;
  if (v->ops == NULL) return -1;
  v->ops->nvlinearcombination = tf ? N_VLinearCombination_Cuda : NULL;
  return 0;
}


int N_VEnableScaleAddMulti_Cuda(N_Vector v, booleantype tf)
{
  if (v == NULL) return -1;
  if (v->ops == NULL) return -1;
  v->ops->nvscaleaddmulti = tf ? N_VScaleAddMulti_Cuda : NULL;
  return 0;
}


int N_VEnableDotProdMulti_Cuda(N_Vector v, booleantype tf)
{
  if (v == NULL) return -1;
  if (v->ops == NULL) return -1;
  v->ops->nvdotprodmulti      = tf ? N_VDotProdMulti_Cuda : NULL;
  v->ops->nvdotprodmultilocal = tf ? N_VDotProdMulti_Cuda : NULL;
  return 0;
}


int N_VEnableLinearSumVectorArray_Cuda(N_Vector v, booleantype tf)
{
  if (v == NULL) return -1;
  if (v->ops == NULL) return -1;
  v->ops->nvlinearsumvectorarray = tf ? N_VLinearSumVectorArray_Cuda : NULL;
  return 0;
}


int N_VEnableScaleVectorArray_Cuda(N_Vector v, booleantype tf)
{
  if (v == NULL) return -1;
  if (v->ops == NULL) return -1;
  v->ops->nvscalevectorarray = tf ? N_VScaleVectorArray_Cuda : NULL;
  return 0;
}


int N_VEnableConstVectorArray_Cuda(N_Vector v, booleantype tf)
{
  if (v == NULL) return -1;
  if (v->ops == NULL) return -1;
  v->ops->nvconstvectorarray = tf ? N_VConstVectorArray_Cuda : NULL;
  return 0;
}


int N_VEnableWrmsNormVectorArray_Cuda(N_Vector v, booleantype tf)
{
  if (v == NULL) return -1;
  if (v->ops == NULL) return -1;
  v->ops->nvwrmsnormvectorarray = tf ? N_VWrmsNormVectorArray_Cuda : NULL;
  return 0;
}


int N_VEnableWrmsNormMaskVectorArray_Cuda(N_Vector v, booleantype tf)
{
  if (v == NULL) return -1;
  if (v->ops == NULL) return -1;
  v->ops->nvwrmsnormmaskvectorarray = tf ?
    N_VWrmsNormMaskVectorArray_Cuda : NULL;
  return 0;
}


int N_VEnableScaleAddMultiVectorArray_Cuda(N_Vector v, booleantype tf)
{
  if (v == NULL) return -1;
  if (v->ops == NULL) return -1;
  v->ops->nvscaleaddmultivectorarray = tf ?
    N_VScaleAddMultiVectorArray_Cuda : NULL;
  return 0;
}


int N_VEnableLinearCombinationVectorArray_Cuda(N_Vector v, booleantype tf)
{
  if (v == NULL) return -1;
  if (v->ops == NULL) return -1;
  v->ops->nvlinearcombinationvectorarray = tf ?
    N_VLinearCombinationVectorArray_Cuda : NULL;
  return 0;
}


} // extern "C"


/*
 * Private helper functions.
 */

static int AllocateData(N_Vector v)
{
  int alloc_fail = 0;
  N_VectorContent_Cuda vc = NVEC_CUDA_CONTENT(v);
  N_PrivateVectorContent_Cuda vcp = NVEC_CUDA_PRIVATE(v);

  if (N_VGetLength_Cuda(v) == 0) return(0);

  if (vcp->use_managed_mem)
  {
    alloc_fail = SUNMemoryHelper_Alloc(NVEC_CUDA_MEMHELP(v), &(vc->device_data),
                                       NVEC_CUDA_MEMSIZE(v), SUNMEMTYPE_UVM,
                                       (void*) NVEC_CUDA_STREAM(v));
    if (alloc_fail)
    {
      SUNDIALS_DEBUG_PRINT("ERROR in AllocateData: SUNMemoryHelper_Alloc failed for SUNMEMTYPE_UVM\n");
    }
    vc->host_data = SUNMemoryHelper_Alias(vc->device_data);
  }
  else
  {
    alloc_fail = SUNMemoryHelper_Alloc(NVEC_CUDA_MEMHELP(v), &(vc->host_data),
                                       NVEC_CUDA_MEMSIZE(v), SUNMEMTYPE_HOST,
                                       (void*) NVEC_CUDA_STREAM(v));
    if (alloc_fail)
    {
      SUNDIALS_DEBUG_PRINT("ERROR in AllocateData: SUNMemoryHelper_Alloc failed to alloc SUNMEMTYPE_HOST\n");
    }

    alloc_fail = SUNMemoryHelper_Alloc(NVEC_CUDA_MEMHELP(v), &(vc->device_data),
                                       NVEC_CUDA_MEMSIZE(v), SUNMEMTYPE_DEVICE,
                                       (void*) NVEC_CUDA_STREAM(v));
    if (alloc_fail)
    {
      SUNDIALS_DEBUG_PRINT("ERROR in AllocateData: SUNMemoryHelper_Alloc failed to alloc SUNMEMTYPE_DEVICE\n");
    }
  }

  return(alloc_fail ? -1 : 0);
}

/*
 * Initializes the internal buffer used for reductions.
 * If the buffer is already allocated, it will only be reallocated
 * if it is no longer large enough. This may occur if the length
 * of the vector is increased. The buffer is initialized to the
 * value given.
 */
static int InitializeReductionBuffer(N_Vector v, realtype value, size_t n)
{
  int         alloc_fail = 0;
  int         copy_fail  = 0;
  booleantype alloc_mem  = SUNFALSE;
  size_t      bytes      = n * sizeof(realtype);

  // Get the vector private memory structure
  N_PrivateVectorContent_Cuda vcp = NVEC_CUDA_PRIVATE(v);

  // Check if the existing reduction memory is not large enough
  if (vcp->reduce_buffer_bytes < bytes)
  {
    FreeReductionBuffer(v);
    alloc_mem = SUNTRUE;
  }

  if (alloc_mem)
  {
    // Allocate pinned memory on the host
    alloc_fail = SUNMemoryHelper_Alloc(NVEC_CUDA_MEMHELP(v),
                                       &(vcp->reduce_buffer_host), bytes,
                                       SUNMEMTYPE_PINNED, (void*) NVEC_CUDA_STREAM(v));
    if (alloc_fail)
    {
      SUNDIALS_DEBUG_PRINT("WARNING in InitializeReductionBuffer: SUNMemoryHelper_Alloc failed to alloc SUNMEMTYPE_PINNED, using SUNMEMTYPE_HOST instead\n");

      // If pinned alloc failed, allocate plain host memory
      alloc_fail = SUNMemoryHelper_Alloc(NVEC_CUDA_MEMHELP(v),
                                         &(vcp->reduce_buffer_host), bytes,
                                         SUNMEMTYPE_HOST, (void*) NVEC_CUDA_STREAM(v));
      if (alloc_fail)
      {
        SUNDIALS_DEBUG_PRINT("ERROR in InitializeReductionBuffer: SUNMemoryHelper_Alloc failed to alloc SUNMEMTYPE_HOST\n");
      }
    }

    // Allocate device memory
    alloc_fail = SUNMemoryHelper_Alloc(NVEC_CUDA_MEMHELP(v),
                                       &(vcp->reduce_buffer_dev), bytes,
                                       SUNMEMTYPE_DEVICE, (void*) NVEC_CUDA_STREAM(v));
    if (alloc_fail)
    {
      SUNDIALS_DEBUG_PRINT("ERROR in InitializeReductionBuffer: SUNMemoryHelper_Alloc failed to alloc SUNMEMTYPE_DEVICE\n");
    }
  }

  if (!alloc_fail)
  {
    // Store the size of the reduction memory buffer
    vcp->reduce_buffer_bytes = bytes;

    // Initialize the host memory with the value
    for (int i = 0; i < n; ++i)
      ((realtype*)vcp->reduce_buffer_host->ptr)[i] = value;

    // Initialize the device memory with the value
    copy_fail = SUNMemoryHelper_CopyAsync(NVEC_CUDA_MEMHELP(v),
                                          vcp->reduce_buffer_dev, vcp->reduce_buffer_host,
                                          bytes, (void*) NVEC_CUDA_STREAM(v));

    if (copy_fail)
    {
      SUNDIALS_DEBUG_PRINT("ERROR in InitializeReductionBuffer: SUNMemoryHelper_CopyAsync failed\n");
    }
  }

  return((alloc_fail || copy_fail) ? -1 : 0);
}

/* Free the reduction buffer
 */
static void FreeReductionBuffer(N_Vector v)
{
  N_PrivateVectorContent_Cuda vcp = NVEC_CUDA_PRIVATE(v);

  if (vcp == NULL) return;

  // Free device mem
  if (vcp->reduce_buffer_dev != NULL)
    SUNMemoryHelper_Dealloc(NVEC_CUDA_MEMHELP(v), vcp->reduce_buffer_dev,
                            (void*) NVEC_CUDA_STREAM(v));
  vcp->reduce_buffer_dev  = NULL;

  // Free host mem
  if (vcp->reduce_buffer_host != NULL)
    SUNMemoryHelper_Dealloc(NVEC_CUDA_MEMHELP(v), vcp->reduce_buffer_host,
                            (void*) NVEC_CUDA_STREAM(v));
  vcp->reduce_buffer_host = NULL;

  // Reset allocated memory size
  vcp->reduce_buffer_bytes = 0;
}

/* Copy the reduction buffer from the device to the host.
 */
static int CopyReductionBufferFromDevice(N_Vector v, size_t n)
{
  int copy_fail;
  hipError_t cuerr;

  copy_fail = SUNMemoryHelper_CopyAsync(NVEC_CUDA_MEMHELP(v),
                                        NVEC_CUDA_PRIVATE(v)->reduce_buffer_host,
                                        NVEC_CUDA_PRIVATE(v)->reduce_buffer_dev,
                                        n * sizeof(realtype),
                                        (void*) NVEC_CUDA_STREAM(v));

  if (copy_fail)
  {
    SUNDIALS_DEBUG_PRINT("ERROR in CopyReductionBufferFromDevice: SUNMemoryHelper_CopyAsync returned nonzero\n");
  }

  /* we synchronize with respect to the host, but only in this stream */
  cuerr = hipStreamSynchronize(*NVEC_CUDA_STREAM(v));
  return (!SUNDIALS_CUDA_VERIFY(cuerr) || copy_fail ? -1 : 0);
}


static int FusedBuffer_Init(N_Vector v, int nreal, int nptr)
{
  int         alloc_fail = 0;
  booleantype alloc_mem  = SUNFALSE;

  // pad buffer with single precision data
#if defined(SUNDIALS_SINGLE_PRECISION)
  size_t bytes = nreal * 2 * sizeof(realtype) + nptr * sizeof(realtype*);
#elif defined(SUNDIALS_DOUBLE_PRECISION)
  size_t bytes = nreal * sizeof(realtype) + nptr * sizeof(realtype*);
#else
#error Incompatible precision for CUDA
#endif

  // Get the vector private memory structure
  N_PrivateVectorContent_Cuda vcp = NVEC_CUDA_PRIVATE(v);

  // Check if the existing memory is not large enough
  if (vcp->fused_buffer_bytes < bytes)
  {
    FusedBuffer_Free(v);
    alloc_mem = SUNTRUE;
  }

  if (alloc_mem)
  {
    // Allocate pinned memory on the host
    alloc_fail = SUNMemoryHelper_Alloc(NVEC_CUDA_MEMHELP(v),
                                       &(vcp->fused_buffer_host), bytes,
                                       SUNMEMTYPE_PINNED, (void*) NVEC_CUDA_STREAM(v));
    if (alloc_fail)
    {
      SUNDIALS_DEBUG_PRINT("WARNING in FusedBuffer_Init: SUNMemoryHelper_Alloc failed to alloc SUNMEMTYPE_PINNED, using SUNMEMTYPE_HOST instead\n");

      // If pinned alloc failed, allocate plain host memory
      alloc_fail = SUNMemoryHelper_Alloc(NVEC_CUDA_MEMHELP(v),
                                         &(vcp->fused_buffer_host), bytes,
                                         SUNMEMTYPE_HOST, (void*) NVEC_CUDA_STREAM(v));
      if (alloc_fail)
      {
        SUNDIALS_DEBUG_PRINT("ERROR in FusedBuffer_Init: SUNMemoryHelper_Alloc failed to alloc SUNMEMTYPE_HOST\n");
        return -1;
      }
    }

    // Allocate device memory
    alloc_fail = SUNMemoryHelper_Alloc(NVEC_CUDA_MEMHELP(v),
                                       &(vcp->fused_buffer_dev), bytes,
                                       SUNMEMTYPE_DEVICE, (void*) NVEC_CUDA_STREAM(v));
    if (alloc_fail)
    {
      SUNDIALS_DEBUG_PRINT("ERROR in FusedBuffer_Init: SUNMemoryHelper_Alloc failed to alloc SUNMEMTYPE_DEVICE\n");
      return -1;
    }

    // Store the size of the fused op buffer
    vcp->fused_buffer_bytes = bytes;
  }

  // Reset the buffer offset
  vcp->fused_buffer_offset = 0;

  return 0;
}


static int FusedBuffer_CopyRealArray(N_Vector v, realtype *rdata, int nval,
                                     realtype **shortcut)
{
  // Get the vector private memory structure
  N_PrivateVectorContent_Cuda vcp = NVEC_CUDA_PRIVATE(v);

  // Check buffer space and fill the host buffer
  if (vcp->fused_buffer_offset >= vcp->fused_buffer_bytes)
  {
    SUNDIALS_DEBUG_PRINT("ERROR in FusedBuffer_CopyRealArray: Buffer offset is exceedes the buffer size\n");
    return -1;
  }

  realtype* h_buffer = (realtype*) ((char*)(vcp->fused_buffer_host->ptr) +
                                    vcp->fused_buffer_offset);

  for (int j = 0; j < nval; j++)
  {
    h_buffer[j] = rdata[j];
  }

  // Set shortcut to the device buffer and update offset
  *shortcut = (realtype*) ((char*)(vcp->fused_buffer_dev->ptr) +
                           vcp->fused_buffer_offset);

  // accounting for buffer padding
#if defined(SUNDIALS_SINGLE_PRECISION)
  vcp->fused_buffer_offset += nval * 2 * sizeof(realtype);
#elif defined(SUNDIALS_DOUBLE_PRECISION)
  vcp->fused_buffer_offset += nval * sizeof(realtype);
#else
#error Incompatible precision for CUDA
#endif

  return 0;
}


static int FusedBuffer_CopyPtrArray1D(N_Vector v, N_Vector *X, int nvec,
                                      realtype ***shortcut)
{
  // Get the vector private memory structure
  N_PrivateVectorContent_Cuda vcp = NVEC_CUDA_PRIVATE(v);

  // Check buffer space and fill the host buffer
  if (vcp->fused_buffer_offset >= vcp->fused_buffer_bytes)
  {
    SUNDIALS_DEBUG_PRINT("ERROR in FusedBuffer_CopyPtrArray1D: Buffer offset is exceedes the buffer size\n");
    return -1;
  }

  realtype** h_buffer = (realtype**) ((char*)(vcp->fused_buffer_host->ptr) +
                                      vcp->fused_buffer_offset);

  for (int j = 0; j < nvec; j++)
  {
    h_buffer[j] = NVEC_CUDA_DDATAp(X[j]);
  }

  // Set shortcut to the device buffer and update offset
  *shortcut = (realtype**) ((char*)(vcp->fused_buffer_dev->ptr) +
                            vcp->fused_buffer_offset);

  vcp->fused_buffer_offset += nvec * sizeof(realtype*);

  return 0;
}


static int FusedBuffer_CopyPtrArray2D(N_Vector v, N_Vector **X, int nvec,
                                      int nsum, realtype ***shortcut)
{
  // Get the vector private memory structure
  N_PrivateVectorContent_Cuda vcp = NVEC_CUDA_PRIVATE(v);

  // Check buffer space and fill the host buffer
  if (vcp->fused_buffer_offset >= vcp->fused_buffer_bytes)
  {
    SUNDIALS_DEBUG_PRINT("ERROR in FusedBuffer_CopyPtrArray2D: Buffer offset is exceedes the buffer size\n");
    return -1;
  }

  realtype** h_buffer = (realtype**) ((char*)(vcp->fused_buffer_host->ptr) +
                                      vcp->fused_buffer_offset);

  for (int j = 0; j < nvec; j++)
  {
    for (int k = 0; k < nsum; k++)
    {
      h_buffer[j * nsum + k] = NVEC_CUDA_DDATAp(X[k][j]);
    }
  }

  // Set shortcut to the device buffer and update offset
  *shortcut = (realtype**) ((char*)(vcp->fused_buffer_dev->ptr) +
                            vcp->fused_buffer_offset);

  // Update the offset
  vcp->fused_buffer_offset += nvec * nsum * sizeof(realtype*);

  return 0;
}


static int FusedBuffer_CopyToDevice(N_Vector v)
{
  // Get the vector private memory structure
  N_PrivateVectorContent_Cuda vcp = NVEC_CUDA_PRIVATE(v);

  // Copy the fused buffer to the device
  int copy_fail = SUNMemoryHelper_CopyAsync(NVEC_CUDA_MEMHELP(v),
                                            vcp->fused_buffer_dev,
                                            vcp->fused_buffer_host,
                                            vcp->fused_buffer_offset,
                                            (void*) NVEC_CUDA_STREAM(v));
  if (copy_fail)
  {
    SUNDIALS_DEBUG_PRINT("ERROR in FusedBuffer_CopyToDevice: SUNMemoryHelper_CopyAsync failed\n");
    return -1;
  }

  // Synchronize with respect to the host, but only in this stream
  SUNDIALS_CUDA_VERIFY(hipStreamSynchronize(*NVEC_CUDA_STREAM(v)));

  return 0;
}


static int FusedBuffer_Free(N_Vector v)
{
  N_PrivateVectorContent_Cuda vcp = NVEC_CUDA_PRIVATE(v);

  if (vcp == NULL) return 0;

  if (vcp->fused_buffer_host)
  {
    SUNMemoryHelper_Dealloc(NVEC_CUDA_MEMHELP(v),
                            vcp->fused_buffer_host, (void*) NVEC_CUDA_STREAM(v));
    vcp->fused_buffer_host = NULL;
  }

  if (vcp->fused_buffer_dev)
  {
    SUNMemoryHelper_Dealloc(NVEC_CUDA_MEMHELP(v),
                            vcp->fused_buffer_dev, (void*) NVEC_CUDA_STREAM(v));
    vcp->fused_buffer_dev = NULL;
  }

  vcp->fused_buffer_bytes  = 0;
  vcp->fused_buffer_offset = 0;

  return 0;
}

static int InitializeDeviceCounter(N_Vector v)
{
  int retval = 0;
  if (NVEC_CUDA_PRIVATE(v)->device_counter == NULL)
  {
    retval = SUNMemoryHelper_Alloc(NVEC_CUDA_MEMHELP(v),
                                   &(NVEC_CUDA_PRIVATE(v)->device_counter), sizeof(unsigned int),
                                   SUNMEMTYPE_DEVICE, (void*) NVEC_CUDA_STREAM(v));
  }
  hipMemsetAsync(NVEC_CUDA_DCOUNTERp(v), 0, sizeof(unsigned int), *NVEC_CUDA_STREAM(v));
  return retval;
}

static int FreeDeviceCounter(N_Vector v)
{
  int retval = 0;
  if (NVEC_CUDA_PRIVATE(v)->device_counter)
    retval = SUNMemoryHelper_Dealloc(NVEC_CUDA_MEMHELP(v), NVEC_CUDA_PRIVATE(v)->device_counter,
                                     (void*) NVEC_CUDA_STREAM(v));
  return retval;
}

/* Get the kernel launch parameters based on the kernel type (reduction or not),
 * using the appropriate kernel execution policy.
 */
static int GetKernelParameters(N_Vector v, booleantype reduction, size_t& grid,
                               size_t& block, size_t& shMemSize,
                               hipStream_t& stream, bool& atomic, size_t n)
{
  n = (n == 0) ? NVEC_CUDA_CONTENT(v)->length : n;
  if (reduction)
  {
    SUNCudaExecPolicy* reduce_exec_policy = NVEC_CUDA_CONTENT(v)->reduce_exec_policy;
    grid      = reduce_exec_policy->gridSize(n);
    block     = reduce_exec_policy->blockSize();
    shMemSize = 0;
    stream    = *(reduce_exec_policy->stream());
    atomic    = reduce_exec_policy->atomic();

    if (!atomic)
    {
      if (InitializeDeviceCounter(v))
      {
  #ifdef SUNDIALS_DEBUG
        throw std::runtime_error("SUNMemoryHelper_Alloc returned nonzero\n");
  #endif
        return(-1);
      }
    }

    if (block % sundials::cuda::WARP_SIZE)
    {
#ifdef SUNDIALS_DEBUG
      throw std::runtime_error("the block size must be a multiple must be of the CUDA warp size");
#endif
      return(-1);
    }
  }
  else
  {
    SUNCudaExecPolicy* stream_exec_policy = NVEC_CUDA_CONTENT(v)->stream_exec_policy;
    grid      = stream_exec_policy->gridSize(n);
    block     = stream_exec_policy->blockSize();
    shMemSize = 0;
    stream    = *(stream_exec_policy->stream());
    atomic    = false;
  }

  if (grid == 0)
  {
#ifdef SUNDIALS_DEBUG
    throw std::runtime_error("the grid size must be > 0");
#endif
    return(-1);
  }
  if (block == 0)
  {
#ifdef SUNDIALS_DEBUG
    throw std::runtime_error("the block size must be > 0");
#endif
    return(-1);
  }

  return(0);
}

static int GetKernelParameters(N_Vector v, booleantype reduction, size_t& grid,
                               size_t& block, size_t& shMemSize, hipStream_t& stream,
                               size_t n)
{
  bool atomic;
  return GetKernelParameters(v, reduction, grid, block, shMemSize, stream, atomic, n);
}

/* Should be called after a kernel launch.
 * If SUNDIALS_DEBUG_CUDA_LASTERROR is not defined, then the function does nothing.
 * If it is defined, the function will synchronize and check the last CUDA error.
 */
static void PostKernelLaunch()
{
#ifdef SUNDIALS_DEBUG_CUDA_LASTERROR
  hipDeviceSynchronize();
  SUNDIALS_CUDA_VERIFY(hipGetLastError());
#endif
}
