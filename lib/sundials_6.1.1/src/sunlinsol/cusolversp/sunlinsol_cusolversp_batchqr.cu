/* ----------------------------------------------------------------------------
 * Programmer(s): Cody J. Balos @ LLNL
 * ----------------------------------------------------------------------------
 * Based on work by Donald Wilcox @ LBNL
 * ----------------------------------------------------------------------------
 * SUNDIALS Copyright Start
 * Copyright (c) 2002-2022, Lawrence Livermore National Security
 * and Southern Methodist University.
 * All rights reserved.
 *
 * See the top-level LICENSE and NOTICE files for details.
 *
 * SPDX-License-Identifier: BSD-3-Clause
 * SUNDIALS Copyright End
 * ----------------------------------------------------------------------------
 * Implementation file for cuSolverSp batched QR SUNLinearSolver interface.
 * ----------------------------------------------------------------------------*/

#include <stdio.h>
#include <stdlib.h>

#include <sunmatrix/sunmatrix_cusparse.h>
#include <sunlinsol/sunlinsol_cusolversp_batchqr.h>

#include "sundials_cuda.h"
#include "sundials_debug.h"

#define ZERO RCONST(0.0)
#define ONE  RCONST(1.0)
#define TWO  RCONST(2.0)

/* macros for handling the different function names based on precision */
#if defined(SUNDIALS_DOUBLE_PRECISION)
#define _cusolverSpXcsrqrBufferInfoBatched cusolverSpDcsrqrBufferInfoBatched
#define _cusolverSpXcsrqrsvBatched cusolverSpDcsrqrsvBatched
#elif defined(SUNDIALS_SINGLE_PRECISION)
#define _cusolverSpXcsrqrBufferInfoBatched cusolverSpScsrqrBufferInfoBatched
#define _cusolverSpXcsrqrsvBatched cusolverSpScsrqrsvBatched
#endif

/*
 * -----------------------------------------------------------------
 * cuSolverSp solver structure accessibility macros:
 * -----------------------------------------------------------------
 */

#define SUN_CUSP_CONTENT(S)        ( (SUNLinearSolverContent_cuSolverSp_batchQR)(S->content) )
#define SUN_CUSP_QRWORKSPACE(S)    ( SUN_CUSP_CONTENT(S)->workspace )
#define SUN_CUSP_FIRSTFACTORIZE(S) ( SUN_CUSP_CONTENT(S)->first_factorize )
#define SUN_CUSP_LASTFLAG(S)       ( SUN_CUSP_CONTENT(S)->last_flag )
#define SUN_CUSOL_HANDLE(S)        ( SUN_CUSP_CONTENT(S)->cusolver_handle )
#define SUN_CUSP_DESC(S)           ( SUN_CUSP_CONTENT(S)->desc )
#define SUN_CUSP_QRINFO(S)         ( SUN_CUSP_CONTENT(S)->info )
#define SUN_CUSP_INTERNAL_SIZE(S)  ( SUN_CUSP_CONTENT(S)->internal_size )
#define SUN_CUSP_WORK_SIZE(S)      ( SUN_CUSP_CONTENT(S)->workspace_size )

/*
 * ----------------------------------------------------------------------------
 *  Implementations of exported functions.
 * ----------------------------------------------------------------------------
 */

SUNLinearSolver SUNLinSol_cuSolverSp_batchQR(N_Vector y, SUNMatrix A, hipsolverSpHandle_t cusol_handle, SUNContext sunctx)
{
  /* Check that required arguments are not NULL */
  if (y == NULL || A == NULL)
  {
    SUNDIALS_DEBUG_PRINT("ERROR in SUNLinSol_cuSolverSp_batchQR: y or A is null\n");
    return NULL;
  }

  /* Check compatibility with supplied SUNMatrix and N_Vector */
  if (SUNMatGetID(A) != SUNMATRIX_CUSPARSE || y->ops->nvgetdevicearraypointer == NULL)
  {
    SUNDIALS_DEBUG_PRINT("ERROR in SUNLinSol_cuSolverSp_batchQR: illegal type for y or A\n");
    return NULL;
  }

  /* Matrix and vector dimensions must agree */
  if (N_VGetLength(y) != SUNMatrix_cuSparse_Columns(A))
  {
    SUNDIALS_DEBUG_PRINT("ERROR in SUNLinSol_cuSolverSp_batchQR: matrix and vector dimensions don't agree\n");
    return NULL;
  }

  /* Create an empty linear solver */
  SUNLinearSolver S;

  S = NULL;
  S = SUNLinSolNewEmpty(sunctx);
  if (S == NULL)
  {
    return NULL;
  }

  /* Attach operations */
  S->ops->gettype    = SUNLinSolGetType_cuSolverSp_batchQR;
  S->ops->getid      = SUNLinSolGetID_cuSolverSp_batchQR;
  S->ops->initialize = SUNLinSolInitialize_cuSolverSp_batchQR;
  S->ops->setup      = SUNLinSolSetup_cuSolverSp_batchQR;
  S->ops->solve      = SUNLinSolSolve_cuSolverSp_batchQR;
  S->ops->lastflag   = SUNLinSolLastFlag_cuSolverSp_batchQR;
  S->ops->free       = SUNLinSolFree_cuSolverSp_batchQR;

  /* Create content */
  SUNLinearSolverContent_cuSolverSp_batchQR content;

  content = NULL;
  content = (SUNLinearSolverContent_cuSolverSp_batchQR) malloc(sizeof(*content));
  if (content == NULL)
  {
    SUNLinSolFree(S);
    return NULL;
  }

  /* Attach content */
  S->content = content;

  /* Fill content */
  content->last_flag       = SUNLS_SUCCESS;
  content->first_factorize = SUNTRUE;
  content->internal_size   = 0;
  content->workspace_size  = 0;
  content->cusolver_handle = cusol_handle;
  content->info            = NULL;
  content->workspace       = NULL;
  content->desc            = NULL;

  return S;
}

/*
 * -----------------------------------------------------------------
 * Implementation of accessor and setter functions.
 * -----------------------------------------------------------------
 */

void SUNLinSol_cuSolverSp_batchQR_GetDescription(SUNLinearSolver S, const char** desc)
{
  *desc = SUN_CUSP_DESC(S);
}

void SUNLinSol_cuSolverSp_batchQR_SetDescription(SUNLinearSolver S, const char* desc)
{
  SUN_CUSP_DESC(S) = desc;
}

void SUNLinSol_cuSolverSp_batchQR_GetDeviceSpace(SUNLinearSolver S,
                                                 size_t* cuSolverInternal,
                                                 size_t* cuSolverWorkspace)
{
  /* size is in bytes */
  *cuSolverInternal  = SUN_CUSP_INTERNAL_SIZE(S); /* buffer for Q and R factors */
  *cuSolverWorkspace = SUN_CUSP_WORK_SIZE(S); /* numerical factorization buffer */
}

/*
 * -----------------------------------------------------------------
 * Implementation of linear solver operations
 * -----------------------------------------------------------------
 */

SUNLinearSolver_Type SUNLinSolGetType_cuSolverSp_batchQR(SUNLinearSolver S)
{
  return(SUNLINEARSOLVER_DIRECT);
}

SUNLinearSolver_ID SUNLinSolGetID_cuSolverSp_batchQR(SUNLinearSolver S)
{
  return(SUNLINEARSOLVER_CUSOLVERSP_BATCHQR);
}

int SUNLinSolInitialize_cuSolverSp_batchQR(SUNLinearSolver S)
{
  SUN_CUSP_FIRSTFACTORIZE(S) = SUNTRUE;
  SUN_CUSP_LASTFLAG(S) = SUNLS_SUCCESS;
  return(SUN_CUSP_LASTFLAG(S));
}

int SUNLinSolSetup_cuSolverSp_batchQR(SUNLinearSolver S, SUNMatrix A)
{
  int blockrows, blockcols, blocknnz, nblock;
  int *d_rowptr, *d_colind;
  realtype *d_data;
  hipsparseMatDescr_t mat_descr;
  hipError_t cuerr;
  hipsolverStatus_t status;

  if (SUN_CUSP_LASTFLAG(S) != SUNLS_SUCCESS)
    return SUN_CUSP_LASTFLAG(S);

  if (SUN_CUSP_FIRSTFACTORIZE(S))
  {

    /* Free old workspace and symbloic analysis */
    if (SUN_CUSP_QRWORKSPACE(S))
    {
      hipFree(SUN_CUSP_QRWORKSPACE(S));
      cusolverSpDestroyCsrqrInfo(SUN_CUSP_QRINFO(S));
    }

    /* We must create a new csrqrinfo_t context every time we want to
       do a symbolic analysis. Trying to reuse it results in a
       HIPSOLVER_STATUS_INVALID_VALUE error. */
    status = cusolverSpCreateCsrqrInfo(&SUN_CUSP_QRINFO(S));
    if (!SUNDIALS_CUSOLVER_VERIFY(status))
    {
      SUN_CUSP_LASTFLAG(S) = SUNLS_PACKAGE_FAIL_UNREC;
      return SUN_CUSP_LASTFLAG(S);
    }

    nblock    = SUNMatrix_cuSparse_NumBlocks(A);
    blocknnz  = SUNMatrix_cuSparse_BlockNNZ(A);
    blockrows = SUNMatrix_cuSparse_BlockRows(A);
    blockcols = SUNMatrix_cuSparse_BlockColumns(A);
    d_data    = SUNMatrix_cuSparse_Data(A);
    d_rowptr  = SUNMatrix_cuSparse_IndexPointers(A);
    d_colind  = SUNMatrix_cuSparse_IndexValues(A);
    mat_descr = SUNMatrix_cuSparse_MatDescr(A);

    /* Perform symbolic analysis of sparsity structure */
    status = cusolverSpXcsrqrAnalysisBatched(SUN_CUSOL_HANDLE(S),
                                             blockrows,
                                             blockcols,
                                             blocknnz,
                                             mat_descr,
                                             d_rowptr,
                                             d_colind,
                                             SUN_CUSP_QRINFO(S));

    if (!SUNDIALS_CUSOLVER_VERIFY(status))
    {
      SUN_CUSP_LASTFLAG(S) = SUNLS_PACKAGE_FAIL_UNREC;
      return SUN_CUSP_LASTFLAG(S);
    }

    /* Compute the workspace we will need */
    status = _cusolverSpXcsrqrBufferInfoBatched(SUN_CUSOL_HANDLE(S),
                                                blockrows,
                                                blockcols,
                                                blocknnz,
                                                mat_descr,
                                                d_data,
                                                d_rowptr,
                                                d_colind,
                                                nblock,
                                                SUN_CUSP_QRINFO(S),
                                                &SUN_CUSP_INTERNAL_SIZE(S),
                                                &SUN_CUSP_WORK_SIZE(S));

    if (!SUNDIALS_CUSOLVER_VERIFY(status))
    {
      SUN_CUSP_LASTFLAG(S) = SUNLS_PACKAGE_FAIL_UNREC;
      return SUN_CUSP_LASTFLAG(S);
    }

    cuerr = hipMalloc((void**) &SUN_CUSP_QRWORKSPACE(S), SUN_CUSP_WORK_SIZE(S));
    if (!SUNDIALS_CUDA_VERIFY(cuerr))
    {
      SUN_CUSP_LASTFLAG(S) = SUNLS_PACKAGE_FAIL_UNREC;
      return SUN_CUSP_LASTFLAG(S);
    }

    SUN_CUSP_FIRSTFACTORIZE(S) = SUNFALSE;
  }

  SUN_CUSP_LASTFLAG(S) = SUNLS_SUCCESS;
  return(SUN_CUSP_LASTFLAG(S));
}

int SUNLinSolSolve_cuSolverSp_batchQR(SUNLinearSolver S, SUNMatrix A,
                                      N_Vector x, N_Vector b, realtype tol)
{
  hipsolverStatus_t status;
  int blockrows, blockcols, blocknnz, nblock;
  int *d_rowptr, *d_colind;
  realtype *d_data;
  hipsparseMatDescr_t mat_descr;

  if ((S == NULL) || (A == NULL) || (x == NULL) || (b == NULL))
    return SUNLS_MEM_NULL;

  SUN_CUSP_LASTFLAG(S) = SUNLS_SUCCESS;

  realtype* device_b = N_VGetDeviceArrayPointer(b);
  realtype* device_x = N_VGetDeviceArrayPointer(x);

  if (SUN_CUSP_LASTFLAG(S) != SUNLS_SUCCESS)
    return SUN_CUSP_LASTFLAG(S);

  /* solve the system */
  nblock    = SUNMatrix_cuSparse_NumBlocks(A);
  blocknnz  = SUNMatrix_cuSparse_BlockNNZ(A);
  blockrows = SUNMatrix_cuSparse_BlockRows(A);
  blockcols = SUNMatrix_cuSparse_BlockColumns(A);
  d_data    = SUNMatrix_cuSparse_Data(A);
  d_rowptr  = SUNMatrix_cuSparse_IndexPointers(A);
  d_colind  = SUNMatrix_cuSparse_IndexValues(A);
  mat_descr = SUNMatrix_cuSparse_MatDescr(A);

  status = _cusolverSpXcsrqrsvBatched(SUN_CUSOL_HANDLE(S),
                                      blockrows,
                                      blockcols,
                                      blocknnz,
                                      mat_descr,
                                      d_data,
                                      d_rowptr,
                                      d_colind,
                                      device_b,
                                      device_x,
                                      nblock,
                                      SUN_CUSP_QRINFO(S),
                                      SUN_CUSP_QRWORKSPACE(S));

  if (!SUNDIALS_CUSOLVER_VERIFY(status))
  {
    SUN_CUSP_LASTFLAG(S) = SUNLS_PACKAGE_FAIL_UNREC;
    return SUN_CUSP_LASTFLAG(S);
  }

  return SUN_CUSP_LASTFLAG(S);
}

sunindextype SUNLinSolLastFlag_cuSolverSp_batchQR(SUNLinearSolver S)
{
  if (S == NULL) return -1;
  return SUN_CUSP_LASTFLAG(S);
}

int SUNLinSolFree_cuSolverSp_batchQR(SUNLinearSolver S)
{
  /* return with success if already freed */
  if (S == NULL) return SUNLS_SUCCESS;

  /* free stuff in the content structure */
  cusolverSpDestroyCsrqrInfo(SUN_CUSP_QRINFO(S));
  hipFree(SUN_CUSP_QRWORKSPACE(S));

  /* free content structure */
  if (S->content) {
    free(S->content);
    S->content = NULL;
  }

  /* free ops structure */
  if (S->ops) {
    free(S->ops);
    S->ops = NULL;
  }

  /* free the actual SUNLinSol */
  free(S);
  S = NULL;

  return(SUNLS_SUCCESS);
}
