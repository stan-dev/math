/*
 * -----------------------------------------------------------------
 * Programmer(s): Cody J. Balos @ LLNL
 * -----------------------------------------------------------------
 * SUNDIALS Copyright Start
 * Copyright (c) 2002-2022, Lawrence Livermore National Security
 * and Southern Methodist University.
 * All rights reserved.
 *
 * See the top-level LICENSE and NOTICE files for details.
 *
 * SPDX-License-Identifier: BSD-3-Clause
 * SUNDIALS Copyright End
 * -----------------------------------------------------------------
 * This is the header file is for the cuSPARSE implementation of the
 * SUNMATRIX module.
 * -----------------------------------------------------------------
 */

#include <stdio.h>
#include <stdlib.h>

#include <sunmemory/sunmemory_cuda.h>
#include <sunmatrix/sunmatrix_cusparse.h>

#include "sundials_cuda.h"
#include "sundials_debug.h"
#include "cusparse_kernels.cuh"


/* Use the namespace for the kernels */
using namespace sundials::cuda;
using namespace sundials::sunmatrix_cusparse;

/* Constants */
#define ZERO RCONST(0.0)
#define ONE  RCONST(1.0)

/* Private function prototypes */
static booleantype SMCompatible_cuSparse(SUNMatrix, SUNMatrix);
static SUNMatrix SUNMatrix_cuSparse_NewEmpty(SUNContext sunctx);
#if CUDART_VERSION >= 11000
static hipsparseStatus_t CreateSpMatDescr(SUNMatrix, hipsparseSpMatDescr_t*);
#endif

/* Macros for handling the different function names based on precision */
#if defined(SUNDIALS_DOUBLE_PRECISION)
#define cusparseXcsrmv hipsparseDcsrmv
#define CUDA_R_XF HIP_R_64F
#elif defined(SUNDIALS_SINGLE_PRECISION)
#define cusparseXcsrmv hipsparseScsrmv
#define CUDA_R_XF HIP_R_32F
#endif

/* Content accessor macros */
#define SMCU_CONTENT(A)     ( (SUNMatrix_Content_cuSparse)(A->content) )
#define SMCU_ROWS(A)        ( SMCU_CONTENT(A)->M )
#define SMCU_COLUMNS(A)     ( SMCU_CONTENT(A)->N )
#define SMCU_NNZ(A)         ( SMCU_CONTENT(A)->NNZ )
#define SMCU_NBLOCKS(A)     ( SMCU_CONTENT(A)->nblocks )
#define SMCU_BLOCKROWS(A)   ( SMCU_CONTENT(A)->blockrows )
#define SMCU_BLOCKCOLS(A)   ( SMCU_CONTENT(A)->blockcols )
#define SMCU_BLOCKNNZ(A)    ( SMCU_CONTENT(A)->blocknnz )
#define SMCU_NP(A)          ( SMCU_CONTENT(A)->NP )
#define SMCU_SPARSETYPE(A)  ( SMCU_CONTENT(A)->sparse_type )
#define SMCU_OWNMATD(A)     ( SMCU_CONTENT(A)->own_matd )
#define SMCU_DATA(A)        ( SMCU_CONTENT(A)->data )
#define SMCU_DATAp(A)       ( (realtype*)SMCU_CONTENT(A)->data->ptr )
#define SMCU_INDEXVALS(A)   ( SMCU_CONTENT(A)->colind )
#define SMCU_INDEXPTRS(A)   ( SMCU_CONTENT(A)->rowptrs )
#define SMCU_INDEXVALSp(A)  ( (int*) SMCU_CONTENT(A)->colind->ptr )
#define SMCU_INDEXPTRSp(A)  ( (int*) SMCU_CONTENT(A)->rowptrs->ptr )
#define SMCU_MEMHELP(A)     ( SMCU_CONTENT(A)->mem_helper )
#define SMCU_MATDESCR(A)    ( SMCU_CONTENT(A)->mat_descr )
#define SMCU_CUSPHANDLE(A)  ( SMCU_CONTENT(A)->cusp_handle )
#define SMCU_FIXEDPATTERN(A)( SMCU_CONTENT(A)->fixed_pattern )
#define SMCU_EXECPOLICY(A)  ( SMCU_CONTENT(A)->exec_policy )


/* ------------------------------------------------------------------
 * Default execution policy definition.
 *
 * This policy tries to help us leverage the structure of the matrix.
 * It will choose block sizes which are a multiple of the warp size,
 * and it will choose a grid size to such that all work elements are
 * covered.
 * ------------------------------------------------------------------ */

class SUNCuSparseMatrixExecPolicy : public ExecPolicy
{
public:
  SUNCuSparseMatrixExecPolicy(const hipStream_t stream = 0)
    : ExecPolicy(stream)
  {}

  SUNCuSparseMatrixExecPolicy(const SUNCuSparseMatrixExecPolicy& ex)
    : ExecPolicy(ex.stream_)
  {}

  virtual size_t gridSize(size_t numWorkElements, size_t blockDim = 0) const
  {
    return(numWorkElements + blockDim - 1)/blockDim;
  }

  virtual size_t blockSize(size_t numWorkElements = 0, size_t gridDim = 0) const
  {
    return(max_block_size(WARP_SIZE*(numWorkElements + WARP_SIZE - 1)/WARP_SIZE));
  }

  virtual const hipStream_t* stream() const
  {
    return(&stream_);
  }

  virtual ExecPolicy* clone() const
  {
    return(static_cast<ExecPolicy*>(new SUNCuSparseMatrixExecPolicy(*this)));
  }

  static size_t max_block_size(int val)
  {
    return((val > MAX_BLOCK_SIZE) ? MAX_BLOCK_SIZE : val );
  }
};

SUNCuSparseMatrixExecPolicy DEFAULT_EXEC_POLICY;

/* ------------------------------------------------------------------
 * Constructors.
 * ------------------------------------------------------------------ */

SUNMatrix SUNMatrix_cuSparse_NewCSR(int M, int N, int NNZ, hipsparseHandle_t cusp, SUNContext sunctx)
{
  SUNMemory d_colind, d_rowptr, d_values;
  int alloc_fail = 0;

  /* return with NULL matrix on illegal input */
  if ( (M <= 0) || (N <= 0) || (NNZ < 0) )
  {
    SUNDIALS_DEBUG_PRINT("ERROR in SUNMatrix_NewCSR_cuSparse: illegal value(s) for M, N, or NNZ\n");
    return(NULL);
  }

  SUNMatrix A = SUNMatrix_cuSparse_NewEmpty(sunctx);
  if (A == NULL)
  {
    SUNDIALS_DEBUG_PRINT("ERROR in SUNMatrix_NewCSR_cuSparse: SUNMatrix_cuSparse_NewEmpty returned NULL\n");
    return(NULL);
  }

  SMCU_MEMHELP(A) = SUNMemoryHelper_Cuda(sunctx);
  if (SMCU_MEMHELP(A) == NULL)
  {
    SUNDIALS_DEBUG_PRINT("ERROR in SUNMatrix_NewCSR_cuSparse: SUNMemoryHelper_Cuda returned NULL\n");
    SUNMatDestroy(A);
    return(NULL);
  }

  /* Allocate device memory for the matrix */
  alloc_fail += SUNMemoryHelper_Alloc(SMCU_MEMHELP(A), &d_colind,
                                      sizeof(int)*NNZ, SUNMEMTYPE_DEVICE,
                                      nullptr);
  alloc_fail += SUNMemoryHelper_Alloc(SMCU_MEMHELP(A), &d_rowptr,
                                      sizeof(int)*(M+1), SUNMEMTYPE_DEVICE,
                                      nullptr);
  alloc_fail += SUNMemoryHelper_Alloc(SMCU_MEMHELP(A), &d_values,
                                      sizeof(realtype)*NNZ, SUNMEMTYPE_DEVICE,
                                      nullptr);
  if (alloc_fail)
  {
    SUNMemoryHelper_Dealloc(SMCU_MEMHELP(A), d_colind, nullptr);
    SUNMemoryHelper_Dealloc(SMCU_MEMHELP(A), d_rowptr, nullptr);
    SUNMemoryHelper_Dealloc(SMCU_MEMHELP(A), d_values, nullptr);
    SUNMatDestroy(A);
    return(NULL);
  }

  /* Choose sensible defaults */
  hipsparseStatus_t cusparse_status = HIPSPARSE_STATUS_SUCCESS;
  hipsparseMatDescr_t mat_descr;
  cusparse_status = hipsparseCreateMatDescr(&mat_descr);
  if (!SUNDIALS_CUSPARSE_VERIFY(cusparse_status))
  {
    SUNMemoryHelper_Dealloc(SMCU_MEMHELP(A), d_colind, nullptr);
    SUNMemoryHelper_Dealloc(SMCU_MEMHELP(A), d_rowptr, nullptr);
    SUNMemoryHelper_Dealloc(SMCU_MEMHELP(A), d_values, nullptr);
    SUNMatDestroy(A);
    return(NULL);
  }

  cusparse_status = hipsparseSetMatType(mat_descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
  if (!SUNDIALS_CUSPARSE_VERIFY(cusparse_status))
  {
    SUNMemoryHelper_Dealloc(SMCU_MEMHELP(A), d_colind, nullptr);
    SUNMemoryHelper_Dealloc(SMCU_MEMHELP(A), d_rowptr, nullptr);
    SUNMemoryHelper_Dealloc(SMCU_MEMHELP(A), d_values, nullptr);
    hipsparseDestroyMatDescr(mat_descr);
    SUNMatDestroy(A);
    return(NULL);
  }

  cusparse_status = hipsparseSetMatIndexBase(mat_descr, HIPSPARSE_INDEX_BASE_ZERO);
  if (!SUNDIALS_CUSPARSE_VERIFY(cusparse_status))
  {
    SUNMemoryHelper_Dealloc(SMCU_MEMHELP(A), d_colind, nullptr);
    SUNMemoryHelper_Dealloc(SMCU_MEMHELP(A), d_rowptr, nullptr);
    SUNMemoryHelper_Dealloc(SMCU_MEMHELP(A), d_values, nullptr);
    hipsparseDestroyMatDescr(mat_descr);
    SUNMatDestroy(A);
    return(NULL);
  }

  hipStream_t stream;
  if (!SUNDIALS_CUSPARSE_VERIFY(hipsparseGetStream(cusp, &stream)))
  {
    SUNMemoryHelper_Dealloc(SMCU_MEMHELP(A), d_colind, nullptr);
    SUNMemoryHelper_Dealloc(SMCU_MEMHELP(A), d_rowptr, nullptr);
    SUNMemoryHelper_Dealloc(SMCU_MEMHELP(A), d_values, nullptr);
    hipsparseDestroyMatDescr(mat_descr);
    SUNMatDestroy(A);
    return(NULL);
  }

  /* Fill the content */
  SMCU_CONTENT(A)->M              = M;
  SMCU_CONTENT(A)->N              = N;
  SMCU_CONTENT(A)->NNZ            = NNZ;
  SMCU_CONTENT(A)->nblocks        = 1;
  SMCU_CONTENT(A)->blockrows      = M;
  SMCU_CONTENT(A)->blockcols      = N;
  SMCU_CONTENT(A)->blocknnz       = NNZ;
  SMCU_CONTENT(A)->own_matd       = SUNTRUE;
  SMCU_CONTENT(A)->matvec_issetup = SUNFALSE;
  SMCU_CONTENT(A)->fixed_pattern  = SUNFALSE;
  SMCU_CONTENT(A)->sparse_type    = SUNMAT_CUSPARSE_CSR;
  SMCU_CONTENT(A)->colind         = d_colind;
  SMCU_CONTENT(A)->rowptrs        = d_rowptr;
  SMCU_CONTENT(A)->data           = d_values;
  SMCU_CONTENT(A)->mat_descr      = mat_descr;
  SMCU_CONTENT(A)->cusp_handle    = cusp;
  SMCU_CONTENT(A)->exec_policy    = DEFAULT_EXEC_POLICY.clone_new_stream(stream);

#if CUDART_VERSION >= 11000
  hipsparseSpMatDescr_t spmat_descr;
  if (!SUNDIALS_CUSPARSE_VERIFY(CreateSpMatDescr(A, &spmat_descr)))
  {
    SUNMatDestroy(A);
    return(NULL);
  }
  SMCU_CONTENT(A)->spmat_descr = spmat_descr;
  SMCU_CONTENT(A)->dBufferMem  = NULL;
  SMCU_CONTENT(A)->bufferSize  = 0;
  SMCU_CONTENT(A)->vecX        = NULL;
  SMCU_CONTENT(A)->vecY        = NULL;
#endif

  return A;
}


SUNMatrix SUNMatrix_cuSparse_MakeCSR(hipsparseMatDescr_t mat_descr, int M, int N, int NNZ,
                                     int *rowptrs , int *colind , realtype *data,
                                     hipsparseHandle_t cusp, SUNContext sunctx)
{
  /* return with NULL matrix on illegal input */
  if ( (M <= 0) || (N <= 0) || (NNZ < 0) )
  {
    SUNDIALS_DEBUG_PRINT("ERROR in SUNMatrix_MakeCSR_cuSparse: illegal value(s) for M, N, or NNZ\n");
    return(NULL);
  }

  if ( (rowptrs == NULL) || (colind == NULL) || (data == NULL) )
  {
    SUNDIALS_DEBUG_PRINT("ERROR in SUNMatrix_MakeCSR_cuSparse: rowptrs, colind, or data is NULL\n");
    return(NULL);
  }

  if (hipsparseGetMatIndexBase(mat_descr) != HIPSPARSE_INDEX_BASE_ZERO)
  {
    SUNDIALS_DEBUG_PRINT("ERROR in SUNMatrix_MakeCSR_cuSparse: the hipsparseMatDescr_t must have index base HIPSPARSE_INDEX_BASE_ZERO\n");
    return(NULL);
  }

  SUNMatrix A = SUNMatrix_cuSparse_NewEmpty(sunctx);
  if (A == NULL)
  {
    SUNDIALS_DEBUG_PRINT("ERROR in SUNMatrix_MakeCSR_cuSparse: SUNMatrix_cuSparse_NewEmpty returned NULL\n");
    return(NULL);
  }

  SMCU_MEMHELP(A) = SUNMemoryHelper_Cuda(sunctx);
  if (SMCU_MEMHELP(A) == NULL)
  {
    SUNDIALS_DEBUG_PRINT("ERROR in SUNMatrix_NewCSR_cuSparse: SUNMemoryHelper_Cuda returned NULL\n");
    SUNMatDestroy(A);
    return(NULL);
  }

  hipStream_t stream;
  if (!SUNDIALS_CUSPARSE_VERIFY(hipsparseGetStream(cusp, &stream)))
  {
    SUNMatDestroy(A);
    return(NULL);
  }

  /* Fill content */
  SMCU_CONTENT(A)->M              = M;
  SMCU_CONTENT(A)->N              = N;
  SMCU_CONTENT(A)->NNZ            = NNZ;
  SMCU_CONTENT(A)->nblocks        = 1;
  SMCU_CONTENT(A)->blockrows      = M;
  SMCU_CONTENT(A)->blockcols      = N;
  SMCU_CONTENT(A)->blocknnz       = NNZ;
  SMCU_CONTENT(A)->own_matd       = SUNFALSE;
  SMCU_CONTENT(A)->matvec_issetup = SUNFALSE;
  SMCU_CONTENT(A)->fixed_pattern  = SUNFALSE;
  SMCU_CONTENT(A)->sparse_type    = SUNMAT_CUSPARSE_CSR;
  SMCU_CONTENT(A)->colind         = SUNMemoryHelper_Wrap(colind, SUNMEMTYPE_DEVICE);
  SMCU_CONTENT(A)->rowptrs        = SUNMemoryHelper_Wrap(rowptrs, SUNMEMTYPE_DEVICE);
  SMCU_CONTENT(A)->data           = SUNMemoryHelper_Wrap(data, SUNMEMTYPE_DEVICE);
  SMCU_CONTENT(A)->mat_descr      = mat_descr;
  SMCU_CONTENT(A)->cusp_handle    = cusp;

  SMCU_CONTENT(A)->exec_policy   = DEFAULT_EXEC_POLICY.clone_new_stream(stream);

  if (SMCU_CONTENT(A)->colind == NULL ||
      SMCU_CONTENT(A)->rowptrs == NULL ||
      SMCU_CONTENT(A)->data == NULL)
  {
    SUNMatDestroy(A);
    return(NULL);
  }

#if CUDART_VERSION >= 11000
  hipsparseSpMatDescr_t spmat_descr;
  if (!SUNDIALS_CUSPARSE_VERIFY(CreateSpMatDescr(A, &spmat_descr)))
  {
    SUNMatDestroy(A);
    return(NULL);
  }
  SMCU_CONTENT(A)->spmat_descr = spmat_descr;
  SMCU_CONTENT(A)->dBufferMem  = NULL;
  SMCU_CONTENT(A)->bufferSize  = 0;
  SMCU_CONTENT(A)->vecX        = NULL;
  SMCU_CONTENT(A)->vecY        = NULL;
#endif

  return(A);
}


SUNMatrix SUNMatrix_cuSparse_NewBlockCSR(int nblocks, int blockrows, int blockcols, int blocknnz, hipsparseHandle_t cusp, SUNContext sunctx)
{
  SUNMemory d_colind, d_rowptr, d_values;
  int M, N, NNZ;
  int alloc_fail = 0;

  /* Return with NULL matrix on illegal input */
  if (blockrows != blockcols)
  {
    SUNDIALS_DEBUG_PRINT("ERROR in SUNMatrix_cuSparse_NewBlockCSR: matrix must be square for the BCSR format\n");
    return(NULL);
  }

  M   = nblocks * blockrows;
  N   = M;
  NNZ = nblocks * blocknnz;

  /* Return with NULL matrix on illegal input */
  if ( (M <= 0) || (N <= 0) || (NNZ < 0) )
  {
    SUNDIALS_DEBUG_PRINT("ERROR in SUNMatrix_cuSparse_NewBlockCSR: illegal value(s) for M, N, or NNZ\n");
    return(NULL);
  }

  /* Allocate the SUNMatrix object */
  SUNMatrix A = SUNMatrix_cuSparse_NewEmpty(sunctx);
  if (A == NULL)
  {
    SUNDIALS_DEBUG_PRINT("ERROR in SUNMatrix_cuSparse_NewBlockCSR: SUNMatrix_cuSparse_NewEmpty returned NULL\n");
    return(NULL);
  }

  SMCU_MEMHELP(A) = SUNMemoryHelper_Cuda(sunctx);
  if (SMCU_MEMHELP(A) == NULL)
  {
    SUNDIALS_DEBUG_PRINT("ERROR in SUNMatrix_NewCSR_cuSparse: SUNMemoryHelper_Cuda returned NULL\n");
    SUNMatDestroy(A);
    return(NULL);
  }

  /* Allocate device memory for the matrix */
  alloc_fail += SUNMemoryHelper_Alloc(SMCU_MEMHELP(A), &d_colind,
                                      sizeof(int)*blocknnz, SUNMEMTYPE_DEVICE,
                                      nullptr);
  alloc_fail += SUNMemoryHelper_Alloc(SMCU_MEMHELP(A), &d_rowptr,
                                      sizeof(int)*(blockrows + 1),
                                      SUNMEMTYPE_DEVICE, nullptr);
  alloc_fail += SUNMemoryHelper_Alloc(SMCU_MEMHELP(A), &d_values,
                                      sizeof(realtype)*blocknnz*nblocks,
                                      SUNMEMTYPE_DEVICE, nullptr);
  if (alloc_fail)
  {
    SUNMemoryHelper_Dealloc(SMCU_MEMHELP(A), d_colind, nullptr);
    SUNMemoryHelper_Dealloc(SMCU_MEMHELP(A), d_rowptr, nullptr);
    SUNMemoryHelper_Dealloc(SMCU_MEMHELP(A), d_values, nullptr);
    SUNMatDestroy(A);
    return(NULL);
  }

  /* Choose sensible defaults */
  hipsparseStatus_t cusparse_status = HIPSPARSE_STATUS_SUCCESS;
  hipsparseMatDescr_t mat_descr;
  cusparse_status = hipsparseCreateMatDescr(&mat_descr);
  if (!SUNDIALS_CUSPARSE_VERIFY(cusparse_status))
  {
    SUNMemoryHelper_Dealloc(SMCU_MEMHELP(A), d_colind, nullptr);
    SUNMemoryHelper_Dealloc(SMCU_MEMHELP(A), d_rowptr, nullptr);
    SUNMemoryHelper_Dealloc(SMCU_MEMHELP(A), d_values, nullptr);
    SUNMatDestroy(A);
    return(NULL);
  }

  cusparse_status = hipsparseSetMatType(mat_descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
  if (!SUNDIALS_CUSPARSE_VERIFY(cusparse_status))
  {
    SUNMemoryHelper_Dealloc(SMCU_MEMHELP(A), d_colind, nullptr);
    SUNMemoryHelper_Dealloc(SMCU_MEMHELP(A), d_rowptr, nullptr);
    SUNMemoryHelper_Dealloc(SMCU_MEMHELP(A), d_values, nullptr);
    hipsparseDestroyMatDescr(mat_descr);
    SUNMatDestroy(A);
    return(NULL);
  }

  cusparse_status = hipsparseSetMatIndexBase(mat_descr, HIPSPARSE_INDEX_BASE_ZERO);
  if (!SUNDIALS_CUSPARSE_VERIFY(cusparse_status))
  {
    SUNMemoryHelper_Dealloc(SMCU_MEMHELP(A), d_colind, nullptr);
    SUNMemoryHelper_Dealloc(SMCU_MEMHELP(A), d_rowptr, nullptr);
    SUNMemoryHelper_Dealloc(SMCU_MEMHELP(A), d_values, nullptr);
    hipsparseDestroyMatDescr(mat_descr);
    SUNMatDestroy(A);
    return(NULL);
  }

  hipStream_t stream;
  if (!SUNDIALS_CUSPARSE_VERIFY(hipsparseGetStream(cusp, &stream)))
  {
    SUNMemoryHelper_Dealloc(SMCU_MEMHELP(A), d_colind, nullptr);
    SUNMemoryHelper_Dealloc(SMCU_MEMHELP(A), d_rowptr, nullptr);
    SUNMemoryHelper_Dealloc(SMCU_MEMHELP(A), d_values, nullptr);
    hipsparseDestroyMatDescr(mat_descr);
    SUNMatDestroy(A);
    return(NULL);
  }

  /* Fill the content */
  SMCU_CONTENT(A)->M              = M;
  SMCU_CONTENT(A)->N              = N;
  SMCU_CONTENT(A)->NNZ            = NNZ;
  SMCU_CONTENT(A)->nblocks        = nblocks;
  SMCU_CONTENT(A)->blockrows      = blockrows;
  SMCU_CONTENT(A)->blockcols      = blockrows;
  SMCU_CONTENT(A)->blocknnz       = blocknnz;
  SMCU_CONTENT(A)->own_matd       = SUNTRUE;
  SMCU_CONTENT(A)->matvec_issetup = SUNFALSE;
  SMCU_CONTENT(A)->cusp_handle    = cusp;
  SMCU_CONTENT(A)->fixed_pattern  = SUNFALSE;
  SMCU_CONTENT(A)->sparse_type    = SUNMAT_CUSPARSE_BCSR;
  SMCU_CONTENT(A)->colind         = d_colind;
  SMCU_CONTENT(A)->rowptrs        = d_rowptr;
  SMCU_CONTENT(A)->data           = d_values;
  SMCU_CONTENT(A)->mat_descr      = mat_descr;
  SMCU_CONTENT(A)->exec_policy    = DEFAULT_EXEC_POLICY.clone_new_stream(stream);

#if CUDART_VERSION >= 11000
  hipsparseSpMatDescr_t spmat_descr;
  if (!SUNDIALS_CUSPARSE_VERIFY(CreateSpMatDescr(A, &spmat_descr)))
  {
    SUNMatDestroy(A);
    return(NULL);
  }
  SMCU_CONTENT(A)->spmat_descr = spmat_descr;
  SMCU_CONTENT(A)->dBufferMem  = NULL;
  SMCU_CONTENT(A)->bufferSize  = 0;
  SMCU_CONTENT(A)->vecX        = NULL;
  SMCU_CONTENT(A)->vecY        = NULL;
#endif

  return(A);
}

/* ------------------------------------------------------------------
 * Implementation specific routines.
 * ------------------------------------------------------------------ */

int SUNMatrix_cuSparse_SparseType(SUNMatrix A)
{
  if (SUNMatGetID(A) == SUNMATRIX_CUSPARSE)
    return(SMCU_SPARSETYPE(A));
  else
    return(SUNMAT_ILL_INPUT);
}

int SUNMatrix_cuSparse_Rows(SUNMatrix A)
{
  if (SUNMatGetID(A) == SUNMATRIX_CUSPARSE)
    return(SMCU_ROWS(A));
  else
    return(SUNMAT_ILL_INPUT);
}

int SUNMatrix_cuSparse_Columns(SUNMatrix A)
{
  if (SUNMatGetID(A) == SUNMATRIX_CUSPARSE)
    return(SMCU_COLUMNS(A));
  else
    return(SUNMAT_ILL_INPUT);
}

int SUNMatrix_cuSparse_NNZ(SUNMatrix A)
{
  if (SUNMatGetID(A) == SUNMATRIX_CUSPARSE)
    return(SMCU_NNZ(A));
  else
    return(SUNMAT_ILL_INPUT);
}

int* SUNMatrix_cuSparse_IndexPointers(SUNMatrix A)
{
  if (SUNMatGetID(A) == SUNMATRIX_CUSPARSE)
    return(SMCU_INDEXPTRSp(A));
  else
    return(NULL);
}

int* SUNMatrix_cuSparse_IndexValues(SUNMatrix A)
{
  if (SUNMatGetID(A) == SUNMATRIX_CUSPARSE)
    return(SMCU_INDEXVALSp(A));
  else
    return(NULL);
}

realtype* SUNMatrix_cuSparse_Data(SUNMatrix A)
{
  if (SUNMatGetID(A) == SUNMATRIX_CUSPARSE)
    return(SMCU_DATAp(A));
  else
    return(NULL);
}

int SUNMatrix_cuSparse_NumBlocks(SUNMatrix A)
{
  if (SUNMatGetID(A) == SUNMATRIX_CUSPARSE)
    return(SMCU_NBLOCKS(A));
  else
    return(SUNMAT_ILL_INPUT);
}

int SUNMatrix_cuSparse_BlockRows(SUNMatrix A)
{
  if (SUNMatGetID(A) == SUNMATRIX_CUSPARSE)
    return(SMCU_BLOCKROWS(A));
  else
    return(SUNMAT_ILL_INPUT);
}

int SUNMatrix_cuSparse_BlockColumns(SUNMatrix A)
{
  if (SUNMatGetID(A) == SUNMATRIX_CUSPARSE)
    return(SMCU_BLOCKCOLS(A));
  else
    return(SUNMAT_ILL_INPUT);
}

int SUNMatrix_cuSparse_BlockNNZ(SUNMatrix A)
{
  if (SUNMatGetID(A) == SUNMATRIX_CUSPARSE)
    return(SMCU_BLOCKNNZ(A));
  else
    return(SUNMAT_ILL_INPUT);
}

realtype* SUNMatrix_cuSparse_BlockData(SUNMatrix A, int blockidx)
{
  realtype *matdata;
  int offset;

  if (SUNMatGetID(A) != SUNMATRIX_CUSPARSE)
    return(NULL);

  if (blockidx >= SMCU_NBLOCKS(A))
    return(NULL);

  matdata = SMCU_DATAp(A);
  offset = SMCU_BLOCKNNZ(A)*blockidx;

  return(&matdata[offset]);
}

hipsparseMatDescr_t SUNMatrix_cuSparse_MatDescr(SUNMatrix A)
{
  if (SUNMatGetID(A) == SUNMATRIX_CUSPARSE)
    return(SMCU_MATDESCR(A));
  else
    return(NULL);
}

int SUNMatrix_cuSparse_SetFixedPattern(SUNMatrix A, booleantype yesno)
{
  if (SUNMatGetID(A) != SUNMATRIX_CUSPARSE)
    return(SUNMAT_ILL_INPUT);

  SMCU_FIXEDPATTERN(A) = yesno;

  return(SUNMAT_SUCCESS);
}


int SUNMatrix_cuSparse_SetKernelExecPolicy(SUNMatrix A, SUNCudaExecPolicy* exec_policy)
{
  if (SUNMatGetID(A) != SUNMATRIX_CUSPARSE)
    return(SUNMAT_ILL_INPUT);

  /* Reset to the default policy if the new one is NULL */
  delete SMCU_EXECPOLICY(A);
  if (exec_policy)
    SMCU_EXECPOLICY(A) = exec_policy->clone();
  else
    SMCU_EXECPOLICY(A) = DEFAULT_EXEC_POLICY.clone_new_stream(*SMCU_EXECPOLICY(A)->stream());

  return(SUNMAT_SUCCESS);
}


int SUNMatrix_cuSparse_CopyToDevice(SUNMatrix dA, realtype* h_data,
                                    int* h_idxptrs, int* h_idxvals)
{
  int retval;
  SUNMemory _h_data, _h_idxptrs, _h_idxvals;
  const hipStream_t* stream;
  int nidxvals, nidxptrs;

  if (SUNMatGetID(dA) != SUNMATRIX_CUSPARSE)
    return(SUNMAT_ILL_INPUT);

  stream  = SMCU_EXECPOLICY(dA)->stream();

  if (h_data != NULL)
  {
    _h_data = SUNMemoryHelper_Wrap(h_data, SUNMEMTYPE_HOST);
    retval  = SUNMemoryHelper_CopyAsync(SMCU_MEMHELP(dA),
                                        SMCU_DATA(dA),
                                        _h_data,
                                        SMCU_NNZ(dA)*sizeof(realtype),
                                        (void*) stream);
    SUNMemoryHelper_Dealloc(SMCU_MEMHELP(dA), _h_data, nullptr);
    if (retval != 0) return(SUNMAT_OPERATION_FAIL);
  }

  switch(SMCU_SPARSETYPE(dA))
  {
    case SUNMAT_CUSPARSE_CSR:
      nidxptrs = SMCU_ROWS(dA)+1;
      nidxvals = SMCU_NNZ(dA);
      break;
    case SUNMAT_CUSPARSE_BCSR:
      nidxptrs = SMCU_BLOCKROWS(dA)+1;
      nidxvals = SMCU_BLOCKNNZ(dA);
      break;
    default:
      SUNDIALS_DEBUG_PRINT("ERROR in SUNMatrix_cuSparse_CopyToDevice: unrecognized sparse type\n");
      return(SUNMAT_ILL_INPUT);
  }

  if (h_idxptrs != NULL)
  {
    _h_idxptrs = SUNMemoryHelper_Wrap(h_idxptrs, SUNMEMTYPE_HOST);
    retval = SUNMemoryHelper_CopyAsync(SMCU_MEMHELP(dA),
                                       SMCU_INDEXPTRS(dA),
                                       _h_idxptrs,
                                       nidxptrs*sizeof(int),
                                       (void*) stream);
    SUNMemoryHelper_Dealloc(SMCU_MEMHELP(dA), _h_idxptrs, nullptr);
    if (retval != 0) return(SUNMAT_OPERATION_FAIL);
  }

  if (h_idxvals != NULL)
  {
    _h_idxvals = SUNMemoryHelper_Wrap(h_idxvals, SUNMEMTYPE_HOST);
    retval = SUNMemoryHelper_CopyAsync(SMCU_MEMHELP(dA),
                                       SMCU_INDEXVALS(dA),
                                       _h_idxvals,
                                       nidxvals*sizeof(int),
                                       (void*) stream);
    SUNMemoryHelper_Dealloc(SMCU_MEMHELP(dA), _h_idxvals, nullptr);
    if (retval != 0) return(SUNMAT_OPERATION_FAIL);
  }

  return(SUNMAT_SUCCESS);
}


int SUNMatrix_cuSparse_CopyFromDevice(SUNMatrix dA, realtype* h_data,
                                      int* h_idxptrs, int* h_idxvals)
{
  int retval;
  SUNMemory _h_data, _h_idxptrs, _h_idxvals;
  const hipStream_t* stream;
  int nidxvals, nidxptrs;

  if (SUNMatGetID(dA) != SUNMATRIX_CUSPARSE)
    return(SUNMAT_ILL_INPUT);

  stream = SMCU_EXECPOLICY(dA)->stream();

  if (h_data != NULL)
  {
    _h_data = SUNMemoryHelper_Wrap(h_data, SUNMEMTYPE_HOST);
    retval  = SUNMemoryHelper_CopyAsync(SMCU_MEMHELP(dA),
                                        _h_data,
                                        SMCU_DATA(dA),
                                        SMCU_NNZ(dA)*sizeof(realtype),
                                        (void*) stream);
    SUNMemoryHelper_Dealloc(SMCU_MEMHELP(dA), _h_data, nullptr);
    if (retval != 0) return(SUNMAT_OPERATION_FAIL);
  }


  switch(SMCU_SPARSETYPE(dA))
  {
    case SUNMAT_CUSPARSE_CSR:
      nidxptrs = SMCU_ROWS(dA)+1;
      nidxvals = SMCU_NNZ(dA);
    case SUNMAT_CUSPARSE_BCSR:
      nidxptrs = SMCU_BLOCKROWS(dA)+1;
      nidxvals = SMCU_BLOCKNNZ(dA);
  }

  if (h_idxptrs != NULL)
  {
    _h_idxptrs = SUNMemoryHelper_Wrap(h_idxptrs, SUNMEMTYPE_HOST);
    retval = SUNMemoryHelper_CopyAsync(SMCU_MEMHELP(dA),
                                       _h_idxptrs,
                                       SMCU_INDEXPTRS(dA),
                                       nidxptrs*sizeof(int),
                                       (void*) stream);
    SUNMemoryHelper_Dealloc(SMCU_MEMHELP(dA), _h_idxptrs, nullptr);
    if (retval != 0) return(SUNMAT_OPERATION_FAIL);
  }

  if (h_idxvals != NULL)
  {
    _h_idxvals = SUNMemoryHelper_Wrap(h_idxvals, SUNMEMTYPE_HOST);
    retval = SUNMemoryHelper_CopyAsync(SMCU_MEMHELP(dA),
                                       _h_idxvals,
                                       SMCU_INDEXVALS(dA),
                                       nidxvals*sizeof(int),
                                       (void*) stream);
    SUNMemoryHelper_Dealloc(SMCU_MEMHELP(dA), _h_idxvals, nullptr);
    if (retval != 0) return(SUNMAT_OPERATION_FAIL);
  }


  return(SUNMAT_SUCCESS);
}

/*
 * -----------------------------------------------------------------
 * implementation of matrix operations
 * -----------------------------------------------------------------
 */


SUNMatrix_ID SUNMatGetID_cuSparse(SUNMatrix A)
{
  return(SUNMATRIX_CUSPARSE);
}

/* Returns a new matrix allocated to have the same structure as A,
   but it does not copy any nonzeros, column vals, or row pointers. */
SUNMatrix SUNMatClone_cuSparse(SUNMatrix A)
{
  SUNMatrix B;

  switch (SMCU_SPARSETYPE(A))
  {
    case SUNMAT_CUSPARSE_CSR:
      B = SUNMatrix_cuSparse_NewCSR(SMCU_ROWS(A), SMCU_COLUMNS(A), SMCU_NNZ(A),
                                    SMCU_CUSPHANDLE(A), A->sunctx);
      break;
    case SUNMAT_CUSPARSE_BCSR:
      B = SUNMatrix_cuSparse_NewBlockCSR(SMCU_NBLOCKS(A), SMCU_BLOCKROWS(A), SMCU_BLOCKCOLS(A),
                                         SMCU_BLOCKNNZ(A), SMCU_CUSPHANDLE(A), A->sunctx);
      break;
    default:
      SUNDIALS_DEBUG_PRINT("ERROR in SUNMatClone_cuSparse: sparse type not recognized\n");
      B = NULL;
  }

  SMCU_FIXEDPATTERN(B) = SMCU_FIXEDPATTERN(A);
  delete SMCU_EXECPOLICY(B);
  SMCU_EXECPOLICY(B) = SMCU_EXECPOLICY(A)->clone();

  return(B);
}


/* Deallocates the SUNMatrix object and all data it owns */
void SUNMatDestroy_cuSparse(SUNMatrix A)
{
  if (A == NULL) return;

  /* free content */
  if (A->content != NULL)
  {
    if (SMCU_MEMHELP(A))
    {
      SUNMemoryHelper_Dealloc(SMCU_MEMHELP(A), SMCU_DATA(A), nullptr);
      SUNMemoryHelper_Dealloc(SMCU_MEMHELP(A), SMCU_INDEXPTRS(A), nullptr);
      SUNMemoryHelper_Dealloc(SMCU_MEMHELP(A), SMCU_INDEXVALS(A), nullptr);
    }
    else
    {
      SUNDIALS_DEBUG_PRINT("WARNING in SUNMatDestroy_cuSparse: mem_helper was NULL when trying to dealloc data, this could result in a memory leak\n");
    }

    if (SMCU_OWNMATD(A))
    {
      /* free hipsparseMatDescr_t */
      SUNDIALS_CUSPARSE_VERIFY( hipsparseDestroyMatDescr(SMCU_MATDESCR(A)) );
    }

#if CUDART_VERSION >= 11000
    SUNDIALS_CUSPARSE_VERIFY( hipsparseDestroyDnVec(SMCU_CONTENT(A)->vecX) );
    SUNDIALS_CUSPARSE_VERIFY( hipsparseDestroyDnVec(SMCU_CONTENT(A)->vecY) );
    SUNDIALS_CUSPARSE_VERIFY( hipsparseDestroySpMat(SMCU_CONTENT(A)->spmat_descr) );
    SUNMemoryHelper_Dealloc(SMCU_MEMHELP(A), SMCU_CONTENT(A)->dBufferMem,
                            nullptr);
#endif

    if (SMCU_EXECPOLICY(A))
    {
      delete SMCU_EXECPOLICY(A);
      SMCU_EXECPOLICY(A) = NULL;
    }

    SUNMemoryHelper_Destroy(SMCU_MEMHELP(A));

    /* free content struct */
    free(A->content);
    A->content = NULL;
  }

  /* free ops and matrix */
  if (A->ops) { free(A->ops); A->ops = NULL; }
  free(A); A = NULL;

  return;
}


/* Performs A_ij = 0 */
int SUNMatZero_cuSparse(SUNMatrix A)
{
  hipError_t cuerr;
  hipStream_t stream;

  stream = *SMCU_EXECPOLICY(A)->stream();

  /* set all data to zero */
  cuerr = hipMemsetAsync(SMCU_DATAp(A), 0, SMCU_NNZ(A)*sizeof(realtype), stream);
  if (!SUNDIALS_CUDA_VERIFY(cuerr)) return(SUNMAT_OPERATION_FAIL);

  /* set all rowptrs to zero unless the sparsity pattern is fixed */
  if (!SMCU_FIXEDPATTERN(A))
  {
    cuerr = hipMemsetAsync(SMCU_INDEXPTRSp(A), 0,
                            (SMCU_BLOCKROWS(A)+1)*sizeof(int),
                            stream);
    if (!SUNDIALS_CUDA_VERIFY(cuerr)) return(SUNMAT_OPERATION_FAIL);

    /* set all colind to zero */
    cuerr = hipMemsetAsync(SMCU_INDEXVALSp(A), 0,
                            SMCU_BLOCKNNZ(A)*sizeof(int),
                            stream);
    if (!SUNDIALS_CUDA_VERIFY(cuerr)) return(SUNMAT_OPERATION_FAIL);
  }

  return(SUNMAT_SUCCESS);
}


/* Copies the nonzeros, column vals, and row pointers into dst */
int SUNMatCopy_cuSparse(SUNMatrix src, SUNMatrix dst)
{
  int retval;
  const hipStream_t* stream;

  /* Verify that src and dst are compatible */
  if (!SMCompatible_cuSparse(src, dst))
    return(SUNMAT_ILL_INPUT);

  stream = SMCU_EXECPOLICY(src)->stream();

  /* Ensure that dst is allocated with at least as
     much memory as we have nonzeros in src */
  if (SMCU_NNZ(dst) < SMCU_NNZ(src))
  {
    SUNDIALS_DEBUG_PRINT("ERROR in SUNMatCopy_cuSparse: the destination matrix has less nonzeros than the source\n");
    return(SUNMAT_ILL_INPUT);
  }

  /* Zero out dst so that copy works correctly */
  if (SUNMatZero_cuSparse(dst) != SUNMAT_SUCCESS)
  {
    SUNDIALS_DEBUG_PRINT("ERROR in SUNMatCopy_cuSparse: SUNMatZero_cuSparse failed\n");
    return(SUNMAT_OPERATION_FAIL);
  }

  /* Copy the data over */
  retval = SUNMemoryHelper_CopyAsync(SMCU_MEMHELP(src),
                                     SMCU_DATA(dst),
                                     SMCU_DATA(src),
                                     SMCU_NNZ(src)*sizeof(realtype),
                                     (void*) stream);
  if (retval) return(SUNMAT_OPERATION_FAIL);

  /* Copy the row pointers over */
  retval = SUNMemoryHelper_CopyAsync(SMCU_MEMHELP(src),
                                     SMCU_INDEXPTRS(dst),
                                     SMCU_INDEXPTRS(src),
                                     (SMCU_BLOCKROWS(src)+1)*sizeof(int),
                                     (void*) stream);
  if (retval) return(SUNMAT_OPERATION_FAIL);

  /* Copy the column indices over */
  retval = SUNMemoryHelper_CopyAsync(SMCU_MEMHELP(src),
                                     SMCU_INDEXVALS(dst),
                                     SMCU_INDEXVALS(src),
                                     SMCU_BLOCKNNZ(src)*sizeof(int),
                                     (void*) stream);
  if (retval) return(SUNMAT_OPERATION_FAIL);

  return(SUNMAT_SUCCESS);
}


/* Performs A = cA + I. Requires the diagonal to be allocated already. */
int SUNMatScaleAddI_cuSparse(realtype c, SUNMatrix A)
{
  unsigned threadsPerBlock, gridSize;
  hipStream_t stream = *SMCU_EXECPOLICY(A)->stream();

  switch (SMCU_SPARSETYPE(A))
  {
    case SUNMAT_CUSPARSE_CSR:
      /* Choose the grid size to be the number of rows in the matrix,
        and then choose threadsPerBlock to be a multiple of the warp size
        that results in enough threads to have one per 2 columns. */
      threadsPerBlock = SMCU_EXECPOLICY(A)->blockSize(SMCU_COLUMNS(A)/2);
      gridSize = SMCU_EXECPOLICY(A)->gridSize(SMCU_ROWS(A)*SMCU_COLUMNS(A)/2, threadsPerBlock);
      scaleAddIKernelCSR<realtype, int>
        <<<gridSize, threadsPerBlock, 0, stream>>>(SMCU_ROWS(A),
                                                   c,
                                                   SMCU_DATAp(A),
                                                   SMCU_INDEXPTRSp(A),
                                                   SMCU_INDEXVALSp(A));
      break;
    case SUNMAT_CUSPARSE_BCSR:
      /* Choose the grid size to be the number of blocks in the matrix,
         and then choose threadsPerBlock to be a multiple of the warp size
         that results in enough threads to have one per row of the block. */
      threadsPerBlock = SMCU_EXECPOLICY(A)->blockSize(SMCU_BLOCKROWS(A));
      gridSize = SMCU_EXECPOLICY(A)->gridSize(SMCU_NBLOCKS(A)*SMCU_BLOCKROWS(A), threadsPerBlock);
      scaleAddIKernelBCSR<realtype, int>
        <<<gridSize, threadsPerBlock, 0, stream>>>(SMCU_BLOCKROWS(A),
                                                   SMCU_NBLOCKS(A),
                                                   SMCU_BLOCKNNZ(A),
                                                   c,
                                                   SMCU_DATAp(A),
                                                   SMCU_INDEXPTRSp(A),
                                                   SMCU_INDEXVALSp(A));
      break;
    default:
      SUNDIALS_DEBUG_PRINT("ERROR in SUNMatScaleAddI_cuSparse: sparse type not recognized\n");
      return(SUNMAT_ILL_INPUT);
  }

#ifdef SUNDIALS_DEBUG_CUDA_LASTERROR
  hipDeviceSynchronize();
  if (!SUNDIALS_CUDA_VERIFY(hipGetLastError())) return(SUNMAT_OPERATION_FAIL);
#endif

  return(SUNMAT_SUCCESS);
}


/* Performs A = cA + B */
int SUNMatScaleAdd_cuSparse(realtype c, SUNMatrix A, SUNMatrix B)
{
  hipStream_t stream;
  unsigned threadsPerBlock, gridSize;

  if (!SMCompatible_cuSparse(A, B))
  {
    SUNDIALS_DEBUG_PRINT("ERROR in SUNMatScaleAdd_cuSparse: SUNMatScaleAdd_cuSparse failed\n");
    return(SUNMAT_ILL_INPUT);
  }

  stream = *SMCU_EXECPOLICY(A)->stream();

  switch (SMCU_SPARSETYPE(A))
  {
    case SUNMAT_CUSPARSE_CSR:
      /* Choose the grid size to be the number of rows in the matrix,
        and then choose threadsPerBlock to be a multiple of the warp size
        that results in enough threads to have one per 2 columns. */
      threadsPerBlock = SMCU_EXECPOLICY(A)->blockSize(SMCU_COLUMNS(A)/2);
      gridSize = SMCU_EXECPOLICY(A)->gridSize(SMCU_ROWS(A)*SMCU_COLUMNS(A)/2, threadsPerBlock);
      scaleAddKernelCSR<realtype, int>
        <<<gridSize, threadsPerBlock, 0, stream>>>(SMCU_NNZ(A),
                                                   c,
                                                   SMCU_DATAp(A),
                                                   SMCU_DATAp(B));
      break;
    case SUNMAT_CUSPARSE_BCSR:
      /* Choose the grid size to be the number of blocks in the matrix,
         and then choose threadsPerBlock to be a multiple of the warp size
         that results in enough threads to have one per row of the block. */
      threadsPerBlock = SMCU_EXECPOLICY(A)->blockSize(SMCU_BLOCKROWS(A));
      gridSize = SMCU_EXECPOLICY(A)->gridSize(SMCU_NBLOCKS(A)*SMCU_BLOCKROWS(A), threadsPerBlock);
      scaleAddKernelCSR<realtype, int>
        <<<gridSize, threadsPerBlock, 0, stream>>>(SMCU_NNZ(A),
                                                   c,
                                                   SMCU_DATAp(A),
                                                   SMCU_DATAp(B));
      break;
    default:
      SUNDIALS_DEBUG_PRINT("ERROR in SUNMatScaleAdd_cuSparse: sparse type not recognized\n");
      return(SUNMAT_ILL_INPUT);
  }

#ifdef SUNDIALS_DEBUG_CUDA_LASTERROR
  hipDeviceSynchronize();
  if (!SUNDIALS_CUDA_VERIFY(hipGetLastError())) return(SUNMAT_OPERATION_FAIL);
#endif

  return(SUNMAT_SUCCESS);
}

/* Setup buffers needed for Matvec */
int SUNMatMatvecSetup_cuSparse(SUNMatrix A)
{
#if CUDART_VERSION >= 11000
  realtype placeholder[1];
  const realtype one = ONE;

  /* Check if setup has already been done */
  if (!(SMCU_CONTENT(A)->matvec_issetup))
  {
    SUNDIALS_CUSPARSE_VERIFY( hipsparseCreateDnVec(&SMCU_CONTENT(A)->vecX,
                                                  SMCU_COLUMNS(A),
                                                  placeholder, CUDA_R_XF) );
    SUNDIALS_CUSPARSE_VERIFY( hipsparseCreateDnVec(&SMCU_CONTENT(A)->vecY,
                                                  SMCU_ROWS(A),
                                                  placeholder, CUDA_R_XF) );

    SUNDIALS_CUSPARSE_VERIFY(
      hipsparseSpMV_bufferSize(SMCU_CUSPHANDLE(A),
                              HIPSPARSE_OPERATION_NON_TRANSPOSE,
                              &one, SMCU_CONTENT(A)->spmat_descr,
                              SMCU_CONTENT(A)->vecX, &one, SMCU_CONTENT(A)->vecY,
                              CUDA_R_XF, HIPSPARSE_MV_ALG_DEFAULT,
                              &SMCU_CONTENT(A)->bufferSize) );

    if ( SUNMemoryHelper_Alloc(SMCU_MEMHELP(A), &SMCU_CONTENT(A)->dBufferMem,
                               SMCU_CONTENT(A)->bufferSize, SUNMEMTYPE_DEVICE,
                               nullptr) )
      return(SUNMAT_OPERATION_FAIL);
  }
#endif
  SMCU_CONTENT(A)->matvec_issetup = SUNTRUE;
  return(SUNMAT_SUCCESS);
}

/* Perform y = Ax */
int SUNMatMatvec_cuSparse(SUNMatrix A, N_Vector x, N_Vector y)
{
  /* Verify that the dimensions of A, x, and y agree */
  if ( (SMCU_COLUMNS(A) != N_VGetLength(x)) ||
       (SMCU_ROWS(A) != N_VGetLength(y)) )
  {
    SUNDIALS_DEBUG_PRINT("ERROR in SUNMatMatvec_cuSparse: dimensions do not agree\n");
    return(SUNMAT_ILL_INPUT);
  }

  realtype *d_xdata = N_VGetDeviceArrayPointer(x);
  realtype *d_ydata = N_VGetDeviceArrayPointer(y);

  if (SMCU_SPARSETYPE(A) == SUNMAT_CUSPARSE_CSR)
  {
    const realtype one = ONE;

    /* Zero result vector */
    N_VConst(ZERO, y);

#if CUDART_VERSION >= 11000
    {
      /* Setup matvec if it has not been done yet */
      if (!SMCU_CONTENT(A)->matvec_issetup && SUNMatMatvecSetup_cuSparse(A))
      {
        return(SUNMAT_OPERATION_FAIL);
      }

      SUNDIALS_CUSPARSE_VERIFY( hipsparseDnVecSetValues(SMCU_CONTENT(A)->vecX,
                                                       d_xdata) );
      SUNDIALS_CUSPARSE_VERIFY( hipsparseDnVecSetValues(SMCU_CONTENT(A)->vecY,
                                                       d_ydata) );

      SUNDIALS_CUSPARSE_VERIFY( hipsparseSpMV(SMCU_CUSPHANDLE(A),
                                             HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                             &one, SMCU_CONTENT(A)->spmat_descr,
                                             SMCU_CONTENT(A)->vecX, &one,
                                             SMCU_CONTENT(A)->vecY, CUDA_R_XF,
                                             HIPSPARSE_MV_ALG_DEFAULT,
                                             SMCU_CONTENT(A)->dBufferMem->ptr) );
    }
#else
    SUNDIALS_CUSPARSE_VERIFY(
      cusparseXcsrmv(SMCU_CUSPHANDLE(A), HIPSPARSE_OPERATION_NON_TRANSPOSE,
                     SMCU_ROWS(A), SMCU_COLUMNS(A), SMCU_NNZ(A),
                     &one, SMCU_MATDESCR(A), SMCU_DATAp(A), SMCU_INDEXPTRSp(A),
                     SMCU_INDEXVALSp(A), d_xdata, &one, d_ydata) );
#endif
  }
  else if (SMCU_SPARSETYPE(A) == SUNMAT_CUSPARSE_BCSR)
  {
    hipStream_t stream;
    unsigned gridSize, threadsPerBlock;

    stream = *SMCU_EXECPOLICY(A)->stream();

    /* Choose the grid size to be the number of blocks in the matrix,
       and then choose threadsPerBlock to be a multiple of the warp size
       that results in enough threads to have one per row of the block. */
    threadsPerBlock = SMCU_EXECPOLICY(A)->blockSize(SMCU_COLUMNS(A)/2);
    gridSize = SMCU_EXECPOLICY(A)->gridSize(SMCU_ROWS(A)*SMCU_COLUMNS(A)/2, threadsPerBlock);
    matvecBCSR<realtype, int>
      <<<gridSize, threadsPerBlock, 0, stream>>>(SMCU_BLOCKROWS(A),
                                                 SMCU_NBLOCKS(A),
                                                 SMCU_BLOCKNNZ(A),
                                                 SMCU_DATAp(A),
                                                 SMCU_INDEXPTRSp(A),
                                                 SMCU_INDEXVALSp(A),
                                                 d_xdata,
                                                 d_ydata);
  }
  else
  {
    SUNDIALS_DEBUG_PRINT("ERROR in SUNMatMatvec_cuSparse: sparse type not recognized\n");
    return(SUNMAT_ILL_INPUT);
  }

#ifdef SUNDIALS_DEBUG_CUDA_LASTERROR
    hipDeviceSynchronize();
    if (!SUNDIALS_CUDA_VERIFY(hipGetLastError())) return(SUNMAT_OPERATION_FAIL);
#endif

  return(SUNMAT_SUCCESS);
}


/*
 * =================================================================
 * private functions
 * =================================================================
 */


/* -----------------------------------------------------------------
 * Function to check compatibility of two sparse SUNMatrix objects
 */
static booleantype SMCompatible_cuSparse(SUNMatrix A, SUNMatrix B)
{
  /* both matrices must be sparse */
  if ( (SUNMatGetID(A) != SUNMATRIX_CUSPARSE) ||
       (SUNMatGetID(B) != SUNMATRIX_CUSPARSE) )
    return(SUNFALSE);

  /* both matrices must have the same shape and sparsity type */
  if (SMCU_ROWS(A) != SMCU_ROWS(B))
    return(SUNFALSE);
  if (SMCU_COLUMNS(A) != SMCU_COLUMNS(B))
    return(SUNFALSE);
  if (SMCU_SPARSETYPE(A) != SMCU_SPARSETYPE(B))
    return(SUNFALSE);

  return(SUNTRUE);
}

/* -----------------------------------------------------------------
 * Function to create empty SUNMatrix with ops attached and
 * the content structure allocated.
 */
SUNMatrix SUNMatrix_cuSparse_NewEmpty(SUNContext sunctx)
{
  /* Create an empty matrix object */
  SUNMatrix A = NULL;
  A = SUNMatNewEmpty(sunctx);
  if (A == NULL)
  {
    SUNDIALS_DEBUG_PRINT("ERROR in SUNMatrix_cuSparse_NewEmpty: SUNMatNewEmpty failed\n");
    return(NULL);
  }

  /* Attach operations */
  A->ops->getid       = SUNMatGetID_cuSparse;
  A->ops->clone       = SUNMatClone_cuSparse;
  A->ops->destroy     = SUNMatDestroy_cuSparse;
  A->ops->zero        = SUNMatZero_cuSparse;
  A->ops->copy        = SUNMatCopy_cuSparse;
  A->ops->scaleadd    = SUNMatScaleAdd_cuSparse;
  A->ops->scaleaddi   = SUNMatScaleAddI_cuSparse;
  A->ops->matvecsetup = SUNMatMatvecSetup_cuSparse;
  A->ops->matvec      = SUNMatMatvec_cuSparse;

  /* Create content */
  SUNMatrix_Content_cuSparse content = NULL;
  content = (SUNMatrix_Content_cuSparse) malloc(sizeof *content);
  if (content == NULL)
  {
    SUNDIALS_DEBUG_PRINT("ERROR in SUNMatrix_cuSparse_NewEmpty: failed to malloc content\n");
    SUNMatDestroy(A);
    return(NULL);
  }

  /* Attach content */
  A->content = content;
  content->mem_helper = NULL;

  return(A);
}

#if CUDART_VERSION >= 11000
hipsparseStatus_t CreateSpMatDescr(SUNMatrix A, hipsparseSpMatDescr_t *spmat_descr)
{
  /* CUDA 11 introduced the "Generic API" and removed the cusparseXcsrmv that
    works on the old hipsparseMatDescr_t and raw data arrays. However,
    cuSolverSp stuff requires the hipsparseMatDescr_t still. So, we have to
    create this hipsparseSpMatDescr_t *and* the hipsparseMatDescr_t. */
  return(hipsparseCreateCsr(spmat_descr, SMCU_ROWS(A), SMCU_COLUMNS(A),
                           SMCU_NNZ(A), SMCU_INDEXPTRSp(A),
                           SMCU_INDEXVALSp(A), SMCU_DATAp(A),
                           HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                           HIPSPARSE_INDEX_BASE_ZERO, CUDA_R_XF));
}
#endif
