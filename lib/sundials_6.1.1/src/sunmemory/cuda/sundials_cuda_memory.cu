/* -----------------------------------------------------------------
 * Programmer(s): Cody J. Balos @ LLNL
 * -----------------------------------------------------------------
 * SUNDIALS Copyright Start
 * Copyright (c) 2002-2022, Lawrence Livermore National Security
 * and Southern Methodist University.
 * All rights reserved.
 *
 * See the top-level LICENSE and NOTICE files for details.
 *
 * SPDX-License-Identifier: BSD-3-Clause
 * SUNDIALS Copyright End
 * -----------------------------------------------------------------
 * SUNDIALS CUDA memory helper implementation.
 * ----------------------------------------------------------------*/

#include <cstdlib>

#include <sunmemory/sunmemory_cuda.h>
#include "sundials_debug.h"
#include "sundials_cuda.h"


SUNMemoryHelper SUNMemoryHelper_Cuda(SUNContext sunctx)
{
  SUNMemoryHelper helper;

  /* Allocate the helper */
  helper = SUNMemoryHelper_NewEmpty(sunctx);

  /* Set the ops */
  helper->ops->alloc     = SUNMemoryHelper_Alloc_Cuda;
  helper->ops->dealloc   = SUNMemoryHelper_Dealloc_Cuda;
  helper->ops->copy      = SUNMemoryHelper_Copy_Cuda;
  helper->ops->copyasync = SUNMemoryHelper_CopyAsync_Cuda;

  /* Attach content and ops */
  helper->content = NULL;

  return helper;
}

int SUNMemoryHelper_Alloc_Cuda(SUNMemoryHelper helper, SUNMemory* memptr,
                               size_t mem_size, SUNMemoryType mem_type,
                               void* queue)
{
  SUNMemory mem = SUNMemoryNewEmpty();

  mem->ptr  = NULL;
  mem->own  = SUNTRUE;
  mem->type = mem_type;

  if (mem_type == SUNMEMTYPE_HOST)
  {
    mem->ptr = malloc(mem_size);
    if (mem->ptr == NULL)
    {
      SUNDIALS_DEBUG_PRINT("ERROR in SUNMemoryHelper_Alloc_Cuda: malloc returned NULL\n");
      free(mem);
      return(-1);
    }
  }
  else if (mem_type == SUNMEMTYPE_PINNED)
  {
    if (!SUNDIALS_CUDA_VERIFY(hipHostMalloc(&(mem->ptr), mem_size)))
    {
      SUNDIALS_DEBUG_PRINT("ERROR in SUNMemoryHelper_Alloc_Cuda: hipHostMalloc failed\n");
      free(mem);
      return(-1);
    }
  }
  else if (mem_type == SUNMEMTYPE_DEVICE)
  {
    if (!SUNDIALS_CUDA_VERIFY(hipMalloc(&(mem->ptr), mem_size)))
    {
      SUNDIALS_DEBUG_PRINT("ERROR in SUNMemoryHelper_Alloc_Cuda: hipMalloc failed\n");
      free(mem);
      return(-1);
    }
  }
  else if (mem_type == SUNMEMTYPE_UVM)
  {
    if (!SUNDIALS_CUDA_VERIFY(hipMallocManaged(&(mem->ptr), mem_size)))
    {
      SUNDIALS_DEBUG_PRINT("ERROR in SUNMemoryHelper_Alloc_Cuda: hipMallocManaged failed\n");
      free(mem);
      return(-1);
    }
  }
  else
  {
    SUNDIALS_DEBUG_PRINT("ERROR in SUNMemoryHelper_Alloc_Cuda: unknown memory type\n");
    free(mem);
    return(-1);
  }

  *memptr = mem;
  return(0);
}

int SUNMemoryHelper_Dealloc_Cuda(SUNMemoryHelper helper, SUNMemory mem,
                                 void *queue)
{
  if (mem == NULL) return(0);

  if (mem->ptr != NULL && mem->own)
  {
    if (mem->type == SUNMEMTYPE_HOST)
    {
      free(mem->ptr);
      mem->ptr = NULL;
    }
    else if (mem->type == SUNMEMTYPE_PINNED)
    {
      if (!SUNDIALS_CUDA_VERIFY(hipHostFree(mem->ptr)))
      {
        SUNDIALS_DEBUG_PRINT("ERROR in SUNMemoryHelper_Dealloc_Cuda: hipHostFree failed\n");
        return(-1);
      }
      mem->ptr = NULL;
    }
    else if (mem->type == SUNMEMTYPE_DEVICE ||
             mem->type == SUNMEMTYPE_UVM)
    {
      if (!SUNDIALS_CUDA_VERIFY(hipFree(mem->ptr)))
      {
        SUNDIALS_DEBUG_PRINT("ERROR in SUNMemoryHelper_Dealloc_Cuda: hipFree failed\n");
        return(-1);
      }
      mem->ptr = NULL;
    }
    else
    {
      SUNDIALS_DEBUG_PRINT("ERROR in SUNMemoryHelper_Dealloc_Cuda: unknown memory type\n");
      return(-1);
    }
  }

  free(mem);
  return(0);
}

int SUNMemoryHelper_Copy_Cuda(SUNMemoryHelper helper, SUNMemory dst,
                              SUNMemory src, size_t memory_size, void* queue)
{
  int retval = 0;
  hipError_t cuerr = hipSuccess;

  switch(src->type)
  {
    case SUNMEMTYPE_HOST:
    case SUNMEMTYPE_PINNED:
      if (dst->type == SUNMEMTYPE_HOST ||
          dst->type == SUNMEMTYPE_PINNED)
      {
        memcpy(dst->ptr, src->ptr, memory_size);
      }
      else if (dst->type == SUNMEMTYPE_DEVICE ||
               dst->type == SUNMEMTYPE_UVM)
      {
        cuerr = hipMemcpy(dst->ptr, src->ptr,
                           memory_size,
                           hipMemcpyHostToDevice);
      }
      if (!SUNDIALS_CUDA_VERIFY(cuerr)) retval = -1;
      break;
    case SUNMEMTYPE_UVM:
    case SUNMEMTYPE_DEVICE:
      if (dst->type == SUNMEMTYPE_HOST ||
          dst->type == SUNMEMTYPE_PINNED)
      {
        cuerr = hipMemcpy(dst->ptr, src->ptr,
                           memory_size,
                           hipMemcpyDeviceToHost);
      }
      else if (dst->type == SUNMEMTYPE_DEVICE ||
               dst->type == SUNMEMTYPE_UVM)
      {
        cuerr = hipMemcpy(dst->ptr, src->ptr,
                           memory_size,
                           hipMemcpyDeviceToDevice);
      }
      if (!SUNDIALS_CUDA_VERIFY(cuerr)) retval = -1;
      break;
    default:
      SUNDIALS_DEBUG_PRINT("ERROR in SUNMemoryHelper_CopyAsync_Cuda: unknown memory type\n");
      retval = -1;
  }

  return(retval);
}

int SUNMemoryHelper_CopyAsync_Cuda(SUNMemoryHelper helper, SUNMemory dst,
                                   SUNMemory src, size_t memory_size,
                                   void* queue)
{
  int retval = 0;
  hipError_t cuerr = hipSuccess;
  hipStream_t stream = 0;

  if (queue != NULL)
  {
    stream = *((hipStream_t*) queue);
  }

  switch(src->type)
  {
    case SUNMEMTYPE_HOST:
    case SUNMEMTYPE_PINNED:
      if (dst->type == SUNMEMTYPE_HOST ||
          dst->type == SUNMEMTYPE_PINNED)
      {
        memcpy(dst->ptr, src->ptr, memory_size);
      }
      else if (dst->type == SUNMEMTYPE_DEVICE ||
               dst->type == SUNMEMTYPE_UVM)
      {
        cuerr = hipMemcpyAsync(dst->ptr, src->ptr,
                                memory_size,
                                hipMemcpyHostToDevice,
                                stream);
      }
      if (!SUNDIALS_CUDA_VERIFY(cuerr)) retval = -1;
      break;
    case SUNMEMTYPE_UVM:
    case SUNMEMTYPE_DEVICE:
      if (dst->type == SUNMEMTYPE_HOST ||
          dst->type == SUNMEMTYPE_PINNED)
      {
        cuerr = hipMemcpyAsync(dst->ptr, src->ptr,
                                memory_size,
                                hipMemcpyDeviceToHost,
                                stream);
      }
      else if (dst->type == SUNMEMTYPE_DEVICE ||
              dst->type == SUNMEMTYPE_UVM)
      {
        cuerr = hipMemcpyAsync(dst->ptr, src->ptr,
                                memory_size,
                                hipMemcpyDeviceToDevice,
                                stream);
      }
      if (!SUNDIALS_CUDA_VERIFY(cuerr)) retval = -1;
      break;
    default:
      SUNDIALS_DEBUG_PRINT("ERROR in SUNMemoryHelper_CopyAsync_Cuda: unknown memory type\n");
      retval = -1;
  }

  return(retval);
}
